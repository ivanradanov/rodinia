#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include "binomialOptions.h"
#include "realtype.h"


//Preprocessed input option data
typedef struct
{
  real S;
  real X;
  real vDt;
  real puByDf;
  real pdByDf;
} __TOptionData;


// Overloaded shortcut functions for different precision modes
#ifndef DOUBLE_PRECISION
__device__ inline float expiryCallValue(float S, float X, float vDt, int i)
{
  float d = S * __expf(vDt * (2.0f * i - NUM_STEPS)) - X;
  return (d > 0.0F) ? d : 0.0F;
}
#else
__device__ inline double expiryCallValue(double S, double X, double vDt, int i)
{
  double d = S * exp(vDt * (2.0 * i - NUM_STEPS)) - X;
  return (d > 0.0) ? d : 0.0;
}
#endif


// GPU kernel
#define THREADBLOCK_SIZE 128
#define ELEMS_PER_THREAD (NUM_STEPS/THREADBLOCK_SIZE)
#if NUM_STEPS % THREADBLOCK_SIZE
#error Bad constants
#endif

__global__ void binomialOptionsKernel(const __TOptionData *__restrict d_OptionData,
                                      real *__restrict d_CallValue)
{
  __shared__ real call_exchange[THREADBLOCK_SIZE + 1];

  const int     tid = threadIdx.x;
  const real      S = d_OptionData[blockIdx.x].S;
  const real      X = d_OptionData[blockIdx.x].X;
  const real    vDt = d_OptionData[blockIdx.x].vDt;
  const real puByDf = d_OptionData[blockIdx.x].puByDf;
  const real pdByDf = d_OptionData[blockIdx.x].pdByDf;

  real call[ELEMS_PER_THREAD + 1];
#pragma unroll
  for(int i = 0; i < ELEMS_PER_THREAD; ++i)
    call[i] = expiryCallValue(S, X, vDt, tid * ELEMS_PER_THREAD + i);

  if (tid == 0)
    call_exchange[THREADBLOCK_SIZE] = expiryCallValue(S, X, vDt, NUM_STEPS);

  int final_it = max(0, tid * ELEMS_PER_THREAD - 1);

#pragma unroll 16
  for(int i = NUM_STEPS; i > 0; --i)
  {
    call_exchange[tid] = call[0];
    __syncthreads();
    call[ELEMS_PER_THREAD] = call_exchange[tid + 1];
    __syncthreads();

    if (i > final_it)
    {
#pragma unroll
      for(int j = 0; j < ELEMS_PER_THREAD; ++j)
        call[j] = puByDf * call[j + 1] + pdByDf * call[j];
    }
  }

  if (tid == 0)
  {
    d_CallValue[blockIdx.x] = call[0];
  }
}

// Host-side interface to GPU binomialOptions
extern "C" void binomialOptionsGPU(
    real *callValue,
    TOptionData  *optionData,
    int optN,
    int numIterations
    )
{
  __TOptionData h_OptionData[MAX_OPTIONS];

  for (int i = 0; i < optN; i++)
  {
    const real      T = optionData[i].T;
    const real      R = optionData[i].R;
    const real      V = optionData[i].V;

    const real     dt = T / (real)NUM_STEPS;
    const real    vDt = V * sqrt(dt);
    const real    rDt = R * dt;
    //Per-step interest and discount factors
    const real     If = exp(rDt);
    const real     Df = exp(-rDt);
    //Values and pseudoprobabilities of upward and downward moves
    const real      u = exp(vDt);
    const real      d = exp(-vDt);
    const real     pu = (If - d) / (u - d);
    const real     pd = (real)1.0 - pu;
    const real puByDf = pu * Df;
    const real pdByDf = pd * Df;

    h_OptionData[i].S      = (real)optionData[i].S;
    h_OptionData[i].X      = (real)optionData[i].X;
    h_OptionData[i].vDt    = (real)vDt;
    h_OptionData[i].puByDf = (real)puByDf;
    h_OptionData[i].pdByDf = (real)pdByDf;
  }

  __TOptionData *d_OptionData;
  hipMalloc ((void**)&d_OptionData, sizeof(__TOptionData) * MAX_OPTIONS);
  hipMemcpy(d_OptionData, h_OptionData, optN * sizeof(__TOptionData), hipMemcpyHostToDevice);

  real *d_CallValue;
  hipMalloc ((void**)&d_CallValue, sizeof(real) * MAX_OPTIONS);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < numIterations; i++)
    binomialOptionsKernel<<<optN, THREADBLOCK_SIZE>>>(d_OptionData, d_CallValue);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time : %f (us)\n", time * 1e-3f / numIterations);

  hipMemcpy(callValue, d_CallValue, optN *sizeof(real), hipMemcpyDeviceToHost);
  hipFree(d_OptionData);
  hipFree(d_CallValue);
}
