#include <iostream>
#include <cstdlib>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include "constants_types.h"
#include "kernel.h"

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  fArray *d_cur_forward;
  fArray *d_next_forward;
  fArray *d_emis;
  tArray *d_trans;
  lArray *d_like;
  sArray *d_start;

  dim3 dimGrid(batch);
  dim3 dimBlock(states-1);

  size_t forward_matrix_size = (x_dim+1)*(y_dim+1)*batch*(states-1)*sizeof(double);
  size_t emissions_size = (x_dim+1)*(y_dim+1)*batch*(states-1)*sizeof(double);
  size_t transitions_size = (x_dim+1)*(states-1)*states*batch*sizeof(double);
  size_t start_transitions_size = batch*(states-1)*sizeof(double);
  size_t likelihood_size = 2*2*(states-1)*batch*sizeof(double);

  fArray *h_cur_forward = (fArray*) malloc (forward_matrix_size); 
  fArray *h_emis = (fArray*) malloc (emissions_size);
  tArray *h_trans = (tArray*) malloc (transitions_size);
  lArray *h_like = (lArray*) malloc (likelihood_size);
  sArray *h_start = (sArray*) malloc (start_transitions_size);

  std::default_random_engine rng (123);
  std::uniform_real_distribution<double> dist (0.0, 1.0);
  for (int i = 0; i < x_dim+1; i++) {
    for (int j = 0; j < y_dim+1; j++) {
      for (int b = 0; b < batch; b++) {
        for (int s = 0; s < states-1; s++) {
           h_cur_forward[i][j][b][s] = dist(rng);
           h_emis[i][j][b][s] = dist(rng);
        }
      }
    }
  }

  for (int i = 0; i < x_dim+1; i++) {
    for (int b = 0; b < batch; b++) {
      for (int s = 0; s < states-1; s++) {
        for (int t = 0; t < states; t++) {
          h_trans[i][b][s][t] = dist(rng);
        }
      }
    }
  }
         
  for (int i = 0; i < batch; i++) {
    for (int s = 0; s < states-1; s++) {
      h_start[i][s] = dist(rng);
    }
  }

  for (int i = 0; i < 2; i++) {
    for (int j = 0; j< 2; j++) {
      for (int b = 0; b < batch; b++) {
        for (int s = 0; s < states-1; s++) {
          h_like[i][j][b][s] = dist(rng);
        }
      }
    }
  }

  hipMalloc((void**)&d_cur_forward, forward_matrix_size); 
  hipMemcpy(d_cur_forward, h_cur_forward, forward_matrix_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_next_forward, forward_matrix_size);  

  hipMalloc((void**)&d_emis, emissions_size);
  hipMemcpy(d_emis, h_emis, forward_matrix_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_trans, transitions_size);
  hipMemcpy(d_trans, h_trans, transitions_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_like, likelihood_size);
  hipMemcpy(d_like, h_like, likelihood_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_start, start_transitions_size);
  hipMemcpy(d_start, h_start, start_transitions_size, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  auto t1 = std::chrono::high_resolution_clock::now();

  for(int count = 0; count < repeat; count++) {
    for (int i = 1; i < x_dim + 1; i++) {
      for (int j = 1; j < y_dim + 1; j++) {
        pair_HMM_forward<<<dimGrid, dimBlock>>>(i, j, d_cur_forward, d_trans, d_emis, d_like, d_start, d_next_forward);
        auto t = d_cur_forward;
        d_cur_forward = d_next_forward;
        d_next_forward = t;
      }
    }
  }
  hipDeviceSynchronize();

  auto t2 = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> milli = (t2 - t1);
  std::cout << "Total execution time " <<  milli.count() << " milliseconds\n" ;

  hipMemcpy(h_cur_forward, d_cur_forward, forward_matrix_size, hipMemcpyDeviceToHost);

  double checkSum = 0.0;
  for (int i = 0; i < x_dim+1; i++) {
    for (int j = 0; j < y_dim+1; j++) {
      for (int b = 0; b < batch; b++) {
        for (int s = 0; s < states-1; s++) {
          #ifdef DEBUG
          std::cout << h_cur_forward[i][j][b][s] << std::endl;
          #endif
          checkSum += h_cur_forward[i][j][b][s];
        }
      }
    }
  }
  std::cout << "Checksum " << checkSum << std::endl;

  hipFree(d_cur_forward);
  hipFree(d_next_forward);
  hipFree(d_emis);
  hipFree(d_trans);
  hipFree(d_like);
  hipFree(d_start);
  free(h_cur_forward);
  free(h_emis);
  free(h_trans);
  free(h_like);
  free(h_start);

  return 0;
}
