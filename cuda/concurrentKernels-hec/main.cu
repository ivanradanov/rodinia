#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also
// illustrates how to introduce dependencies between CUDA streams with the
// hipStreamWaitEvent function.
//

// Devices of compute capability 2.0 or higher can overlap the kernels
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000) + tv.tv_usec;
}

// This is a kernel that does no real work but runs at least for a specified
// number
__global__ void clock_block(long *d_o, long clock_count) {
  long clock_offset = 0;
  for (int i = 0; i < clock_count; i++)
    clock_offset += i % 3;
  d_o[0] = clock_offset;
}

// Single warp reduction kernel
__global__ void sum(long *d_clocks, int N) {
  // Handle to thread block group
  __shared__ long s_clocks[32];

  long my_sum = 0;

  for (int i = threadIdx.x; i < N; i += blockDim.x) {
    my_sum += d_clocks[i];
  }

  s_clocks[threadIdx.x] = my_sum;
  __syncthreads();

  for (int i = 16; i > 0; i /= 2) {
    if (threadIdx.x < i) {
      s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
    }
    __syncthreads();
  }

  d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <number of concurrent kernels>\n", argv[0]);
    return 1;
  }
    
  int nkernels = atoi(argv[1]);         // number of concurrent kernels (at least 1)
  int nstreams = nkernels + 1;          // use one more stream than concurrent kernel
  int nbytes = nkernels * sizeof(long); // number of data bytes
  float kernel_time = 20;               // time the kernel should run
  int cuda_device = 0;

  printf("[%s] - Starting...\n", argv[0]);

  long start = get_time();

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_device);

  // allocate host memory
  long *a;  // pointer to the array data in host memory
  hipHostMalloc((void **)&a, nbytes);

  // allocate device memory
  long *d_a;  // pointers to data and init value in the device memory
  hipMalloc((void **)&d_a, nbytes);

  // allocate and initialize an array of stream handles
  hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  for (int i = 0; i < nstreams; i++) {
    hipStreamCreate(&(streams[i]));
  }

  // the events are used for synchronization only and hence do not need to
  // record timings this also makes events not introduce global sync points when
  // recorded which is critical to get overlap
  hipEvent_t *kernelEvent;
  kernelEvent = (hipEvent_t *)malloc(nkernels * sizeof(hipEvent_t));

  for (int i = 0; i < nkernels; i++) {
    hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming);
  }

  // time execution with nkernels streams
  long total_clocks = 0;
  long time_clocks = (long)(kernel_time * deviceProp.clockRate);
  printf("time clocks = %ld\n", time_clocks);

  // queue nkernels in separate streams and record when they are done
  for (int i = 0; i < nkernels; ++i) {
    clock_block<<<1, 1, 0, streams[i]>>>(&d_a[i], time_clocks);
    total_clocks += time_clocks;
    hipEventRecord(kernelEvent[i], streams[i]);

    // make the last stream wait for the kernel event to be recorded
    hipStreamWaitEvent(streams[nstreams - 1], kernelEvent[i], 0);
  }

  // queue a sum kernel and a copy back to host in the last stream.
  // the commands in this stream get dispatched as soon as all the kernel events
  // have been recorded
  sum<<<1, 32, 0, streams[nstreams - 1]>>>(d_a, nkernels);
  hipMemcpyAsync(a, d_a, sizeof(long), hipMemcpyDeviceToHost, streams[nstreams - 1]);

  // at this point the CPU has dispatched all work for the GPU and can continue
  // processing other tasks in parallel

  // wait until the GPU is done
  hipDeviceSynchronize();

  long end = get_time();
  printf("Measured time for sample = %.3fs\n", (end-start) / 1e6f);

  // check the result
  long sum = 0;
  for (int i = 0; i < time_clocks; i++) sum += i % 3;
  printf("%s\n", a[0] == nkernels * sum ? "PASS" : "FAIL");

  // release resources
  for (int i = 0; i < nkernels; i++) {
    hipStreamDestroy(streams[i]);
    hipEventDestroy(kernelEvent[i]);
  }

  free(streams);
  free(kernelEvent);

  hipHostFree(a);
  hipFree(d_a);

  return 0;
}
