#include "hip/hip_runtime.h"
#include "distance.h"

__global__ void compute_haversine_distance(
  const double4 *__restrict__ p,
        double*__restrict__ distance,
  const int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    auto ay = p[i].x * DEGREE_TO_RADIAN;  // a_lat
    auto ax = p[i].y * DEGREE_TO_RADIAN;  // a_lon
    auto by = p[i].z * DEGREE_TO_RADIAN;  // b_lat
    auto bx = p[i].w * DEGREE_TO_RADIAN;  // b_lon

    // haversine formula
    auto x        = (bx - ax) / 2.0;
    auto y        = (by - ay) / 2.0;
    auto sinysqrd = sin(y) * sin(y);
    auto sinxsqrd = sin(x) * sin(x);
    auto scale    = cos(ay) * cos(by);
    distance[i] = 2.0 * EARTH_RADIUS_KM * asin(sqrt(sinysqrd + sinxsqrd * scale));
  }
}

void distance_device(const double4* loc, double* dist, const int n, const int iteration) {

  dim3 grids ((n+255)/256);
  dim3 threads (256);

  double4 *d_loc;
  double *d_dist;
  hipMalloc((void**)&d_loc, sizeof(double4)*n);
  hipMemcpy(d_loc, loc, sizeof(double4)*n, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_dist, sizeof(double)*n);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < iteration; i++) {
    compute_haversine_distance<<<grids, threads>>>(d_loc, d_dist, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (s)\n", (time * 1e-9f) / iteration);

  hipMemcpy(dist, d_dist, sizeof(double)*n, hipMemcpyDeviceToHost);
  hipFree(d_loc);
  hipFree(d_dist);
}
