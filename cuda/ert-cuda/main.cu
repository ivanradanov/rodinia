#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>

#define ERT_ALIGN           256
#define ERT_NUM_EXPERIMENTS 1
#define ERT_MEMORY_MAX      33554432
#define ERT_WORKING_SET_MIN 128
#define ERT_TRIALS_MIN      1
#define ERT_WSS_MULT        1.3

#include "kernel.h"

double getTime()
{
  double time;
  struct timeval tm;
  gettimeofday(&tm, NULL);
  time = tm.tv_sec + (tm.tv_usec / 1000000.0);
  return time;
}

template <typename T>
T *alloc(uint64_t psize)
{
  T* buffer = (T *)calloc(psize/sizeof(T), sizeof(T));
  if (buffer == nullptr) {
    fprintf(stderr, "Out of memory!\n");
    exit(1);
  }
  return buffer;
}

template <typename T>
inline void launchKernel(uint64_t n, uint64_t t, T *buf, T *d_buf, int *bytes_per_elem_ptr,
    int *mem_accesses_per_elem_ptr)
{
  gpuKernel<T>(n, t, d_buf, bytes_per_elem_ptr, mem_accesses_per_elem_ptr);
}

template <typename T>
void run(uint64_t PSIZE, T *buf)
{

  uint64_t nsize = PSIZE;
  nsize          = nsize & (~(ERT_ALIGN - 1));
  nsize          = nsize / sizeof(T);

  T *d_buf;
  hipMalloc((void **)&d_buf, nsize * sizeof(T));
  hipMemset(d_buf, 0, nsize * sizeof(T));

  uint64_t n, nNew;
  uint64_t t;
  int bytes_per_elem;
  int mem_accesses_per_elem;

  n = ERT_WORKING_SET_MIN;
  while (n <= nsize) { // working set - nsize

    uint64_t ntrials = nsize / n;
    if (ntrials < ERT_TRIALS_MIN)
      ntrials = ERT_TRIALS_MIN;

    // initialize small chunck of buffer within each thread
    float value = -1.f;
    initialize<T>(nsize, buf, value);

    hipMemcpy(d_buf, buf, n * sizeof(T), hipMemcpyHostToDevice);

    for (t = 1; t <= ntrials; t = t * 2) { // working set - ntrials
      launchKernel<T>(n, t, buf, d_buf, &bytes_per_elem, &mem_accesses_per_elem);
    } // working set - ntrials

    hipMemcpy(buf, d_buf, n * sizeof(T), hipMemcpyDeviceToHost);

    nNew = ERT_WSS_MULT * n;
    if (nNew == n) {
      nNew = n + 1;
    }

    n = nNew;
  } // working set - nsize

  hipFree(d_buf);

  if (hipGetLastError() != hipSuccess) {
    printf("Last GPU error: %s\n", hipGetErrorString(hipGetLastError()));
  }
}

int main(int argc, char *argv[])
{
  if (argc != 3) {
    fprintf(stderr, "Usage: %s gpu_blocks gpu_threads\n", argv[0]);
    return -1;
  }

  gpu_blocks  = atoi(argv[1]);
  gpu_threads = atoi(argv[2]);
  printf("\n");
  printf("GPU_BLOCKS     %d\n", gpu_blocks);
  printf("GPU_THREADS    %d\n", gpu_threads);

  uint64_t TSIZE = ERT_MEMORY_MAX;
  uint64_t PSIZE = TSIZE;
  double start, checksum;

  // FP16
  half2 *hlfbuf = alloc<half2>(PSIZE);
  start = getTime();
  run<half2>(PSIZE, hlfbuf);
  printf("runtime (half2): %lf (s)\n", getTime() - start);
  checksum = 0; 
  for (uint64_t i = 0; i < PSIZE / sizeof(half2); i++) {
    float2 t = __half22float2(hlfbuf[i]);
    checksum += t.x + t.y;
  }
  printf("checksum: %lf\n", checksum);
  free(hlfbuf);

  // FP32
  float *sglbuf = alloc<float>(PSIZE);
  start = getTime();
  run<float>(PSIZE, sglbuf);
  printf("runtime (float): %lf (s)\n", getTime() - start);
  checksum = 0; 
  for (uint64_t i = 0; i < PSIZE/sizeof(float); i++) {
    checksum += sglbuf[i];
  }
  printf("checksum: %lf\n", checksum);
  free(sglbuf);

  // FP64
  double *dblbuf = alloc<double>(PSIZE);
  start = getTime();
  run<double>(PSIZE, dblbuf);
  printf("runtime (double): %lf (s)\n", getTime() - start);
  checksum = 0; 
  for (uint64_t i = 0; i < PSIZE/sizeof(double); i++) {
    checksum += dblbuf[i];
  }
  printf("checksum: %lf\n", checksum);
  free(dblbuf);

  return 0;
}
