#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "kernels.cu"

const int HIGHEST = 3;
const int ITER = 100;
const int WORKLOAD = 1;
int sizepernode;

// global var
float preScore = -99999999999.f;
float score = 0.f;
float maxScore[HIGHEST] = {-999999999.f};
bool orders[NODE_N][NODE_N];
bool preOrders[NODE_N][NODE_N];
bool preGraph[NODE_N][NODE_N];
bool bestGraph[HIGHEST][NODE_N][NODE_N];
bool graph[NODE_N][NODE_N];
float *localscore, *scores;
float *LG;
int *parents;

void initial();  // initial orders and data
int genOrders(); // swap
int ConCore();   // discard new order or not
// get every possible set of parents for a node
void incr(int *bit, int n);  // binary code increases 1 each time
void incrS(int *bit, int n); // STATE_N code increases 1 each time
// get every possible combination of state for a parent set
bool getState( int parN, int *state, int time); 
float logGamma(int N); // log and gamma
float findBestGraph(float* D_localscore, int* D_resP, float* D_Score, bool *D_parent);
void genScore();
void sortGraph();
void swap(int a, int b);
void Pre_logGamma();
int findindex(int *arr, int size);
int C(int n, int a);

int main(int argc, char** argv) {

  if (argc != 3) {
    printf("Usage: ./%s <path to output file> <repeat>\n", argv[0]);
    return 1;
  }

  // save output in a file
  FILE *fpout = fopen(argv[1], "w");
  if (fpout == NULL) {
    printf("Error: failed to open %s. Exit..\n", argv[1]);
    return -1;
  }

  const int repeat = atoi(argv[2]);

  int i, j, c = 0, tmp, a, b;
  float tmpd;

  printf("NODE_N=%d\nInitialization...\n", NODE_N);

  srand(2);

  initial(); // update sizepernode
  scores = (float*) malloc ((sizepernode / (256 * WORKLOAD) + 1) * sizeof(float));
  parents = (int*) malloc ((sizepernode / (256 * WORKLOAD) + 1) * 4 * sizeof(int));

  Pre_logGamma();

  int *D_data;
  float *D_LG;
  float *D_localscore;
  float *D_Score;
  bool *D_parent;
  int *D_resP;
  hipMalloc((void **)&D_data, NODE_N * DATA_N * sizeof(int));
  hipMalloc((void **)&D_localscore, NODE_N * sizepernode * sizeof(float));
  hipMalloc((void **)&D_LG, (DATA_N + 2) * sizeof(float));
  hipMalloc((void **)&D_Score, (sizepernode / (256 * WORKLOAD) + 1) * sizeof(float));
  hipMalloc((void **)&D_parent, NODE_N * sizeof(bool)); 
  hipMalloc((void **)&D_resP, (sizepernode / (256 * WORKLOAD) + 1) * 4 * sizeof(int));

  dim3 grid(sizepernode / 256 + 1, 1, 1);
  dim3 threads(256, 1, 1);

  hipMemset(D_localscore, 0.f, NODE_N * sizepernode * sizeof(float));
  hipMemcpy(D_data, data, NODE_N * DATA_N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(D_LG, LG, (DATA_N + 2) * sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (i = 0; i < repeat; i++)
    genScoreKernel<<<grid, threads>>>(sizepernode, D_localscore, D_data, D_LG);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of genScoreKernel: %f (s)\n", time * 1e-9f / repeat);

  hipMemcpy(localscore, D_localscore, NODE_N * sizepernode * sizeof(float), hipMemcpyDeviceToHost);

  long findBestGraph_time = 0;
  i = 0;
  while (i != ITER) {

    i++;
    score = 0;

    for (a = 0; a < NODE_N; a++) {
      for (j = 0; j < NODE_N; j++) {
        orders[a][j] = preOrders[a][j];
      }
    }

    tmp = rand() % 6;
    for (j = 0; j < tmp; j++)
      genOrders();

    start = std::chrono::steady_clock::now();

    score = findBestGraph(D_localscore, D_resP, D_Score, D_parent);

    end = std::chrono::steady_clock::now();
    findBestGraph_time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    ConCore();

    // store the top HIGHEST highest orders
    if (c < HIGHEST) {
      tmp = 1;
      for (j = 0; j < c; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
        }
      }
      if (tmp != 0) {
        maxScore[c] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[c][a][b] = preGraph[a][b];
          }
        }
        c++;
      }

    } else if (c == HIGHEST) {
      sortGraph();
      c++;
    } else {

      tmp = 1;
      for (j = 0; j < HIGHEST; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
          break;
        }
      }
      if (tmp != 0 && preScore > maxScore[HIGHEST - 1]) {
        maxScore[HIGHEST - 1] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[HIGHEST - 1][a][b] = preGraph[a][b];
          }
        }
        b = HIGHEST - 1;
        for (a = HIGHEST - 2; a >= 0; a--) {
          if (maxScore[b] > maxScore[a]) {
            swap(a, b);
            tmpd = maxScore[a];
            maxScore[a] = maxScore[b];
            maxScore[b] = tmpd;
            b = a;
          }
        }
      }
    }

  } // endwhile

  printf("Find best graph time %lf (s)\n", findBestGraph_time * 1e-9);

  free(localscore);
  free(scores);
  free(parents);
  free(LG);
  hipFree(D_LG);
  hipFree(D_data);
  hipFree(D_localscore);
  hipFree(D_parent);
  hipFree(D_Score);
  hipFree(D_resP);

  for(j=0;j<HIGHEST;j++){
    fprintf(fpout,"score:%f\n",maxScore[j]);
    fprintf(fpout,"Best Graph:\n");
    for(int a=0;a<NODE_N;a++){
      for(int b=0;b<NODE_N;b++)
        fprintf(fpout,"%d ",bestGraph[j][a][b]);
      fprintf(fpout,"\n");
    }
    fprintf(fpout,"--------------------------------------------------------------------\n");
  }

  return 0;
}


float findBestGraph(float* D_localscore, int* D_resP, float* D_Score, bool *D_parent) {
  float bestls = -99999999.f;
  int bestparent[5];
  int bestpN, total;
  int node, index;
  int pre[NODE_N] = {0};
  int parent[NODE_N] = {0};
  int posN = 0, i, j, parN, tmp, k, l;
  float ls = -99999999999.f, score = 0;
  int blocknum;

  for (i = 0; i < NODE_N; i++)
    for (j = 0; j < NODE_N; j++)
      graph[i][j] = 0;

  for (node = 0; node < NODE_N; node++) {

    bestls = -99999999.f;
    posN = 0;

    for (i = 0; i < NODE_N; i++) {
      if (orders[node][i] == 1) {
        pre[posN++] = i;
      }
    }

    if (posN >= 0) {
      total = C(posN, 4) + C(posN, 3) + C(posN, 2) + posN + 1;
      blocknum = total / (256 * WORKLOAD) + 1;

      hipMemset(D_resP, 0, blocknum * 4 * sizeof(int));
      hipMemset(D_Score, -999999.f, blocknum * sizeof(float));
      hipMemcpy(D_parent, orders[node], NODE_N * sizeof(bool), hipMemcpyHostToDevice);

      computeKernel<<<blocknum, 256, 256 * sizeof(float)>>>(
          WORKLOAD, sizepernode, D_localscore, D_parent, node, total, D_Score,
          D_resP);
      hipMemcpy(parents, D_resP, blocknum * 4 * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(scores, D_Score, blocknum * sizeof(float), hipMemcpyDeviceToHost);

      for (i = 0; i < blocknum; i++) {

        if (scores[i] > bestls) {

          bestls = scores[i];

          parN = 0;
          for (tmp = 0; tmp < 4; tmp++) {
            if (parents[i * 4 + tmp] < 0)
              break;

            bestparent[tmp] = parents[i * 4 + tmp];

            parN++;
          }

          bestpN = parN;
        }
      }
    } else {
      if (posN >= 4) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {
              for (l = k + 1; l < posN; l++) {
                parN = 4;
                if (pre[i] > node)
                  parent[1] = pre[i];
                else
                  parent[1] = pre[i] + 1;
                if (pre[j] > node)
                  parent[2] = pre[j];
                else
                  parent[2] = pre[j] + 1;
                if (pre[k] > node)
                  parent[3] = pre[k];
                else
                  parent[3] = pre[k] + 1;
                if (pre[l] > node)
                  parent[4] = pre[l];
                else
                  parent[4] = pre[l] + 1;

                index = findindex(parent, parN);
                index += sizepernode * node;
                ls = localscore[index];

                if (ls > bestls) {
                  bestls = ls;
                  bestpN = parN;
                  for (tmp = 0; tmp < parN; tmp++)
                    bestparent[tmp] = parent[tmp + 1];
                }
              }
            }
          }
        }
      }

      if (posN >= 3) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {

              parN = 3;
              if (pre[i] > node)
                parent[1] = pre[i];
              else
                parent[1] = pre[i] + 1;
              if (pre[j] > node)
                parent[2] = pre[j];
              else
                parent[2] = pre[j] + 1;
              if (pre[k] > node)
                parent[3] = pre[k];
              else
                parent[3] = pre[k] + 1;

              index = findindex(parent, parN);
              index += sizepernode * node;
              ls = localscore[index];

              if (ls > bestls) {
                bestls = ls;
                bestpN = parN;
                for (tmp = 0; tmp < parN; tmp++)
                  bestparent[tmp] = parent[tmp + 1];
              }
            }
          }
        }
      }

      if (posN >= 2) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {

            parN = 2;
            if (pre[i] > node)
              parent[1] = pre[i];
            else
              parent[1] = pre[i] + 1;
            if (pre[j] > node)
              parent[2] = pre[j];
            else
              parent[2] = pre[j] + 1;

            index = findindex(parent, parN);
            index += sizepernode * node;
            ls = localscore[index];

            if (ls > bestls) {
              bestls = ls;
              bestpN = parN;
              for (tmp = 0; tmp < parN; tmp++)
                bestparent[tmp] = parent[tmp + 1];
            }
          }
        }
      }

      if (posN >= 1) {
        for (i = 0; i < posN; i++) {

          parN = 1;
          if (pre[i] > node)
            parent[1] = pre[i];
          else
            parent[1] = pre[i] + 1;

          index = findindex(parent, parN);
          index += sizepernode * node;
          ls = localscore[index];

          if (ls > bestls) {
            bestls = ls;
            bestpN = parN;
            for (tmp = 0; tmp < parN; tmp++)
              bestparent[tmp] = parent[tmp + 1];
          }
        }
      }

      parN = 0;
      index = sizepernode * node;

      ls = localscore[index];

      if (ls > bestls) {
        bestls = ls;
        bestpN = 0;
      }
    }
    if (bestls > -99999999.f) {

      for (i = 0; i < bestpN; i++) {
        if (bestparent[i] < node)
          graph[node][bestparent[i] - 1] = 1;
        else
          graph[node][bestparent[i]] = 1;
      }
      score += bestls;
    }
  }

  return score;
}


void sortGraph() {
  float max = -99999999999999.f;
  int maxi, i, j;
  float tmp;

  for (j = 0; j < HIGHEST - 1; j++) {
    max = maxScore[j];
    maxi = j;
    for (i = j + 1; i < HIGHEST; i++) {
      if (maxScore[i] > max) {
        max = maxScore[i];
        maxi = i;
      }
    }

    swap(j, maxi);
    tmp = maxScore[j];
    maxScore[j] = max;
    maxScore[maxi] = tmp;
  }
}

void swap(int a, int b) {
  int i, j;
  bool tmp;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {

      tmp = bestGraph[a][i][j];
      bestGraph[a][i][j] = bestGraph[b][i][j];
      bestGraph[b][i][j] = tmp;
    }
  }
}

void initial() {
  int i, j, tmp, a, b, r;
  bool tmpd;
  tmp = 1;
  for (i = 1; i <= 4; i++) {
    tmp += C(NODE_N - 1, i);
  }
  sizepernode = tmp;
  tmp *= NODE_N;

  localscore = (float*) malloc(tmp * sizeof(float));

  for (i = 0; i < tmp; i++)
    localscore[i] = 0;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++)
      orders[i][j] = 0;
  }
  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < i; j++)
      orders[i][j] = 1;
  }
  r = rand() % 10000;
  for (i = 0; i < r; i++) {
    a = rand() % NODE_N;
    b = rand() % NODE_N;
    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[j][a];
      orders[j][a] = orders[j][b];
      orders[j][b] = tmpd;
    }

    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[a][j];
      orders[a][j] = orders[b][j];
      orders[b][j] = tmpd;
    }
  }

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {
      preOrders[i][j] = orders[i][j];
    }
  }
}

// generate ramdom order
int genOrders() {

  int a, b, j;
  bool tmp;
  a = rand() % NODE_N;
  b = rand() % NODE_N;

  for (j = 0; j < NODE_N; j++) {
    tmp = orders[a][j];
    orders[a][j] = orders[b][j];
    orders[b][j] = tmp;
  }
  for (j = 0; j < NODE_N; j++) {
    tmp = orders[j][a];
    orders[j][a] = orders[j][b];
    orders[j][b] = tmp;
  }

  return 1;
}

// decide leave or discard an order
int ConCore() {
  int i, j;
  float tmp;
  tmp = log((rand() % 100000) / 100000.0);
  if (tmp < (score - preScore)) {

    for (i = 0; i < NODE_N; i++) {
      for (j = 0; j < NODE_N; j++) {
        preOrders[i][j] = orders[i][j];
        preGraph[i][j] = graph[i][j];
      }
    }
    preScore = score;

    return 1;
  }

  return 0;
}

void genScore() {
}

void Pre_logGamma() {

  LG = (float*) malloc ((DATA_N + 2) * sizeof(float));

  LG[1] = log(1.0);
  float i;
  for (i = 2; i <= DATA_N + 1; i++) {
    LG[(int)i] = LG[(int)i - 1] + log((float)i);
  }
}

void incr(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= 2) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

void incrS(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= STATE_N) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

bool getState(int parN, int *state, int time) {
  int j = 1;

  j = pow(STATE_N, (float)parN) - 1;

  if (time > j)
    return false;

  if (time >= 1)
    incrS(state, 0);

  return true;
}

int findindex(int *arr, int size) { // reminder: arr[0] has to be 0 && size ==
  // array size-1 && index start from 0
  int i, j, index = 0;

  for (i = 1; i < size; i++) {
    index += C(NODE_N - 1, i);
  }

  for (i = 1; i <= size - 1; i++) {
    for (j = arr[i - 1] + 1; j <= arr[i] - 1; j++) {
      index += C(NODE_N - 1 - j, size - i);
    }
  }

  index += arr[size] - arr[size - 1];

  return index;
}

int C(int n, int a) {
  int i, res = 1, atmp = a;

  for (i = 0; i < atmp; i++) {
    res *= n;
    n--;
  }

  for (i = 0; i < atmp; i++) {
    res /= a;
    a--;
  }

  return res;
}
