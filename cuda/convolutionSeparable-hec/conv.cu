#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include "conv.h"

#define ROWS_BLOCKDIM_X       16
#define COLUMNS_BLOCKDIM_X    16
#define ROWS_BLOCKDIM_Y       4
#define COLUMNS_BLOCKDIM_Y    8
#define ROWS_RESULT_STEPS     8
#define COLUMNS_RESULT_STEPS  8
#define ROWS_HALO_STEPS       1
#define COLUMNS_HALO_STEPS    1

__global__ void conv_rows(
    float *__restrict__ dst,
    const float *__restrict__ src,
    const float *__restrict__ kernel,
    const int imageW,
    const int imageH,
    const int pitch)
{
  __shared__ float l_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

  int gidX = blockIdx.x;
  int gidY = blockIdx.y;
  int lidX = threadIdx.x;
  int lidY = threadIdx.y;
  //Offset to the left halo edge
  const int baseX = (gidX * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + lidX;
  const int baseY = gidY * ROWS_BLOCKDIM_Y + lidY;

  src += baseY * pitch + baseX;
  dst += baseY * pitch + baseX;

  //Load main data
  #pragma unroll
  for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    l_Data[lidY][lidX + i * ROWS_BLOCKDIM_X] = src[i * ROWS_BLOCKDIM_X];

  //Load left halo
  #pragma unroll
  for(int i = 0; i < ROWS_HALO_STEPS; i++)
    l_Data[lidY][lidX + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X >= 0) ? src[i * ROWS_BLOCKDIM_X] : 0;

  //Load right halo
  #pragma unroll
  for(int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    l_Data[lidY][lidX + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X < imageW) ? src[i * ROWS_BLOCKDIM_X] : 0;

  //Compute and store results
  __syncthreads();

  #pragma unroll
  for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++) {
    float sum = 0;

    #pragma unroll
    for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
      sum += kernel[KERNEL_RADIUS - j] * l_Data[lidY][lidX + i * ROWS_BLOCKDIM_X + j];

    dst[i * ROWS_BLOCKDIM_X] = sum;
  }
}

__global__ void conv_cols(
    float *__restrict__ dst,
    const float *__restrict__ src,
    const float *__restrict__ kernel,
    const int imageW,
    const int imageH,
    const int pitch)
{
  __shared__ float l_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

  int gidX = blockIdx.x;
  int gidY = blockIdx.y;
  int lidX = threadIdx.x;
  int lidY = threadIdx.y;

  //Offset to the upper halo edge
  const int baseX = gidX * COLUMNS_BLOCKDIM_X + lidX;
  const int baseY = (gidY * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + lidY;
  src += baseY * pitch + baseX;
  dst += baseY * pitch + baseX;

  //Load main data
  #pragma unroll
  for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    l_Data[lidX][lidY + i * COLUMNS_BLOCKDIM_Y] = src[i * COLUMNS_BLOCKDIM_Y * pitch];

  //Load upper halo
  #pragma unroll
  for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
    l_Data[lidX][lidY + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y >= 0) ? src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

  //Load lower halo
  #pragma unroll
  for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    l_Data[lidX][lidY + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < imageH) ? src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

  //Compute and store results
  __syncthreads();

  #pragma unroll
  for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++) {
    float sum = 0;

    #pragma unroll
    for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
      sum += kernel[KERNEL_RADIUS - j] * l_Data[lidX][lidY + i * COLUMNS_BLOCKDIM_Y + j];

    dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
  }
}

void convolutionRows(
    float* dst,
    const float* src,
    const float* kernel,
    const int imageW,
    const int imageH,
    const int pitch)
{
  assert ( ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS );
  assert ( imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0 );
  assert ( imageH % ROWS_BLOCKDIM_Y == 0 );

  dim3 block (ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
  dim3 grid (imageW / ROWS_RESULT_STEPS / ROWS_BLOCKDIM_X, imageH/ROWS_BLOCKDIM_Y );

  conv_rows<<<grid, block>>>(
      dst,
      src,
      kernel,
      imageW,
      imageH,
      imageW );

}

void convolutionColumns(
    float* dst,
    const float* src,
    const float* kernel,
    const int imageW,
    const int imageH,
    const int pitch)
{
  assert ( COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS );
  assert ( imageW % COLUMNS_BLOCKDIM_X == 0 );
  assert ( imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0 );

  dim3 block (COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
  dim3 grid (imageW / COLUMNS_BLOCKDIM_X, imageH / COLUMNS_RESULT_STEPS / COLUMNS_BLOCKDIM_Y);

  conv_cols<<<grid, block>>>(
      dst,
      src,
      kernel,
      imageW,
      imageH,
      imageW );
}
