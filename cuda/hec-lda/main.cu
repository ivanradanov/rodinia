#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>
#include "kernel.h"

int main(int argc, char* argv[]) {

  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }

  // repeat kernel execution
  const int repeat = atoi(argv[1]);

  int i;
  srand(123);

  const int num_topics = 1000;
  const int num_words  = 10266;
  const int block_cnt  = 500;
  const int num_indptr = block_cnt; // max: num_words
  const int block_dim  = 256;
  const int num_iters  = 64;
 
  std::vector<float> alpha(num_topics);
  for (i = 0; i < num_topics; i++)  alpha[i] = (float) rand() / (float) RAND_MAX;

  std::vector<float> beta(num_topics * num_words);
  for (i = 0; i < num_topics * num_words; i++)  beta[i] = (float) rand() / (float) RAND_MAX;

  std::vector<float> grad_alpha(num_topics * block_cnt, 0.0f);
  std::vector<float> new_beta(num_topics * num_words, 0.0f);
  std::vector<int> h_locks(num_words, 0);
  std::vector<float> gamma (num_indptr * num_topics);

  std::vector<int> indptr (num_indptr+1, 0);
  indptr[num_indptr] = num_words-1;
  for (i = num_indptr; i >= 1; i--) {
    int t = indptr[i] - 1 - (rand() % (num_words/num_indptr));
    if (t < 0) break;
    indptr[i-1] = t;
  }
  const int num_cols = num_words;

  std::vector<int> cols (num_cols);
  std::vector<float> counts (num_cols);

  for (i = 0; i < num_cols; i++) {
    cols[i] = i;
    counts[i] = 0.5f; // arbitrary
  }

  float *d_alpha, *d_beta, *d_grad_alpha, *d_new_beta;
  float *d_counts, *d_gamma, *d_vali_losses, *d_train_losses;
  int *d_locks, *d_cols, *d_indptr;
  bool *d_vali;

  hipMalloc((void**)&d_alpha, sizeof(float) * num_topics);
  hipMemcpy(d_alpha, alpha.data(), sizeof(float) * num_topics, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_beta, sizeof(float) * num_topics * num_words);
  hipMemcpy(d_beta, beta.data(), sizeof(float) * num_topics * num_words, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_grad_alpha, sizeof(float) * num_topics * block_cnt);
  hipMemcpy(d_grad_alpha, grad_alpha.data(), sizeof(float) * block_cnt * num_topics, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_new_beta, sizeof(float) * num_topics * num_words);
  hipMemcpy(d_new_beta, new_beta.data(), sizeof(float) * num_topics * num_words, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_locks, sizeof(int) * num_words);
  hipMemcpy(d_locks, h_locks.data(), sizeof(int) * num_words, hipMemcpyHostToDevice);
  
  hipMalloc((void**)&d_cols, sizeof(int) * num_cols);
  hipMemcpy(d_cols, cols.data(), sizeof(int) * num_cols, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_indptr, sizeof(int) * (num_indptr + 1));
  hipMemcpy(d_indptr, indptr.data(), sizeof(int) * (num_indptr + 1), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_vali, sizeof(bool) * num_cols);

  hipMalloc((void**)&d_counts, sizeof(float) * num_cols);
  hipMemcpy(d_counts, counts.data(), sizeof(float) * num_cols, hipMemcpyHostToDevice);

  // gamma will be initialized in the kernel
  hipMalloc((void**)&d_gamma, sizeof(float) * num_indptr * num_topics);

  // reset losses
  hipMalloc((void**)&d_train_losses, sizeof(float) * block_cnt);
  hipMemset(d_train_losses, 0, sizeof(float) * block_cnt);

  hipMalloc((void**)&d_vali_losses, sizeof(float) * block_cnt);
  hipMemset(d_vali_losses, 0, sizeof(float) * block_cnt);

  // store device results
  std::vector<float> train_losses(block_cnt), vali_losses(block_cnt);

  // training
  hipMemset(d_vali, 0, sizeof(bool) * num_cols); 
  bool init_gamma = false;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (i = 0; i < repeat; i++) {
    init_gamma = (i == 0) ? true : false;
    EstepKernel<<<block_cnt, block_dim, 4 * num_topics * sizeof(float)>>>(
      d_cols,
      d_indptr,
      d_vali,
      d_counts,
      init_gamma, num_cols, num_indptr, num_topics, num_iters,
      d_alpha,
      d_beta,
      d_gamma,
      d_grad_alpha,
      d_new_beta,
      d_train_losses,
      d_vali_losses,
      d_locks);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (training): %f (s)\n", (time * 1e-9f) / repeat);

  // validation
  hipMemset(d_vali, 0xFFFFFFFF, sizeof(bool) * num_cols); 

  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (i = 0; i < repeat; i++) {
    EstepKernel<<<block_cnt, block_dim, 4 * num_topics * sizeof(float)>>>(
      d_cols,
      d_indptr,
      d_vali,
      d_counts,
      init_gamma, num_cols, num_indptr, num_topics, num_iters,
      d_alpha,
      d_beta,
      d_gamma,
      d_grad_alpha,
      d_new_beta,
      d_train_losses,
      d_vali_losses,
      d_locks);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (validation): %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(vali_losses.data(), d_vali_losses, sizeof(float) * block_cnt, hipMemcpyDeviceToHost);
  hipMemcpy(train_losses.data(), d_train_losses, sizeof(float) * block_cnt, hipMemcpyDeviceToHost);

  float total_train_loss = std::accumulate(train_losses.begin(), train_losses.end(), 0.0f);
  float total_vali_loss = std::accumulate(vali_losses.begin(), vali_losses.end(), 0.0f);
  printf("Total train and validate loss: %f %f\n", total_train_loss, total_vali_loss);

  hipFree(d_cols);
  hipFree(d_indptr);
  hipFree(d_vali);
  hipFree(d_counts);
  hipFree(d_alpha);
  hipFree(d_beta);
  hipFree(d_gamma);
  hipFree(d_grad_alpha);
  hipFree(d_new_beta);
  hipFree(d_train_losses);
  hipFree(d_vali_losses);
  hipFree(d_locks);

  return 0;
}
