#include "hip/hip_runtime.h"
//=====================================================================
//  MAIN FUNCTION
//=====================================================================

double master(fp timeinst,
    fp* initvalu,
    fp* parameter,
    fp* finavalu,
    fp* com,

    fp* d_initvalu,
    fp* d_finavalu,
    fp* d_params,
    fp* d_com)
{

  //=====================================================================
  //  VARIABLES
  //=====================================================================

  // counters
  int i;

  // offset pointers
  int initvalu_offset_ecc;
  int initvalu_offset_Dyad;
  int initvalu_offset_SL;
  int initvalu_offset_Cyt;

  dim3 threads;
  dim3 blocks;

  //=====================================================================
  //  execute ECC&CAM kernel - it runs ECC and CAMs in parallel
  //=====================================================================

  int d_initvalu_mem;
  d_initvalu_mem = EQUATIONS * sizeof(fp);
  int d_finavalu_mem;
  d_finavalu_mem = EQUATIONS * sizeof(fp);
  int d_params_mem;
  d_params_mem = PARAMETERS * sizeof(fp);
  int d_com_mem;
  d_com_mem = 3 * sizeof(fp);

#ifdef DEBUG
  for (int i = 0; i < EQUATIONS; i++)
    printf("initvalu %d %f\n", i, initvalu[i]);
  for (int i = 0; i < PARAMETERS; i++)
    printf("params %d %f\n", i, parameter[i]);
  printf("\n");
#endif

  hipMemcpy(d_initvalu, initvalu, d_initvalu_mem, hipMemcpyHostToDevice);
  hipMemcpy(d_params, parameter, d_params_mem, hipMemcpyHostToDevice);

  threads.x = NUMBER_THREADS;
  threads.y = 1;
  blocks.x = 2;
  blocks.y = 1;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  kernel<<<blocks, threads>>>(
      timeinst,
      d_initvalu,
      d_finavalu,
      d_params,
      d_com);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  hipMemcpy(finavalu, d_finavalu, d_finavalu_mem, hipMemcpyDeviceToHost);
  hipMemcpy(com, d_com, d_com_mem, hipMemcpyDeviceToHost);

#ifdef DEBUG
  for (int i = 0; i < EQUATIONS; i++)
    printf("finavalu %d %f\n", i, finavalu[i]);
  for (int i = 0; i < 3; i++)
    printf("%f ", com[i]);
  printf("\n");

#endif

  //=====================================================================
  //  FINAL KERNEL
  //=====================================================================

  initvalu_offset_ecc = 0;
  initvalu_offset_Dyad = 46;
  initvalu_offset_SL = 61;
  initvalu_offset_Cyt = 76;

  kernel_fin(
      initvalu,
      initvalu_offset_ecc,
      initvalu_offset_Dyad,
      initvalu_offset_SL,
      initvalu_offset_Cyt,
      parameter,
      finavalu,
      com[0],
      com[1],
      com[2]);

  //=====================================================================
  //  COMPENSATION FOR NANs and INFs
  //=====================================================================

  for(i=0; i<EQUATIONS; i++){
    if (isnan(finavalu[i])){ 
      finavalu[i] = 0.0001;                        // for NAN set rate of change to 0.0001
    }
    else if (isinf(finavalu[i])){ 
      finavalu[i] = 0.0001;                        // for INF set rate of change to 0.0001
    }
  }

  return time;
}
