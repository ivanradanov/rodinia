#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "reference.h"

__global__ void car (
    const float *__restrict__ img,
    const float *__restrict__ kernels,
    const float *__restrict__ offsets_h,
    const float *__restrict__ offsets_v,
          float *__restrict__ output,
    const params p,
    const int offset_unit,
    const int padding,
    const size_t n)
{
  size_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(global_idx >= n) return;

  const int dim_b = p.output_dim_b;
  const int dim_c = p.output_dim_c;
  const int dim_h = p.output_dim_h;
  const int dim_w = p.output_dim_w;
  const int kernels_size = p.kernel_size;
  const int img_w = p.image_w;
  const int img_h = p.image_h;

  const size_t vol_size = (size_t)dim_c * dim_h * dim_w;
  const size_t img_size = (size_t)dim_h * dim_w;

  const int idb = (global_idx / vol_size) % dim_b;
  const int idc = (global_idx / img_size) % dim_c;
  const int idy = (global_idx / dim_w) % dim_h;
  const int idx = global_idx % dim_w;

  const int k_size = (int)sqrtf(float(kernels_size));
  const int w = img_w - 2 * padding;
  const int h = img_h - 2 * padding;

  float result = 0;
  for(int k_y = 0; k_y < k_size; ++k_y)
  {
    for(int k_x = 0; k_x < k_size; ++k_x)
    {
      const float offset_h = offsets_h(idb,k_size * k_y + k_x,idy,idx) * offset_unit;
      const float offset_v = offsets_v(idb,k_size * k_y + k_x,idy,idx) * offset_unit;

      const float p_x = static_cast<float>(idx + 0.5f) / dim_w * w + k_x + offset_h - 0.5f;
      const float p_y = static_cast<float>(idy + 0.5f) / dim_h * h + k_y + offset_v - 0.5f;
      const float alpha = p_x - floorf(p_x);
      const float beta = p_y - floorf(p_y);

      const int xL = max(min(int(floorf(p_x)), w + 2 * padding - 1), 0);
      const int xR = max(min(xL + 1, w + 2 * padding - 1), 0);
      const int yT = max(min(int(floorf(p_y)), h + 2 * padding - 1), 0);
      const int yB = max(min(yT + 1, h + 2 * padding - 1), 0);

      float val = (1.f - alpha) * (1.f - beta) * img(idb,idc,yT,xL);
      val += alpha * (1.f - beta) * img(idb,idc,yT,xR);
      val += (1.f - alpha) * beta * img(idb,idc,yB,xL);
      val += alpha * beta * img(idb,idc,yB,xR);
      result += val * kernels(idb,k_size * k_y + k_x,idy,idx);
    }
  }
  output(idb,idc,idy,idx) = result;
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  params p = {128, 3, 480, 640, 9, 1024, 1024};
  const int dim_b = p.output_dim_b;
  const int dim_c = p.output_dim_c;
  const int dim_h = p.output_dim_h;
  const int dim_w = p.output_dim_w;
  const int kernels_size = p.kernel_size;
  const int img_w = p.image_w;
  const int img_h = p.image_h;

  const int padding = 1;

  size_t image_size = (size_t)dim_b * dim_c * (img_w + padding) * (img_h + padding);
  size_t offset_size = (size_t)dim_b * kernels_size * dim_w * dim_h;
  size_t kernel_size = (size_t)dim_b * kernels_size * dim_w * dim_h;
  size_t output_size = (size_t)dim_b * dim_c * dim_w * dim_h;

  size_t image_size_byte = sizeof(float) * image_size;
  size_t offset_size_byte = sizeof(float) * offset_size;
  size_t kernel_size_byte = sizeof(float) * kernel_size;
  size_t output_size_byte = sizeof(float) * output_size;

  float *img = (float*) malloc (image_size_byte);
  float *offsets_h = (float*) malloc (offset_size_byte);
  float *offsets_v = (float*) malloc (offset_size_byte);
  float *kernel = (float*) malloc (kernel_size_byte);
  float *output = (float*) malloc (output_size_byte);
  float *output_ref = (float*) malloc (output_size_byte);

  unsigned long long seed = 123;
  for (size_t i = 0; i < image_size; i++) img[i] = (unsigned char)(256*LCG_random_double(&seed));
  for (size_t i = 0; i < kernel_size; i++) kernel[i] = (unsigned char)(256*LCG_random_double(&seed));
  for (size_t i = 0; i < offset_size; i++) {
    offsets_h[i] = LCG_random_double(&seed);
    offsets_v[i] = LCG_random_double(&seed);
  }

  float *d_img, *d_offsets_h, *d_offsets_v, *d_kernel, *d_output;
  hipMalloc((void**)&d_img, image_size_byte);
  hipMemcpy(d_img, img, image_size_byte, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_offsets_h, offset_size_byte);
  hipMemcpy(d_offsets_h, offsets_h, offset_size_byte, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_offsets_v, offset_size_byte);
  hipMemcpy(d_offsets_v, offsets_v, offset_size_byte, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_kernel, kernel_size_byte);
  hipMemcpy(d_kernel, kernel, kernel_size_byte, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_output, output_size_byte);

  dim3 grid ((output_size + 255) / 256);
  dim3 block (256);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    car <<<grid, block>>> (
        d_img,
        d_kernel,
        d_offsets_h,
        d_offsets_v,
        d_output,
        p,
        1, // offset_unit,
        padding,
        output_size);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (s)\n", time * 1e-9f / repeat);

  reference (img, kernel, offsets_h, offsets_v, output_ref, p, 1, padding);

  hipMemcpy(output, d_output, output_size_byte, hipMemcpyDeviceToHost);

  float rmse = 0;
  for (size_t i = 0; i < output_size; i++)
    rmse += (output_ref[i] - output[i]) * (output_ref[i] - output[i]);
  printf("RMSE: %f\n", sqrtf(rmse/output_size));

  hipFree(d_img);
  hipFree(d_offsets_h);
  hipFree(d_offsets_v);
  hipFree(d_kernel);
  hipFree(d_output);

  free(img);
  free(offsets_h);
  free(offsets_v);
  free(kernel);
  free(output);
  free(output_ref);
  return 0;
}
