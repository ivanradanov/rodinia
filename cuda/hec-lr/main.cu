#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "linear.h"

double gettime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + t.tv_usec * 1e-6;
}

clock_t start;
clock_t end;

extern int cpu_offset;

/* Read file */
static void create_dataset(linear_param_t * params, data_t * dataset) {
  FILE *ptr_file = fopen(params->filename, "r");
  if (ptr_file == NULL) {
    perror("Failed to load dataset file");
    exit(1);
  }

  char *token;
  char buf[1024];

  for (size_t i = 0; i < params->size && fgets(buf, 1024, ptr_file) != NULL; i++) {
    token = strtok(buf, "\t");
    dataset[i].x = atof(token);
    token = strtok(NULL, "\t");
    dataset[i].y = atof(token);
  }

  fclose(ptr_file);
}

static void temperature_regression(results_t * results, int repeat) {
  linear_param_t params;
  params.repeat = repeat;
  params.filename = TEMP_FILENAME;
  params.size = TEMP_SIZE;
  params.wg_size = TEMP_WORKGROUP_SIZE;
  params.wg_count = TEMP_WORKGROUP_NBR;

  data_t *dataset = (data_t*) malloc (sizeof(data_t) * params.size);
  create_dataset(&params, dataset);

  results->parallelized.ktime = 0;

  parallelized_regression(&params, dataset, &results->parallelized);
  iterative_regression(&params, dataset, &results->iterative);

  free(dataset);
}

static void print_results(results_t * results) {
  PRINT_RESULT("Parallelized", results->parallelized);
  PRINT_RESULT("Iterative", results->iterative);
}

static void write_results(results_t * results, const char * restricts) {
  FILE* file = fopen(RESULT_FILENAME, restricts);
  WRITE_RESULT(file, results->parallelized);
  WRITE_RESULT(file, results->iterative);
  fclose(file);
}

int main(int argc, char* argv[]) {
  results_t results = {{0}};
  if (argc != 3) {
    printf("Usage: linear <repeat> <cpu offset>\n");
    printf("Device execution only when cpu offset is 0\n");
    printf("Host execution only when cpu offset is 100\n");
    exit(0);
  }

  int repeat = atoi(argv[1]);
  cpu_offset = atoi(argv[2]);
  printf("CPU offset: %d\n", cpu_offset);

  double starttime = gettime();

  temperature_regression(&results, repeat);

  double endtime = gettime();
  printf("Total execution time: %lf ms\n", 1000.0 * (endtime - starttime));

  printf("Average kernel execution time: %lf us\n",
         results.parallelized.ktime * 1e-3 / repeat);

  write_results(&results, "a");

  if (argc == 1 || strcmp(argv[1], "-no_print") > 0) {
    printf("\n> TEMPERATURE REGRESSION (%d)\n\n", TEMP_SIZE);
    print_results(&results);
  }

  return 0;
}
