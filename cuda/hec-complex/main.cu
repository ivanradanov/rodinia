#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "complex.h"
#include "kernels.h"

bool check (const char *cs, int n)
{
  bool ok = true;
  for (int i = 0; i < n; i++) {
    if (cs[i] != 5) {
      ok = false; 
      break;
    }
  }
  return ok;
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <size> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

  char* cs = (char*) malloc (n);

  char* d_cs;
  hipMalloc((void**)&d_cs, n);

  dim3 grids ((n + 255)/256); 
  dim3 blocks (256);

  // warmup
  complex_float<<<grids, blocks>>>(d_cs, n);
  complex_double<<<grids, blocks>>>(d_cs, n);
  hipDeviceSynchronize();

  auto start = std::chrono::steady_clock::now();

  // complex numbers in single precision
  for (int i = 0; i < repeat; i++) {
    complex_float<<<grids, blocks>>>(d_cs, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (float) %f (s)\n", time * 1e-9f / repeat);

  hipMemcpy(cs, d_cs, n, hipMemcpyDeviceToHost);
  bool complex_float_check = check(cs, n);

  start = std::chrono::steady_clock::now();

  // complex numbers in double precision
  for (int i = 0; i < repeat; i++) {
    complex_double<<<grids, blocks>>>(d_cs, n);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (double) %f (s)\n", time * 1e-9f / repeat);

  hipMemcpy(cs, d_cs, n, hipMemcpyDeviceToHost);
  bool complex_double_check = check(cs, n);

  printf("%s\n", (complex_float_check && complex_double_check)
                 ? "PASS" : "FAIL");

  hipFree(d_cs);
  free(cs);

  return 0;
}
