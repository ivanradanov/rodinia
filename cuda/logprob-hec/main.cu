#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <chrono>
#include <random>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "reduce.h"

template<typename T>
__global__ void log_probs_kernel(
    float*       log_probs,
    const T*     logits,
    const int*   ids,
    const int*   lengths,
    const int    max_input_length,
    const int    batch_size,
    const int    vocab_size,
    const int    vocab_size_padded,
    bool         batch_first)
{
  // Calculate the log probability from logits.
  //   log_probs[t, :] = log(softmax(logits))[ids[t + 1, :]]
  //
  // log_probs: [max_length - 1, batch_size] or [batch_size, max_length -1],
  //     log probabilities of each token.
  // logits: [max_length, batch_size, vocab_size_padded] or [batch_size, max_length, vocab_size_padded]
  // lengths: [batch_size], sequence lengths
  // ids: [max_length, batch_size], token ids.
  // batch_size: [1], batch_size. in case of beam > 1, batch x beam.
  // vocab_size: [1], vocab_size,
  // vocab_size: [1], vocab_size_padded, padded vocab size.

  const bool IS_FP16   = std::is_same<T, half>::value;
  const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

  int tidx = threadIdx.x;                            // vocab dim
  int bidx = batch_first ? blockIdx.x : blockIdx.y;  // batch dim
  int step = batch_first ? blockIdx.y : blockIdx.x;  // step dim

  __shared__ float s_max_logit;

  if (bidx < batch_size && step < lengths[bidx] - 1) {
    // Compute the address of logits to data for the current batch
    int step_offset  = batch_first ? step * vocab_size_padded : step * batch_size * vocab_size_padded;
    int batch_offset = batch_first ? bidx * max_input_length * vocab_size_padded : bidx * vocab_size_padded;
    logits += step_offset + batch_offset;

    // Find max(logits)
    float local_max = -MAX_T_VAL;
    float val       = -MAX_T_VAL;
    for (int i = tidx; i < vocab_size; i += blockDim.x) {
      val       = static_cast<float>(logits[i]);
      local_max = fmaxf(local_max, val);
    }

    float max_val = blockReduceMax<float>(local_max);
    if (tidx == 0) {
      s_max_logit = max_val;
    }
    __syncthreads();

    // Calculate the denominator: sum_i exp(logits[i])
    float local_sum_exp = 0.0f;
    for (int i = tidx; i < vocab_size; i += blockDim.x) {
      val = __expf(static_cast<float>(logits[i]) - s_max_logit);
      local_sum_exp += val;
    }

    float sum_exp = blockReduceSum<float>(local_sum_exp);
    if (tidx == 0) {
      int idx = batch_first ? step + bidx * (max_input_length - 1) : step * batch_size + bidx;
      // log_probs[step, ...] is the log probability of a token at step t + 1.
      int token_idx = batch_first ? step + 1 + bidx * max_input_length : (step + 1) * batch_size + bidx;
      log_probs[idx] = static_cast<float>(logits[ids[token_idx]]) - s_max_logit - __logf(sum_exp + 1e-9f);
    }
  }
}

__global__ void accumulate_log_probs(
          float* cum_log_probs,
    const float* log_probs,
    const int*   lengths,
    const int    max_input_length,
    const int    batch_size,
    const bool   batch_first)
{
  // Accumulate the log probability along the sequence dimension.
  //   cum_log_probs[j] = sum_i log(softmax(logits))[ids[i,j]]
  //
  // cum_log_probs: [batch_size], cumulative log probability
  // log_probs: [max_length - 1, batch_size] or [batch_size, max_length - 1],
  //   log probability of each token
  // lengths: [batch_size], sequence lengths
  // batch_size: [1], batch_size. in case of beam > 1, batch x beam.

  int bidx = blockIdx.x;   // batch dim
  int tidx = threadIdx.x;  // step dim

  int length = lengths[bidx];
  // reposition logits to data for the current batch.
  log_probs += batch_first ? bidx * (max_input_length - 1) : bidx;
  int stride = batch_first ? 1 : batch_size;  // stride along with seq dim.
  float local_accum = 0.0f;
  for (int step = tidx; step < length - 1; step += blockDim.x) {
    local_accum += static_cast<float>(log_probs[step * stride]);
  }
  float accum = blockReduceSum<float>(local_accum);
  if (tidx == 0) {
    cum_log_probs[bidx] = accum;
  }
}


int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: %s <maximum sequence length> <batch size> <vocabulary size> <repeat>\n", argv[0]);
    return 1;
  }
  const int max_length = atoi(argv[1]);  // max input length
  const int batch_size = atoi(argv[2]);
  const int vocab_size = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  const int vocab_size_padded = (vocab_size + 31) / 32 * 32;

  size_t logits_size = (size_t)batch_size * max_length * vocab_size_padded;
  size_t logits_size_bytes = logits_size * sizeof(float);

  size_t log_probs_size = (size_t)batch_size * (max_length - 1);
  size_t log_probs_size_bytes = log_probs_size * sizeof(float);

  size_t batch_size_bytes = batch_size * sizeof(float);
  size_t length_size_bytes = batch_size * sizeof(int);

  float *h_logits = (float*) malloc (logits_size_bytes);

  std::default_random_engine g (123);
  std::uniform_real_distribution<float> distr (-6.f, 6.f);

  for (size_t i = 0; i < logits_size; i++)
    h_logits[i] = distr(g);

  float *d_logits;
  hipMalloc((void**)&d_logits, logits_size_bytes);
  hipMemcpy(d_logits, h_logits, logits_size_bytes, hipMemcpyHostToDevice);

  float *h_log_probs = (float*) malloc (log_probs_size_bytes);
  float *d_log_probs;
  hipMalloc((void**)&d_log_probs, log_probs_size_bytes);

  float *h_cum_log_probs = (float*) malloc (batch_size_bytes);
  float *d_cum_log_probs;
  hipMalloc((void**)&d_cum_log_probs, batch_size_bytes);

  int *h_lengths = (int*) malloc (length_size_bytes);

  srand(123);
  for (int i = 0; i < batch_size; i++)
    h_lengths[i] = rand() % max_length + max_length / 2;

  int *d_lengths;
  hipMalloc((void**)&d_lengths, length_size_bytes);
  hipMemcpy(d_lengths, h_lengths, length_size_bytes, hipMemcpyHostToDevice);

  size_t ids_size = batch_size * max_length;
  size_t ids_size_bytes = ids_size * sizeof(int);

  int *h_ids = (int*) malloc (ids_size_bytes);
  for (size_t i = 0; i < ids_size; i++)
    h_ids[i] = rand() % vocab_size;

  int *d_ids;
  hipMalloc((void**)&d_ids, ids_size_bytes);
  hipMemcpy(d_ids, h_ids, ids_size_bytes, hipMemcpyHostToDevice);

  // A batched version of log prob computation.
  //
  // cum_log_probs: [batch_size]
  // logits: [max_input_length, batch_size, vocab_size] or [batch_size, max_input_length, vocab_size]
  // input_ids: [max_input_length, batch_size] or [max_input_length, batch_size]
  // input_lengths: [batch_size]

  // TODO: batch_first = 0 does not produce the same results for each run
  bool batch_first = 1;

  const int block_size = vocab_size < 1024 ? (vocab_size + 31) / 32 * 32 : 1024;
  const int gx = batch_first ? batch_size : max_length - 1;
  const int gy = batch_first ? max_length - 1 : batch_size;

  dim3 grid(gx, gy);
  dim3 block(block_size, 1);

  hipDeviceSynchronize();

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    log_probs_kernel<float><<<grid, block>>>(
      d_log_probs,
      d_logits,
      d_ids,
      d_lengths,
      max_length,
      batch_size,
      vocab_size,
      vocab_size_padded,
      batch_first);

    accumulate_log_probs<<<batch_size, block>>>(
      d_cum_log_probs,
      d_log_probs,
      d_lengths,
      max_length,
      batch_size,
      batch_first);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of kernels: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(h_log_probs, d_log_probs, log_probs_size_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_cum_log_probs, d_cum_log_probs, batch_size_bytes, hipMemcpyDeviceToHost);

  float checksum = 0, checkmax = FLT_MIN, checkmin = FLT_MAX;
  for (int i = 0; i < batch_size; i++) {
    checksum += h_cum_log_probs[i];
    checkmax = fmax(checkmax, h_cum_log_probs[i]);
    checkmin = fmin(checkmin, h_cum_log_probs[i]);
  }
  printf("Checksum = %f\n", checksum / batch_size);
  printf("Max cumulative log probs = %f\n", checkmax);
  printf("Min cumulative log probs = %f\n", checkmin);

  hipFree(d_cum_log_probs);
  hipFree(d_log_probs);
  hipFree(d_logits);
  hipFree(d_ids);
  hipFree(d_lengths);

  free(h_cum_log_probs);
  free(h_log_probs);
  free(h_logits);
  free(h_ids);
  free(h_lengths);

  return 0;
}
