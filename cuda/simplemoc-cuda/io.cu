#include "hip/hip_runtime.h"
#include "SimpleMOC-kernel_header.h"

// Prints program logo
void logo(int version)
{
	border_print();
	printf(
"   __           __        ___        __   __           ___  __        ___     \n"
"  /__` |  |\\/| |__) |    |__   |\\/| /  \\ /  ` __ |__/ |__  |__) |\\ | |__  |   \n"
"  .__/ |  |  | |    |___ |___  |  | \\__/ \\__,    |  \\ |___ |  \\ | \\| |___ |___\n" 
	);
	printf("\n");
	border_print();
	printf("\n");

	center_print("Developed at", 79);
	center_print("The Massachusetts Institute of Technology", 79);
	center_print("and", 79);
	center_print("Argonne National Laboratory", 79);
	printf("\n");
	char v[100];
	sprintf(v, "Version: %d", version);
	center_print(v, 79);
	printf("\n");
	border_print();
}

// Prints Section titles in center of 80 char terminal
void center_print(const char *s, int width)
{
	int length = strlen(s);
	int i;
	for (i=0; i<=(width-length)/2; i++) {
		fputs(" ", stdout);
	}
	fputs(s, stdout);
	fputs("\n", stdout);
}

// Prints a border
void border_print(void)
{
	printf(
	"==================================================================="
	"=============\n");
}

// Prints comma separated integers - for ease of reading
void fancy_int( int a )
{
    if( a < 1000 )
        printf("%d\n",a);

    else if( a >= 1000 && a < 1000000 )
        printf("%d,%03d\n", a / 1000, a % 1000);

    else if( a >= 1000000 && a < 1000000000 )
        printf("%d,%03d,%03d\n", a / 1000000, (a % 1000000) / 1000, a % 1000 );

    else if( a >= 1000000000 )
        printf("%d,%03d,%03d,%03d\n",
               a / 1000000000,
               (a % 1000000000) / 1000000,
               (a % 1000000) / 1000,
               a % 1000 );
    else
        printf("%d\n",a);
}

// Prints out the summary of User input
void print_input_summary(Input * I)
{
	center_print("INPUT SUMMARY", 79);
	border_print();
	#ifdef OPENMP
	printf("%-25s%d\n", "Number of Threads:", I->nthreads);
	#endif
	printf("%-25s%d\n", "Kernel execution times:", I->repeat);
	printf("%-25s%d\n", "Energy Groups:", I->egroups);
	printf("%-25s%d\n", "2D Source Regions:", I->source_2D_regions);
	printf("%-25s%d\n", "Coarse Axial Intervals:", I->coarse_axial_intervals);
	printf("%-25s%d\n", "Fine Axial Intervals:", I->fine_axial_intervals);
	printf("%-25s%d\n", "Axial Decomposition:", I->decomp_assemblies_ax);
	printf("%-25s%d\n", "3D Source Regions:", I->source_3D_regions);
	printf("%-25s", "Segments:"); fancy_int(I->segments);
	printf("%-25s%.2f\n", "Memory Estimate (MB):", I->nbytes/1024.0/1024.0);
	border_print();
}

// reads command line inputs and applies options
void read_CLI( int argc, char * argv[], Input * input )
{
	// defaults to max threads on the system	
	#ifdef OPENMP
	input->nthreads = omp_get_num_procs();
	#else
	input->nthreads = 1;
	#endif
	
	// Collect Raw Input
	for( int i = 1; i < argc; i++ )
	{
		char * arg = argv[i];

		// nthreads (-t)
		if( strcmp(arg, "-t") == 0 )
		{
			if( ++i < argc )
				input->nthreads = atoi(argv[i]);
			else
				print_CLI_error();
		}

		// segments (-s)
		else if( strcmp(arg, "-s") == 0 )
		{
			if( ++i < argc )
#ifdef VERIFY
				input->segments = 1;
#else
				input->segments = atoi(argv[i]);
#endif
			else
				print_CLI_error();
		}
		
		// egroups (-e)
		else if( strcmp(arg, "-e") == 0 )
		{
			if( ++i < argc )
				input->egroups = atoi(argv[i]);
			else
				print_CLI_error();
		}
		else if( strcmp(arg, "-n") == 0 )
		{
			if( ++i < argc )
#ifdef VERIFY
				input->repeat = 1;
#else
				input->repeat = atoi(argv[i]);
#endif
			else
				print_CLI_error();
		}
		else
			print_CLI_error();
	}


	// Validate nthreads
	if( input->nthreads < 1 )
		print_CLI_error();
}

// print error to screen, inform program options
void print_CLI_error(void)
{
	printf("Usage: ./SimpleMOC <options>\n");
	printf("Options include:\n");
	printf("  -t <threads>        Number of OpenMP threads to run\n");
	printf("  -s <segments>       Number of segments to process\n");
	printf("  -e <energy groups>  Number of energy groups\n");
	printf("  -n <kernel runs>    Number of kernel execution on a device (GPU)\n");
	exit(1);
}

