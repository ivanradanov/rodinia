//////////////////////////////////////////////////////////////////
//                                                              //
// This software was written by Mike Giles in 2007 based on     //
// C code written by Zhao and Glasserman at Columbia University //
//                                                              //
// It is copyright University of Oxford, and provided under     //
// the terms of the BSD3 license:                               //
// https://opensource.org/licenses/BSD-3-Clause                 //
//                                                              //
// It is provided along with an informal report on              //
// https://people.maths.ox.ac.uk/~gilesm/cuda_old.html          //
//                                                              //
// Note: this was written for CUDA 1.0 and optimised for        //
// execution on an NVIDIA 8800 GTX GPU                          //
//                                                              //
// Mike Giles, 29 April 2021                                    //
//                                                              //
//////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

// parameters for device execution

#define BLOCK_SIZE 64
#define GRID_SIZE 1500

// parameters for LIBOR calculation

#define NN 80
#define NMAT 40
#define L2_SIZE 3280 //NN*(NMAT+1)
#define NOPT 15
#define NPATH 96000

// Monte Carlo LIBOR path calculation

__device__
void path_calc(float *L, 
               const float *z, 
               const float *lambda, 
               const float delta,
               const int Nmat, 
               const int N)
{
  int   i, n;
  float sqez, lam, con1, v, vrat;

  for(n=0; n<Nmat; n++) {
    sqez = sqrtf(delta)*z[n];
    v = 0.f;

    for (i=n+1; i<N; i++) {
      lam  = lambda[i-n-1];
      con1 = delta*lam;
      v   += __fdividef(con1*L[i],1.f+delta*L[i]);
      vrat = __expf(con1*v + lam*(sqez-0.5f*con1));
      L[i] = L[i]*vrat;
    }
  }
}


// forward path calculation storing data
// for subsequent reverse path calculation

__device__
void path_calc_b1(float *L, 
                  const float *z, 
                  float *L2,
                  const float *lambda,
                  const float delta,
                  const int Nmat,
                  const int N)
{
  int   i, n;
  float sqez, lam, con1, v, vrat;

  for (i=0; i<N; i++) L2[i] = L[i];
   
  for(n=0; n<Nmat; n++) {
    sqez = sqrtf(delta)*z[n];
    v = 0.f;

    for (i=n+1; i<N; i++) {
      lam  = lambda[i-n-1];
      con1 = delta*lam;
      v   += __fdividef(con1*L[i],1.f+delta*L[i]);
      vrat = __expf(con1*v + lam*(sqez-0.5f*con1));
      L[i] = L[i]*vrat;

      // store these values for reverse path
      L2[i+(n+1)*N] = L[i];
    }
  }
}


// reverse path calculation of deltas using stored data

__device__
void path_calc_b2(float *L_b, 
                  const float *z, 
                  const float *L2, 
                  const float *lambda, 
                  const float delta,
                  const int Nmat,
                  const int N)
{
  int   i, n;
  float faci, v1;

  for (n=Nmat-1; n>=0; n--) {
    v1 = 0.f;
    for (i=N-1; i>n; i--) {
      v1    += lambda[i-n-1]*L2[i+(n+1)*N]*L_b[i];
      faci   = __fdividef(delta,1.f+delta*L2[i+n*N]);
      L_b[i] = L_b[i]*__fdividef(L2[i+(n+1)*N],L2[i+n*N])
              + v1*lambda[i-n-1]*faci*faci;
 
    }
  }
}

// calculate the portfolio value v, and its sensitivity to L
// hand-coded reverse mode sensitivity

__device__
float portfolio_b(float *L, 
                  float *L_b,
                  const float *lambda, 
                  const   int *maturities, 
                  const float *swaprates, 
                  const float delta,
                  const int Nmat,
                  const int N,
                  const int Nopt)
{
  int   m, n;
  float b, s, swapval,v;
  float B[NMAT], S[NMAT], B_b[NMAT], S_b[NMAT];

  b = 1.f;
  s = 0.f;
  for (m=0; m<N-Nmat; m++) {
    n    = m + Nmat;
    b    = __fdividef(b,1.f+delta*L[n]);
    s    = s + delta*b;
    B[m] = b;
    S[m] = s;
  }

  v = 0.f;

  for (m=0; m<NMAT; m++) {
    B_b[m] = 0.f;
    S_b[m] = 0.f;
  }

  for (n=0; n<Nopt; n++){
    m = maturities[n] - 1;
    swapval = B[m] + swaprates[n]*S[m] - 1.f;
    if (swapval<0) {
      v     += -100.f*swapval;
      S_b[m] += -100.f*swaprates[n];
      B_b[m] += -100.f;
    }
  }

  for (m=N-Nmat-1; m>=0; m--) {
    n = m + Nmat;
    B_b[m] += delta*S_b[m];
    L_b[n]  = -B_b[m]*B[m]*__fdividef(delta,1.f+delta*L[n]);
    if (m>0) {
      S_b[m-1] += S_b[m];
      B_b[m-1] += __fdividef(B_b[m],1.f+delta*L[n]);
    }
  }

  // apply discount

  b = 1.f;
  for (n=0; n<Nmat; n++) b = b/(1.f+delta*L[n]);

  v = b*v;

  for (n=0; n<Nmat; n++){
    L_b[n] = -v*delta/(1.f+delta*L[n]);
  }

  for (n=Nmat; n<N; n++){
    L_b[n] = b*L_b[n];
  }

  return v;
}


// calculate the portfolio value v

__device__
float portfolio(float *L,
                const float *lambda, 
                const   int *maturities, 
                const float *swaprates, 
                const float delta,
                const int Nmat,
                const int N,
                const int Nopt)
{
  int   n, m, i;
  float v, b, s, swapval, B[40], S[40];
	
  b = 1.f;
  s = 0.f;

  for(n=Nmat; n<N; n++) {
    b = b/(1.f+delta*L[n]);
    s = s + delta*b;
    B[n-Nmat] = b;
    S[n-Nmat] = s;
  }

  v = 0.f;

  for(i=0; i<Nopt; i++){
    m = maturities[i] - 1;
    swapval = B[m] + swaprates[i]*S[m] - 1.f;
    if(swapval<0)
      v += -100.f*swapval;
  }

  // apply discount

  b = 1.f;
  for (n=0; n<Nmat; n++) b = b/(1.f+delta*L[n]);

  v = b*v;

  return v;
}


__global__
void Pathcalc_Portfolio_KernelGPU(
  float *__restrict__ d_v, 
  float *__restrict__ d_Lb,
  const float *__restrict__ lambda, 
  const   int *__restrict__ maturities, 
  const float *__restrict__ swaprates, 
  const float delta,
  const int Nmat,
  const int N,
  const int Nopt)
{
  const int     tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int threadN = blockDim.x * gridDim.x;

  int   i,path;
  float L[NN], L2[L2_SIZE], z[NN];
  float *L_b = L;
  
  // Monte Carlo LIBOR path calculation

  for(path = tid; path < NPATH; path += threadN){
    // initialise the data for current thread
    for (i=0; i<N; i++) {
      // for real application, z should be randomly generated
      z[i] = 0.3f;
      L[i] = 0.05f;
    }
    path_calc_b1(L, z, L2, lambda, delta, Nmat, N);
    d_v[path] = portfolio_b(L, L_b, lambda, maturities, swaprates, delta, Nmat, N, Nopt);
    path_calc_b2(L_b, z, L2, lambda, delta, Nmat, N);
    d_Lb[path] = L_b[NN-1];
  }
}

__global__
void Pathcalc_Portfolio_KernelGPU2(
  float *__restrict__ d_v, 
  const float *__restrict__ lambda, 
  const   int *__restrict__ maturities, 
  const float *__restrict__ swaprates, 
  const float delta,
  const int Nmat,
  const int N,
  const int Nopt)
{
  const int     tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int threadN = blockDim.x * gridDim.x;

  int   i, path;
  float L[NN], z[NN];
  
  // Monte Carlo LIBOR path calculation

  for(path = tid; path < NPATH; path += threadN){
    // initialise the data for current thread
    for (i=0; i<N; i++) {
      // for real application, z should be randomly generated
      z[i] = 0.3f;
      L[i] = 0.05f;
    }	   
    path_calc(L, z, lambda, delta, Nmat, N);
    d_v[path] = portfolio(L, lambda, maturities, swaprates, delta, Nmat, N, Nopt);
  }
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);
    
  // 'h_' prefix - CPU (host) memory space

  float  *h_v, *h_Lb, h_lambda[NN], h_delta=0.25f;
  int     h_N=NN, h_Nmat=NMAT, h_Nopt=NOPT, i;
  int     h_maturities[] = {4,4,4,8,8,8,20,20,20,28,28,28,40,40,40};
  float   h_swaprates[]  = {.045f,.05f,.055f,.045f,.05f,.055f,.045f,.05f,
                            .055f,.045f,.05f,.055f,.045f,.05f,.055f };
  double  v, Lb; 
  bool    ok = true;

  // 'd_' prefix - GPU (device) memory space

  float *d_v;
  float *d_Lb;
  float *d_swaprates;
  float *d_lambda;
    int *d_maturities;

  for (i=0; i<NN; i++) h_lambda[i] = 0.2f;

  h_v      = (float *)malloc(sizeof(float)*NPATH);
  h_Lb     = (float *)malloc(sizeof(float)*NPATH);

  hipMalloc((void**)&d_maturities, sizeof(h_maturities));

  hipMalloc((void**)&d_swaprates, sizeof(h_swaprates));

  hipMalloc((void**)&d_lambda, sizeof(h_lambda));

  hipMalloc((void **)&d_v, sizeof(float)*NPATH);

  hipMalloc((void **)&d_Lb, sizeof(float)*NPATH);

  // Execute GPU kernel -- no Greeks

  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(GRID_SIZE);

  hipMemcpy(d_maturities, h_maturities, sizeof(h_maturities), hipMemcpyHostToDevice);
  hipMemcpy(d_swaprates, h_swaprates, sizeof(h_swaprates), hipMemcpyHostToDevice);
  hipMemcpy(d_lambda, h_lambda, sizeof(h_lambda), hipMemcpyHostToDevice);

  // Launch the device computation threads
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    Pathcalc_Portfolio_KernelGPU2
      <<<dimGrid, dimBlock>>>(d_v,
                              d_lambda, 
                              d_maturities, 
                              d_swaprates, 
                              h_delta,
                              h_Nmat,
                              h_N,
                              h_Nopt);
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time : %f (s)\n", (time * 1e-9f) / repeat);

  // Read back GPU results and compute average
  hipMemcpy(h_v, d_v, sizeof(float)*NPATH, hipMemcpyDeviceToHost);

  v = 0.0;
  for (i=0; i<NPATH; i++) v += h_v[i];
  v = v / NPATH;

  if (fabs(v - 224.323) > 1e-3) {
    ok = false;
    printf("Expected: 224.323 Actual %15.3f\n", v);
  }

  // Execute GPU kernel -- Greeks

  // Launch the device computation threads
  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    Pathcalc_Portfolio_KernelGPU
      <<<dimGrid, dimBlock>>>(d_v,
                              d_Lb,
                              d_lambda, 
                              d_maturities, 
                              d_swaprates, 
                              h_delta,
                              h_Nmat,
                              h_N,
                              h_Nopt);

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time : %f (s)\n", (time * 1e-9f) / repeat);

  // Read back GPU results and compute average
  hipMemcpy(h_v, d_v, sizeof(float)*NPATH, hipMemcpyDeviceToHost);
  hipMemcpy(h_Lb, d_Lb, sizeof(float)*NPATH, hipMemcpyDeviceToHost);

  v = 0.0;
  for (i=0; i<NPATH; i++) v += h_v[i];
  v = v / NPATH;

  Lb = 0.0;
  for (i=0; i<NPATH; i++) Lb += h_Lb[i];
  Lb = Lb / NPATH;

  if (fabs(v - 224.323) > 1e-3) {
    ok = false;
    printf("Expected: 224.323 Actual %15.3f\n", v);
  }
  if (fabs(Lb - 21.348) > 1e-3) {
    ok = false;
    printf("Expected:  21.348 Actual %15.3f\n", Lb);
  }

  hipFree(d_v);
  hipFree(d_Lb);
  hipFree(d_maturities);
  hipFree(d_swaprates);
  hipFree(d_lambda);
       
  free(h_v);
  free(h_Lb);

  printf("%s\n", ok ? "PASS" : "FAIL");

  return 0;
}
