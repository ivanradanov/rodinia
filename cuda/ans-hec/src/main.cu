#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 * MULTIANS - Massively parallel ANS decoding on GPUs
 *
 * released under LGPL-3.0
 *
 * 2017-2019 André Weißenberger
 *
 *****************************************************************************/

#include <cassert>
#include <chrono>
#include <random>
#include <algorithm>
#include <iostream>
#include <iomanip>

#include "multians.h"

// encoder configuration //
#define NUM_SYMBOLS 256
#define NUM_STATES 1024

// seed for PRNG to generate random test data
#define SEED 5

// decoder configuration //

// SUBSEQUENCE_SIZE must be a multiple of 4
#define SUBSEQUENCE_SIZE 4

// number of GPU threads per thread block //
#define THREADS_PER_BLOCK 128

void run(long int input_size) {

  // print column headers
  std::cout << "\u03BB | compressed size (bytes) | ";
  std::cout << std::endl << std::endl;

  auto start = std::chrono::steady_clock::now();

  for(float lambda = 0.1f; lambda < 2.5f; lambda += 0.16) {

    // vectors to record timings
    std::cout << std::left << std::setw(5) << lambda << std::setfill(' ');

    // generate random, exponentially distributed data
    auto dist = ANSTableGenerator::generate_distribution(
        SEED, NUM_SYMBOLS, NUM_STATES,
        [&](double x) {return lambda * exp(-lambda * x);});

    auto random_data = ANSTableGenerator::generate_test_data(
          dist.dist, input_size, NUM_STATES, SEED);

    // create an ANS table, based on the distribution
    auto table = ANSTableGenerator::generate_table(
        dist.prob, dist.dist, nullptr, NUM_SYMBOLS,
        NUM_STATES);

    // derive an encoder table from the ANS table
    auto encoder_table = ANSTableGenerator::generate_encoder_table(table);

    // derive a decoder table from the ANS table
    auto decoder_table = ANSTableGenerator::get_decoder_table(encoder_table);

    // tANS-encode the generated data using the encoder table
    auto input_buffer = ANSEncoder::encode(
        random_data->data(), input_size, encoder_table);

    // allocate buffer for the decoded output
    auto output_buffer = std::make_shared<CUHDOutputBuffer>(input_size);

    // allocate device buffer for compressed input
    size_t compressed_size = input_buffer->get_compressed_size();
    size_t input_buffer_bytes = sizeof(UNIT_TYPE) * (compressed_size + 4);
    UNIT_TYPE* d_input_buffer;
    hipMalloc((void**)&d_input_buffer, input_buffer_bytes);
    hipMemcpy(d_input_buffer, input_buffer->get_compressed_data(), 
        input_buffer_bytes, hipMemcpyHostToDevice);

    // allocate device buffer for coding table
    size_t decoder_table_size = decoder_table->get_size() * sizeof(CUHDCodetableItem);
    std::uint32_t *d_decoder_table;
    hipMalloc((void**)&d_decoder_table, decoder_table_size);
    hipMemcpy(d_decoder_table, reinterpret_cast<std::uint32_t*>(decoder_table->get()), 
               decoder_table_size, hipMemcpyHostToDevice); //cast the type CUHDCodetableItem 

    // allocate device buffer for decompressed output
    size_t output_buffer_bytes = sizeof(SYMBOL_TYPE) * output_buffer->get_uncompressed_size();
    SYMBOL_TYPE* d_output_buffer;
    hipMalloc((void**)&d_output_buffer, output_buffer_bytes);

    size_t num_subseq = SDIV(compressed_size, SUBSEQUENCE_SIZE);
    size_t num_blocks = SDIV(num_subseq, THREADS_PER_BLOCK);

    // allocate device buffer for subsequence synchronization
    // Note the original type is cuhd::CUHDSubsequenceSyncPoint (uint4 is equivalent)
    uint4 *d_sync_info;
    hipMalloc((void**) &d_sync_info, num_subseq * sizeof(uint4));
    hipMemset(d_sync_info, 0, num_subseq * sizeof(uint4));

    // allocate device buffer for size of output for each subsequence
    std::uint32_t *d_output_sizes;
    hipMalloc((void**) &d_output_sizes, num_subseq * sizeof(std::uint32_t));

    // allocate device buffer for indicating inter-sequence synchronisation
    std::uint8_t *d_sequence_synced;
    hipMalloc((void**)&d_sequence_synced, num_blocks * sizeof(std::uint8_t));
    hipMemset(d_sequence_synced, 0, num_blocks * sizeof(std::uint8_t));

    std::uint8_t* h_sequence_synced = (std::uint8_t*) malloc(num_blocks * sizeof(std::uint8_t));

    // decode the compressed data on a GPU

    cuhd::CUHDGPUDecoder::decode(
        d_input_buffer, input_buffer->get_compressed_size(),
        d_output_buffer, output_buffer->get_uncompressed_size(),
        d_decoder_table,
        d_sync_info,
        d_output_sizes,
        d_sequence_synced,
        h_sequence_synced,
        input_buffer->get_first_state(),
        input_buffer->get_first_bit(), 
        decoder_table->get_num_entries(),
        11, 
        SUBSEQUENCE_SIZE, 
        THREADS_PER_BLOCK);

      // copy decompressed output from the GPU to the host system
    hipMemcpy(output_buffer->get_decompressed_data().get(), d_output_buffer, 
          output_buffer_bytes, hipMemcpyDeviceToHost);

    // reverse all bytes
    output_buffer->reverse();

    // check for errors in decompressed data
    if(cuhd::CUHDUtil::equals(random_data->data(),
          output_buffer->get_decompressed_data().get(), input_size));
    else std::cout << "********* MISMATCH ************" << std::endl;

    // print compressed size (bytes)
    std::cout << std::left << std::setw(10)
      << input_buffer->get_compressed_size() * sizeof(UNIT_TYPE)
      << std::setfill(' ') << std::endl;

    hipFree(d_input_buffer);
    hipFree(d_output_buffer);
    hipFree(d_decoder_table);
    hipFree(d_sync_info);
    hipFree(d_output_sizes);
    hipFree(d_sequence_synced);
    free(h_sequence_synced);
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Total elapsed time " << time * 1e-9f << " (s)\n";
}

int main(int argc, char **argv) {

  // name of the binary file
  const char* bin = argv[0];

  auto print_help = [&]() {
    std::cout << "USAGE: " << bin << "<size of input in megabytes> " << std::endl;
  };

  if(argc < 2) {print_help(); return 1;}

  // input size in MB
  const long int size = atoi(argv[1]) * 1024 * 1024;

  if(size < 1) {
    print_help();
    return 1;
  }

  // SUBSEQUENCE_SIZE must be a multiple of 4
  assert(SUBSEQUENCE_SIZE % 4 == 0);

  // run the test
  run(size);

  return 0;
}

