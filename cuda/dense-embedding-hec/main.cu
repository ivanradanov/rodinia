#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>

template <typename T>
void reference(
    const T* input,
    const T* dense,
    T* output,
    int embedding_dim,
    int batch_size,
    const int* offset)
{
  for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
    const int range = offset[batch_idx + 1] - offset[batch_idx];
    for (int idx = 0; idx < embedding_dim; idx++) {
      const T dense_elem = dense[batch_idx * embedding_dim + idx];
      for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
        output[offset[batch_idx] + nested_idx] = 
          input[offset[batch_idx] + nested_idx] + dense_elem;
      }
    }
  }
}

template <typename T>
__global__ void dense_esuhm(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset)
{
  const int batch_idx  = blockIdx.x; // each batch is handled by a block
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = offset[batch_idx + 1] - offset[batch_idx];
  for (int idx = tid; idx < embedding_dim; idx += grain_size) {
    const T dense_elem = dense[batch_idx * embedding_dim + idx];
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[offset[batch_idx] + nested_idx] = input[offset[batch_idx] + nested_idx] + dense_elem;
    }
  }
}

template <typename T>
__global__ void dense_esuhm2(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset)
{
  const int batch_idx  = blockIdx.x;
  const int start = offset[batch_idx];
  const int range = offset[batch_idx + 1] - start;
  for (int idx = threadIdx.x; idx < embedding_dim; idx += blockDim.x) {
    const T dense_elem = dense[batch_idx * embedding_dim + idx];
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[start + nested_idx] = input[start + nested_idx] + dense_elem;
    }
  }
}

int main(int argc, char* argv[])
{
  if (argc != 4) {
    printf("Usage: %s <number of rows> <batch size> <repeat>\n", argv[0]);
    return 1;
  }
  const int nrows = atoi(argv[1]);
  const int batch_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);
  assert(nrows > batch_size * batch_size);

  printf("Number of rows in the embedding table: %d\n", nrows);
  printf("Batch size: %d\n", batch_size);

  for (int ncols = 64; ncols <= 2048; ncols = ncols * 2) {

    printf("\nEmbedding dimension: %d\n", ncols);

    int input_size = nrows * ncols;  // same as output size
    size_t input_size_bytes = input_size * sizeof(float);

    int dense_size = batch_size * ncols ;
    int dense_size_bytes = dense_size * sizeof(float);

    int batch_size_bytes = (batch_size + 1) * sizeof(float);

    float *input, *dense, *output, *output_ref;
    input = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    dense = (float*) malloc (dense_size_bytes); // [batch_size x embedding_dim]
    output = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    output_ref = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    int *input_offset = (int*) malloc (batch_size_bytes);  // [batch_size]

    // meet a few conditions:
    // range > ncols
    // range + offset[i] < input_size
    // offset[i] > offset[i-1]
    srand(123);
    input_offset[0] = 0;
    for (int i = 1; i <= batch_size; i++)
      input_offset[i] = input_offset[i-1] + (rand() % batch_size + 1) * ncols;

    std::default_random_engine g (123);
    std::uniform_real_distribution<float> distr (-1.f, 1.f);
    for (int i = 0; i < dense_size; i++) {
      dense[i] = distr(g);
    }

    for (int i = 0; i < input_size; i++) {
      input[i] = distr(g);
      output_ref[i] = 0;
    }

    reference(input, dense, output_ref, ncols, batch_size, input_offset);

    float *d_input, *d_dense, *d_output;
    hipMalloc((void**)&d_input, input_size_bytes);
    hipMemcpy(d_input, input, input_size_bytes, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_dense, dense_size_bytes);
    hipMemcpy(d_dense, dense, dense_size_bytes, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_output, input_size_bytes);
    hipMemset(d_output, 0, input_size_bytes);

    int* d_input_offset;
    hipMalloc((void**)&d_input_offset, batch_size_bytes);
    hipMemcpy(d_input_offset, input_offset, batch_size_bytes, hipMemcpyHostToDevice);
    
    dim3 grid (batch_size);
    dim3 block (256);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++) 
      dense_esuhm<<<grid, block>>>(d_input, d_dense, d_output, ncols, d_input_offset);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("Average execution time of dense embedding kernel (k1): %f (us)\n", (time * 1e-3f) / repeat);

    hipDeviceSynchronize();
    start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++) 
      dense_esuhm2<<<grid, block>>>(d_input, d_dense, d_output, ncols, d_input_offset);

    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("Average execution time of dense embedding kernel (k2): %f (us)\n", (time * 1e-3f) / repeat);

    hipMemcpy(output, d_output, input_size_bytes, hipMemcpyDeviceToHost);

    bool ok = true;
    for (int i = 0; i < input_size; i++) {
      if (fabsf(output[i] - output_ref[i]) > 1e-3f) {
        ok = false;
        break;
      }
    }
    printf("%s\n", ok ? "PASS" : "FAIL");

    hipFree(d_input);
    hipFree(d_dense);
    hipFree(d_output);
    hipFree(d_input_offset);

    free(input);
    free(dense);
    free(output);
    free(output_ref);
    free(input_offset);
  }

  return 0;
}
