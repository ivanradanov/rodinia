#include "hip/hip_runtime.h"
#include <stdio.h>
#include "grid.h"
#include "constants.h"

struct grid_t init_grid(llint nx, llint ny, llint nz, llint tsx, llint tsy,
                        llint ngpu)
{
    struct grid_t grid;
    grid.nx = nx; grid.ny = ny; grid.nz = nz;
    grid.dx = 20;  grid.dy = 20;  grid.dz = 20;
    grid.lx = 4; grid.ly = 4; grid.lz = 4;
    grid.ntaperx = 3; grid.ntapery = 3; grid.ntaperz = 3;

    const float lambdamax = vmax/_fmax;
    grid.ndampx = grid.ntaperx * lambdamax / grid.dx;
    grid.ndampy = grid.ntapery * lambdamax / grid.dy;
    grid.ndampz = grid.ntaperz * lambdamax / grid.dz;

    grid.x1 = 0;
    grid.x2 = grid.ndampx;
    grid.x3 = grid.ndampx;
    grid.x4 = grid.nx-grid.ndampx;
    grid.x5 = grid.nx-grid.ndampx;
    grid.x6 = grid.nx;

    grid.y1 = 0;
    grid.y2 = grid.ndampy;
    grid.y3 = grid.ndampy;
    grid.y4 = grid.ny-grid.ndampy;
    grid.y5 = grid.ny-grid.ndampy;
    grid.y6 = grid.ny;

    grid.z1 = 0;
    grid.z2 = grid.ndampz;
    grid.z3 = grid.ndampz;
    grid.z4 = grid.nz-grid.ndampz;
    grid.z5 = grid.nz-grid.ndampz;
    grid.z6 = grid.nz;

    grid.tsx = tsx;
    grid.tsy = tsy;
    grid.ntx = nx/tsx;
    grid.nty = ny/tsy;

    // For multi-gpu targets
    grid.ngpu = ngpu;

    printf("ndamp = %lld %lld %lld\n", grid.ndampx, grid.ndampy, grid.ndampz);
    return grid;
}
