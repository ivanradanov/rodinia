/*------------------------------------------------------------------------------
* Copyright 2015-16: Tom Deakin, Simon McIntosh-Smith, University of Bristol HPC
* Based on John D. McCalpin’s original STREAM benchmark for CPUs
*------------------------------------------------------------------------------
* License:
*  1. You are free to use this program and/or to redistribute
*     this program.
*  2. You are free to modify this program for your own use,
*     including commercial use, subject to the publication
*     restrictions in item 3.
*  3. You are free to publish results obtained from running this
*     program, or from works that you derive from this program,
*     with the following limitations:
*     3a. In order to be referred to as "BabelStream benchmark results",
*         published results must be in conformance to the BabelStream
*         Run Rules published at
*         http://github.com/UoB-HPC/BabelStream/wiki/Run-Rules
*         and incorporated herein by reference.
*         The copyright holders retain the
*         right to determine conformity with the Run Rules.
*     3b. Results based on modified source code or on runs not in
*         accordance with the BabelStream Run Rules must be clearly
*         labelled whenever they are published.  Examples of
*         proper labelling include:
*         "tuned BabelStream benchmark results"
*         "based on a variant of the BabelStream benchmark code"
*         Other comparable, clear and reasonable labelling is
*         acceptable.
*     3c. Submission of results to the BabelStream benchmark web site
*         is encouraged, but not required.
*  4. Use of this program or creation of derived works based on this
*     program constitutes acceptance of these licensing restrictions.
*  5. Absolutely no warranty is expressed or implied.
*/

#include <iostream>
#include <vector>
#include <numeric>
#include <cmath>
#include <limits>
#include <chrono>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <hip/hip_runtime.h>


// Thread block size
#define TBSIZE 256

// Number of blocks for the DOT kernel 
#define DOT_NUM_BLOCKS 256

// scalar constanst for the mul, triad and nstream kernels
#define SCALAR (0.4)

int ARRAY_SIZE = 33554432; // Default size of 2^25
unsigned int num_times = 100;

void check_error(void)
{
#ifdef DEBUG
  cudaError_t err = cudaGetLastError();
  if (err != cudaSuccess)
  {
    std::cerr << "Error: " << cudaGetErrorString(err) << std::endl;
    exit(err);
  }
#endif
}

template <typename T>
__global__ void init_kernel(
  T *__restrict__ a,
  T *__restrict__ b,
  T *__restrict__ c,
  T initA, T initB, T initC)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;
}

template <class T>
void init_arrays(T *da, T *db, T *dc, T initA, T initB, T initC)
{
  const int array_size = ARRAY_SIZE; 
  init_kernel<<<array_size/TBSIZE, TBSIZE>>>(da, db, dc, initA, initB, initC);
  check_error();
  hipDeviceSynchronize();
  check_error();
}


template <typename T>
__global__ void copy_kernel(
  const T *__restrict__ a,
        T *__restrict__ c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i];
}

template <class T>
void copy(const T *da, T *dc)
{
  const int array_size = ARRAY_SIZE;
  copy_kernel<<<array_size/TBSIZE, TBSIZE>>>(da, dc);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void mul_kernel(
        T *__restrict__ b,
  const T *__restrict__ c)
{
  const T scalar = SCALAR;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  b[i] = scalar * c[i];
}

template <class T>
void mul(T *db, const T *dc)
{
  const int array_size = ARRAY_SIZE;
  mul_kernel<<<array_size/TBSIZE, TBSIZE>>>(db, dc);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void add_kernel(
  const T *__restrict__ a,
  const T *__restrict__ b,
        T *__restrict__ c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

template <class T>
void add(const T *da, const T *db, T *dc)
{
  const int array_size = ARRAY_SIZE;
  add_kernel<<<array_size/TBSIZE, TBSIZE>>>(da, db, dc);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void triad_kernel(
        T *__restrict__ a,
  const T *__restrict__ b,
  const T *__restrict__ c)
{
  const T scalar = SCALAR;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = b[i] + scalar * c[i];
}

template <class T>
void triad(T *da, const T *db, const T *dc)
{
  const int array_size = ARRAY_SIZE;
  triad_kernel<<<array_size/TBSIZE, TBSIZE>>>(da, db, dc);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void nstream_kernel(
        T *__restrict__ a,
  const T *__restrict__ b,
  const T *__restrict__ c)
{
  const T scalar = SCALAR;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] += b[i] + scalar * c[i];
}

template <class T>
void nstream(T * da, const T * db, const T * dc)
{
  const int array_size = ARRAY_SIZE;
  nstream_kernel<<<array_size/TBSIZE, TBSIZE>>>(da, db, dc);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
__global__ void dot_kernel(
  const T *__restrict__ a,
  const T *__restrict__ b,
        T *__restrict__ sum,
  int array_size)
{
  __shared__ T tb_sum[TBSIZE];

  const size_t local_i = threadIdx.x;

  tb_sum[local_i] = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x;
       i < array_size; i += blockDim.x*gridDim.x)
    tb_sum[local_i] += a[i] * b[i];

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
  {
    __syncthreads();
    if (local_i < offset)
    {
      tb_sum[local_i] += tb_sum[local_i+offset];
    }
  }

  if (local_i == 0)
    sum[blockIdx.x] = tb_sum[local_i];
}

template <class T>
T dot(const T * da, const T * db, T * dsum, T *sums)
{
  const int array_size = ARRAY_SIZE;
  dot_kernel<<<DOT_NUM_BLOCKS, TBSIZE>>>(da, db, dsum, array_size);
  check_error();

  // sum up partial sums on a host
  hipMemcpy(sums, dsum, DOT_NUM_BLOCKS*sizeof(T), hipMemcpyDeviceToHost);
  check_error();

  T sum = 0.0;
  for (int i = 0; i < DOT_NUM_BLOCKS; i++)
    sum += sums[i];
  return sum;
}


// Runs the kernel(s) and prints output.
template <typename T>
void run()
{
  std::streamsize ss = std::cout.precision();

  std::cout << "Running kernels " << num_times << " times" << std::endl;

  // The array size must be divisible by TBSIZE for kernel launches
  if (ARRAY_SIZE % TBSIZE != 0)
  {
    std::stringstream ss;
    ss << "Array size must be a multiple of " << TBSIZE;
    throw std::runtime_error(ss.str());
  }

  T *da;
  hipMalloc(&da, ARRAY_SIZE*sizeof(T));
  check_error();

  T *db;
  hipMalloc(&db, ARRAY_SIZE*sizeof(T));
  check_error();

  T *dc;
  hipMalloc(&dc, ARRAY_SIZE*sizeof(T));
  check_error();

  T *dsum;
  hipMalloc(&dsum, DOT_NUM_BLOCKS*sizeof(T));
  check_error();

  // Allocate the host array for partial sums for the dot kernel
  T *sums = (T*)malloc(sizeof(T) * DOT_NUM_BLOCKS);

  if (sizeof(T) == sizeof(float))
    std::cout << "Precision: float" << std::endl;
  else
    std::cout << "Precision: double" << std::endl;

  // MB = 10^6
  std::cout << std::setprecision(1) << std::fixed
    << "Array size: " << ARRAY_SIZE*sizeof(T)*1.0E-6 << " MB"
    << " (=" << ARRAY_SIZE*sizeof(T)*1.0E-9 << " GB)" << std::endl;
  std::cout << "Total size: " << 3.0*ARRAY_SIZE*sizeof(T)*1.0E-6 << " MB"
    << " (=" << 3.0*ARRAY_SIZE*sizeof(T)*1.0E-9 << " GB)" << std::endl;
  std::cout.precision(ss);

  // Initialize device arrays
  init_arrays(da, db, dc, (T)0.1, (T)0.2, T(0.0));

  // List of times
  std::vector<std::vector<double>> timings(6);

  // Declare timers
  std::chrono::high_resolution_clock::time_point t1, t2;

  // Main loop
  for (unsigned int k = 0; k < num_times; k++)
  {
    // Execute Copy
    t1 = std::chrono::high_resolution_clock::now();
    copy(da, dc);
    t2 = std::chrono::high_resolution_clock::now();
    timings[0].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());


    // Execute Mul
    t1 = std::chrono::high_resolution_clock::now();
    mul(db, dc);
    t2 = std::chrono::high_resolution_clock::now();
    timings[1].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

    // Execute Add
    t1 = std::chrono::high_resolution_clock::now();
    add(da, db, dc);
    t2 = std::chrono::high_resolution_clock::now();
    timings[2].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

    // Execute Triad
    t1 = std::chrono::high_resolution_clock::now();
    triad(da, db, dc);
    t2 = std::chrono::high_resolution_clock::now();
    timings[3].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

    // Execute Dot
    t1 = std::chrono::high_resolution_clock::now();
    dot(da, db, dsum, sums);
    t2 = std::chrono::high_resolution_clock::now();
    timings[4].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());

    // Execute NStream
    t1 = std::chrono::high_resolution_clock::now();
    nstream(da, db, dc);
    t2 = std::chrono::high_resolution_clock::now();
    timings[5].push_back(std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count());
  }

  // Display timing results
  std::cout
    << std::left << std::setw(12) << "Function"
    << std::left << std::setw(12) << "MBytes/sec"
    << std::left << std::setw(12) << "Min (sec)"
    << std::left << std::setw(12) << "Max"
    << std::left << std::setw(12) << "Average"
    << std::endl
    << std::fixed;

  std::vector<std::string> labels;
  std::vector<size_t> sizes;

  labels = {"Copy", "Mul", "Add", "Triad", "Dot", "Nstream"};
  sizes = {
    2 * sizeof(T) * ARRAY_SIZE,
    2 * sizeof(T) * ARRAY_SIZE,
    3 * sizeof(T) * ARRAY_SIZE,
    3 * sizeof(T) * ARRAY_SIZE,
    2 * sizeof(T) * ARRAY_SIZE,
    4 * sizeof(T) * ARRAY_SIZE};

  for (size_t i = 0; i < timings.size(); ++i)
  {
    // Get min/max; ignore the first result
    auto minmax = std::minmax_element(timings[i].begin()+1, timings[i].end());

    // Calculate average; ignore the first result
    double average = std::accumulate(timings[i].begin()+1, timings[i].end(), 0.0) / (double)(num_times - 1);

    double bandwidth = 1.0E-6 * sizes[i] / (*minmax.first);

    std::cout
      << std::left << std::setw(12) << labels[i]
      << std::left << std::setw(12) << std::setprecision(3) << bandwidth
      << std::left << std::setw(12) << std::setprecision(5) << *minmax.first
      << std::left << std::setw(12) << std::setprecision(5) << *minmax.second
      << std::left << std::setw(12) << std::setprecision(5) << average
      << std::endl;
  }
  // Add a blank line
  std::cout << std::endl;

  hipFree(da);
  check_error();
  hipFree(db);
  check_error();
  hipFree(dc);
  check_error();
  hipFree(dsum);
  check_error();
  free(sums);
}


int parseUInt(const char *str, unsigned int *output)
{
  char *next;
  *output = strtoul(str, &next, 10);
  return !strlen(next);
}

int parseInt(const char *str, int *output)
{
  char *next;
  *output = strtol(str, &next, 10);
  return !strlen(next);
}

void parseArguments(int argc, char *argv[])
{
  for (int i = 1; i < argc; i++)
  {
    if (!std::string("--arraysize").compare(argv[i]) ||
        !std::string("-s").compare(argv[i]))
    {
      if (++i >= argc || !parseInt(argv[i], &ARRAY_SIZE) || ARRAY_SIZE <= 0)
      {
        std::cerr << "Invalid array size." << std::endl;
        exit(EXIT_FAILURE);
      }
    }
    else if (!std::string("--numtimes").compare(argv[i]) ||
        !std::string("-n").compare(argv[i]))
    {
      if (++i >= argc || !parseUInt(argv[i], &num_times))
      {
        std::cerr << "Invalid number of times." << std::endl;
        exit(EXIT_FAILURE);
      }
      if (num_times < 2)
      {
        std::cerr << "Number of times must be 2 or more" << std::endl;
        exit(EXIT_FAILURE);
      }
    }
    else if (!std::string("--help").compare(argv[i]) ||
        !std::string("-h").compare(argv[i]))
    {
      std::cout << std::endl;
      std::cout << "Usage: " << argv[0] << " [OPTIONS]" << std::endl << std::endl;
      std::cout << "Options:" << std::endl;
      std::cout << "  -h  --help               Print the message" << std::endl;
      std::cout << "  -s  --arraysize  SIZE    Use SIZE elements in the array" << std::endl;
      std::cout << "  -n  --numtimes   NUM     Run the test NUM times (NUM >= 2)" << std::endl;
      std::cout << std::endl;
      exit(EXIT_SUCCESS);
    }
    else
    {
      std::cerr << "Unrecognized argument '" << argv[i] << "' (try '--help')"
        << std::endl;
      exit(EXIT_FAILURE);
    }
  }
}

int main(int argc, char *argv[])
{
  parseArguments(argc, argv);
  run<float>();
  run<double>();
}


