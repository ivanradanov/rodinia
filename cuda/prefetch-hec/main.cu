#include "hip/hip_runtime.h"
/*
 Copyright (c) 2015 - 2021 Advanced Micro Devices, Inc. All rights reserved.
 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:
 The above copyright notice and this permission notice shall be included in
 all copies or substantial portions of the Software.
 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 THE SOFTWARE.
*/

#include <stdio.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define CUDACHECK(error)                                                                       \
{                                                                                              \
    hipError_t localError = error;                                                            \
    if (localError != hipSuccess) {                                                           \
        printf("error: %s at %s:%d\n", hipGetErrorString(localError),  __FILE__, __LINE__);   \
    }                                                                                          \
}

__global__
void add(int n, const float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] += x[i];
}

void prefetch (const int gpuDeviceId, const int numElements, const int repeat)
{
  printf("Concurrent managed access with prefetch\n");

  float *A, *B;

  CUDACHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  CUDACHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  CUDACHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    CUDACHECK(hipMemAdvise(A, numElements*sizeof(float), hipMemAdviseSetReadMostly, hipCpuDeviceId));
    CUDACHECK(hipMemPrefetchAsync(A, numElements*sizeof(float), gpuDeviceId));
    CUDACHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), gpuDeviceId));

    add <<< dimGrid, dimBlock >>> (numElements, A, B);

    CUDACHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), hipCpuDeviceId));
    CUDACHECK(hipDeviceSynchronize());
  }

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  CUDACHECK(hipFree(A));
  CUDACHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

void naive (const int numElements, const int repeat)
{
  printf("Concurrent managed access without prefetch\n");

  float *A, *B;

  CUDACHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  CUDACHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  CUDACHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    add <<< dimGrid, dimBlock >>> (numElements, A, B);

    CUDACHECK(hipDeviceSynchronize());
  }

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  CUDACHECK(hipFree(A));
  CUDACHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

int main(int argc, char *argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int p_gpuDevice = 0;
  CUDACHECK(hipSetDevice(p_gpuDevice));
  printf("info: set device to %d\n", p_gpuDevice);

  int concurrentManagedAccess = 0;
  CUDACHECK(hipDeviceGetAttribute(&concurrentManagedAccess,
        hipDeviceAttributeConcurrentManagedAccess,
        p_gpuDevice));
  if(!concurrentManagedAccess) {
    printf("info: concurrent managed access not supported on device %d\n Skipped\n", p_gpuDevice);
    return 0;
  }

  const int numElements = 64 * 1024 * 1024;

  printf("------------\n");
  printf("   Warmup   \n");
  printf("------------\n");
  prefetch(p_gpuDevice, numElements, repeat);
  naive(numElements, repeat);
  printf("------------\n");
  printf("   Done     \n");
  printf("------------\n");

  prefetch(p_gpuDevice, numElements, repeat);
  naive(numElements, repeat);
  return 0;
}
