#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define GridDimMaxY 65536

template <typename T, bool kNFirst>
__global__ void ChannelShuffleNCHWKernel(
    const int G,
    const int K,
    const int HxW,
    const T* X,
          T* Y)
{
  const int C = G * K;
  const int n = kNFirst ? blockIdx.x : blockIdx.y;
  const int s = kNFirst ? blockIdx.y : blockIdx.x;
  const int g = blockIdx.z % G;
  const int k = blockIdx.z / G;
  const int offset = s * NUM_THREADS + threadIdx.x;
  if (offset < HxW) {
    Y[(n * C + blockIdx.z) * HxW + offset] =
        __ldg(X + (n * C + g * K + k) * HxW + offset);
  }
}

template <typename T, int kSharedSize>
__global__ void
ChannelShuffleNHWCKernel(const int G, const int K, const T* X, T* Y)
{
  __shared__ T sdata[kSharedSize];
  const int C = G * K;
  const int offset = blockIdx.x * C;
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
    sdata[i] = __ldg(X + offset + i);
  }
  __syncthreads();
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
    const int g = i % G;
    const int k = i / G;
    Y[offset + i] = sdata[g * K + k];
  }
}

template <typename T>
bool ChannelShuffleNCHW (T *X, int N, int C, int G, int numel, T *Y,
                         long &time, int repeat)
{
  if (C % G != 0 || numel < N * C) return false;

  const int K = C / G;
  const int HxW = numel / (N * C);
  const int S = (HxW + NUM_THREADS - 1) / NUM_THREADS;

  auto start = std::chrono::steady_clock::now();

  if (N <= GridDimMaxY) {
    const dim3 dim_grid(S, N, C);
    for (int i = 0; i < repeat; i++)
      ChannelShuffleNCHWKernel<float, false>
        <<<dim_grid, NUM_THREADS>>>(G, K, HxW, X, Y);
  } else {
    const dim3 dim_grid(N, S, C);
    for (int i = 0; i < repeat; i++)
      ChannelShuffleNCHWKernel<float, true>
        <<<dim_grid, NUM_THREADS>>>(G, K, HxW, X, Y);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  return true;
}

template <typename T>
bool ChannelShuffleNHWC (T *X, int N, int C, int G, int numel, T *Y,
                         long &time, int repeat)
{
  if (C % G != 0 || numel < N * C) return false;

  const int K = C / G;
  const int HxW = numel / (N * C);
  const int outer_size = N * HxW;

  auto start = std::chrono::steady_clock::now();

  if (C <= 32) {
    for (int i = 0; i < repeat; i++)
      ChannelShuffleNHWCKernel<float, 32>
        <<<outer_size, NUM_THREADS>>>(G, K, X, Y);
  } else if (C <= 128) {
    for (int i = 0; i < repeat; i++)
      ChannelShuffleNHWCKernel<float, 128>
        <<<outer_size, NUM_THREADS>>>(G, K, X, Y);
  } else if (C <= 512) {
    for (int i = 0; i < repeat; i++)
      ChannelShuffleNHWCKernel<float, 512>
        <<<outer_size, NUM_THREADS>>>(G, K, X, Y);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  return true;
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: %s <group size> <width> <height> <repeat>\n", argv[0]);
    return 1;
  }
  const int G = atoi(argv[1]);
  const int W = atoi(argv[2]);
  const int H = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  long time;

  // limited by the global device memory
  for (int N = 1; N <= 64; N = N * 4) {
    for (int C = 32; C <= 512; C = C * 4) {

      printf("\n(N=%d C=%d W=%d H=%d)\n", N, C, W, H);

      const int numel = N * C * W * H; // assume no integer overflow
      size_t data_size_bytes = numel * sizeof(float);

      float *d_X, *d_Y;
      auto errorX = hipMalloc((void**)&d_X, data_size_bytes);
      auto errorY = hipMalloc((void**)&d_Y, data_size_bytes);
      if (errorX != hipSuccess || errorY != hipSuccess) {
        if (errorX == hipSuccess) hipFree(d_X);
        if (errorY == hipSuccess) hipFree(d_Y);
        printf("Device memory allocation failed. Exit\n");
        goto end;
      }

      auto ok = ChannelShuffleNHWC (d_X, N, C, G, numel, d_Y, time, repeat);
      if (ok)
        printf("Average time of channel shuffle (nhwc): %f (ms)\n", (time * 1e-6f) / repeat);
      else
        printf("Failed to execute channel shuffle (nhwc)\n");

      ok = ChannelShuffleNCHW (d_X, N, C, G, numel, d_Y, time, repeat);
      if (ok)
        printf("Average time of channel shuffle (nchw): %f (ms)\n", (time * 1e-6f) / repeat);
      else
        printf("Failed to execute channel shuffle (nchw)\n");

      hipFree(d_X);
      hipFree(d_Y);
    }
  }
  
  end: return 0;
}
