#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "kernels.h"

template<typename T>
void invokeAddBiasResidualLayerNorm(
          T*     out,
    const T*     input,
    const T*     bias,
    const T*     gamma,
    const T*     beta,
    const float  layernorm_eps,
    int          m,
    int          n)
{
  dim3 grid(m);
  dim3 block(std::min(n, 256));

  if (m >= 512 && (n == 768 || n == 1024)) {
    addBiasResidualPostLayerNormV2<T><<<grid, n / 8>>>(out, input, bias, gamma, beta, layernorm_eps, n);
  }
  else {
    int num_trips = (n + block.x - 1) / block.x;
    if (num_trips == 1) {
      addBiasResidualPostLayerNorm<T, 1>
        <<<grid, block>>>(out, input, bias, gamma, beta, layernorm_eps, n);
    }
    else if (num_trips == 2) {
      addBiasResidualPostLayerNorm<T, 2>
        <<<grid, block>>>(out, input, bias, gamma, beta, layernorm_eps, n);
    }
    else {
      generalAddBiasResidualPostLayerNorm<T>
        <<<grid, block>>>(out, input, bias, gamma, beta, layernorm_eps, n);
    }
  }
}

template<typename T>
void layer(int repeat) {

  std::mt19937 gen (19937);
  std::uniform_real_distribution<float> dis(0.f, 1.f);

  const int m = 4096;  // batch size

  // n-dimensional data
  for (int n = 512; n <= 4096; n = n * 2) {
    const int input_size = m * n;
    const int output_size = m * n;
    const int input_size_bytes = input_size * sizeof(T);
    const int output_size_bytes = output_size * sizeof(T);
    const int bias_size_bytes = n * sizeof(T);
    const int beta_size_bytes = n * sizeof(T);
    const int gamma_size_bytes = n * sizeof(T);

    T *h_input = (T*) malloc (input_size_bytes);
    T *h_output = (T*) malloc (output_size_bytes);
    T *h_bias = (T*) malloc (bias_size_bytes);
    T *h_gamma = (T*) malloc (gamma_size_bytes);
    T *h_beta = (T*) malloc (beta_size_bytes);

    for (int i = 0; i < input_size; i++) {
      h_input[i] = (T) dis(gen);
    }
    for (int i = 0; i < n; i++) {
      h_bias[i] = (T) dis(gen);
      h_gamma[i] = (T) dis(gen);
      h_beta[i] = (T) dis(gen);
    }

    float layernorm_eps = 1e-6;
    T *d_input, *d_output, *d_bias, *d_gamma, *d_beta;
    hipMalloc((void**)&d_input, input_size_bytes);
    hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_output,  output_size_bytes);
    hipMemset(d_output,  0, output_size_bytes); // need to reset output

    hipMalloc((void**)&d_bias,  bias_size_bytes);
    hipMemcpy(d_bias, h_bias, bias_size_bytes, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_gamma,  gamma_size_bytes);
    hipMemcpy(d_gamma, h_gamma, gamma_size_bytes, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_beta,  beta_size_bytes);
    hipMemcpy(d_beta, h_beta, beta_size_bytes, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++) {
      invokeAddBiasResidualLayerNorm(d_output,
                                     d_input,
                                     d_bias,
                                     d_gamma,
                                     d_beta,
                                     layernorm_eps,
                                     m,
                                     n);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("Average execution time of AddBiasResidualLayerNorm (%d x %d): %f (us)\n",
           m, n, (time * 1e-3f) / repeat);

    hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);

    float s = 0;
    for (int i = 0; i < output_size; i++)
      s += float(h_output[i]);

    printf("Checksum = %f\n", s / (n * n));
    
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_bias);
    hipFree(d_gamma);
    hipFree(d_beta);

    free(h_input);
    free(h_output);
    free(h_bias);
    free(h_gamma);
    free(h_beta);
  }
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }

  const int repeat = atoi(argv[1]);
  layer<half>(repeat);

  return 0;
}
