#include <chrono>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "kernels.h"

// thread block size
#define BLOCK_SIZE 256

template <typename T>
inline void memcpyH2D(T *d, const T *h, const int n) {
  hipMemcpy(d, h, n * sizeof(T), hipMemcpyHostToDevice);
}

template <typename T>
void test (const int repeat, const int numFloats) 
{
  // Initialize host data, with the first half the same as the second
  T *hostMem = (T*) malloc (sizeof(T) * numFloats);

  srand48(123);
  for (int j = 0; j < numFloats/2 ; ++j)
    hostMem[j] = hostMem[numFloats-j-1] = (T)(drand48()*10.0);

  T *deviceMem;
  hipMalloc((void**)&deviceMem, numFloats * sizeof(T));

  dim3 threads(BLOCK_SIZE);
  dim3 blocks((numFloats)/BLOCK_SIZE);

  // warmup
  for (int i = 0; i < 4; i++) {
    Add1<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
    Add2<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
    Add4<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
    Add8<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
    hipDeviceSynchronize();
  }

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,0);
  Add1<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
  hipDeviceSynchronize();
  auto k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,0);
  printf("kernel execution time (Add1): %f (s)\n", (k_time * 1e-9f));

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,1);
  Add2<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,1);
  printf("kernel execution time (Add2): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,2);
  Add4<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,2);
  printf("kernel execution time (Add4): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,3);
  Add8<T><<< blocks, threads >>>(deviceMem, repeat, 10.0);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,3);
  printf("kernel execution time (Add8): %f (s)\n", k_time * 1e-9f);

  // warmup
  for (int i = 0; i < 4; i++) {
    Mul1<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
    Mul2<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
    Mul4<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
    Mul8<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
    hipDeviceSynchronize();
  }

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,4);
  Mul1<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,4);
  printf("kernel execution time (Mul1): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,5);
  Mul2<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,5);
  printf("kernel execution time (Mul2): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,6);
  Mul4<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,6);
  printf("kernel execution time (Mul4): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,7);
  Mul8<T><<< blocks, threads >>>(deviceMem, repeat, 1.01);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,7);
  printf("kernel execution time (Mul8): %f (s)\n", k_time * 1e-9f);

  // warmup
  for (int i = 0; i < 4; i++) {
    MAdd1<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
    MAdd2<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
    MAdd4<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
    MAdd8<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
    hipDeviceSynchronize();
  }

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,8);
  MAdd1<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,8);
  printf("kernel execution time (MAdd1): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,9);
  MAdd2<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,9);
  printf("kernel execution time (MAdd2): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
MY_START_CLOCK(maxFlops-cuda main.cu,10);
  MAdd4<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
MY_STOP_CLOCK(maxFlops-cuda main.cu,10);
  printf("kernel execution time (MAdd4): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
  k_start = std::chrono::high_resolution_clock::now(); 
  MAdd8<T><<< blocks, threads >>>(deviceMem, repeat, 10.0, 0.9899);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
  k_time = std::chrono::duration_cast<std::chrono::nanoseconds>(k_end - k_start).count();
  printf("kernel execution time (MAdd8): %f (s)\n", k_time * 1e-9f);

  // warmup
  for (int i = 0; i < 4; i++) {
    MulMAdd1<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
    MulMAdd2<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
    MulMAdd4<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
    MulMAdd8<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
    hipDeviceSynchronize();
  }

  memcpyH2D(deviceMem, hostMem, numFloats);
  k_start = std::chrono::high_resolution_clock::now(); 
  MulMAdd1<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
  k_time = std::chrono::duration_cast<std::chrono::nanoseconds>(k_end - k_start).count();
  printf("kernel execution time (MulMAdd1): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
  k_start = std::chrono::high_resolution_clock::now(); 
  MulMAdd2<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
  k_time = std::chrono::duration_cast<std::chrono::nanoseconds>(k_end - k_start).count();
  printf("kernel execution time (MulMAdd2): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
  k_start = std::chrono::high_resolution_clock::now(); 
  MulMAdd4<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
  k_time = std::chrono::duration_cast<std::chrono::nanoseconds>(k_end - k_start).count();
  printf("kernel execution time (MulMAdd4): %f (s)\n", k_time * 1e-9f);

  memcpyH2D(deviceMem, hostMem, numFloats);
  k_start = std::chrono::high_resolution_clock::now(); 
  MulMAdd8<T><<< blocks, threads >>>(deviceMem, repeat, 3.75, 0.355);
  hipDeviceSynchronize();
  k_end = std::chrono::high_resolution_clock::now(); 
  k_time = std::chrono::duration_cast<std::chrono::nanoseconds>(k_end - k_start).count();
  printf("kernel execution time (MulMAdd8): %f (s)\n", k_time * 1e-9f);

  hipFree(deviceMem);
  free(hostMem);
}

int main(int argc, char* argv[]) 
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  // the number of loop iterations inside kernels
  const int repeat = atoi(argv[1]);

  // a multiple of BLOCK_SIZE
  const int numFloats = 2*1024*1024;

  printf("=== Single-precision floating-point kernels ===\n");
  test<float>(repeat, numFloats);

  // comment out when double-precision is not supported by a device
  printf("=== Double-precision floating-point kernels ===\n");
  test<double>(repeat, numFloats);

  return 0;
}
