#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "ecdh.h"

#define P_x 5
#define P_y 1
#define MODULUS 17
#define A 2

int main(int argc, char **argv)
{
  if (argc != 3) {
    printf("Usage: %s <positive number of keys> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_pk = atoi(argv[1]); 
  const int repeat = atoi(argv[2]); 

  // timing fast and slow versions for making public key
  struct timeval start_fast, end_fast, start_slow, end_slow;

  const int pk_x_size = num_pk * sizeof(int);
  const int pk_y_size = num_pk * sizeof(int);

  int *pk_slow_x = (int*) malloc (pk_x_size);
  int *pk_slow_y = (int*) malloc (pk_y_size);
  int *pk_fast_x = (int*) malloc (pk_x_size);
  int *pk_fast_y = (int*) malloc (pk_y_size);

  int *d_pk_x, *d_pk_y;
  hipMalloc((void**)&d_pk_x, pk_x_size);
  hipMalloc((void**)&d_pk_y, pk_y_size);
  
  dim3 grids = (num_pk + 255) / 256;

  gettimeofday(&start_slow,NULL);

  for (int i = 0; i < repeat; i++)
    k_slow <<<grids, 256>>> (18, P_x, P_y, d_pk_x, d_pk_y, MODULUS, A, num_pk);  

  hipDeviceSynchronize();
  gettimeofday(&end_slow,NULL);
  double elapsed_slow = (((end_slow.tv_sec*1000000.0 + end_slow.tv_usec) -
                        (start_slow.tv_sec*1000000.0 + start_slow.tv_usec)) / 1000000.00);

  printf("Average time (slow kernel): %f s\n", elapsed_slow / repeat);

  hipMemcpy(pk_slow_x, d_pk_x, pk_x_size, hipMemcpyDeviceToHost);
  hipMemcpy(pk_slow_y, d_pk_y, pk_y_size, hipMemcpyDeviceToHost);

  gettimeofday(&start_fast,NULL);

  for (int i = 0; i < repeat; i++)
    k_fast <<<grids, 256>>> (18, P_x, P_y, d_pk_x, d_pk_y, MODULUS, A, num_pk);  

  hipDeviceSynchronize();
  gettimeofday(&end_fast,NULL);
  double elapsed_fast = (((end_fast.tv_sec*1000000.0 + end_fast.tv_usec) -
                        (start_fast.tv_sec*1000000.0 + start_fast.tv_usec)) / 1000000.00);

  printf("Average time (fast kernel): %f s\n", elapsed_fast / repeat);

  hipMemcpy(pk_fast_x, d_pk_x, pk_x_size, hipMemcpyDeviceToHost);
  hipMemcpy(pk_fast_y, d_pk_y, pk_y_size, hipMemcpyDeviceToHost);


  bool fail_pk_x = memcmp(pk_slow_x, pk_fast_x, pk_x_size);
  bool fail_pk_y = memcmp(pk_slow_y, pk_fast_y, pk_x_size);
  printf("%s\n", (fail_pk_x || fail_pk_y) ? "FAIL" : "PASS");
  
  hipFree(d_pk_x);
  hipFree(d_pk_y);
  free(pk_slow_x);
  free(pk_slow_y);
  free(pk_fast_x);
  free(pk_fast_y);
  return 0;
}
