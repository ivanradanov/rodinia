//blackScholesAnalyticEngine.cu
//Scott Grauer-Gray
//Functions for running black scholes using the analytic engine (from Quantlib) on the GPU

#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
//needed for optionInputStruct
#include "blackScholesAnalyticEngineStructs.cuh"

//needed for the kernel(s) to run on the GPU
#include "blackScholesAnalyticEngineKernels.cu"

#include "blackScholesAnalyticEngineKernelsCpu.cu"

#define NUM_DIFF_SETTINGS 37

//function to run the black scholes analytic engine on the gpu
void runBlackScholesAnalyticEngine(const int repeat)
{
  int numberOfSamples = 50000000;
  {
    int numVals = numberOfSamples;//nSamplesArray[numTime];

    optionInputStruct* values = new optionInputStruct[numVals];

    for (int numOption = 0; numOption < numVals; numOption++)
    {
      if ((numOption % NUM_DIFF_SETTINGS) == 0)
      {
        optionInputStruct currVal = { CALL,  40.00,  42.00, 0.08, 0.04, 0.75, 0.35,  5.0975, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 1)
      {
        optionInputStruct currVal = { CALL, 100.00,  90.00, 0.10, 0.10, 0.10, 0.15,  0.0205, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 2)
      {
        optionInputStruct currVal = { CALL, 100.00, 100.00, 0.10, 0.10, 0.10, 0.15,  1.8734, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 3)
      {
        optionInputStruct currVal = { CALL, 100.00, 110.00, 0.10, 0.10, 0.10, 0.15,  9.9413, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 4)
      {
        optionInputStruct currVal = { CALL, 100.00,  90.00, 0.10, 0.10, 0.10, 0.25,  0.3150, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 5)
      {
        optionInputStruct currVal = { CALL, 100.00, 100.00, 0.10, 0.10, 0.10, 0.25,  3.1217, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 6)
      {
        optionInputStruct currVal = { CALL, 100.00, 110.00, 0.10, 0.10, 0.10, 0.25, 10.3556, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 7)
      {
        optionInputStruct currVal =  { CALL, 100.00,  90.00, 0.10, 0.10, 0.10, 0.35,  0.9474, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 8)
      {
        optionInputStruct currVal = { CALL, 100.00, 100.00, 0.10, 0.10, 0.10, 0.35,  4.3693, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 9)
      {
        optionInputStruct currVal = { CALL, 100.00, 110.00, 0.10, 0.10, 0.10, 0.35, 11.1381, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 10)
      {
        optionInputStruct currVal =  { CALL, 100.00,  90.00, 0.10, 0.10, 0.50, 0.15,  0.8069, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 11)
      {
        optionInputStruct currVal =  { CALL, 100.00, 100.00, 0.10, 0.10, 0.50, 0.15,  4.0232, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 12)
      {
        optionInputStruct currVal =  { CALL, 100.00, 110.00, 0.10, 0.10, 0.50, 0.15, 10.5769, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 13)
      {
        optionInputStruct currVal =   { CALL, 100.00,  90.00, 0.10, 0.10, 0.50, 0.25,  2.7026, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 14)
      {
        optionInputStruct currVal =   { CALL, 100.00, 100.00, 0.10, 0.10, 0.50, 0.25,  6.6997, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 15)
      {
        optionInputStruct currVal =   { CALL, 100.00, 110.00, 0.10, 0.10, 0.50, 0.25, 12.7857, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 16)
      {
        optionInputStruct currVal =   { CALL, 100.00,  90.00, 0.10, 0.10, 0.50, 0.35,  4.9329, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 17)
      {
        optionInputStruct currVal =  { CALL, 100.00, 100.00, 0.10, 0.10, 0.50, 0.35,  9.3679, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 18)
      {
        optionInputStruct currVal = { CALL, 100.00, 110.00, 0.10, 0.10, 0.50, 0.35, 15.3086, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 19)
      {
        optionInputStruct currVal =  { PUT,  100.00,  90.00, 0.10, 0.10, 0.10, 0.15,  9.9210, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 20)
      {
        optionInputStruct currVal =   { PUT,  100.00, 100.00, 0.10, 0.10, 0.10, 0.15,  1.8734, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 21)
      {
        optionInputStruct currVal =   { PUT,  100.00, 110.00, 0.10, 0.10, 0.10, 0.15,  0.0408, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 22)
      {
        optionInputStruct currVal =  { PUT,  100.00,  90.00, 0.10, 0.10, 0.10, 0.25, 10.2155, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 23)
      {
        optionInputStruct currVal =   { PUT,  100.00, 100.00, 0.10, 0.10, 0.10, 0.25,  3.1217, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 24)
      {
        optionInputStruct currVal =    { PUT,  100.00, 110.00, 0.10, 0.10, 0.10, 0.25,  0.4551, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 25)
      {
        optionInputStruct currVal =  { PUT,  100.00,  90.00, 0.10, 0.10, 0.10, 0.35, 10.8479, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 26)
      {
        optionInputStruct currVal =   { PUT,  100.00, 100.00, 0.10, 0.10, 0.10, 0.35,  4.3693, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 27)
      {
        optionInputStruct currVal =  { PUT,  100.00, 110.00, 0.10, 0.10, 0.10, 0.35,  1.2376, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 28)
      {
        optionInputStruct currVal =  { PUT,  100.00,  90.00, 0.10, 0.10, 0.50, 0.15, 10.3192, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 29)
      {
        optionInputStruct currVal =   { PUT,  100.00, 100.00, 0.10, 0.10, 0.50, 0.15,  4.0232, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 30)
      {
        optionInputStruct currVal =  { PUT,  100.00, 110.00, 0.10, 0.10, 0.50, 0.15,  1.0646, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 31)
      {
        optionInputStruct currVal =  { PUT,  100.00,  90.00, 0.10, 0.10, 0.50, 0.25, 12.2149, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 32)
      {
        optionInputStruct currVal =   { PUT,  100.00, 100.00, 0.10, 0.10, 0.50, 0.25,  6.6997, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 33)
      {
        optionInputStruct currVal =   { PUT,  100.00, 110.00, 0.10, 0.10, 0.50, 0.25,  3.2734, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 34)
      {
        optionInputStruct currVal =   { PUT,  100.00,  90.00, 0.10, 0.10, 0.50, 0.35, 14.4452, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 35)
      {
        optionInputStruct currVal =  { PUT,  100.00, 100.00, 0.10, 0.10, 0.50, 0.35,  9.3679, 1.0e-4};
        values[numOption] = currVal;
      }
      if ((numOption % NUM_DIFF_SETTINGS) == 36)
      {
        optionInputStruct currVal =   { PUT,  100.00, 110.00, 0.10, 0.10, 0.50, 0.35,  5.7963, 1.0e-4};
        values[numOption] = currVal;
      }
    }

    // Run GPU code

    //initialize the arrays

    //declare and allocate the input and output data on the CPU
    float* outputVals = (float*)malloc(numVals * sizeof(float));

    printf("Number of options: %d\n\n", numVals);
    long seconds, useconds, kseconds, kuseconds;
    float mtimeCpu, mtimeGpu, ktimeGpu;
    struct timeval start;
    gettimeofday(&start, NULL);

    //declare the data on the GPU
    optionInputStruct* optionsGpu;
    float* outputValsGpu;

    //allocate space for data on GPU
    hipMalloc((void**)&optionsGpu, numVals * sizeof(optionInputStruct));
    hipMalloc((void**)&outputValsGpu, numVals * sizeof(float));

    //copy the data from the CPU to the GPU
    hipMemcpy(optionsGpu, values, numVals * sizeof(optionInputStruct), hipMemcpyHostToDevice);

    // setup execution parameters
    dim3  grid((numVals + THREAD_BLOCK_SIZE - 1)/THREAD_BLOCK_SIZE, 1, 1);
    dim3  threads( THREAD_BLOCK_SIZE, 1, 1);

    struct timeval kstart;
    gettimeofday(&kstart, NULL);

    for (int i = 0; i < repeat; i++)
      getOutValOption <<< dim3(grid), dim3(threads) >>> (optionsGpu, outputValsGpu, numVals);

    hipDeviceSynchronize();
    struct timeval kend;
    gettimeofday(&kend, NULL);
    kseconds  = kend.tv_sec  - kstart.tv_sec;
    kuseconds = kend.tv_usec - kstart.tv_usec;
    ktimeGpu = ((kseconds) * 1000 + ((float)kuseconds)/1000.0) + 0.5f;

    //copy the resulting option values back to the CPU
    hipMemcpy(outputVals, outputValsGpu, numVals * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(optionsGpu);
    hipFree(outputValsGpu);

    struct timeval end;
    gettimeofday(&end, NULL);
    seconds  = end.tv_sec  - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;
    mtimeGpu = ((seconds) * 1000 + ((float)useconds)/1000.0) + 0.5f;

    printf("Run on GPU\n");
    printf("Average kernel execution time on GPU: %f (ms)\n", ktimeGpu / repeat);

    mtimeGpu -= ktimeGpu + ktimeGpu / repeat;
    printf("Processing time on GPU: %f (ms)\n", mtimeGpu);

    float totResult = 0.0f;
    for (int i=0; i<numVals; i++)
    {
      totResult += outputVals[i];
    }

    printf("Summation of output prices on GPU: %f\n", totResult);
    printf("Output price at index %d on GPU: %f\n\n", numVals/2, outputVals[numVals/2]);

    //run on CPU
    gettimeofday(&start, NULL);
    for (int numOption=0; numOption < numVals; numOption++)
    {
      getOutValOptionCpu(values, outputVals, numOption, numVals);  
    }
    gettimeofday(&end, NULL);
    seconds  = end.tv_sec  - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;

    mtimeCpu = ((seconds) * 1000 + ((float)useconds)/1000.0) + 0.5f;

    printf("Run on CPU\n");
    printf("Processing time on CPU: %f (ms)\n", mtimeCpu);

    totResult = 0.0f;
    for (int i=0; i<numVals; i++)
    {
      totResult += outputVals[i];
    }

    printf("Summation of output prices on CPU: %f\n", totResult);
    printf("Output price at index %d on CPU: %f\n\n", numVals/2, outputVals[numVals/2]);

    printf("Speedup on GPU: %f\n", mtimeCpu / mtimeGpu);

    delete [] values;
    free(outputVals);
  }
}

int main( int argc, char** argv) 
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }

  const int repeat = atoi(argv[1]);
  runBlackScholesAnalyticEngine(repeat);
  return 0;
}
