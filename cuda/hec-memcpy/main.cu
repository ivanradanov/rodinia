#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

#define NUM_SIZE 16

void setup(size_t *size) {
  for (int i = 0; i < NUM_SIZE; i++) {
    size[i] = 1 << (i + 6);
  }
}

void valSet(int* A, int val, size_t size) {
  size_t len = size / sizeof(int);
  for (size_t i = 0; i < len; i++) {
    A[i] = val;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    std::cerr << "Usage: " << argv[0] << " <repeat>\n";
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int *d_A;
  size_t size[NUM_SIZE];
  hipError_t err;

  setup(size);
  for (int i = 0; i < NUM_SIZE; i++) {
    int* A = (int*)malloc(size[i]);
    if (A == nullptr) {
      std::cerr << "Host memory allocation failed\n";
      return -1;
    }	
    valSet(A, 1, size[i]);

    err = hipMalloc((void**)&d_A, size[i]);
    if (err != hipSuccess) {
      std::cerr << "Device memory allocation failed\n";
      free(A);
      return -1;
    }

    // warmup
    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(d_A, A, size[i], hipMemcpyHostToDevice, 0);
    }
    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(d_A, A, size[i], hipMemcpyHostToDevice, 0);
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    std::cout << "Copy " << size[i] << " btyes from host to device takes " 
              << (time * 1e-3f) / repeat <<  " us" << std::endl;

    // warmup
    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(A, d_A, size[i], hipMemcpyDeviceToHost, 0);
    }
    hipDeviceSynchronize();

    start = std::chrono::steady_clock::now();

    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(A, d_A, size[i], hipMemcpyDeviceToHost, 0);
    }
    hipDeviceSynchronize();

    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    std::cout << "Copy " << size[i] << " btyes from device to host takes " 
              << (time * 1e-3f) / repeat <<  " us" << std::endl;

    hipFree(d_A);
    free(A);
  }
  return 0;
}
