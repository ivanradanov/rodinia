#include "hip/hip_runtime.h"
#include <unistd.h>  // access, F_OK
#include "sgd.h"

using namespace std;

SGDIndex* gen_random_map(SGDIndex size)
{
  srand(123);
  vector<SGDIndex> map(size, 0);
  for(SGDIndex i = 0; i < size; i++) map[i] = i;

  random_device rd;
  mt19937 g(rd());
  shuffle(map.begin(), map.end(), g);

  int*map_ptr = new int[size];
  for(int i = 0;i < size;i++)map_ptr[i] = map[i];

  return map_ptr;
}

SGDIndex* gen_inv_map(SGDIndex*map,int size)
{
  int*inv_map = new int[size];
  for(int i = 0;i < size;i++)inv_map[map[i]] = i;
  return inv_map;
}

struct sort_node_by_p
{
  bool operator() (mf_node const &lhs, mf_node const& rhs)
  {
    return tie(lhs.u, lhs.v) < tie(rhs.u, rhs.v);
  }
};

struct sort_node_by_q
{
  bool operator() (mf_node const &lhs, mf_node const &rhs)
  {
    return tie(lhs.v, lhs.u) < tie(rhs.v, rhs.u);
  }
};

void collect_data(mf_problem *prob, SGDRate& ave, SGDRate& std_dev)
{
  double ex = 0;
  double ex2 = 0;

  for(long long i = 0; i < prob->nnz; i++)
  {
    SGDRate r = prob->R[i].rate;
    ex += (double)r;
    ex2 += (double)r*r;
  }
  ex  = ex/(double)prob->nnz;
  ex2 = ex2/(double)prob->nnz;

  ave = (SGDRate)ex;
  std_dev = (SGDRate)sqrt(ex2-ex*ex);
}

void scale_problem(mf_problem*prob, float scale, long long u_seg, long long v_seg)
{
  if(prob->ux*prob->vy == 1)
  {
    for(long long i = 0;i < prob->nnz; i++)
    {   
      prob->R[i].rate = prob->R[i].rate*scale;
    }
  }
  else
  {
    for(long long i = 0;i < prob->nnz; i++)
    {   
      prob->R[i].rate = prob->R[i].rate*scale;

      long long tmp_u = prob->R[i].u;
      while(tmp_u >= u_seg)tmp_u = tmp_u - u_seg;
      prob->R[i].u = tmp_u;

      long long tmp_v = prob->R[i].v;
      while(tmp_v >= v_seg)tmp_v = tmp_v - v_seg;
      prob->R[i].v = tmp_v;
    }
  }
}

void shuffle_problem(mf_problem*prob, SGDIndex*p_map, SGDIndex*q_map)
{
  for(long long i = 0; i < prob->nnz; i++)
  {
    mf_node &N = prob->R[i];
    N.u = p_map[N.u];
    N.v = q_map[N.v];
  }
}

struct pthread_arg
{
  int thread_id; 
  string path;
  mf_node *R;
  long long offset;
  long long size;
  int max_m;
  int max_n;
};

void *read_problem_thread(void *argument)
{
  pthread_arg *arg = (pthread_arg*)argument;

  FILE*fptr = fopen(arg->path.c_str(), "rb");
  if(fptr == NULL)
  {
    printf("file %s open failed\n", arg->path.c_str());
    exit(0);
  }

  int max_m = -1;
  int max_n = -1;

  for(long long idx = 0;idx < arg->size;idx ++)
  {
    int flag = 0;
    int u,v;
    float r;

    flag += fread(&u, sizeof(int), 1, fptr); 
    flag += fread(&v, sizeof(int), 1, fptr); 
    flag += fread(&r, sizeof(float), 1, fptr); 

    if(flag != 3)break;

    if(u + 1 > max_m)max_m = u + 1;
    if(v + 1 > max_n)max_n = v + 1;

    arg->R[idx + arg->offset].u = u;
    arg->R[idx + arg->offset].v = v;
    arg->R[idx + arg->offset].rate = r;

  }
  fclose(fptr);

  arg->max_m = max_m;
  arg->max_n = max_n;
  return NULL;
}

mf_problem read_problem(string path)
{
  printf("read problem called\n");
  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  mf_problem prob;
  prob.m = 1;
  prob.n = 1;
  prob.nnz = 0;
  prob.R = NULL;

  int num_files = 0;
  vector<string> file_names;
  for(int i = 0; i < 80; i++)
  {
    stringstream tmp_name_stream;
    tmp_name_stream << path << i;
    string tmp_name = tmp_name_stream.str();

    if(access(tmp_name.c_str(), F_OK) != -1)file_names.push_back(tmp_name);
  }
  num_files = file_names.size();

  if(num_files <= 0)
  {
    if(path.empty())
    {
      printf("file %s open failed\n", path.c_str());
      exit(0);
      return prob;
    }

    FILE*fptr = fopen(path.c_str(), "rb");
    if(fptr == NULL)
    {
      printf("file %s open failed\n", path.c_str());
      exit(0);
      return prob;
    }
    fseek(fptr, 0L, SEEK_END);
    prob.nnz = ftell(fptr)/12;
    printf("prob.nnz = %lld\n", prob.nnz);

    mf_node *R;
    hipHostMalloc((void**)&R,sizeof(mf_node)*prob.nnz); 

    rewind(fptr);

    long long idx = 0;
    while(true)
    {
      int flag = 0;
      int u,v;
      float r;

      flag += fread(&u, sizeof(int), 1, fptr); 
      flag += fread(&v, sizeof(int), 1, fptr); 
      flag += fread(&r, sizeof(float), 1, fptr); 
      if(flag != 3)break;

      if(u + 1 > prob.m)prob.m = u + 1;
      if(v + 1 > prob.n)prob.n = v + 1;

      R[idx].u = u;
      R[idx].v = v;
      R[idx].rate = r;
      idx ++;
      //if(idx > 0 && idx%100000000 == 0)printf("progress: %%%.3f\n",100.0*idx/prob.nnz);
    }
    prob.R = R;

    fclose(fptr);

    printf("m:%d, n:%d, nnz:%lld\n",prob.m, prob.n, prob.nnz);
  }
  else
  {
    //data
    long long size_list[128];
    long long offset_list[128];
    pthread_t threads[128];
    pthread_arg pthread_arg_list[128];

    //get nnz & size_list
    FILE*fptrs[80];
    prob.nnz = 0;
    for(int i = 0;i < num_files;i++)
    {
      fptrs[i] = fopen(file_names[i].c_str(), "rb");
      fseek(fptrs[i], 0L, SEEK_END);
      size_list[i] = ftell(fptrs[i])/12;
      prob.nnz +=  size_list[i];
      fclose(fptrs[i]);
    }

    //get offset_list
    for(int i = 1;i < num_files;i++)
    {
      offset_list[i] = offset_list[i-1] + size_list[i-1];
    }

    //malloc
    mf_node *R;
    hipHostMalloc((void**)&R,sizeof(mf_node)*prob.nnz); 
    prob.R = R;

    //launch
    for(int i = 0;i < num_files; i++)
    {
      pthread_arg_list[i].thread_id = i;
      pthread_arg_list[i].path = file_names[i];
      pthread_arg_list[i].R = prob.R;
      pthread_arg_list[i].offset = offset_list[i];
      pthread_arg_list[i].size = size_list[i];
      pthread_create(&(threads[i]), NULL, read_problem_thread, (void*)(&(pthread_arg_list[i])));
    }

    for(int i = 0;i < num_files;i++)
    {
      pthread_join(threads[i], NULL);
    }
    prob.m = -1;
    prob.n = -1;
    for(int i = 0;i < num_files;i++)
    {
      if(pthread_arg_list[i].max_m >= prob.m) prob.m = pthread_arg_list[i].max_m;
      if(pthread_arg_list[i].max_n >= prob.n) prob.n = pthread_arg_list[i].max_n;
    }
    printf("m:%d, n:%d, nnz:%lld\n",prob.m, prob.n, prob.nnz);
  }

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);

  return prob;
}

void grid_problem(mf_problem* prob)
{
  printf("grid problem ...\n");

  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  //grid the problem into several grids
  long long u_seg, v_seg;
  if(prob->ux == 1)u_seg = prob->m;
  else u_seg = (long long)ceil((double)prob->m/prob->ux);
  if(prob->vy == 1)v_seg = prob->n;
  else v_seg = (long long)ceil((double)prob->n/prob->vy);

  prob->u_seg = u_seg;
  prob->v_seg = v_seg;

  auto get_grid_id = [=](int u, int v)
  {
    return ((u/u_seg)*prob->vy + v/v_seg);
  };

  //count the size of each grid
  prob->gridSize = new long long[prob->ux*prob->vy]();

  long long *gridSize = prob->gridSize;
  for(long long i = 0;i < prob->nnz;i++)
  {
    int tmp_u = prob->R[i].u;
    int tmp_v = prob->R[i].v;
    gridSize[get_grid_id(tmp_u, tmp_v)] ++;
  }

  long long max_grid_size = 0;
  for(int i = 0;i < prob->ux*prob->vy; i++)
  {
    //printf("gridSize[%d]:%lld\n",i,prob->gridSize[i]);
    if(max_grid_size < prob->gridSize[i])max_grid_size = prob->gridSize[i];
  }
  prob->maxGridSize = max_grid_size;

  //generate the pointer to each grid.
  mf_node**R2D = new mf_node*[prob->ux*prob->vy + 1];
  mf_node* R = prob->R;
  R2D[0] = R;
  for(int grid = 0;grid < prob->ux*prob->vy; grid++)R2D[grid + 1] = R2D[grid] + gridSize[grid];

  prob->R2D = R2D;

  //swap
  mf_node**pivots = new mf_node*[prob->ux*prob->vy];
  for(int i = 0;i < prob->ux*prob->vy; i++)pivots[i] = R2D[i];

  for(int grid = 0; grid < prob->ux*prob->vy; grid++)
  {
    for(mf_node*pivot = pivots[grid]; pivot != R2D[grid + 1];)
    {
      int corre_grid = get_grid_id(pivot->u, pivot->v);
      if(corre_grid == grid)
      {  
        pivot ++;
        continue;
      }
      mf_node *next = pivots[corre_grid];
      swap(*pivot, *next);
      pivots[corre_grid] ++;
    }
  }

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);
}

__device__
float LCG_random(unsigned int * seed) {
  const unsigned int m = 2147483648;
  const unsigned int a = 26757677;
  const unsigned int c = 1;
  *seed = (a * (*seed) + c) % m;
  return (float) (*seed) / (float) m;
}

__device__
void LCG_random_init(unsigned int * seed) {
  const unsigned int m = 2147483648;
  const unsigned int a = 26757677;
  const unsigned int c = 1;
  *seed = (a * (*seed) + c) % m;
}

__global__ void init_rand_state(unsigned int seed, unsigned int *state)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  state[i] = seed ^ i;
  LCG_random_init(state+i);
}

__global__ void random_init(
    unsigned int *__restrict__ state,
    int state_size,
    half *__restrict__ array,
    long long array_size,
    long long k, 
    float scale)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int state_id = tid % state_size;
  for(int i = 0;i < array_size;i += gridDim.x*blockDim.x)
  {
    int idx = i + tid;
    if(idx >= array_size) break;
    array[idx] = __float2half(LCG_random(state+state_id)*scale);
  }
}

void init_feature(short *feature_vec, int grid, long long seg, int k)
{
  float scale = (float)sqrt(1.0/k);

  half *gpu_vec;
  hipMalloc((void**)&gpu_vec, seg*k*sizeof(half));

  int state_size = (seg/256 + 1)*256;
  printf("state_size (a multiple of 256):%d\n", state_size);
  unsigned int* d_state;
  hipMalloc((void**)&d_state, sizeof(unsigned int)*state_size);

  init_rand_state<<<state_size/256, 256>>>(5551212, d_state);

  const int blockSize = 256;
  const int blockNum = (seg*k + 255)/256;
  printf("\tnumber of thread blocks:%d\n", blockNum);
  printf("\tarraysize:%lld\n", seg*k);

  for(int i = 0;i < grid; i++)
  {
    printf("grid:%d\n",i);
    random_init<<<blockNum, blockSize>>>(d_state, state_size, gpu_vec, seg*k, k, scale);
    hipMemcpy(feature_vec + i*seg*k,gpu_vec,sizeof(half)*seg*k, hipMemcpyDeviceToHost);
  }

  hipFree(d_state);
  hipFree(gpu_vec);
}

mf_model* init_model(mf_problem*prob, int k, float ave)
{
  printf("init model ...\n");
  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  mf_model *model = new mf_model;
  model->fun = 0;
  model->m = prob->m;
  model->n = prob->n;

  model->u_grid = prob->u_grid;
  model->v_grid = prob->v_grid;

  model->x_grid = prob->x_grid;
  model->y_grid = prob->y_grid;

  model->ux = prob->ux;
  model->vy = prob->vy;

  model->u_seg = prob->u_seg;
  model->v_seg = prob->v_seg;
  model->k = k;
  model->b = ave;

  //allocate memory
  hipHostMalloc((void**)&model->floatp, sizeof(float)*model->ux*model->u_seg*k);
  hipHostMalloc((void**)&model->floatq, sizeof(float)*model->vy*model->v_seg*k);

  hipHostMalloc((void**)&model->halfp, sizeof(short)*model->ux*model->u_seg*k);
  hipHostMalloc((void**)&model->halfq, sizeof(short)*model->vy*model->v_seg*k);

  gpuErr(hipPeekAtLastError());

  //random init
  init_feature(model->halfp, model->ux, model->u_seg, k);
  init_feature(model->halfq, model->vy, model->v_seg, k);

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);

  return model;
}

#include "sgd_k128_kernel_hogwild_warp32.h"

__global__ void init_rand_state(unsigned int seed, unsigned int *state, int size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  state[i] = seed ^ i;
  if(i < size) LCG_random_init(state+i);
}


__global__ void transform_half(
  const half *__restrict__ gpu_half_feature,
  float *__restrict__ gpu_float_feature,
  long long vec_size)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int number_threads = gridDim.x*blockDim.x;

  for(long long i = tid;i < vec_size;i += number_threads)
  {
    gpu_float_feature[i] = __half2float(gpu_half_feature[i]); 
  }
}

void transform_feature_vector(short *half_feature, float *float_feature,
                              int m, int grid, long long seg, int k)
{
  half *gpu_half_feature;
  float *gpu_float_feature;

  hipMalloc((void**)&gpu_half_feature, sizeof(half)*seg*k);
  hipMalloc((void**)&gpu_float_feature, sizeof(float)*seg*k);
  gpuErr(hipPeekAtLastError());

  for(int i = 0;i < grid;i++)
  {
    hipMemcpy(gpu_half_feature, half_feature + i*seg*k, sizeof(half)*seg*k, hipMemcpyHostToDevice);
    gpuErr(hipPeekAtLastError());

    int num_blocks = (seg*k+255)/256;
    if(num_blocks > 8*24)num_blocks = 8*24;

    transform_half<<<num_blocks,256>>>(gpu_half_feature, gpu_float_feature, seg*k);

    gpuErr(hipPeekAtLastError());
    hipMemcpy(float_feature + i*seg*k, gpu_float_feature, sizeof(float)*seg*k, hipMemcpyDeviceToHost);
    gpuErr(hipPeekAtLastError());
  }

  hipFree(gpu_half_feature);
  hipFree(gpu_float_feature);
  gpuErr(hipPeekAtLastError());
}

void sgd_update_k128(Parameter para, mf_model *model, mf_problem *prob, float scale)
{
  printf("sgd_update_k128 ...\n");

  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  //generate the random state for the hogwild scheduling policy.
  unsigned int *rand_state;
  hipMalloc((void**)&rand_state, sizeof(unsigned int)*para.num_workers);
  gpuErr(hipPeekAtLastError());

  init_rand_state<<<((para.num_workers+255)/256),256>>>(5551212, rand_state, para.num_workers);
  gpuErr(hipPeekAtLastError());

  //generate the dynamic learning rate
  float dynamic_rate[1024];
  float alpha = para.alpha;
  float beta  = para.beta;
  float lrate = para.lrate;

  for(int i = 0;i < (para.num_iters + 4);i++)
  {
    double tmp_rate = alpha/(1 + beta*pow(i, 1.5)) + lrate;
    dynamic_rate[i] = tmp_rate;
  }
  float *gpu_dynamic_rate;
  hipMalloc((void**)&gpu_dynamic_rate, sizeof(float)*1024);
  gpuErr(hipPeekAtLastError());
  hipMemcpy(gpu_dynamic_rate, dynamic_rate, sizeof(float)*1024, hipMemcpyHostToDevice);
  gpuErr(hipPeekAtLastError());

  //malloc a problem grid on GPU
  if(prob->x_grid*prob->y_grid == 1)
  {
    hipMalloc((void**)&(prob->gpuR), sizeof(mf_node)*prob->maxGridSize);
    prob->cur_u_id = -1;
    prob->cur_v_id = -1;
  }
  else
  {
    hipMalloc((void**)&(prob->gpuRptrs[0]), sizeof(mf_node)*prob->maxGridSize);
    hipMalloc((void**)&(prob->gpuRptrs[1]), sizeof(mf_node)*prob->maxGridSize);
    prob->cur_global_x_id[0] = -1;
    prob->cur_global_x_id[1] = -1;
    prob->cur_global_y_id[0] = -1;
    prob->cur_global_y_id[1] = -1;
  }

  //malloc feature vectors on GPU
  if(prob->x_grid*prob->y_grid == 1)
  {
    hipMalloc((void**)&model->gpuHalfp, sizeof(half)*model->u_seg*model->k);
    hipMalloc((void**)&model->gpuHalfq, sizeof(half)*model->v_seg*model->k);
    model->cur_u_id = -1;
    model->cur_v_id = -1;
  }
  else
  {
    hipMalloc((void**)&model->gpuHalfPptrs[0], sizeof(half)*model->u_seg*model->k);
    hipMalloc((void**)&model->gpuHalfPptrs[1], sizeof(half)*model->u_seg*model->k);
    hipMalloc((void**)&model->gpuHalfQptrs[0], sizeof(half)*model->v_seg*model->k);
    hipMalloc((void**)&model->gpuHalfQptrs[1], sizeof(half)*model->v_seg*model->k);

    model->cur_global_x_id[0] = -1;
    model->cur_global_x_id[1] = -1;
    model->cur_global_y_id[0] = -1;
    model->cur_global_y_id[1] = -1;
  }   

  //set update count
  int update_vector_size = 128;
  int *update_count_per_block = new int[prob->ux*prob->vy]();
  int max_update_count_per_block = -1;
  for(int cur_grid_id = 0;cur_grid_id < prob->ux*prob->vy; cur_grid_id ++)
  {
    update_count_per_block[cur_grid_id] = (ceil)(1.0*prob->gridSize[cur_grid_id]/(para.num_workers*update_vector_size));   
    if(max_update_count_per_block < update_count_per_block[cur_grid_id])
    {
      max_update_count_per_block = update_count_per_block[cur_grid_id];
    }
  }

  // random shuffle
  random_device rd;
  mt19937 g(rd());

  //run the update kernel
  if(prob->u_grid*prob->v_grid == 1)
  {
    hipMemcpy(prob->gpuR, prob->R2D[0], sizeof(mf_node)*prob->gridSize[0], hipMemcpyHostToDevice);
    hipMemcpy(model->gpuHalfp, model->halfp, sizeof(half)*model->u_seg*model->k, hipMemcpyHostToDevice);
    hipMemcpy(model->gpuHalfq, model->halfq, sizeof(half)*model->v_seg*model->k, hipMemcpyHostToDevice);

    sgd_k128_kernel_hogwild_warp32_lrate<<<para.num_workers/4,128>>>(
        prob->gpuR,
        prob->gridSize[0],
        model->gpuHalfp,
        model->gpuHalfq,
        rand_state,
        gpu_dynamic_rate,
        model->u_seg,
        model->v_seg,
        model->k,
        para.num_iters,
        0,
        max_update_count_per_block,
        update_count_per_block[0],
        update_vector_size,
        para.lambda_p,
        para.lambda_q,
        prob->u_grid,
        prob->v_grid,
        0,
        0);
    hipMemcpy(model->halfp, model->gpuHalfp, sizeof(half)*model->u_seg*model->k, hipMemcpyDeviceToHost);
    hipMemcpy(model->halfq, model->gpuHalfq, sizeof(half)*model->v_seg*model->k, hipMemcpyDeviceToHost);
  }
  else if(prob->x_grid*prob->y_grid == 1)
  {
    //random shuffle
    vector<int> u_id_vec(prob->u_grid, 0);
    vector<int> v_id_vec(prob->v_grid, 0);
    for(int i = 0;i < prob->u_grid;i++) u_id_vec[i] = i;
    for(int i = 0;i < prob->v_grid;i++) v_id_vec[i] = i;

    for(int iter = 0;iter < para.num_iters; iter ++)
    {
      shuffle(u_id_vec.begin(), u_id_vec.end(), g);
      for(int u_ite = 0;u_ite < prob->u_grid; u_ite ++)
      {

        shuffle(v_id_vec.begin(), v_id_vec.end(), g);
        for(int v_ite = 0;v_ite < prob->v_grid; v_ite ++)
        {
          int cur_u_id = u_id_vec[u_ite];
          int cur_v_id = v_id_vec[v_ite];

          int cur_grid_id = cur_u_id*prob->v_grid + cur_v_id;
          //transfer problem grid to gpu.
          if(prob->cur_u_id != cur_u_id || prob->cur_v_id != cur_v_id)
          {
            hipMemcpy(prob->gpuR, prob->R2D[cur_grid_id], sizeof(mf_node)*prob->gridSize[cur_grid_id], hipMemcpyHostToDevice);
          }
          gpuErr(hipPeekAtLastError());
          prob->cur_u_id = cur_u_id;
          prob->cur_v_id = cur_v_id;

          //transfer p grid to gpu
          if(model->cur_u_id == -1)
          {
            short *p_tmp = model->halfp + model->u_seg*model->k*cur_u_id; 
            hipMemcpy(model->gpuHalfp, p_tmp, sizeof(half)*model->u_seg*model->k, hipMemcpyHostToDevice);
            gpuErr(hipPeekAtLastError());
          }
          else if(model->cur_u_id != cur_u_id)
          {
            short *p_tmp = model->halfp + model->u_seg*model->k*model->cur_u_id;
            hipMemcpy(p_tmp, model->gpuHalfp, sizeof(half)*model->u_seg*model->k, hipMemcpyDeviceToHost);
            gpuErr(hipPeekAtLastError());

            p_tmp = model->halfp + model->u_seg*model->k*cur_u_id;
            hipMemcpy(model->gpuHalfp, p_tmp, sizeof(half)*model->u_seg*model->k, hipMemcpyHostToDevice);
            gpuErr(hipPeekAtLastError());
          }
          model->cur_u_id = cur_u_id;
          gpuErr(hipPeekAtLastError());

          //transfer q grid to gpu
          if(model->cur_v_id == -1)
          {
            short *q_tmp = model->halfq + model->v_seg*model->k*cur_v_id;
            hipMemcpy(model->gpuHalfq, q_tmp, sizeof(half)*model->v_seg*model->k, hipMemcpyHostToDevice);
            gpuErr(hipPeekAtLastError());
          }
          else if(model->cur_v_id != cur_v_id)
          {
            short *q_tmp = model->halfq + model->v_seg*model->k*model->cur_v_id;
            hipMemcpy(q_tmp, model->gpuHalfq, sizeof(half)*model->v_seg*model->k, hipMemcpyDeviceToHost);
            gpuErr(hipPeekAtLastError());

            q_tmp = model->halfq + model->v_seg*model->k*cur_v_id;
            hipMemcpy(model->gpuHalfq, q_tmp, sizeof(half)*model->v_seg*model->k, hipMemcpyHostToDevice);
            gpuErr(hipPeekAtLastError());
          }
          model->cur_v_id = cur_v_id;
          gpuErr(hipPeekAtLastError());

          //call the kernel
          sgd_k128_kernel_hogwild_warp32_lrate<<<para.num_workers/4,128>>>(
              prob->gpuR,
              prob->gridSize[cur_grid_id],
              model->gpuHalfp,
              model->gpuHalfq,
              rand_state,
              gpu_dynamic_rate,
              model->u_seg,
              model->v_seg,
              model->k,
              1,
              iter,
              max_update_count_per_block,
              update_count_per_block[cur_grid_id],
              update_vector_size,
              para.lambda_p,
              para.lambda_q,
              prob->u_grid,
              prob->v_grid,
              cur_u_id,
              cur_v_id);
          gpuErr(hipPeekAtLastError());
        }
      }
      hipDeviceSynchronize();

    }
    hipDeviceSynchronize();

    //printf("%d,%d\n", model->cur_u_id, model->cur_v_id);

    //transfer p back to CPU
    if(model->cur_u_id >= 0)
    {
      short *p_tmp = model->halfp + model->u_seg*model->k*model->cur_u_id;
      hipMemcpy(p_tmp, model->gpuHalfp, sizeof(half)*model->u_seg*model->k, hipMemcpyDeviceToHost);
      gpuErr(hipPeekAtLastError());
    }
    //transfer q back to CPU
    if(model->cur_v_id >= 0)
    {
      short *q_tmp = model->halfq + model->v_seg*model->k*model->cur_v_id;
      hipMemcpy(q_tmp, model->gpuHalfq, sizeof(half)*model->v_seg*model->k, hipMemcpyDeviceToHost);
      gpuErr(hipPeekAtLastError());
    }
  }
  else
  {
    //scheduling info
    int *global_x_list = new int[prob->x_grid*prob->y_grid];
    int *global_y_list = new int[prob->x_grid*prob->y_grid];
    int *global_id_list = new int[prob->x_grid*prob->y_grid];

    //create stream
    hipStream_t stream_com, stream_mem_d2h, stream_mem_h2d;
    hipStreamCreate(&stream_com);
    hipStreamCreate(&stream_mem_d2h);
    hipStreamCreate(&stream_mem_h2d);

    //random shuffle
    vector<int> u_id_vec(prob->u_grid, 0);
    vector<int> v_id_vec(prob->v_grid, 0);
    for(int i = 0;i < prob->u_grid;i++)u_id_vec[i] = i;
    for(int i = 0;i < prob->v_grid;i++)v_id_vec[i] = i;

    vector<int> x_id_vec(prob->x_grid, 0);
    vector<int> y_id_vec(prob->y_grid, 0);
    for(int i = 0;i < prob->x_grid;i++)x_id_vec[i] = i;
    for(int i = 0;i < prob->y_grid;i++)y_id_vec[i] = i;

    //fully random
    vector<int> uv_id_vec(prob->u_grid*prob->v_grid, 0);
    for(int i = 0;i < prob->u_grid*prob->v_grid; i++)uv_id_vec[i] = i;
    vector<int> xy_id_vec(prob->x_grid*prob->y_grid, 0);
    for(int i = 0;i < prob->x_grid*prob->y_grid; i++)xy_id_vec[i] = i;

    for(int iter = 0;iter < para.num_iters; iter ++)
    {
      shuffle(uv_id_vec.begin(), uv_id_vec.end(), g);
      shuffle(u_id_vec.begin(), u_id_vec.end(), g);

      for(int u_ite = 0;u_ite < prob->u_grid; u_ite ++)
      {
        shuffle(v_id_vec.begin(), v_id_vec.begin(), g);
        for(int v_ite = 0;v_ite < prob->v_grid; v_ite ++)
        {

          //fully random
          int tmp_uv_id = u_ite*prob->v_grid + v_ite;
          int cur_u_id = uv_id_vec[tmp_uv_id]/prob->v_grid;
          int cur_v_id = uv_id_vec[tmp_uv_id]%prob->v_grid;

          //set information
          shuffle(x_id_vec.begin(), x_id_vec.end(), g);
          shuffle(xy_id_vec.begin(), xy_id_vec.end(), g);

          for(int local_x_ite = 0;local_x_ite < prob->x_grid;local_x_ite ++)
          {
            shuffle(y_id_vec.begin(),y_id_vec.end(), g);
            for(int local_y_ite = 0;local_y_ite < prob->y_grid;local_y_ite ++)
            {

              //fully random
              int tmp_xy_id = local_x_ite*prob->y_grid + local_y_ite;
              int cur_x_id = xy_id_vec[tmp_xy_id]/prob->y_grid;
              int cur_y_id = xy_id_vec[tmp_xy_id]%prob->y_grid;

              int local_id = cur_x_id*prob->y_grid + cur_y_id;

              int global_x = cur_u_id*prob->x_grid + cur_x_id;
              int global_y = cur_v_id*prob->y_grid + cur_y_id;
              int global_id = global_x*prob->vy + global_y;

              global_x_list[local_id] = global_x;
              global_y_list[local_id] = global_y;
              global_id_list[local_id] = global_id;

            }
          }

          //run
          for(int i = -1;i < prob->x_grid*prob->y_grid;i++)
          {
            //compute
            if(i >= 0)
            {

              sgd_k128_kernel_hogwild_warp32_lrate<<<para.num_workers/4,128, 0, stream_com>>>(
                  prob->gpuRptrs[i%2],
                  prob->gridSize[global_id_list[i]],
                  model->gpuHalfPptrs[i%2],
                  model->gpuHalfQptrs[i%2],
                  rand_state,
                  gpu_dynamic_rate,
                  model->u_seg,
                  model->v_seg,
                  model->k,
                  1,
                  iter,
                  max_update_count_per_block,
                  update_count_per_block[global_id_list[i]],
                  update_vector_size,
                  para.lambda_p,
                  para.lambda_q,
                  prob->ux,
                  prob->vy,
                  global_x_list[i],
                  global_y_list[i]);
            }

            //memcpy for the next block
            if(i != (prob->x_grid*prob->y_grid - 1))
            {
              int next_global_x = global_x_list[i+1];
              int next_global_y = global_y_list[i+1];
              int next_global_id = global_id_list[i+1];

              //transfer problem grid to gpu
              if(prob->cur_global_x_id[(i+1)%2] !=  next_global_x || prob->cur_global_y_id[(i+1)%2] != next_global_y)
              {
                hipMemcpyAsync(prob->gpuRptrs[(i+1)%2], 
                    prob->R2D[next_global_id], 
                    sizeof(mf_node)*prob->gridSize[next_global_id],
                    hipMemcpyHostToDevice,
                    stream_mem_h2d);
              }

              //transfer feature p
              if(model->cur_global_x_id[(i+1)%2] == -1)
              {
                if(model->cur_global_x_id[(i+2)%2] == next_global_x)
                {
                  model->cur_global_x_id[(i+2)%2] = -1;
                  model->cur_global_x_id[(i+1)%2] = next_global_x;

                  half *tmp_ptr = model->gpuHalfPptrs[(i+1)%2];
                  model->gpuHalfPptrs[(i+1)%2] = model->gpuHalfPptrs[(i+2)%2];
                  model->gpuHalfPptrs[(i+2)%2] = tmp_ptr;
                }
                else
                {
                  short *p_tmp = model->halfp + model->u_seg*model->k*next_global_x;
                  hipMemcpyAsync(model->gpuHalfPptrs[(i+1)%2],
                      p_tmp,    
                      sizeof(half)*model->u_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);
                  model->cur_global_x_id[(i+1)%2] = next_global_x;
                }
              }
              else if(model->cur_global_x_id[(i+1)%2] != next_global_x)
              {
                if(model->cur_global_x_id[(i+2)%2] == -1)
                {
                  //swap value
                  int tmp = model->cur_global_x_id[(i+1)%2];
                  model->cur_global_x_id[(i+1)%2] = next_global_x;
                  model->cur_global_x_id[(i+2)%2] = tmp;

                  //swap pointer
                  half *p_tmp = model->gpuHalfPptrs[(i+1)%2];
                  model->gpuHalfPptrs[(i+1)%2] = model->gpuHalfPptrs[(i+2)%2];
                  model->gpuHalfPptrs[(i+2)%2] = p_tmp;

                  //transfer
                  short *p_tmp_trans = model->halfp + model->u_seg*model->k*next_global_x;
                  hipMemcpyAsync(model->gpuHalfPptrs[(i+1)%2],
                      p_tmp_trans,    
                      sizeof(half)*model->u_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);
                  model->cur_global_x_id[(i+1)%2] = next_global_x;
                }
                else if(model->cur_global_x_id[(i+2)%2] == next_global_x)
                {
                  //swap value
                  int tmp = model->cur_global_x_id[(i+1)%2];
                  model->cur_global_x_id[(i+1)%2] = next_global_x;
                  model->cur_global_x_id[(i+2)%2] = tmp;

                  //swap pointer
                  half *p_tmp = model->gpuHalfPptrs[(i+1)%2];
                  model->gpuHalfPptrs[(i+1)%2] = model->gpuHalfPptrs[(i+2)%2];
                  model->gpuHalfPptrs[(i+2)%2] = p_tmp;
                }
                else
                {
                  short *p_tmp = model->halfp + model->u_seg*model->k*model->cur_global_x_id[(i+1)%2];
                  hipMemcpyAsync(p_tmp,
                      model->gpuHalfPptrs[(i+1)%2],
                      sizeof(half)*model->u_seg*model->k,
                      hipMemcpyDeviceToHost,
                      stream_mem_d2h);

                  p_tmp = model->halfp + model->u_seg*model->k*next_global_x;
                  hipMemcpyAsync(model->gpuHalfPptrs[(i+1)%2],
                      p_tmp,
                      sizeof(half)*model->u_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);

                  model->cur_global_x_id[(i+1)%2] = next_global_x;
                }
              }

              //transfer feature q
              if(model->cur_global_y_id[(i+1)%2] == -1)
              {
                if(model->cur_global_y_id[(i+2)%2] == next_global_y)
                {
                  model->cur_global_y_id[(i+2)%2] = -1;
                  model->cur_global_y_id[(i+1)%2] = next_global_y;

                  half *tmp_ptr = model->gpuHalfQptrs[(i+1)%2];
                  model->gpuHalfQptrs[(i+1)%2] = model->gpuHalfQptrs[(i+2)%2];
                  model->gpuHalfQptrs[(i+2)%2] = tmp_ptr;
                }
                else
                {
                  short *q_tmp = model->halfq + model->v_seg*model->k*next_global_y;
                  hipMemcpyAsync(model->gpuHalfQptrs[(i+1)%2],
                      q_tmp,
                      sizeof(half)*model->v_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);
                  model->cur_global_y_id[(i+1)%2] = next_global_y;
                }
              }
              else if(model->cur_global_y_id[(i+1)%2] != next_global_y)
              {
                if(model->cur_global_y_id[(i+2)%2] == -1)
                {
                  //swap value
                  int tmp = model->cur_global_y_id[(i+1)%2];
                  model->cur_global_y_id[(i+1)%2] = model->cur_global_y_id[(i+2)%2];
                  model->cur_global_y_id[(i+2)%2] = tmp;

                  //swap pointer
                  half *q_tmp = model->gpuHalfQptrs[(i+1)%2];
                  model->gpuHalfQptrs[(i+1)%2] = model->gpuHalfQptrs[(i+2)%2];
                  model->gpuHalfQptrs[(i+2)%2] = q_tmp;

                  short *q_tmp_trans = model->halfq + model->v_seg*model->k*next_global_y;
                  hipMemcpyAsync(model->gpuHalfQptrs[(i+1)%2],
                      q_tmp_trans,
                      sizeof(half)*model->v_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);
                  model->cur_global_y_id[(i+1)%2] = next_global_y;
                }
                else if(model->cur_global_y_id[(i+2)%2] == next_global_y)
                {
                  //swap value
                  int tmp = model->cur_global_y_id[(i+1)%2];
                  model->cur_global_y_id[(i+1)%2] = model->cur_global_y_id[(i+2)%2];
                  model->cur_global_y_id[(i+2)%2] = tmp;

                  //swap pointer
                  half *q_tmp = model->gpuHalfQptrs[(i+1)%2];
                  model->gpuHalfQptrs[(i+1)%2] = model->gpuHalfQptrs[(i+2)%2];
                  model->gpuHalfQptrs[(i+2)%2] = q_tmp;
                }
                else
                {
                  short *q_tmp = model->halfq + model->v_seg*model->k*model->cur_global_y_id[(i+1)%2];
                  hipMemcpyAsync(q_tmp,
                      model->gpuHalfQptrs[(i+1)%2],
                      sizeof(half)*model->v_seg*model->k,
                      hipMemcpyDeviceToHost,
                      stream_mem_d2h);

                  q_tmp = model->halfq + model->v_seg*model->k*next_global_y;
                  hipMemcpyAsync(model->gpuHalfQptrs[(i+1)%2],
                      q_tmp,
                      sizeof(half)*model->v_seg*model->k,
                      hipMemcpyHostToDevice,
                      stream_mem_h2d);
                  model->cur_global_y_id[(i+1)%2] = next_global_y;
                }
              }
            }
            hipDeviceSynchronize();
          }   
        }
      }
      hipDeviceSynchronize();
    }
    hipDeviceSynchronize();

    //transfer p back
    if(model->cur_global_x_id[0] != -1)
    {
      short *p_tmp = model->halfp + model->u_seg*model->k*model->cur_global_x_id[0];
      hipMemcpy(p_tmp, model->gpuHalfPptrs[0], sizeof(half)*model->u_seg*model->k, hipMemcpyDeviceToHost);
    }
    if(model->cur_global_x_id[1] != -1)
    {
      short *p_tmp = model->halfp + model->u_seg*model->k*model->cur_global_x_id[1];
      hipMemcpy(p_tmp, model->gpuHalfPptrs[1], sizeof(half)*model->u_seg*model->k, hipMemcpyDeviceToHost);
    }

    //transfer q back
    if(model->cur_global_y_id[0] != -1)
    {
      short *q_tmp = model->halfq + model->v_seg*model->k*model->cur_global_y_id[0];
      hipMemcpy(q_tmp, model->gpuHalfQptrs[0], sizeof(half)*model->v_seg*model->k, hipMemcpyDeviceToHost);
    }
    if(model->cur_global_y_id[1] != -1)
    {
      short *q_tmp = model->halfq + model->v_seg*model->k*model->cur_global_y_id[1];
      hipMemcpy(q_tmp, model->gpuHalfQptrs[1], sizeof(half)*model->v_seg*model->k, hipMemcpyDeviceToHost);
    }
  }   

  if(prob->x_grid*prob->y_grid == 1)
  {
    hipFree(model->gpuHalfp);
    hipFree(model->gpuHalfq);
    hipFree(prob->gpuR);
  }
  else
  {
    hipFree(model->gpuHalfPptrs[0]);
    hipFree(model->gpuHalfPptrs[1]);
    hipFree(model->gpuHalfQptrs[0]);
    hipFree(model->gpuHalfQptrs[1]);
    hipFree(prob->gpuRptrs[0]);
    hipFree(prob->gpuRptrs[1]);
  }

  gpuErr(hipPeekAtLastError());

  //transform halfp & halfq to floatp & floatq.
  hipDeviceSynchronize();
  transform_feature_vector(model->halfp, model->floatp, model->m, model->ux, model->u_seg, model->k);
  transform_feature_vector(model->halfq, model->floatq, model->n, model->vy, model->v_seg, model->k);

  hipFree(gpu_dynamic_rate);
  hipFree(rand_state);

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);
}

void scale_model(mf_model *model, float scale)
{
  printf("scale model ...\n");

  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  float factor_scale = sqrt(scale);
  for(long long i = 0; i < ((long long)model->m)*model->k; i++)model->floatp[i] = model->floatp[i]*factor_scale;


  for(long long i = 0; i < model->n*model->k; i++)model->floatq[i] = model->floatq[i]*factor_scale;

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);
}


void shuffle_model(mf_model *model, int* inv_p_map, int* inv_q_map)
{
  printf("shuffle model ...\n");

  struct timespec begin, end;
  double elapsed;
  clock_gettime(CLOCK_MONOTONIC, &begin);

  auto inv_shuffle1 = [] (float *vec, int *map, int size, int k)
  {
    for(int pivot = 0; pivot < size;)
    {
      if(pivot == map[pivot])
      {
        ++pivot;
        continue;
      }

      int next = map[pivot];

      for(SGDIndex d = 0; d < k; d++)swap(*(vec + (long long)pivot*k+d), *(vec+(long long)next*k+d));

      map[pivot] = map[next];
      map[next] = next;
    }
  };

  inv_shuffle1(model->floatp, inv_p_map, model->m, model->k);
  inv_shuffle1(model->floatq, inv_q_map, model->n, model->k);

  clock_gettime(CLOCK_MONOTONIC, &end);
  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("time elapsed:%.8fs\n\n\n",elapsed);
}

//the core computation function
mf_model*sgd_train(mf_problem*tr, mf_problem*te, Parameter para)
{
  printf("sgd_train called\n");

  //collect the factor. scaling is used to make sure every rating is around 1.
  SGDRate ave;
  SGDRate std_dev;
  SGDRate scale = 1.0;

  collect_data(tr, ave, std_dev);
  scale = max((SGDRate)1e-4, std_dev);

  //shuffle the u & v randomly to: 1) increase randomness. 2) block balance.
  int* p_map = gen_random_map(tr->m);
  int* q_map = gen_random_map(tr->n);
  int* inv_p_map = gen_inv_map(p_map, tr->m);
  int* inv_q_map = gen_inv_map(q_map, tr->n);

  shuffle_problem(tr, p_map, q_map);

  grid_problem(tr); 

  //scale problem
  scale_problem(tr, 1.0/scale, tr->u_seg, tr->v_seg);
  para.lambda_p = para.lambda_p/scale;
  para.lambda_q = para.lambda_q/scale;

  //init model
  mf_model*model = init_model(tr, para.k, ave/std_dev);

  //train
  sgd_update_k128(para, model, tr, scale);

  //scale model
  scale_model(model, scale);

  //shuffle model
  shuffle_model(model, inv_p_map, inv_q_map);

  return model;
}
