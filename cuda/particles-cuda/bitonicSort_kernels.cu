#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

__device__
inline void ComparatorPrivate(
    unsigned int *keyA,
    unsigned int *valA,
    unsigned int *keyB,
    unsigned int *valB,
    unsigned int dir)
{
  if( (*keyA > *keyB) == dir ){
    unsigned int t;
    t = *keyA; *keyA = *keyB; *keyB = t;
    t = *valA; *valA = *valB; *valB = t;
  }
}

__device__
inline void ComparatorLocal(
    unsigned int* keyA,
    unsigned int* valA,
    unsigned int* keyB,
    unsigned int* valB,
    const unsigned int dir)
{
  if( (*keyA > *keyB) == dir ){
    unsigned int t;
    t = *keyA; *keyA = *keyB; *keyB = t;
    t = *valA; *valA = *valB; *valB = t;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into local memory
////////////////////////////////////////////////////////////////////////////////
__global__ void bitonicSortLocal(
    unsigned int*__restrict__ d_DstKey,
    unsigned int*__restrict__ d_DstVal,
    const unsigned int*__restrict__ d_SrcKey,
    const unsigned int*__restrict__ d_SrcVal,
    const unsigned int arrayLength,
    const unsigned int dir)
{
  __shared__  unsigned int l_key[LOCAL_SIZE_LIMIT];
  __shared__  unsigned int l_val[LOCAL_SIZE_LIMIT];

  //Offset to the beginning of subbatch and load data
  d_SrcKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  l_key[threadIdx.x +                      0] = d_SrcKey[                     0];
  l_val[threadIdx.x +                      0] = d_SrcVal[                     0];
  l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcKey[(LOCAL_SIZE_LIMIT / 2)];
  l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcVal[(LOCAL_SIZE_LIMIT / 2)];

  for(unsigned int size = 2; size < arrayLength; size <<= 1){
    //Bitonic merge
    unsigned int ddd = dir ^ ( (threadIdx.x & (size / 2)) != 0 );
    for(unsigned int stride = size / 2; stride > 0; stride >>= 1){
      __syncthreads();
      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      ComparatorLocal(
          &l_key[pos +      0], &l_val[pos +      0],
          &l_key[pos + stride], &l_val[pos + stride],
          ddd);
    }
  }

  //ddd == dir for the last bitonic merge step
  {
    for(unsigned int stride = arrayLength / 2; stride > 0; stride >>= 1){
      __syncthreads();
      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      ComparatorLocal(
          &l_key[pos +      0], &l_val[pos +      0],
          &l_key[pos + stride], &l_val[pos + stride],
          dir);
    }
  }

  __syncthreads();
  d_DstKey[                     0] = l_key[threadIdx.x +                      0];
  d_DstVal[                     0] = l_val[threadIdx.x +                      0];
  d_DstKey[(LOCAL_SIZE_LIMIT / 2)] = l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
  d_DstVal[(LOCAL_SIZE_LIMIT / 2)] = l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
}

////////////////////////////////////////////////////////////////////////////////
// Bitonic sort kernel for large arrays (not fitting into local memory)
////////////////////////////////////////////////////////////////////////////////
//Bottom-level bitonic sort
//Almost the same as bitonicSortLocal with the only exception
//of even / odd subarrays (of LOCAL_SIZE_LIMIT points) being
//sorted in opposite directions
__global__ void bitonicSortLocal1(
    unsigned int*__restrict__ d_DstKey,
    unsigned int*__restrict__ d_DstVal,
    const unsigned int*__restrict__ d_SrcKey,
    const unsigned int*__restrict__ d_SrcVal)
{
  __shared__ unsigned int l_key[LOCAL_SIZE_LIMIT];
  __shared__ unsigned int l_val[LOCAL_SIZE_LIMIT];

  //Offset to the beginning of subarray and load data
  d_SrcKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  l_key[threadIdx.x +                      0] = d_SrcKey[                     0];
  l_val[threadIdx.x +                      0] = d_SrcVal[                     0];
  l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcKey[(LOCAL_SIZE_LIMIT / 2)];
  l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcVal[(LOCAL_SIZE_LIMIT / 2)];

  unsigned int comparatorI = (blockIdx.x * blockDim.x + threadIdx.x) & ((LOCAL_SIZE_LIMIT / 2) - 1);

  for(unsigned int size = 2; size < LOCAL_SIZE_LIMIT; size <<= 1){
    //Bitonic merge
    unsigned int ddd = (comparatorI & (size / 2)) != 0;
    for(unsigned int stride = size / 2; stride > 0; stride >>= 1){
      __syncthreads();
      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      ComparatorLocal(
          &l_key[pos +      0], &l_val[pos +      0],
          &l_key[pos + stride], &l_val[pos + stride],
          ddd
               );
    }
  }

  //Odd / even arrays of LOCAL_SIZE_LIMIT elements
  //sorted in opposite directions
  {
    unsigned int ddd = (blockIdx.x & 1);
    for(unsigned int stride = LOCAL_SIZE_LIMIT / 2; stride > 0; stride >>= 1){
      __syncthreads();
      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      ComparatorLocal(
          &l_key[pos +      0], &l_val[pos +      0],
          &l_key[pos + stride], &l_val[pos + stride],
          ddd
               );
    }
  }

  __syncthreads();
  d_DstKey[                     0] = l_key[threadIdx.x +                      0];
  d_DstVal[                     0] = l_val[threadIdx.x +                      0];
  d_DstKey[(LOCAL_SIZE_LIMIT / 2)] = l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
  d_DstVal[(LOCAL_SIZE_LIMIT / 2)] = l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
}

//Bitonic merge iteration for 'stride' >= LOCAL_SIZE_LIMIT
__global__ void bitonicMergeGlobal(
    unsigned int*__restrict__ d_DstKey,
    unsigned int*__restrict__ d_DstVal,
    const unsigned int*__restrict__ d_SrcKey,
    const unsigned int*__restrict__ d_SrcVal,
    const unsigned int arrayLength,
    const unsigned int size,
    const unsigned int stride,
    const unsigned int dir)
{
  unsigned int global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int        comparatorI = global_comparatorI & (arrayLength / 2 - 1);

  //Bitonic merge
  unsigned int ddd = dir ^ ( (comparatorI & (size / 2)) != 0 );
  unsigned int pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

  unsigned int keyA = d_SrcKey[pos +      0];
  unsigned int valA = d_SrcVal[pos +      0];
  unsigned int keyB = d_SrcKey[pos + stride];
  unsigned int valB = d_SrcVal[pos + stride];

  ComparatorPrivate(
      &keyA, &valA,
      &keyB, &valB,
      ddd);

  d_DstKey[pos +      0] = keyA;
  d_DstVal[pos +      0] = valA;
  d_DstKey[pos + stride] = keyB;
  d_DstVal[pos + stride] = valB;
}

//Combined bitonic merge steps for
//'size' > LOCAL_SIZE_LIMIT and 'stride' = [1 .. LOCAL_SIZE_LIMIT / 2]
__global__ void bitonicMergeLocal(
    unsigned int*__restrict__ d_DstKey,
    unsigned int*__restrict__ d_DstVal,
    const unsigned int*__restrict__ d_SrcKey,
    const unsigned int*__restrict__ d_SrcVal,
    const unsigned int arrayLength,
    const unsigned int size,
    unsigned int stride,
    const unsigned int dir)
{
  __shared__ unsigned int l_key[LOCAL_SIZE_LIMIT];
  __shared__ unsigned int l_val[LOCAL_SIZE_LIMIT];

  d_SrcKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * LOCAL_SIZE_LIMIT + threadIdx.x;
  l_key[threadIdx.x +                      0] = d_SrcKey[                     0];
  l_val[threadIdx.x +                      0] = d_SrcVal[                     0];
  l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcKey[(LOCAL_SIZE_LIMIT / 2)];
  l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)] = d_SrcVal[(LOCAL_SIZE_LIMIT / 2)];

  //Bitonic merge
  unsigned int comparatorI = (blockIdx.x * blockDim.x + threadIdx.x) & ((arrayLength / 2) - 1);
  unsigned int         ddd = dir ^ ( (comparatorI & (size / 2)) != 0 );
  for(; stride > 0; stride >>= 1){
    __syncthreads();
    unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    ComparatorLocal(
        &l_key[pos +      0], &l_val[pos +      0],
        &l_key[pos + stride], &l_val[pos + stride],
        ddd);
  }

  __syncthreads();
  d_DstKey[                     0] = l_key[threadIdx.x +                      0];
  d_DstVal[                     0] = l_val[threadIdx.x +                      0];
  d_DstKey[(LOCAL_SIZE_LIMIT / 2)] = l_key[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
  d_DstVal[(LOCAL_SIZE_LIMIT / 2)] = l_val[threadIdx.x + (LOCAL_SIZE_LIMIT / 2)];
}
