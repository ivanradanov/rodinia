/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

#define LOCAL_SIZE_LIMIT 512U

#include "bitonicSort_kernels.cu"

static unsigned int factorRadix2(unsigned int& log2L, unsigned int L){
  if(!L){
    log2L = 0;
    return 0;
  }else{
    for(log2L = 0; (L & 1) == 0; L >>= 1, log2L++);
    return L;
  }
}

void bitonicSort(
    unsigned int *d_dstKey,
    unsigned int *d_dstVal,
    unsigned int *d_srcKey,
    unsigned int *d_srcVal,
    unsigned int batch,
    unsigned int arrayLength,
    unsigned int dir)
{
  if(arrayLength < 2) return;

  //Only power-of-two array lengths are supported so far
  unsigned int log2L;
  unsigned int factorizationRemainder = factorRadix2(log2L, arrayLength);
  assert(factorizationRemainder == 1);

  dir = (dir != 0);

  size_t localWorkSize, globalWorkSize;

  if(arrayLength <= LOCAL_SIZE_LIMIT)
  {
    assert( (batch * arrayLength) % LOCAL_SIZE_LIMIT == 0 );

    //Launch bitonicSortLocal
    localWorkSize  = LOCAL_SIZE_LIMIT / 2;
    globalWorkSize = batch * arrayLength / 2;
    dim3 bs_gws (globalWorkSize/localWorkSize);
    dim3 bs_lws (localWorkSize);

    bitonicSortLocal<<<bs_gws, bs_lws>>>(
        d_dstKey,  
        d_dstVal,  
        d_srcKey,  
        d_srcVal,  
        arrayLength,
        dir);
  }
  else
  {
    //Launch bitonicSortLocal1
    localWorkSize  = LOCAL_SIZE_LIMIT / 2;
    globalWorkSize = batch * arrayLength / 2;
    dim3 bs1_gws (globalWorkSize/localWorkSize);
    dim3 bs1_lws (localWorkSize);
    bitonicSortLocal1<<<bs1_gws, bs1_lws>>>(
        d_dstKey,
        d_dstVal,
        d_srcKey,
        d_srcVal);

    for(unsigned int size = 2 * LOCAL_SIZE_LIMIT; size <= arrayLength; size <<= 1)
    {
      for(unsigned stride = size / 2; stride > 0; stride >>= 1)
      {
        if(stride >= LOCAL_SIZE_LIMIT)
        {
          //Launch bitonicMergeGlobal
          localWorkSize  = LOCAL_SIZE_LIMIT / 4;
          globalWorkSize = batch * arrayLength / 2;
          dim3 bmg_gws (globalWorkSize/localWorkSize);
          dim3 bmg_lws (localWorkSize);

          bitonicMergeGlobal<<<bmg_gws, bmg_lws>>>(
              d_dstKey,
              d_dstVal,
              d_dstKey,
              d_dstVal,
              arrayLength,
              size,
              stride,
              dir);
        }
        else
        {
          //Launch bitonicMergeLocal
          localWorkSize  = LOCAL_SIZE_LIMIT / 2;
          globalWorkSize = batch * arrayLength / 2;

          dim3 bml_gws (globalWorkSize/localWorkSize);
          dim3 bml_lws (localWorkSize);

          assert(stride < LOCAL_SIZE_LIMIT);
          bitonicMergeLocal<<<bml_gws, bml_lws>>>(
              d_dstKey,
              d_dstVal,
              d_dstKey,
              d_dstVal,
              arrayLength,
              size,
              stride,
              dir);
          break;
        }
      }
    }
  }
}
