#include "hip/hip_runtime.h"
//
// An implementation of Parallel Marching Blocks algorithm
//

#include <cstdio>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include "tables.h"

// problem size
constexpr unsigned int N(1024);
constexpr unsigned int Nd2(N / 2);
constexpr unsigned int voxelXLv1(16);
constexpr unsigned int voxelYLv1(16);
constexpr unsigned int voxelZLv1(64);
constexpr unsigned int gridXLv1((N - 1) / (voxelXLv1 - 1));
constexpr unsigned int gridYLv1((N - 1) / (voxelYLv1 - 1));
constexpr unsigned int gridZLv1((N - 1) / (voxelZLv1 - 1));
constexpr unsigned int countingThreadNumLv1(128);
constexpr unsigned int blockNum(gridXLv1* gridYLv1* gridZLv1);
constexpr unsigned int countingBlockNumLv1(blockNum / countingThreadNumLv1);

constexpr unsigned int voxelXLv2(4);
constexpr unsigned int voxelYLv2(4);
constexpr unsigned int voxelZLv2(8);
constexpr unsigned int blockXLv2(5);
constexpr unsigned int blockYLv2(5);
constexpr unsigned int blockZLv2(9);
constexpr unsigned int voxelNumLv2(blockXLv2* blockYLv2* blockZLv2);

constexpr unsigned int countingThreadNumLv2(1024);
constexpr unsigned int gridXLv2(gridXLv1* blockXLv2);
constexpr unsigned int gridYLv2(gridYLv1* blockYLv2);
//constexpr unsigned int gridZLv2(gridZLv1* blockZLv2);

__inline__ __device__ float f(unsigned int x, unsigned int y, unsigned int z)
{
  constexpr float d(2.0f / N);
  float xf((int(x - Nd2)) * d);//[-1, 1)
  float yf((int(z - Nd2)) * d);
  float zf((int(z - Nd2)) * d);
  return 1.f - 16.f * xf * yf * zf - 4.f * (xf * xf + yf * yf + zf * zf);
}

__inline__ __device__ float zeroPoint(unsigned int x, float v0, float v1, float isoValue)
{
  return ((x * (v1 - isoValue) + (x + 1) * (isoValue - v0)) / (v1 - v0) - Nd2) * (2.0f / N);
}

__inline__ __device__ float transformToCoord(unsigned int x)
{
  return (int(x) - int(Nd2)) * (2.0f / N);
}

__global__ void computeMinMaxLv1(float*__restrict__ minMax)
{
  __shared__ float sminMax[64];
  constexpr unsigned int threadNum(voxelXLv1 * voxelYLv1);
  constexpr unsigned int warpNum(threadNum / 32);
  unsigned int x(blockIdx.x * (voxelXLv1 - 1) + threadIdx.x);
  unsigned int y(blockIdx.y * (voxelYLv1 - 1) + threadIdx.y);
  unsigned int z(blockIdx.z * (voxelZLv1 - 1));
  unsigned int tid(threadIdx.x + voxelXLv1 * threadIdx.y);
  unsigned int laneid = tid % 32;
  unsigned int blockid(blockIdx.x + gridXLv1 * (blockIdx.y + gridYLv1 * blockIdx.z));
  unsigned int warpid(tid >> 5);
  float v(f(x, y, z));
  float minV(v), maxV(v);
  for (int c0(1); c0 < voxelZLv1; ++c0)
  {
    v = f(x, y, z + c0);
    if (v < minV)minV = v;
    if (v > maxV)maxV = v;
  }
#pragma unroll
  for (int c0(16); c0 > 0; c0 /= 2)
  {
    float t0, t1;
    t0 = __shfl_down_sync(0xffffffffu, minV, c0);
    t1 = __shfl_down_sync(0xffffffffu, maxV, c0);
    if (t0 < minV)minV = t0;
    if (t1 > maxV)maxV = t1;
  }
  if (laneid == 0)
  {
    sminMax[warpid] = minV;
    sminMax[warpid + warpNum] = maxV;
  }
  __syncthreads();
  if (warpid == 0)
  {
    minV = sminMax[laneid];
    maxV = sminMax[laneid + warpNum];
#pragma unroll
    for (int c0(warpNum / 2); c0 > 0; c0 /= 2)
    {
      float t0, t1;
      t0 = __shfl_down_sync(0xffffffffu, minV, c0);
      t1 = __shfl_down_sync(0xffffffffu, maxV, c0);
      if (t0 < minV)minV = t0;
      if (t1 > maxV)maxV = t1;
    }
    if (laneid == 0)
    {
      minMax[blockid * 2] = minV;
      minMax[blockid * 2 + 1] = maxV;
    }
  }
}

__global__ void compactLv1(
  float isoValue, 
  const float*__restrict__ minMax,
  unsigned int*__restrict__ blockIndices,
  unsigned int*__restrict__ countedBlockNum)
{
  __shared__ unsigned int sums[32];
  constexpr unsigned int warpNum(countingThreadNumLv1 / 32);
  unsigned int tid(threadIdx.x);
  unsigned int laneid = tid % 32;
  unsigned int bIdx(blockIdx.x * countingThreadNumLv1 + tid);
  unsigned int warpid(tid >> 5);
  unsigned int test;
  if (minMax[2 * bIdx] <= isoValue && minMax[2 * bIdx + 1] >= isoValue)test = 1;
  else test = 0;
  unsigned int testSum(test);
#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp(__shfl_up_sync(0xffffffffu, testSum, c0));
    if (laneid >= c0)testSum += tp;
  }
  if (laneid == 31)sums[warpid] = testSum;
  __syncthreads();
  if (warpid == 0)
  {
    unsigned int warpSum = sums[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp(__shfl_up_sync(0xffffffffu, warpSum, c0));
      if (laneid >= c0) warpSum += tp;
    }
    sums[laneid] = warpSum;
  }
  __syncthreads();
  if (warpid != 0)testSum += sums[warpid - 1];
  if (tid == countingThreadNumLv1 - 1 && testSum != 0)
    sums[31] = atomicAdd(countedBlockNum, testSum);
  __syncthreads();
  if (test)blockIndices[testSum + sums[31] - 1] = bIdx;
}

__global__ void computeMinMaxLv2(
  const unsigned int*__restrict__ blockIndicesLv1,
  float*__restrict__ minMax)
{
  unsigned int tid(threadIdx.x);
  unsigned int voxelOffset(threadIdx.y);
  unsigned int blockIndex(blockIndicesLv1[blockIdx.x]);
  unsigned int tp(blockIndex);
  unsigned int x((blockIndex % gridXLv1) * (voxelXLv1 - 1) + (voxelOffset % 5) * (voxelXLv2 - 1) + (tid & 3));
  tp /= gridXLv1;
  unsigned int y((tp % gridYLv1) * (voxelYLv1 - 1) + (voxelOffset / 5) * (voxelYLv2 - 1) + (tid >> 2));
  tp /= gridYLv1;
  unsigned int z(tp * (voxelZLv1 - 1));
  float v(f(x, y, z));
  float minV(v), maxV(v);
  unsigned int idx(2 * (voxelOffset + voxelNumLv2 * blockIdx.x));
  for (int c0(0); c0 < blockZLv2; ++c0)
  {
    for (int c1(1); c1 < voxelZLv2; ++c1)
    {
      v = f(x, y, z + c1);
      if (v < minV)minV = v;
      if (v > maxV)maxV = v;
    }
    z += voxelZLv2 - 1;
#pragma unroll
    for (int c1(8); c1 > 0; c1 /= 2)
    {
      float t0, t1;
      t0 = __shfl_down_sync(0xffffffffu, minV, c1);
      t1 = __shfl_down_sync(0xffffffffu, maxV, c1);
      if (t0 < minV)minV = t0;
      if (t1 > maxV)maxV = t1;
    }
    if (tid == 0)
    {
      minMax[idx] = minV;
      minMax[idx + 1] = maxV;
      constexpr unsigned int offsetSize(2 * blockXLv2 * blockYLv2);
      idx += offsetSize;
    }
    minV = v;
    maxV = v;
  }
}

__global__ void compactLv2(
  float isoValue,
  const float*__restrict__ minMax,
  const unsigned int*__restrict__ blockIndicesLv1,
  unsigned int*__restrict__ blockIndicesLv2,
  unsigned int counterBlockNumLv1,
  unsigned int*__restrict__ countedBlockNumLv2)
{
  __shared__ unsigned int sums[32];
  constexpr unsigned int warpNum(countingThreadNumLv2 / 32);
  unsigned int tid(threadIdx.x);
  unsigned int laneid = tid % 32;
  unsigned int warpid(tid >> 5);
  unsigned int id0(tid + blockIdx.x * countingThreadNumLv2);
  unsigned int id1(id0 / voxelNumLv2);
  unsigned int test;
  if (id1 < counterBlockNumLv1)
  {
    if (minMax[2 * id0] <= isoValue && minMax[2 * id0 + 1] >= isoValue)
      test = 1;
    else
      test = 0;
  }
  else test = 0;
  unsigned int testSum(test);
#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp(__shfl_up_sync(0xffffffffu, testSum, c0));
    if (laneid >= c0)testSum += tp;
  }
  if (laneid == 31)sums[warpid] = testSum;
  __syncthreads();
  if (warpid == 0)
  {
    unsigned warpSum = sums[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp(__shfl_up_sync(0xffffffffu, warpSum, c0));
      if (laneid >= c0)warpSum += tp;
    }
    sums[laneid] = warpSum;
  }
  __syncthreads();
  if (warpid != 0)testSum += sums[warpid - 1];
  if (tid == countingThreadNumLv2 - 1)
    sums[31] = atomicAdd(countedBlockNumLv2, testSum);
  __syncthreads();

  if (test)
  {
    unsigned int bIdx1(blockIndicesLv1[id1]);
    unsigned int bIdx2;
    unsigned int x1, y1, z1;
    unsigned int x2, y2, z2;
    unsigned int tp1(bIdx1);
    unsigned int tp2((tid + blockIdx.x * countingThreadNumLv2) % voxelNumLv2);
    x1 = tp1 % gridXLv1;
    x2 = tp2 % blockXLv2;
    tp1 /= gridXLv1;
    tp2 /= blockXLv2;
    y1 = tp1 % gridYLv1;
    y2 = tp2 % blockYLv2;
    z1 = tp1 / gridYLv1;
    z2 = tp2 / blockYLv2;
    bIdx2 = x2 + blockXLv2 * (x1 + gridXLv1 * (y2 + blockYLv2 * (y1 + gridYLv1 * (z1 * blockZLv2 + z2))));
    blockIndicesLv2[testSum + sums[31] - 1] = bIdx2;
  }
}

__global__ void generatingTriangles(
  float isoValue, 
  const unsigned int*__restrict__ blockIndicesLv2,
  const unsigned short *__restrict__ distinctEdgesTable,
  const int *__restrict__ triTable,
  const uchar4 *__restrict__ edgeIDTable,
  unsigned int*__restrict__ countedVerticesNum,
  unsigned int*__restrict__ countedTrianglesNum,
  unsigned long long*__restrict__ triangles,
  float*__restrict__ coordX,
  float*__restrict__ coordY,
  float*__restrict__ coordZ,
  float*__restrict__ coordZP)
{
  __shared__ unsigned short vertexIndices[voxelZLv2][voxelYLv2][voxelXLv2];
  __shared__ float value[voxelZLv2 + 1][voxelYLv2 + 1][voxelXLv2 + 1];
  __shared__ unsigned int sumsVertices[32];
  __shared__ unsigned int sumsTriangles[32];

  unsigned int blockId(blockIndicesLv2[blockIdx.x]);
  unsigned int tp(blockId);
  unsigned int x((tp % gridXLv2) * (voxelXLv2 - 1) + threadIdx.x);
  tp /= gridXLv2;
  unsigned int y((tp % gridYLv2) * (voxelYLv2 - 1) + threadIdx.y);
  unsigned int z((tp / gridYLv2) * (voxelZLv2 - 1) + threadIdx.z);
  unsigned int eds(7);
  float v(value[threadIdx.z][threadIdx.y][threadIdx.x] = f(x, y, z));
  if (threadIdx.x == voxelXLv2 - 1)
  {
    eds &= 6;
    value[threadIdx.z][threadIdx.y][voxelXLv2] = f(x + 1, y, z);
    if (threadIdx.y == voxelYLv2 - 1)
      value[threadIdx.z][voxelYLv2][voxelXLv2] = f(x + 1, y + 1, z);
  }
  if (threadIdx.y == voxelYLv2 - 1)
  {
    eds &= 5;
    value[threadIdx.z][voxelYLv2][threadIdx.x] = f(x, y + 1, z);
    if (threadIdx.z == voxelZLv2 - 1)
      value[voxelZLv2][voxelYLv2][threadIdx.x] = f(x, y + 1, z + 1);
  }
  if (threadIdx.z == voxelZLv2 - 1)
  {
    eds &= 3;
    value[voxelZLv2][threadIdx.y][threadIdx.x] = f(x, y, z + 1);
    if (threadIdx.x == voxelXLv2 - 1)
      value[voxelZLv2][threadIdx.y][voxelXLv2] = f(x + 1, y, z + 1);
  }
  eds <<= 13;
  __syncthreads();
  unsigned int cubeCase(0);
  if (value[threadIdx.z][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 1;
  if (value[threadIdx.z][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 2;
  if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 4;
  if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 8;
  if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 16;
  if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 32;
  if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 64;
  if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 128;

  unsigned int distinctEdges(eds ? distinctEdgesTable[cubeCase] : 0);
  unsigned int numTriangles(eds != 0xe000 ? 0 : distinctEdges & 7);
  unsigned int numVertices(__popc(distinctEdges &= eds));
  unsigned int laneid = (threadIdx.x + voxelXLv2 * (threadIdx.y + voxelYLv2 * threadIdx.z)) % 32;
  unsigned warpid((threadIdx.x + voxelXLv2 * (threadIdx.y + voxelYLv2 * threadIdx.z)) >> 5);
  constexpr unsigned int threadNum(voxelXLv2 * voxelYLv2 * voxelZLv2);
  constexpr unsigned int warpNum(threadNum / 32);
  unsigned int sumVertices(numVertices);
  unsigned int sumTriangles(numTriangles);

#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp0(__shfl_up_sync(0xffffffffu, sumVertices, c0));
    unsigned int tp1(__shfl_up_sync(0xffffffffu, sumTriangles, c0));
    if (laneid >= c0)
    {
      sumVertices += tp0;
      sumTriangles += tp1;
    }
  }
  if (laneid == 31)
  {
    sumsVertices[warpid] = sumVertices;
    sumsTriangles[warpid] = sumTriangles;
  }
  __syncthreads();
  if (warpid == 0)
  {
    unsigned warpSumVertices = sumsVertices[laneid];
    unsigned warpSumTriangles = sumsTriangles[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp0(__shfl_up_sync(0xffffffffu, warpSumVertices, c0));
      unsigned int tp1(__shfl_up_sync(0xffffffffu, warpSumTriangles, c0));
      if (laneid >= c0)
      {
        warpSumVertices += tp0;
        warpSumTriangles += tp1;
      }
    }
    sumsVertices[laneid] = warpSumVertices;
    sumsTriangles[laneid] = warpSumTriangles;
  }
  __syncthreads();
  if (warpid != 0)
  {
    sumVertices += sumsVertices[warpid - 1];
    sumTriangles += sumsTriangles[warpid - 1];
  }
  if (eds == 0)
  {
    sumsVertices[31] = atomicAdd(countedVerticesNum, sumVertices);
    sumsTriangles[31] = atomicAdd(countedTrianglesNum, sumTriangles);
  }

  unsigned int interOffsetVertices(sumVertices - numVertices);
  sumVertices = interOffsetVertices + sumsVertices[31];//exclusive offset
  sumTriangles = sumTriangles + sumsTriangles[31] - numTriangles;//exclusive offset
  vertexIndices[threadIdx.z][threadIdx.y][threadIdx.x] = interOffsetVertices | distinctEdges;
  __syncthreads();

  for (unsigned int c0(0); c0 < numTriangles; ++c0)
  {
#pragma unroll
    for (unsigned int c1(0); c1 < 3; ++c1)
    {
      int edgeID(triTable[16 * cubeCase + 3 * c0 + c1]);
      uchar4 edgePos(edgeIDTable[edgeID]);
      unsigned short vertexIndex(vertexIndices[threadIdx.z + edgePos.z][threadIdx.y + edgePos.y][threadIdx.x + edgePos.x]);
      unsigned int tp(__popc(vertexIndex >> (16 - edgePos.w)) + (vertexIndex & 0x1fff));
      atomicAdd(triangles, (unsigned long long)(sumsVertices[31] + tp));
    }
  }

  // sumVertices may be too large for a GPU memory
  float zp = 0.f, cx = 0.f, cy = 0.f, cz = 0.f;

  if (distinctEdges & (1 << 15))
  {
    zp = zeroPoint(x, v, value[threadIdx.z][threadIdx.y][threadIdx.x + 1], isoValue);
    cy = transformToCoord(y);
    cz = transformToCoord(z);
  }
  if (distinctEdges & (1 << 14))
  {
    cx = transformToCoord(x);
    zp += zeroPoint(y, v, value[threadIdx.z][threadIdx.y + 1][threadIdx.x], isoValue);
    cz += transformToCoord(z);
  }
  if (distinctEdges & (1 << 13))
  {
    cx += transformToCoord(x);
    cy += transformToCoord(y);
    zp += zeroPoint(z, v, value[threadIdx.z + 1][threadIdx.y][threadIdx.x], isoValue);
  }
  atomicAdd(coordX, cx);
  atomicAdd(coordY, cy);
  atomicAdd(coordZ, cz);
  atomicAdd(coordZP, zp);
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  unsigned int repeat = atoi(argv[1]);

  std::uniform_real_distribution<float>rd(0, 1);
  std::mt19937 mt(123);

  float* minMaxLv1Device;
  float* minMaxLv2Device;
  unsigned int* blockIndicesLv1Device;
  unsigned int* blockIndicesLv2Device;
  unsigned int* countedBlockNumLv1Device;
  unsigned int* countedBlockNumLv2Device;
  unsigned short* distinctEdgesTableDevice;
  int* triTableDevice;
  uchar4* edgeIDTableDevice;
  unsigned int* countedVerticesNumDevice;
  unsigned int* countedTrianglesNumDevice;
  unsigned long long* trianglesDevice;
  float *coordXDevice;
  float *coordYDevice;
  float *coordZDevice;
  float *coordZPDevice;

  hipMalloc(&minMaxLv1Device, blockNum * 2 * sizeof(float));
  hipMalloc(&blockIndicesLv1Device, blockNum * sizeof(unsigned int));
  hipMalloc(&countedBlockNumLv1Device, sizeof(unsigned int));
  hipMalloc(&countedBlockNumLv2Device, sizeof(unsigned int));
  hipMalloc(&distinctEdgesTableDevice, sizeof(distinctEdgesTable));
  hipMalloc(&triTableDevice, sizeof(triTable));
  hipMalloc(&edgeIDTableDevice, sizeof(edgeIDTable));
  hipMalloc(&countedVerticesNumDevice, sizeof(unsigned int));
  hipMalloc(&countedTrianglesNumDevice, sizeof(unsigned int));
  hipMemcpy(distinctEdgesTableDevice, distinctEdgesTable, sizeof(distinctEdgesTable), hipMemcpyHostToDevice);
  hipMemcpy(triTableDevice, triTable, sizeof(triTable), hipMemcpyHostToDevice);
  hipMemcpy(edgeIDTableDevice, edgeIDTable, sizeof(edgeIDTable), hipMemcpyHostToDevice);

  // simulate rendering without memory allocation for vertices and triangles 
  hipMalloc(&trianglesDevice, sizeof(unsigned long long));
  hipMalloc(&coordXDevice, sizeof(float));
  hipMalloc(&coordYDevice, sizeof(float));
  hipMalloc(&coordZDevice, sizeof(float));
  hipMalloc(&coordZPDevice, sizeof(float));

  const dim3 BlockSizeLv1{ voxelXLv1, voxelYLv1, 1 };
  const dim3 GridSizeLv1{ gridXLv1, gridYLv1, gridZLv1 };
  
  const dim3 BlockSizeLv2{ voxelXLv2 * voxelYLv2, blockXLv2 * blockYLv2, 1 };
  const dim3 BlockSizeGenerating{ voxelXLv2, voxelYLv2, voxelZLv2 };

  float isoValue(-0.9f);

  unsigned int countedBlockNumLv1;
  unsigned int countedBlockNumLv2;
  unsigned int countedVerticesNum;
  unsigned int countedTrianglesNum;

  float time(0.f);

  for (unsigned int c0(0); c0 < repeat; ++c0)
  {
    hipDeviceSynchronize();
    hipMemset(countedBlockNumLv1Device, 0, sizeof(unsigned int));
    hipMemset(countedBlockNumLv2Device, 0, sizeof(unsigned int));
    hipMemset(countedVerticesNumDevice, 0, sizeof(unsigned int));
    hipMemset(countedTrianglesNumDevice,0, sizeof(unsigned int));
    hipMemset(trianglesDevice, 0, sizeof(unsigned long long));
    hipMemset(coordXDevice, 0, sizeof(float));
    hipMemset(coordYDevice, 0, sizeof(float));
    hipMemset(coordZDevice, 0, sizeof(float));
    hipMemset(coordZPDevice, 0, sizeof(float));

    computeMinMaxLv1 <<< GridSizeLv1, BlockSizeLv1 >>> (minMaxLv1Device);
    compactLv1 <<< countingBlockNumLv1, countingThreadNumLv1 >>> (
      isoValue, minMaxLv1Device, blockIndicesLv1Device, countedBlockNumLv1Device);

    hipMemcpy(&countedBlockNumLv1, countedBlockNumLv1Device, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMalloc(&minMaxLv2Device, countedBlockNumLv1 * voxelNumLv2 * 2 * sizeof(float));

    computeMinMaxLv2 <<< countedBlockNumLv1, BlockSizeLv2 >>> (blockIndicesLv1Device, minMaxLv2Device);

    hipMalloc(&blockIndicesLv2Device, countedBlockNumLv1 * voxelNumLv2 * sizeof(unsigned int));
    unsigned int countingBlockNumLv2((countedBlockNumLv1 * voxelNumLv2 + countingThreadNumLv2 - 1) / countingThreadNumLv2);

    compactLv2 <<< countingBlockNumLv2, countingThreadNumLv2 >>> (
      isoValue, minMaxLv2Device, blockIndicesLv1Device, blockIndicesLv2Device, countedBlockNumLv1, countedBlockNumLv2Device);

    hipMemcpy(&countedBlockNumLv2, countedBlockNumLv2Device, sizeof(unsigned int), hipMemcpyDeviceToHost);

    auto start = std::chrono::steady_clock::now();

    generatingTriangles <<< countedBlockNumLv2, BlockSizeGenerating >>> (
        isoValue, blockIndicesLv2Device,
        distinctEdgesTableDevice, triTableDevice, edgeIDTableDevice,
        countedVerticesNumDevice, countedTrianglesNumDevice, trianglesDevice,
        coordXDevice, coordYDevice, coordZDevice, coordZPDevice);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto ktime = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time += ktime;

    hipMemcpy(&countedVerticesNum, countedVerticesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(&countedTrianglesNum, countedTrianglesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(minMaxLv2Device);
    hipFree(blockIndicesLv2Device);
  }

  printf("Block Lv1: %u\nBlock Lv2: %u\n", countedBlockNumLv1, countedBlockNumLv2);
  printf("Vertices Size: %u\n", countedBlockNumLv2 * 304);
  printf("Triangles Size: %u\n", countedBlockNumLv2 * 315 * 3);
  printf("Vertices: %u\nTriangles: %u\n", countedVerticesNum, countedTrianglesNum);
  printf("Average kernel execution time (generatingTriangles): %f (s)\n", (time * 1e-9f) / repeat);

  // specific to the problem size
  bool ok = (countedBlockNumLv1 == 8296 && countedBlockNumLv2 == 240380 &&
             countedVerticesNum == 4856560 && countedTrianglesNum == 6101640);
  printf("%s\n", ok ? "PASS" : "FAIL");

  hipFree(minMaxLv1Device);
  hipFree(blockIndicesLv1Device);
  hipFree(countedBlockNumLv1Device);
  hipFree(countedBlockNumLv2Device);
  hipFree(distinctEdgesTableDevice);
  hipFree(triTableDevice);
  hipFree(edgeIDTableDevice);
  hipFree(countedVerticesNumDevice);
  hipFree(countedTrianglesNumDevice);
  hipFree(trianglesDevice);
  hipFree(coordXDevice);
  hipFree(coordYDevice);
  hipFree(coordZDevice);
  hipFree(coordZPDevice);
  return 0;
}
