#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define m1  0x5555555555555555
#define m2  0x3333333333333333 
#define m4  0x0f0f0f0f0f0f0f0f 
#define h01 0x0101010101010101

#define BLOCK_SIZE 256

// reference implementation
int popcount_ref(unsigned long x)
{
  int count;
  for (count=0; x; count++)
    x &= x - 1;
  return count;
}

// CUDA kernels
__global__ void pc1 (const unsigned long* __restrict__ data,int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  x -= (x >> 1) & m1;             //put count of each 2 bits into those 2 bits
  x = (x & m2) + ((x >> 2) & m2); //put count of each 4 bits into those 4 bits 
  x = (x + (x >> 4)) & m4;        //put count of each 8 bits into those 8 bits 
  x += x >>  8;  //put count of each 16 bits into their lowest 8 bits
  x += x >> 16;  //put count of each 32 bits into their lowest 8 bits
  x += x >> 32;  //put count of each 64 bits into their lowest 8 bits
  r[i] = x & 0x7f;
}

__global__ void pc2 (const unsigned long* __restrict__ data, int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  x -= (x >> 1) & m1;             //put count of each 2 bits into those 2 bits
  x = (x & m2) + ((x >> 2) & m2); //put count of each 4 bits into those 4 bits 
  x = (x + (x >> 4)) & m4;        //put count of each 8 bits into those 8 bits 
  r[i] = (x * h01) >> 56;  //returns left 8 bits of x + (x<<8) + (x<<16) + (x<<24) + ... 
}

__global__ void pc3 (const unsigned long* __restrict__ data, int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  char count;
  unsigned long x = data[i];
  for (count=0; x; count++) x &= x - 1;
  r[i] = count;
}

__global__ void pc4 (const unsigned long* __restrict__ data, int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  char cnt = 0;
  for (char i = 0; i < 64; i++)
  {
    cnt = cnt + (x & 0x1);
    x = x >> 1;
  }
  r[i] = cnt;
}

__global__ void pc5 (const unsigned long* __restrict__ data, int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  const unsigned char a[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char b[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char c[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char d[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};

  unsigned char i1 = a[(x & 0xFF)];
  unsigned char i2 = a[(x >> 8) & 0xFF];
  unsigned char i3 = b[(x >> 16) & 0xFF];
  unsigned char i4 = b[(x >> 24) & 0xFF];
  unsigned char i5 = c[(x >> 32) & 0xFF];
  unsigned char i6 = c[(x >> 40) & 0xFF];
  unsigned char i7 = d[(x >> 48) & 0xFF];
  unsigned char i8 = d[(x >> 56) & 0xFF];
  r[i] = (i1+i2)+(i3+i4)+(i5+i6)+(i7+i8);
}

__global__ void pc6 (const unsigned long* __restrict__ data, int* __restrict__ r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  r[i] = __popcll(data[i]);
}

void checkResults(const unsigned long *d, const int *r, const int length)
{
  int error = 0;
  for (int i=0;i<length;i++)
    if (popcount_ref(d[i]) != r[i]) {
      error = 1;
      break;
    }

  if (error)
    printf("Fail\n");
  else
    printf("Success\n");
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <length> <repeat>\n", argv[0]);
    return 1;
  }
  const int length = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

  unsigned long *data = NULL;
  int *result = NULL;
  int s1 = posix_memalign((void**)&data, 1024, length*sizeof(unsigned long));
  int s2 = posix_memalign((void**)&result, 1024, length*sizeof(int));
  if (s1 != 0 || s2 != 0) {
    printf("Error: posix_memalign fails\n");
    if (s1 == 0) free(data);
    if (s2 == 0) free(result);
    return 1;
  }

  // initialize input
  srand(2);
  for (int i = 0; i < length; i++) {
    unsigned long t = (unsigned long)rand() << 32;
    data[i] = t | rand();
  }

  // run each popcount implementation repeat times

  unsigned long* d_data;
  hipMalloc((void**)&d_data, sizeof(unsigned long)*length);
  hipMemcpy(d_data, data, sizeof(unsigned long)*length, hipMemcpyHostToDevice);

  int* d_result;
  hipMalloc((void**)&d_result, sizeof(int)*length);

  dim3 grids ((length+BLOCK_SIZE-1)/BLOCK_SIZE);
  dim3 threads (BLOCK_SIZE);

  auto start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc1<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc1): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc2<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc2): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc3<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc3): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc4<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc4): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc5<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc5): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  start = std::chrono::steady_clock::now();
  for (int n = 0; n < repeat; n++) {
    pc6<<<grids, threads>>>(d_data, d_result, length);
  }
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (pc6): %f (us)\n", (time * 1e-3) / repeat);

  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  hipFree(d_data);
  hipFree(d_result);
  free(data);
  free(result);
  return 0;
}
