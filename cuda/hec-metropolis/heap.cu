#include "hip/hip_runtime.h"
/*
    File:   minHeap.c
    Desc:   Program showing various operations on a binary min heap
    Author: Robin Thomas <robinthomas2591@gmail.com>
*/

#include <stdio.h>
#include <stdlib.h>
#include "heap.h"
/*
    Function to initialize the min heap with size = 0
*/
minHeap initMinHeap(int size) {
    minHeap hp ;
    hp.size = 0 ;
    return hp ;
}



/*
    Function to swap data within two nodes of the min heap using pointers
*/
void swap(node *n1, node *n2) {
    node temp = *n1 ;
    *n1 = *n2 ;
    *n2 = temp ;
}


void printNode(node n){
    printf("[%f, {%i, %i}]\n", n.data, n.coord.f, n.coord.i);
}


/*
    Heapify function is used to make sure that the heap property is never violated
    In case of deletion of a node, or creating a min heap from an array, heap property
    may be violated. In such cases, heapify function can be called to make sure that
    heap property is never violated
*/
void heapify(minHeap *hp, int i) {
    int smallest = (LCHILD(i) < hp->size && hp->elem[LCHILD(i)].data < hp->elem[i].data) ? LCHILD(i) : i ;
    if(RCHILD(i) < hp->size && hp->elem[RCHILD(i)].data < hp->elem[smallest].data) {
        smallest = RCHILD(i) ;
    }
    if(smallest != i) {
        swap(&(hp->elem[i]), &(hp->elem[smallest])) ;
        heapify(hp, smallest) ;
    }
}


/* 
    Build a Min Heap given an array of numbers
    Instead of using insertNode() function n times for total complexity of O(nlogn),
    we can use the buildMinHeap() function to build the heap in O(n) time
*/
void buildMinHeap(minHeap *hp, int *arr, int size) {
    int i ;

    // Insertion into the heap without violating the shape property
    for(i = 0; i < size; i++) {
        if(hp->size) {
            hp->elem = (node*)realloc(hp->elem, (hp->size + 1) * sizeof(node)) ;
        } else {
            hp->elem = (node*)malloc(sizeof(node)) ;
        }
        node nd ;
        nd.data = arr[i] ;
        hp->elem[(hp->size)++] = nd ;
    }

    // Making sure that heap property is also satisfied
    for(i = (hp->size - 1) / 2; i >= 0; i--) {
        heapify(hp, i) ;
    }
}


/*
    Function to insert a node into the min heap, by allocating space for that node in the
    heap and also making sure that the heap property and shape propety are never violated.
*/
void insertNode(minHeap *hp, float data, findex_t frag) {
    if(hp->size) {
        hp->elem = (node*)realloc(hp->elem, (hp->size + 1) * sizeof(node)) ;
    } else {
        hp->elem = (node*)malloc(sizeof(node)) ;
    }

    node nd ;
    nd.data = data;
    nd.coord = frag;

    int i = (hp->size)++ ;
    while(i && nd.data < hp->elem[PARENT(i)].data) {
        hp->elem[i] = hp->elem[PARENT(i)] ;
        i = PARENT(i) ;
    }
    hp->elem[i] = nd ;
}


/*
    Function to delete a node from the min heap
    It shall remove the root node, and place the last node in its place
    and then call heapify function to make sure that the heap property
    is never violated
*/
void deleteNode(minHeap *hp) {
    if(hp->size) {
        printf("Deleting node [%f, {%i ,%i}] \n\n", hp->elem[0].data, hp->elem[0].coord.f, hp->elem[0].coord.i);
        hp->elem[0] = hp->elem[--(hp->size)] ;
        hp->elem = (node*)realloc(hp->elem, hp->size * sizeof(node)) ;
        heapify(hp, 0) ;
    } else {
        printf("\nMin Heap is empty!\n") ;
        free(hp->elem) ;
    }
}

/*
    Function to pop the min value form the root and heapify accordingly
*/
node popRoot(minHeap *hp) {
    node mynode;
    if(hp->size) {
        //printf("Popping node:"); printNode(hp->elem[0]);
        mynode = hp->elem[0];
        hp->elem[0] = hp->elem[--(hp->size)] ;
        hp->elem = (node*)realloc(hp->elem, hp->size * sizeof(node)) ;
        heapify(hp, 0) ;
    } else {
        //printf("\nMin Heap is empty!\n") ;
        free(hp->elem) ;
    }
    return mynode;
}

/*
    Function to get maximum node from a min heap
    The maximum node shall always be one of the leaf nodes. So we shall recursively
    move through both left and right child, until we find their maximum nodes, and
    compare which is larger. It shall be done recursively until we get the maximum
    node
*/
int getMaxNode(minHeap *hp, int i) {
    if(LCHILD(i) >= hp->size) {
        return hp->elem[i].data ;
    }

    int l = getMaxNode(hp, LCHILD(i)) ;
    int r = getMaxNode(hp, RCHILD(i)) ;

    if(l >= r) {
        return l ;
    } else {
        return r ;
    }
}


/*
    Function to clear the memory allocated for the min heap
*/
void deleteMinHeap(minHeap *hp) {
    free(hp->elem) ;
}


/*
    Function to display all the nodes in the min heap by doing a inorder traversal
*/
void inorderTraversal(minHeap *hp, int i) {
    if(LCHILD(i) < hp->size) {
        inorderTraversal(hp, LCHILD(i)) ;
    }
    printNode(hp->elem[i]);
    if(RCHILD(i) < hp->size) {
        inorderTraversal(hp, RCHILD(i)) ;
    }
}

/*
    Function to display all the nodes in the min heap by doing a preorder traversal
*/
void preorderTraversal(minHeap *hp, int i) {
    if(LCHILD(i) < hp->size) {
        preorderTraversal(hp, LCHILD(i)) ;
    }
    if(RCHILD(i) < hp->size) {
        preorderTraversal(hp, RCHILD(i)) ;
    }
    printNode(hp->elem[i]);
}


/*
    Function to display all the nodes in the min heap by doing a post order traversal
*/
void postorderTraversal(minHeap *hp, int i) {
    printNode(hp->elem[i]);
    if(LCHILD(i) < hp->size) {
        postorderTraversal(hp, LCHILD(i)) ;
    }
    if(RCHILD(i) < hp->size) {
        postorderTraversal(hp, RCHILD(i)) ;
    }
}


/*
    Function to display all the nodes in the min heap by doing a level order traversal
*/
void levelorderTraversal(minHeap *hp) {
    int i ;
    for(i = 0; i < hp->size; i++) {
        printNode(hp->elem[i]);
    }
}

