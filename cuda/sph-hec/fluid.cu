#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include "common.h"

////////////////////////////////////////////////////////////////////////////
// B spline smoothing kernel
////////////////////////////////////////////////////////////////////////////

__device__
double W(double3 p_pos, double3 q_pos, double h)
{
    double r = sqrt((p_pos.x-q_pos.x)*(p_pos.x-q_pos.x)
                  + (p_pos.y-q_pos.y)*(p_pos.y-q_pos.y)
                  + (p_pos.z-q_pos.z)*(p_pos.z-q_pos.z));
    double C = 1.0/(M_PI*h*h*h);
    double u = r/h;
    double val = 0.0;
    if(u >= 2.0)
        return val;
    else if(u < 1.0 )
        val = 1.0 - (3.0/2.0)*u*u + (3.0/4.0)*u*u*u;
    else if(u >= 1.0 && u < 2.0)
        val = (1.0/4.0) * pow(2.0-u,3.0);

    val *= C;
    return val;
}

// Gradient of B spline kernel
__device__
double del_W(double3 p_pos, double3 q_pos, double h)
{
    double r = sqrt((p_pos.x-q_pos.x)*(p_pos.x-q_pos.x)
                  + (p_pos.y-q_pos.y)*(p_pos.y-q_pos.y)
                  + (p_pos.z-q_pos.z)*(p_pos.z-q_pos.z));
    double C = 1.0/(M_PI * h*h*h);
    double u = r/h;
    double val = 0.0;
    if(u >= 2.0)
        return val;
    else if(u < 1.0 )
        val = -1.0/(h*h) * (3.0 - 9.0/4.0*u);
    else if(u >= 1.0 && u < 2.0)
        val = -3.0/(4.0*h*r) * pow(2.0-u,2.0);

    val *= C;
    return val;
}

////////////////////////////////////////////////////////////////////////////
// Boundary particle force
// http://iopscience.iop.org/0034-4885/68/8/R01/pdf/0034-4885_68_8_R01.pdf
////////////////////////////////////////////////////////////////////////////

__device__
double boundaryGamma(double3 p_pos, double3 k_pos, double3 k_n, double h, double speed_sound)
{
    // Radial distance between p,q
    double r = sqrt((p_pos.x-k_pos.x)*(p_pos.x-k_pos.x)
                  + (p_pos.y-k_pos.y)*(p_pos.y-k_pos.y)
                  + (p_pos.z-k_pos.z)*(p_pos.z-k_pos.z));
    // Distance to p normal to surface particle
    double y = sqrt((p_pos.x-k_pos.x)*(p_pos.x-k_pos.x)*(k_n.x*k_n.x)
                  + (p_pos.y-k_pos.y)*(p_pos.y-k_pos.y)*(k_n.y*k_n.y)
                  + (p_pos.z-k_pos.z)*(p_pos.z-k_pos.z)*(k_n.z*k_n.z));
    // Tangential distance
    double x = r-y;

    double u = y/h;
    double xi = (1-x/h)?x<h:0.0;
    double C = xi*2.0*0.02 * speed_sound * speed_sound / y;
    double val = 0.0;

    if(u > 0.0 && u < 2.0/3.0)
        val = 2.0/3.0;
    else if(u < 1.0 && u > 2.0/3.0 )
        val = (2*u - 3.0/2.0*u*u);
    else if (u < 2.0 && u > 1.0)
        val = 0.5*(2.0-u)*(2.0-u);
    else
        val = 0.0;

    val *= C;

    return val;
}

////////////////////////////////////////////////////////////////////////////
// Particle attribute computations
////////////////////////////////////////////////////////////////////////////

__device__
double computeDensity(double3 p_pos, double3 p_v, double3 q_pos, double3 q_v,
                      const param *params)
{
    double v_x = (p_v.x - q_v.x);
    double v_y = (p_v.y - q_v.y);
    double v_z = (p_v.z - q_v.z);

    double density = params->mass_particle * del_W(p_pos,q_pos,
                                                   params->smoothing_radius);
    double density_x = density * v_x * (p_pos.x - q_pos.x);
    double density_y = density * v_y * (p_pos.y - q_pos.y);
    double density_z = density * v_z * (p_pos.z - q_pos.z);

    density = (density_x + density_y + density_z)*params->time_step;

    return density;
}

__device__
double computePressure(double p_density, const param *params)
{
    double gam = 7.0;
    double B = params->rest_density * params->speed_sound*params->speed_sound / gam;
    double pressure =  B * (pow((p_density/params->rest_density),gam) - 1.0);

    return pressure;
}

__global__
void updatePressures(fluid_particle *__restrict__ fluid_particles,
                     const param *__restrict__ params)
{
    int num_fluid_particles = params->number_fluid_particles;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_fluid_particles) return;
    double3 p_pos = fluid_particles[i].pos;
    double3 p_v   = fluid_particles[i].v;
    double density = fluid_particles[i].density;

    for(int j=0; j< num_fluid_particles; j++) {
        double3 q_pos = fluid_particles[j].pos;
        double3 q_v   = fluid_particles[j].v;
        density += computeDensity(p_pos,p_v,q_pos,q_v, params);
    }
    fluid_particles[i].density = density;
    fluid_particles[i].pressure = computePressure(density, params);
}

__device__
double3 computeBoundaryAcceleration(double3 p_pos, double3 k_pos, double3 k_n,
                                    double h, double speed_sound)
{
    double3 p_a;
    double bGamma = boundaryGamma(p_pos,k_pos,k_n,h,speed_sound);
    p_a.x = bGamma * k_n.x;
    p_a.y = bGamma * k_n.y;
    p_a.z = bGamma * k_n.z;

    return p_a;
}

__device__
double3 computeAcceleration(double3 p_pos, double3 p_v, double p_density,
                            double p_pressure, double3 q_pos, double3 q_v,
                            double q_density, double q_pressure, const param *const params)
{
    double3 a;
    double accel;
    double h = params->smoothing_radius;
    double alpha = params->alpha;
    double speed_sound = params->speed_sound;
    double mass_particle = params->mass_particle;
    double surface_tension = params->surface_tension;

    // Pressure force
    accel = (p_pressure/(p_density*p_density) + q_pressure/(q_density*q_density))
            * mass_particle * del_W(p_pos,q_pos,h);
    a.x = -accel * (p_pos.x - q_pos.x);
    a.y = -accel * (p_pos.y - q_pos.y);
    a.z = -accel * (p_pos.z - q_pos.z);

    // Viscosity force
    double VdotR = (p_v.x-q_v.x)*(p_pos.x-q_pos.x)
                 + (p_v.y-q_v.y)*(p_pos.y-q_pos.y)
                 + (p_v.z-q_v.z)*(p_pos.z-q_pos.z);
    if(VdotR < 0.0)
    {
        double nu = 2.0 * alpha * h * speed_sound / (p_density + q_density);
        double r2 = (p_pos.x-q_pos.x)*(p_pos.x-q_pos.x)
                  + (p_pos.y-q_pos.y)*(p_pos.y-q_pos.y)
                  + (p_pos.z-q_pos.z)*(p_pos.z-q_pos.z);
        double eps = h/10.0;
        double stress = nu * VdotR / (r2 + eps*h*h);
        accel = mass_particle * stress * del_W(p_pos, q_pos, h);
        a.x += accel * (p_pos.x - q_pos.x);
        a.y += accel * (p_pos.y - q_pos.y);
        a.z += accel * (p_pos.z - q_pos.z);
    }

    //Surface tension
    // BT 07 http://cg.informatik.uni-freiburg.de/publications/2011_GRAPP_airBubbles.pdf
    accel = surface_tension * W(p_pos,q_pos,h);
    a.x += accel * (p_pos.x - q_pos.x);
    a.y += accel * (p_pos.y - q_pos.y);
    a.z += accel * (p_pos.z - q_pos.z);

    return a;
}

// Update particle acclerations
__global__
void updateAccelerationsFP(fluid_particle *__restrict__ fluid_particles,
                           const param *__restrict__ params)
{
    int num_fluid_particles = params->number_fluid_particles;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_fluid_particles) return;

        double ax = 0.0;
        double ay = 0.0;
        double az = -9.8;

        double3 p_pos = fluid_particles[i].pos;
        double3 p_v   = fluid_particles[i].v;
        double p_density = fluid_particles[i].density;
        double p_pressure = fluid_particles[i].pressure;

        for(int j=0; j<num_fluid_particles; j++) {
            if (i!=j) {
                double3 q_pos = fluid_particles[j].pos;
                double3 q_v   = fluid_particles[j].v;
                double q_density = fluid_particles[j].density;
                double q_pressure = fluid_particles[j].pressure;
                double3 tmp_a = computeAcceleration(p_pos, p_v, p_density,
                                                    p_pressure, q_pos, q_v,
                                                    q_density, q_pressure, params);

                ax += tmp_a.x;
                ay += tmp_a.y;
                az += tmp_a.z;
            }
        }

        fluid_particles[i].a.x = ax;
        fluid_particles[i].a.y = ay;
        fluid_particles[i].a.z = az;
}

__global__
void updateAccelerationsBP(fluid_particle *__restrict__ fluid_particles,
                           const boundary_particle *__restrict__ boundary_particles, 
                           const param *__restrict__ params)
{
    int num_fluid_particles = params->number_fluid_particles;
    int num_boundary_particles = params->number_boundary_particles;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_fluid_particles) return;

    double ax = fluid_particles[i].a.x;
    double ay = fluid_particles[i].a.y;
    double az = fluid_particles[i].a.z;
    double3 p_pos = fluid_particles[i].pos;

    for (int j=0; j<num_boundary_particles; j++) {
      double3 k_pos = boundary_particles[j].pos;
      double3 k_n   = boundary_particles[j].n;
      double3 tmp_a = computeBoundaryAcceleration(p_pos,k_pos,k_n,
          params->smoothing_radius,
          params->speed_sound);
      ax += tmp_a.x;
      ay += tmp_a.y;
      az += tmp_a.z;
    }

    fluid_particles[i].a.x = ax;
    fluid_particles[i].a.y = ay;
    fluid_particles[i].a.z = az;
}

// Update particle positions
// Leap Frog integration with v(t+1) estimated
__global__
void updatePositions(fluid_particle *__restrict__ fluid_particles,
                     const param *__restrict__ params)
{
    double dt = params->time_step;

    int num_fluid_particles = params->number_fluid_particles;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_fluid_particles) return;

    // Velocity at t + dt/2
    double3 v_half = fluid_particles[i].v_half;
    double3 v      = fluid_particles[i].v;
    double3 pos    = fluid_particles[i].pos;
    double3 a      = fluid_particles[i].a;

    v_half.x = v_half.x + dt * a.x;
    v_half.y = v_half.y + dt * a.y;
    v_half.z = v_half.z + dt * a.z;

    // Velocity at t + dt, must estimate for foce calc
    v.x = v_half.x + a.x * (dt / 2.0);
    v.y = v_half.y + a.y * (dt / 2.0);
    v.z = v_half.z + a.z * (dt / 2.0);

    // Position at time t + dt
    pos.x = pos.x + dt * v_half.x;
    pos.y = pos.y + dt * v_half.y;
    pos.z = pos.z + dt * v_half.z;

    fluid_particles[i].v_half = v_half;
    fluid_particles[i].v      = v;
    fluid_particles[i].pos    = pos;
}

// Seed simulation with Euler step v(t-dt/2) needed by leap frog integrator
// Should calculate all accelerations but assuming just g simplifies acc port
void eulerStart(fluid_particle* fluid_particles,
                boundary_particle* boundary_particles, param *params)
{
    // Set V (t0 - dt/2)
    double dt_half = params->time_step/2.0;

    for(int i=0; i<params->number_fluid_particles; i++)
    {
        // Velocity at t + dt/2
        double3 v      = fluid_particles[i].v;
        double3 v_half;

        v_half.x = v.x;
        v_half.y = v.y;
        v_half.z = v.z - params->g * dt_half;

        fluid_particles[i].v_half = v_half;
    }
}

// Initialize particles
void initParticles(fluid_particle** fluid_particles, boundary_particle** boundary_particles,
                   AABB* water, AABB* boundary, param* params)
{
    // Allocate fluid particles array
    *fluid_particles = (fluid_particle*) malloc(params->number_fluid_particles * sizeof(fluid_particle));
    // Allocate boundary particles array
    *boundary_particles = (boundary_particle*) malloc(params->number_boundary_particles * sizeof(boundary_particle));

    double spacing = params->spacing_particle;

    // Initialize particle values
    for(int i=0; i<params->number_fluid_particles; i++) {
        (*fluid_particles)[i].a.x = 0.0;
        (*fluid_particles)[i].a.y = 0.0;
        (*fluid_particles)[i].a.z = 0.0;
        (*fluid_particles)[i].v.x = 0.0;
        (*fluid_particles)[i].v.y = 0.0;
        (*fluid_particles)[i].v.z = 0.0;
        (*fluid_particles)[i].density = params->rest_density;
    }

    // Place particles inside bounding volume
    double x,y,z;
    int i = 0;
    for(z=water->min_z; z<=water->max_z; z+=spacing) {
        for(y=water->min_y; y<=water->max_y; y+=spacing) {
            for(x=water->min_x; x<=water->max_x; x+=spacing) {
                if(i < params->number_fluid_particles) {
                    (*fluid_particles)[i].pos.x = x;
                    (*fluid_particles)[i].pos.y = y;
                    (*fluid_particles)[i].pos.z = z;
                    i++;
                }
            }
        }
    }
    params->number_fluid_particles = i;

    // Construct bounding box
    constructBoundaryBox(*boundary_particles, boundary, params);
}

void initParams(AABB* water_volume, AABB* boundary_volume, param* params)
{
    // Boundary box
    boundary_volume->min_x = 0.0;
    boundary_volume->max_x = 1.1;
    boundary_volume->min_y = 0.0;
    boundary_volume->max_y = 1.1;
    boundary_volume->min_z = 0.0;
    boundary_volume->max_z = 1.1;

    // water volume
    water_volume->min_x = 0.1;
    water_volume->max_x = 0.5;
    water_volume->min_y = 0.1;
    water_volume->max_y = 0.5;
    water_volume->min_z = 0.08;
    water_volume->max_z = 0.8;

    // Simulation parameters
    params->number_fluid_particles = 2048;
    params->rest_density = 1000.0;
    params->g = 9.8;
    params->alpha = 0.02;
    params->surface_tension =  0.01;
    params->number_steps = 500; // reduce from 5000
    params->time_step = 0.00035;

    // Mass of each particle
    double volume = (water_volume->max_x - water_volume->min_x)
                  * (water_volume->max_y - water_volume->min_y)
                  * (water_volume->max_z - water_volume->min_z);
    params->mass_particle = params->rest_density * (volume/params->number_fluid_particles);

    // Cube calculated spacing
    params->spacing_particle = pow(volume/params->number_fluid_particles,1.0/3.0);

    // Smoothing radius, h
    params->smoothing_radius = params->spacing_particle;

    // Boundary particles
    int num_x = ceil((boundary_volume->max_x - boundary_volume->min_x)/params->spacing_particle);
    int num_y = ceil((boundary_volume->max_y - boundary_volume->min_y)/params->spacing_particle);
    int num_z = ceil((boundary_volume->max_z - boundary_volume->min_z)/params->spacing_particle);
    int num_boundary_particles = (2 * num_x * num_z) + (2 * num_y * num_z) + (2* num_y * num_z);
    params->number_boundary_particles = num_boundary_particles;

    // Total number of particles
    params->number_particles = params->number_boundary_particles + params->number_fluid_particles;

    // Number of steps before frame needs to be written for 30 fps
    params->steps_per_frame = (int)(1.0/(params->time_step*30.0));

    // Calculate speed of sound for simulation
    double max_height = water_volume->max_y;
    double max_velocity = sqrt(2.0*params->g*max_height);
    params->speed_sound = max_velocity/sqrt(0.01);

    // Minimum stepsize from Courant-Friedrichs-Lewy condition
    double recomend_step = 0.4 * params->smoothing_radius / (params->speed_sound
                                                         * (1+ 0.6*params->alpha));
    printf("Using time step: %f, Minimum recomended %f\n",params->time_step, recomend_step);
}

void finalizeParticles(fluid_particle *fluid_particles, boundary_particle *boundary_particles)
{
    free(fluid_particles);
    free(boundary_particles);
}

int main(int argc, char *argv[])
{
    param params;
    AABB water_volume;
    AABB boundary_volume;
    fluid_particle *fluid_particles = NULL;
    boundary_particle *boundary_particles = NULL;
    initParams(&water_volume, &boundary_volume, &params);

    initParticles(&fluid_particles, &boundary_particles, &water_volume,
                  &boundary_volume, &params);

    eulerStart(fluid_particles, boundary_particles, &params);

    int num_fluid_particles = params.number_fluid_particles;
    int num_boundary_particles = params.number_boundary_particles;

    fluid_particle *d_fluid_particles;
    boundary_particle *d_boundary_particles;
    param *d_params;
    hipMalloc((void**)&d_fluid_particles, num_fluid_particles * sizeof(fluid_particle));
    hipMalloc((void**)&d_boundary_particles, num_boundary_particles * sizeof(boundary_particle)); 
    hipMalloc((void**)&d_params, sizeof(param));
    hipMemcpy(d_fluid_particles, fluid_particles, 
        num_fluid_particles * sizeof(fluid_particle), hipMemcpyHostToDevice);
    hipMemcpy(d_boundary_particles, boundary_particles, 
        num_boundary_particles * sizeof(boundary_particle), hipMemcpyHostToDevice);
    hipMemcpy(d_params, &params, sizeof(param), hipMemcpyHostToDevice);

    dim3 block1D(256);
    dim3 grid1D_FP((num_fluid_particles + 255)/256);
    dim3 grid1D_BP((num_boundary_particles + 255)/256);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    // Main simulation loop
    for(int n=0; n<params.number_steps; n++) {
        updatePressures <<< dim3(grid1D_FP), dim3(block1D) >>> (d_fluid_particles, d_params);
        updateAccelerationsFP <<< dim3(grid1D_FP), dim3(block1D) >>> (d_fluid_particles, d_params);
        updateAccelerationsBP <<< dim3(grid1D_BP), dim3(block1D) >>> (d_fluid_particles, d_boundary_particles, d_params);
        updatePositions <<< dim3(grid1D_FP), dim3(block1D) >>> (d_fluid_particles, d_params);
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("Average execution time of sph kernels: %f (ms)\n", (time * 1e-6f) / params.number_steps);

    hipMemcpy(fluid_particles, d_fluid_particles, 
        num_fluid_particles * sizeof(fluid_particle), hipMemcpyDeviceToHost);

    writeFile(fluid_particles, &params);

    finalizeParticles(fluid_particles, boundary_particles);
    hipFree(d_fluid_particles);
    hipFree(d_boundary_particles);
    hipFree(d_params);
    return 0;
}
