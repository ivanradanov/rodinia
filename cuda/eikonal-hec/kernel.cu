#include "hip/hip_runtime.h"
//
// CUDA implementation of FIM (Fast Iterative Method) for Eikonal equations
//
// Copyright (c) Won-Ki Jeong (wkjeong@unist.ac.kr)
//
// 2016. 2. 4
//

#include "kernel.h"

__device__ DOUBLE get_time_eikonal(DOUBLE a, DOUBLE b, DOUBLE c, DOUBLE s)
{
  DOUBLE ret, tmp;

  // a > b > c
  if(a < b) { tmp = a; a = b; b = tmp; }
  if(b < c) { tmp = b; b = c; c = tmp; }
  if(a < b) { tmp = a; a = b; b = tmp; }

  ret = INF;

  if(c < INF)
  {
    ret = c + s;

    if(ret > b) 
    {  
      tmp = ((b+c) + sqrtf(2.0f*s*s-(b-c)*(b-c)))*0.5f;

      if(tmp > b) ret = tmp; 

      if(ret > a)  {      
        tmp = (a+b+c)/3.0f + sqrtf(2.0f*(a*(b-a)+b*(c-b)+c*(a-c))+3.0f*s*s)/3.0f; 

        if(tmp > a) ret = tmp;
      }
    }
  }

  return ret;
}

__global__ void run_solver(
  const double*__restrict__ spd,
  const bool*__restrict__ mask,
  const DOUBLE *__restrict__ sol_in,
  DOUBLE *__restrict__ sol_out,
  bool *__restrict__ con,
  const uint*__restrict__ list,
  int xdim, int ydim, int zdim,
  int nIter, uint nActiveBlock)
{
  uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

  if(list_idx < nActiveBlock)
  {
    // retrieve actual block index from the active list
    uint block_idx = list[list_idx];

    double F;
    bool isValid;
    uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH;
    uint base_addr = block_idx*blocksize;

    uint xgridlength = xdim/BLOCK_LENGTH;
    uint ygridlength = ydim/BLOCK_LENGTH;
    uint zgridlength = zdim/BLOCK_LENGTH;

    // compute block index
    uint bx = block_idx%xgridlength;
    uint tmpIdx = (block_idx - bx)/xgridlength;
    uint by = tmpIdx%ygridlength;
    uint bz = (tmpIdx-by)/ygridlength;

    uint tx = threadIdx.x;
    uint ty = threadIdx.y;
    uint tz = threadIdx.z;
    uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

    __shared__ DOUBLE _sol[BLOCK_LENGTH+2][BLOCK_LENGTH+2][BLOCK_LENGTH+2];

    // copy global to shared memory
    dim3 idx(tx+1,ty+1,tz+1);

    SOL(idx.x,idx.y,idx.z) = sol_in[base_addr + tIdx];
    F = spd[base_addr + tIdx];
    if(F > 0) F = 1.0/F; // F = 1/f
    isValid = mask[base_addr + tIdx];

    uint new_base_addr, new_tIdx;

    // 1-neighborhood values
    if(tx == 0) 
    {
      if(bx == 0) // end of the grid
      {  
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx + BLOCK_LENGTH-1;
        new_base_addr = (block_idx - 1)*blocksize;  
      }

      SOL(tx,idx.y,idx.z) = sol_in[new_base_addr + new_tIdx];  
    }

    if(tx == BLOCK_LENGTH-1)
    {
      if(bx == xgridlength-1) // end of the grid
      {
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx - (BLOCK_LENGTH-1);
        new_base_addr = (block_idx + 1)*blocksize;  
      }
      SOL(tx+2,idx.y,idx.z) = sol_in[new_base_addr + new_tIdx];  
    }

    if(ty == 0)
    {
      if(by == 0)
      {
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH;
        new_base_addr = (block_idx - xgridlength)*blocksize;
      }

      SOL(idx.x,ty,idx.z) = sol_in[new_base_addr + new_tIdx];
    }

    if(ty == BLOCK_LENGTH-1)
    {
      if(by == ygridlength-1) 
      {
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH;
        new_base_addr = (block_idx + xgridlength)*blocksize;
      }

      SOL(idx.x,ty+2,idx.z) = sol_in[new_base_addr + new_tIdx];
    }

    if(tz == 0)
    {
      if(bz == 0)
      {
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
        new_base_addr = (block_idx - xgridlength*ygridlength)*blocksize;
      }

      SOL(idx.x,idx.y,tz) = sol_in[new_base_addr + new_tIdx];
    }

    if(tz == BLOCK_LENGTH-1)
    {
      if(bz == zgridlength-1) 
      {
        new_tIdx = tIdx;
        new_base_addr = base_addr;
      }
      else
      {
        new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
        new_base_addr = (block_idx + xgridlength*ygridlength)*blocksize;
      }

      SOL(idx.x,idx.y,tz+2) = sol_in[new_base_addr + new_tIdx];
    }

    __syncthreads();

    DOUBLE a,b,c,oldT,newT;

    for(int iter=0; iter<nIter; iter++)  
    {
      //
      // compute new value
      //
      oldT = newT = SOL(idx.x,idx.y,idx.z);

      if(isValid)
      {
        a = min(SOL(tx,idx.y,idx.z),SOL(tx+2,idx.y,idx.z));
        b = min(SOL(idx.x,ty,idx.z),SOL(idx.x,ty+2,idx.z));
        c = min(SOL(idx.x,idx.y,tz),SOL(idx.x,idx.y,tz+2));

        DOUBLE tmp = (DOUBLE) get_time_eikonal(a, b, c, F);

        newT = min(tmp,oldT);
      }
      __syncthreads();  

      if(isValid) SOL(idx.x,idx.y,idx.z) = newT;

      __syncthreads(); // this may not required    
    }

    DOUBLE residue = oldT - newT;

    // write back to global memory
    con[base_addr + tIdx] = (residue < EPS) ? true : false;
    sol_out[base_addr + tIdx] = newT;    
  }
}

__global__ void run_reduction(
  const bool *__restrict__ con,
  bool *__restrict__ listVol,
  const uint *__restrict__ list,
  uint nActiveBlock)
{
  uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

  if(list_idx < nActiveBlock)
  {
    uint block_idx = list[list_idx];

    __shared__ bool conv[BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH];

    uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH/2;
    uint base_addr = block_idx*blocksize*2;
    uint tx = threadIdx.x;
    uint ty = threadIdx.y;
    uint tz = threadIdx.z;
    uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

    conv[tIdx] = con[base_addr + tIdx];
    conv[tIdx + blocksize] = con[base_addr + tIdx + blocksize];

    __syncthreads();

    for(uint i=blocksize; i>0; i/=2)
    {
      if(tIdx < i)
      {
        bool b1, b2;
        b1 = conv[tIdx];
        b2 = conv[tIdx+i];
        conv[tIdx] = (b1 && b2) ? true : false ;
      }
      __syncthreads();
    }

    if(tIdx == 0) 
    {    
      listVol[block_idx] = !conv[0]; // active list is negation of tile convergence (active = not converged)
    }
  }
}

__global__ void run_check_neighbor(
  const double*__restrict__ spd,
  const bool*__restrict__ mask,
  const DOUBLE *__restrict__ sol_in,
  DOUBLE *__restrict__ sol_out,
  bool *__restrict__ con,
  const uint*__restrict__ list,
  int xdim, int ydim, int zdim,
  uint nActiveBlock, uint nTotalBlock)
{

  uint list_idx = blockIdx.y*gridDim.x + blockIdx.x;

  if(list_idx < nTotalBlock)
  {
    double F;
    bool isValid;
    __shared__ DOUBLE _sol[BLOCK_LENGTH+2][BLOCK_LENGTH+2][BLOCK_LENGTH+2];

    uint block_idx = list[list_idx];
    uint blocksize = BLOCK_LENGTH*BLOCK_LENGTH*BLOCK_LENGTH;
    uint base_addr = block_idx*blocksize;

    uint tx = threadIdx.x;
    uint ty = threadIdx.y;
    uint tz = threadIdx.z;
    uint tIdx = tz*BLOCK_LENGTH*BLOCK_LENGTH + ty*BLOCK_LENGTH + tx;

    if(list_idx < nActiveBlock) // copy value
    {
      sol_out[base_addr + tIdx] = sol_in[base_addr + tIdx];
    } 
    else
    {
      uint xgridlength = xdim/BLOCK_LENGTH;
      uint ygridlength = ydim/BLOCK_LENGTH;
      uint zgridlength = zdim/BLOCK_LENGTH;

      // compute block index
      uint bx = block_idx%xgridlength;
      uint tmpIdx = (block_idx - bx)/xgridlength;
      uint by = tmpIdx%ygridlength;
      uint bz = (tmpIdx-by)/ygridlength;

      // copy global to shared memory
      dim3 idx(tx+1,ty+1,tz+1);
      _sol[idx.x][idx.y][idx.z] = sol_in[base_addr + tIdx];
      F = spd[base_addr + tIdx];
      if(F > 0) F = 1.0/F;
      isValid = mask[base_addr + tIdx];

      uint new_base_addr, new_tIdx;

      // 1-neighborhood values
      if(tx == 0) 
      {
        if(bx == 0) // end of the grid
        {  
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx + BLOCK_LENGTH-1;
          new_base_addr = (block_idx - 1)*blocksize;  
        }
        _sol[tx][idx.y][idx.z] = sol_in[new_base_addr + new_tIdx];  
      }

      if(tx == BLOCK_LENGTH-1)
      {
        if(bx == xgridlength-1) // end of the grid
        {
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx - (BLOCK_LENGTH-1);
          new_base_addr = (block_idx + 1)*blocksize;  
        }
        _sol[tx+2][idx.y][idx.z] = sol_in[new_base_addr + new_tIdx];  
      }

      if(ty == 0)
      {
        if(by == 0)
        {
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH;
          new_base_addr = (block_idx - xgridlength)*blocksize;
        }
        _sol[idx.x][ty][idx.z] = sol_in[new_base_addr + new_tIdx];
      }

      if(ty == BLOCK_LENGTH-1)
      {
        if(by == ygridlength-1) 
        {
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH;
          new_base_addr = (block_idx + xgridlength)*blocksize;
        }
        _sol[idx.x][ty+2][idx.z] = sol_in[new_base_addr + new_tIdx];
      }

      if(tz == 0)
      {
        if(bz == 0)
        {
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx + (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
          new_base_addr = (block_idx - xgridlength*ygridlength)*blocksize;
        }
        _sol[idx.x][idx.y][tz] = sol_in[new_base_addr + new_tIdx];
      }

      if(tz == BLOCK_LENGTH-1)
      {
        if(bz == zgridlength-1) 
        {
          new_tIdx = tIdx;
          new_base_addr = base_addr;
        }
        else
        {
          new_tIdx = tIdx - (BLOCK_LENGTH-1)*BLOCK_LENGTH*BLOCK_LENGTH;
          new_base_addr = (block_idx + xgridlength*ygridlength)*blocksize;
        }
        _sol[idx.x][idx.y][tz+2] = sol_in[new_base_addr + new_tIdx];
      }

      __syncthreads();


      DOUBLE a,b,c,oldT,newT;

      //
      // compute new value
      //
      oldT = newT = _sol[idx.x][idx.y][idx.z];

      if(isValid)
      {
        a = min(_sol[tx][idx.y][idx.z],_sol[tx+2][idx.y][idx.z]);
        b = min(_sol[idx.x][ty][idx.z],_sol[idx.x][ty+2][idx.z]);
        c = min(_sol[idx.x][idx.y][tz],_sol[idx.x][idx.y][tz+2]);

        DOUBLE tmp = (DOUBLE) get_time_eikonal(a, b, c, F);
        newT = min(tmp,oldT);

        sol_out[base_addr + tIdx] = newT;
      }
      // write back to global memory
      DOUBLE residue = oldT - newT;
      con[base_addr + tIdx] = (residue < EPS) ? true : false;  
    }
  }
}
