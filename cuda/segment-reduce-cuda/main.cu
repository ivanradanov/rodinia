#include <chrono>
#include <cstdio>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#define VALUE 1

void segreduce (const size_t num_elements, const int repeat ) {
  printf("num_elements = %zu\n", num_elements);

  int *h_in = new int[num_elements];
  int *h_keys = new int[num_elements];
  for (size_t i = 0; i < num_elements; i++) h_in[i] = VALUE;

  int *d_in, *d_keys;
  hipMalloc(&d_keys, num_elements * sizeof(int));
  hipMalloc(&d_in, num_elements * sizeof(int));
  hipMemcpy(d_in, h_in, num_elements * sizeof(int), hipMemcpyHostToDevice);

  for (size_t segment_size = 16;
              segment_size <= 16384;
              segment_size = segment_size * 2) {

    // initialize input keys which depend on the segment size
    for (size_t i = 0; i < num_elements; i++) h_keys[i] = i / segment_size;
    hipMemcpy(d_keys, h_keys, num_elements * sizeof(int), hipMemcpyHostToDevice);

    // allocate output keys and values for each segment size
    const size_t num_segments = num_elements / segment_size;
    int *h_out = new int[num_segments];
    int *d_out, *d_keys_out;
    hipMalloc(&d_keys_out, num_segments * sizeof(int));
    hipMalloc(&d_out, num_segments * sizeof(int));

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++)
      thrust::reduce_by_key(thrust::device, d_keys, d_keys + num_elements, d_in,
                            d_keys_out, d_out);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("num_segments = %zu ", num_segments);
    printf("segment_size = %zu ", segment_size);
    printf("Throughput = %f (G/s)\n", 1.f * num_elements * repeat / time);

    hipMemcpy(h_out, d_out, num_segments * sizeof(int), hipMemcpyDeviceToHost);

    int correct_segment_sum = 0;
    for (size_t i = 0; i < segment_size; i++) {
      correct_segment_sum += h_in[i];
    }

    int errors = 0;
    for (size_t i = 0; i < num_segments; i++) {
      if (h_out[i] != correct_segment_sum) {
        errors++;
        if (errors < 10) {
          printf("segment %zu has sum %d (expected %d)\n", i,
                 h_out[i], correct_segment_sum);
        }
      }
    }

    if (errors > 0) {
      printf("segmented reduction does not agree with the reference! %d "
             "errors!\n", errors);
    }

    hipFree(d_out);
    hipFree(d_keys_out);
    delete[] h_out;
  }

  delete[] h_in;
  delete[] h_keys;
  hipFree(d_in);
  hipFree(d_keys);
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <multiplier> <repeat>\n", argv[0]);
    printf("The total number of elements is 16384 x multiplier\n");
    return 1;
  }
  const int multiplier = atoi(argv[1]);  
  const int repeat = atoi(argv[2]);  

  size_t num_elements = 16384 * size_t(multiplier);
  segreduce(num_elements, repeat);
  return 0;
}
