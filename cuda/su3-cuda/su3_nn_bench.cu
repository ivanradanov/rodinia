#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/resource.h>
#include <math.h>
#include <vector>
#include <iostream>
#include <string>
#include <cassert>
#include <cmath>
#include <complex>
#include <chrono>
typedef std::chrono::system_clock Clock;

#ifndef ITERATIONS
#  define ITERATIONS 100
#endif
#ifndef LDIM
#  define LDIM 32       // Lattice size = LDIM^4
#endif
#ifndef PRECISION
#  define PRECISION 1  // 1->single, 2->double
#endif

// Global variables
unsigned int verbose=1;
int          warmups=1;
// global argc and argv for parsing model specific parameters 
int  g_argc;
char **g_argv;

#include "lattice.hpp"

#ifndef MILC_COMPLEX
template<class T>
bool almost_equal(thrust::complex<T> x, thrust::complex<T> y, double tol)
{
  if (std::isnan(x.real()) || std::isnan(x.imag())
  ||  std::isnan(y.real()) || std::isnan(y.imag()) )
	  return (0);
  return thrust::abs( x - y ) < tol ;
}
#else
template<class T>
bool almost_equal(T x, T y, double tol)
{
  if (std::isnan(x) || std::isnan(y))
	  return (0);
  return std::abs( x - y ) < tol ;
}

// std::isnan() lacks complex support, so need a complex template
template<class T>
bool almost_equal(std::complex<T> x, std::complex<T> y, double tol)
{
  if (std::isnan(x.real()) || std::isnan(x.imag())
  ||  std::isnan(y.real()) || std::isnan(y.imag()) )
	  return (0);
  return std::abs( x - y ) < tol ;
}
#endif

// initializes su3_matrix to a given value
void init_link(su3_matrix *s, Complx val) {
  for(int j=0; j<4; ++j) for(int k=0; k<3; ++k) for(int l=0; l<3; ++l) {
    s[j].e[k][l] = val;
  }
}

// initializes a lattice site 
void make_lattice(site *s, size_t n, Complx val) {
  int nx=n;
  int ny=n;
  int nz=n;
  int nt=n;
  for(int t=0;t<nt;t++) {
    int i=t*nz*ny*nx;
    for(int z=0;z<nz;z++)for(int y=0;y<ny;y++)for(int x=0;x<nx;x++,i++){
      s[i].x=x; s[i].y=y; s[i].z=z; s[i].t=t;
      s[i].index = x+nx*(y+ny*(z+nz*t));
      if( (x+y+z+t)%2 == 0)
        s[i].parity=EVEN;
      else
        s[i].parity=ODD;
      init_link(&s[i].link[0], val);
    }
  }
}

// Include the programming model specific function for su3_mat_nn()
#ifdef USE_THRUST
#include <thrust/host_vector.h>
#endif
#include "mat_nn_cuda.hpp"

// Main
int main(int argc, char **argv)
{
  int iterations = ITERATIONS;
  size_t ldim = LDIM;
  int threads_per_group = 128; // nominally works well across implementations
  int device = -1;             // Let implementation choose the device

  int opt;
  g_argc = argc;
  g_argv = argv;
  // parse command line for parameters
	// the options list must include flags used by the various
  //   su3_mat_nn() implementations internally,
  //   as getopt rearrages the order of arguments and
  //   can screw things up for unknown options
  while ((opt=getopt(argc, argv, ":hi:l:t:v:d:w:n:")) != -1) {
    switch (opt) {
    case 'i':
      iterations = atoi(optarg);
      break;
    case 'l':
      ldim = atoi(optarg);
      break;
    case 't':
      threads_per_group = atoi(optarg);
      break;
    case 'v':
      verbose = atoi(optarg);
      break;
    case 'd':
      device = atoi(optarg);
      break;
    case 'w':
      warmups = atoi(optarg);
      break;
    case 'h':
      fprintf(stderr, "Usage: %s [-i iterations] [-l lattice dimension] \
[-t threads per workgroup] [-d device] [-v verbosity level [0,1,2,3]] [-w warmups]\n", argv[0]);
      exit (1);
    }
  }

  // allocate and initialize the working lattices and B su3 matrices
  size_t total_sites = ldim*ldim*ldim*ldim;
#ifdef MILC_COMPLEX
  std::vector<site> a(total_sites);
  std::vector<su3_matrix> b(4);
  std::vector<site> c(total_sites);
#else
  thrust::host_vector<site> a(total_sites);
  thrust::host_vector<su3_matrix> b(4);
  thrust::host_vector<site> c(total_sites);
#endif

  // initialize the lattices
  make_lattice(a.data(), ldim, Complx{1.0,0.0});
  init_link(b.data(), Complx{1.0/3.0,0.0});

  if (verbose >= 1) {
    printf("Number of sites = %zu^4\n", ldim);
    printf("Executing %d iterations with %d warmups\n", iterations, warmups);
    if (threads_per_group != 0)
      printf("Threads per group = %d\n", threads_per_group);
  }

  // benchmark call
  const double ttotal = su3_mat_nn(a, b, c, total_sites, iterations, threads_per_group, device);
  if (verbose >= 1)
    printf("Total kernel execution time = %f (s)\n", ttotal);

  // calculate flops/s, etc.
  // each matrix multiply is (3*3)*4*(12 mult + 12 add) = 4*(108 mult + 108 add) = 4*216 ops
  const double tflop = (double)iterations * total_sites * 864.0;
  printf("Total GFLOP/s = %.3f\n", tflop / ttotal / 1.0e9);

  const double memory_usage = (double)sizeof(site)*(a.capacity()+c.capacity())+sizeof(su3_matrix)*b.capacity();
  printf("Total GByte/s (GPU memory)  = %.3f\n", iterations * memory_usage / ttotal / 1.0e9);
  fflush(stdout);

  // Verification of the result
  for (size_t i=0;i<total_sites;++i) for(int j=0;j<4;++j)  for(int k=0;k<3;++k)  for(int l=0;l<3;++l) {
    Complx cc = {0.0, 0.0};
    for(int m=0;m<3;m++) {
      #ifdef MILC_COMPLEX
        CMULSUM( a[i].link[j].e[k][m], b[j].e[m][l], cc)
      #else
        cc += a[i].link[j].e[k][m] * b[j].e[m][l];
      #endif
    }

    #ifdef MILC_COMPLEX
       assert(almost_equal(c[i].link[j].e[k][l].real, cc.real, 1E-6));
       assert(almost_equal(c[i].link[j].e[k][l].imag, cc.imag, 1E-6));
    #else
       assert(almost_equal(c[i].link[j].e[k][l], cc, 1E-6));
    #endif
  }

  // check memory usage
  if (verbose >= 2) {
    printf("Total allocation for matrices = %.3f MiB\n", memory_usage / 1048576.0);
    struct rusage usage;
    if (getrusage(RUSAGE_SELF, &usage) == 0)
      printf("Approximate memory usage = %.3f MiB\n", (float)usage.ru_maxrss/1024.0);
  }
}
