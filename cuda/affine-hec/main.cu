#include "hip/hip_runtime.h"
/**********
  Copyright (c) 2017, Xilinx, Inc.
  All rights reserved.

  Redistribution and use in source and binary forms, with or without modification,
  are permitted provided that the following conditions are met:

  1. Redistributions of source code must retain the above copyright notice,
  this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.

  3. Neither the name of the copyright holder nor the names of its contributors
  may be used to endorse or promote products derived from this software
  without specific prior written permission.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,
  THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
  IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
  HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 **********/
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstring>
#include <cmath>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"
#include "kernel.h"

int main(int argc, char** argv)
{
  if (argc != 4)
  {
    printf("Usage: %s <input image> <output image> <iterations>\n", argv[0]) ;
    return -1 ;
  }

  unsigned short input_image[Y_SIZE*X_SIZE] __attribute__((aligned(1024)));
  unsigned short output_image[Y_SIZE*X_SIZE] __attribute__((aligned(1024)));
  unsigned short output_image_ref[Y_SIZE*X_SIZE] __attribute__((aligned(1024)));

  // Read the bit map file into memory and allocate memory for the final image
  std::cout << "Reading input image...\n";

  // Load the input image
  const char *inputImageFilename = argv[1];
  FILE *input_file = fopen(inputImageFilename, "rb");
  if (!input_file)
  {
    printf("Error: Unable to open input image file %s!\n", inputImageFilename);
    return 1;
  }

  printf("\n");
  printf("   Reading RAW Image\n");
  size_t items_read = fread(input_image, sizeof(input_image), 1, input_file);
  printf("   Bytes read = %d\n\n", (int)(items_read * sizeof(input_image)));
  fclose(input_file);

  const int iterations = atoi(argv[3]);

  size_t image_size_bytes = sizeof(unsigned short) * X_SIZE * Y_SIZE;

  unsigned short *d_input_image;
  hipMalloc((void**)&d_input_image, image_size_bytes);
  hipMemcpy(d_input_image, input_image, image_size_bytes, hipMemcpyHostToDevice);

  unsigned short *d_output_image;
  hipMalloc((void**)&d_output_image, image_size_bytes);

  dim3 grids (X_SIZE/16,Y_SIZE/16);
  dim3 threads (16,16);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < iterations; i++) {
    affine<<<grids, threads>>>(d_input_image, d_output_image);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "   Average kernel execution time " << (time * 1e-9f) / iterations << " (s)\n";

  hipMemcpy(output_image, d_output_image, image_size_bytes, hipMemcpyDeviceToHost);
  hipFree(d_input_image);
  hipFree(d_output_image);

  // verify
  affine_reference(input_image, output_image_ref);
  int max_error = 0;
  for (int y = 0; y < Y_SIZE; y++) {
    for (int x = 0; x < X_SIZE; x++) {
      max_error = std::max(max_error, std::abs(output_image[y*X_SIZE+x] - output_image_ref[y*X_SIZE+x]));
    }
  }
  printf("   Max output error is %d\n\n", max_error);

  printf("   Writing RAW Image\n");
  const char *outputImageFilename = argv[2];
  FILE *output_file = fopen(outputImageFilename, "wb");
  if (!output_file)
  {
    printf("Error: Unable to write  image file %s!\n", outputImageFilename);
    return 1;
  }
  size_t items_written = fwrite(output_image, sizeof(output_image), 1, output_file);
  printf("   Bytes written = %d\n\n", (int)(items_written * sizeof(output_image)));
  fclose(output_file);

  return 0 ;
}
