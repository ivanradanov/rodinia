#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#ifndef _PETRINET_KERNEL_H_
#define _PETRINET_KERNEL_H_

#include <stdio.h>
#include "petri.h"

#define BLOCK_SIZE 256
#define BLOCK_SIZE_BITS 8

__device__ 
void fire_transition(char* g_places, int* conflict_array, int tr, 
		     int tc, int step, int N, int thd_thrd) 
{
  int val1, val2, val3, to_update;
  int mark1, mark2;
	
  to_update = 0;
  if (threadIdx.x<thd_thrd) 
    {
      // check if the transition is enabled and conflict-free
      val1 = (tr==0)? (N+N)-1: tr-1;
      val2 = (tr & 0x1)? (tc==N-1? 0: tc+1): tc;
      val3 = (tr==(N+N)-1)? 0: tr+1;
      mark1 = g_places[val1*N+val2];
      mark2 = g_places[tr*N+tc];
      if ( (mark1>0) && (mark2>0) ) 
	{
	  to_update = 1;
	  conflict_array[tr*N+tc] = step;
	}
    }
  __syncthreads();

  if (to_update) 
    {
      // If there are conflicts, transitions on even/odd rows are 
      // kept when the step is even/odd
      to_update = ((step & 0x01) == (tr & 0x01) ) || 
	( (conflict_array[val1*N+val2]!=step) && 
	  (conflict_array[val3*N+((val2==0)? N-1: val2-1)]!=step) );
    }

  // now update state
  // 6 kernel memory accesses 
  if (to_update) 
    {
      g_places[val1*N+val2] = mark1-1;  // the place above
      g_places[tr*N+tc] = mark2-1; // the place on the left
    }
  __syncthreads();
  if (to_update) 
    {
      g_places[val3*N+val2]++;  // the place below
      g_places[tr*N+(tc==N-1? 0: tc+1)]++; // the place on the right
    }
  __syncthreads();
}


__device__ 
void initialize_grid(int* g_places, int NSQUARE2, int seed) 
{
  // N is an even number
  int i;
  int loop_num = NSQUARE2 >> (BLOCK_SIZE_BITS+2);
	
  for (i=0; i<loop_num; i++) 
    {
      g_places[threadIdx.x+(i<<BLOCK_SIZE_BITS)] = 0x01010101;
    }
    
  if (threadIdx.x < (NSQUARE2>>2)-(loop_num<<BLOCK_SIZE_BITS)) 
    {
      g_places[threadIdx.x+(loop_num<<BLOCK_SIZE_BITS)] = 0x01010101;
    }
	
  RandomInit(blockIdx.x+seed);
}

__device__ 
void run_trajectory(int* g_places, int N, int max_steps) 
{
  int step, NSQUARE2, val;

  step = 0;
  NSQUARE2 = (N+N)*N;
	
  while (step<max_steps) 
    {
      BRandom(); // select the next MERS_N (624) transitions

      // process 256 transitions
      val = mt[threadIdx.x]%NSQUARE2;
      fire_transition((char*)g_places, g_places+(NSQUARE2>>2), 
		      val/N, val%N, step+7, N, BLOCK_SIZE);
      
      // process 256 transitions
      val = mt[threadIdx.x+BLOCK_SIZE]%NSQUARE2;
      fire_transition((char*)g_places, g_places+(NSQUARE2>>2), 
		      val/N, val%N, step+11, N, BLOCK_SIZE);
		                
      // process 112 transitions
      if (  threadIdx.x < MERS_N-(BLOCK_SIZE<<1)  ) 
	{
	  val = mt[threadIdx.x+(BLOCK_SIZE<<1)]%NSQUARE2;
	}
      fire_transition((char*)g_places, g_places+(NSQUARE2>>2), 
		      val/N, val%N, step+13, N, MERS_N-(BLOCK_SIZE<<1));

      step += MERS_N>>1; 
      // experiments show that for N>2000 and max_step<20000, 
      // the step increase is larger than 320
    }
}


__device__ 
void compute_reward_stat(int *__restrict__ g_places,
                         float* __restrict__ g_vars,
                         int* __restrict__ g_maxs, 
			 int NSQUARE2) 
{
  float sum = 0;
  int i;
  int max = 0;
  int temp, data; 
  int loop_num = NSQUARE2 >> (BLOCK_SIZE_BITS+2);
  for (i=0; i<=loop_num-1; i++) 
    {  // a bug. i<loop_num should be changed to i<=loop_num-1
      data = g_places[threadIdx.x+(i<<BLOCK_SIZE_BITS)];
	    
      temp = data & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>8) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>16) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>24) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
    }

  i = NSQUARE2>>2;
  i &= 0x0FF;
  loop_num *= BLOCK_SIZE; 
  // I do not know why loop_num<<=BLOCK_SIZE_BITS does not work
  if (threadIdx.x <= i-1) 
    {
      data = g_places[threadIdx.x+loop_num];
	    
      temp = data & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>8) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>16) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
      temp = (data>>24) & 0x0FF;
      sum += temp*temp;
      max = max<temp? temp: max;
    }
	
  ((float*)mt)[threadIdx.x] = (float)sum;
  mt[threadIdx.x+BLOCK_SIZE] = (uint32)max;
  __syncthreads();
		
  for (i=(BLOCK_SIZE>>1); i>0; i = (i>>1) ) 
    {
      if (threadIdx.x<i) 
	{
	  ((float*)mt)[threadIdx.x] += ((float*)mt)[threadIdx.x+i];
	  if (mt[threadIdx.x+BLOCK_SIZE]<mt[threadIdx.x+i+BLOCK_SIZE])
	    mt[threadIdx.x+BLOCK_SIZE] = mt[threadIdx.x+i+BLOCK_SIZE];
	}
      __syncthreads();
    }
		
  if (threadIdx.x==0) 
    {
      g_vars[blockIdx.x] = (((float*)mt)[0])/NSQUARE2-1; 
      // D(X)=E(X^2)-E(X)^2, E(X)=1
      g_maxs[blockIdx.x] = (int)mt[BLOCK_SIZE];
    }
}

// Kernel function for simulating Petri Net for a defined grid
// n: the grid has 2nX2n places and transitions together
// s: steps in each trajectory
// t: number of trajectories
__global__ 
void PetrinetKernel(int* __restrict__ g_s,
                    float* __restrict__ g_v,
                    int*__restrict__ g_m,
                    int n, int s, int seed) 
{
  // block size must be 256
  // n is an even number
  int NSQUARE2 = n*n*2;
  int* g_places = g_s+blockIdx.x*((NSQUARE2>>2)+NSQUARE2);   
  // place numbers, conflict_array
  initialize_grid(g_places, NSQUARE2, seed);
  
  run_trajectory(g_places, n, s);
  compute_reward_stat(g_places, g_v, g_m, NSQUARE2);
}

#endif // #ifndef _PETRINET_KERNEL_H_
