#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void rotate_matrix_parallel (float *matrix, const int n) {
  int layer = blockIdx.x * blockDim.x + threadIdx.x;
  if (layer < n/2) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;

      float top = matrix[first*n+i]; // save top
      // left -> top
      matrix[first*n+i] = matrix[(last-offset)*n+first];

      // bottom -> left
      matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

      // right -> bottom
      matrix[last*n+(last-offset)] = matrix[i*n+last];

      // top -> right
      matrix[i*n+last] = top; // right <- saved top
    }
  }
}

void rotate_matrix_serial(float *matrix, int n) {

  for (int layer = 0; layer < n / 2; ++layer) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;
        float top = matrix[first*n+i]; // save top
        // left -> top
        matrix[first*n+i] = matrix[(last-offset)*n+first];

        // bottom -> left
        matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

        // right -> bottom
        matrix[last*n+(last-offset)] = matrix[i*n+last];

        // top -> right
        matrix[i*n+last] = top; // right <- saved top
    }
  }
}

int main(int argc, char** argv) {
  if (argc != 3) {
    printf("Usage: %s <matrix size> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

  float *serial_res = (float*) aligned_alloc(1024, n*n*sizeof(float));
  float *parallel_res = (float*) aligned_alloc(1024, n*n*sizeof(float));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      serial_res[i*n+j] = parallel_res[i*n+j] = i*n+j;

  for (int i = 0; i < repeat; i++) {
    rotate_matrix_serial(serial_res, n);
  }

  float *d_parallel_res;
  hipMalloc((void**)&d_parallel_res, n*n*sizeof(float));
  hipMemcpy(d_parallel_res, parallel_res, n*n*sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();
  
  for (int i = 0; i < repeat; i++) {
    rotate_matrix_parallel<<<(n/2+255)/256, 256>>>(d_parallel_res, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(parallel_res, d_parallel_res, n*n*sizeof(float), hipMemcpyDeviceToHost);

  bool ok = true;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (serial_res[i*n+j] != parallel_res[i*n+j]) {
        ok = false;
        break;
      }
    }
  }

  printf("%s\n", ok ? "PASS" : "FAIL");

  free(serial_res);
  free(parallel_res);
  hipFree(d_parallel_res);
  return 0;
}
