#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tc.h"

template <typename T, unsigned int blockSize, unsigned int dataLength>
__device__ void conditionalWarpReduce(volatile T *sharedData)
{
  if(blockSize >= dataLength)
  {
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

template <typename T, unsigned int blockSize>
__device__ void warpReduce(T* __restrict__ outDataPtr,
    volatile T* __restrict__ sharedData)
{
  conditionalWarpReduce<T, blockSize, 64>(sharedData);
  conditionalWarpReduce<T, blockSize, 32>(sharedData);
  conditionalWarpReduce<T, blockSize, 16>(sharedData);
  conditionalWarpReduce<T, blockSize, 8>(sharedData);
  conditionalWarpReduce<T, blockSize, 4>(sharedData);
  if(threadIdx.x == 0)
    *outDataPtr = sharedData[0] + sharedData[1];
}

template <typename T, unsigned int blockSize, unsigned int dataLength>
__device__ void conditionalReduce(volatile T* __restrict__ sharedData)
{
  if(blockSize >= dataLength)
  {
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }

  if((blockSize < dataLength) && (blockSize > (dataLength/2)))
  {
    if(threadIdx.x+(dataLength/2) < blockSize)
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

template <typename T, unsigned int blockSize>
__device__ void blockReduce(T* __restrict__ outGlobalDataPtr,
    volatile T* __restrict__ sharedData)
{
  __syncthreads();
  conditionalReduce<T, blockSize, 256>(sharedData);
  conditionalReduce<T, blockSize, 128>(sharedData);

  warpReduce<T, blockSize>(outGlobalDataPtr, sharedData);
  __syncthreads();
}

// 
template <typename T>
__device__ void initialize(const T diag_id,
    const T u_len, T v_len,
    T* const __restrict__ u_min, T* const __restrict__ u_max,
    T* const __restrict__ v_min, T* const __restrict__ v_max,
    T* const __restrict__ found)
{
  if (diag_id == 0)
  {
    *u_min=*u_max=*v_min=*v_max=0;
    *found=1;
  }
  else if (diag_id < u_len)
  {
    *u_min=0; *u_max=diag_id;
    *v_max=diag_id;*v_min=0;
  }
  else if (diag_id < v_len)
  {
    *u_min=0; *u_max=u_len;
    *v_max=diag_id;*v_min=diag_id-u_len;
  }
  else
  {
    *u_min=diag_id-v_len; *u_max=u_len;
    *v_min=diag_id-u_len; *v_max=v_len;
  }
}

template <typename T>
__device__ void calcWorkPerThread(const T uLength,
    const T vLength, const T threadsPerIntersection,
    const T threadId,
    T * const __restrict__ outWorkPerThread,
    T * const __restrict__ outDiagonalId)
{
  T totalWork = uLength + vLength;
  T remainderWork = totalWork%threadsPerIntersection;
  T workPerThread = totalWork/threadsPerIntersection;

  T longDiagonals = (threadId > remainderWork) ? remainderWork:threadId;
  T shortDiagonals = (threadId > remainderWork) ? threadId - remainderWork:0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

template <typename T>
__device__ void bSearch(
    unsigned int found,
    const T diagonalId,
    T const * const __restrict__ uNodes,
    T const * const __restrict__ vNodes,
    T const * const __restrict__ uLength,
    T * const __restrict__ outUMin,
    T * const __restrict__ outUMax,
    T * const __restrict__ outVMin,
    T * const __restrict__ outVMax,
    T * const __restrict__ outUCurr,
    T * const __restrict__ outVCurr)
{
  T length;
  while(!found)
  {
    *outUCurr = (*outUMin + *outUMax)>>1;
    *outVCurr = diagonalId - *outUCurr;
    if(*outVCurr >= *outVMax)
    {
      length = *outUMax - *outUMin;
      if(length == 1)
      {
        found = 1;
        continue;
      }
    }

    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
    if(comp1 && !comp2)
    {
      found = 1;
    }
    else if(comp1)
    {
      *outVMin = *outVCurr;
      *outUMax = *outUCurr;
    }
    else
    {
      *outVMax = *outVCurr;
      *outUMin = *outUCurr;
    }
  }

  if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
      (*outUCurr > 0) && (*outUCurr < (*uLength - 1)))
  {
    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
    unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
    if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
  }
}

template <typename T>
__device__ T fixThreadWorkEdges(const T uLength, const T vLength,
    T * const __restrict__ uCurr, T * const __restrict__ vCurr,
    T const * const __restrict__ uNodes, T const * const __restrict__ vNodes)
{
  unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) &&
    (uNodes[*uCurr-1] == vNodes[*vCurr]);
  unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) &&
    (vNodes[*vCurr-1] == uNodes[*uCurr]);
  *uCurr += vBigger;
  *vCurr += uBigger;

  return (uBigger + vBigger);
}

template <typename T>
__device__ void intersectCount(const T uLength, const T vLength,
    T const * const __restrict__ uNodes, T const * const __restrict__ vNodes,
    T * const __restrict__ uCurr, T * const __restrict__ vCurr,
    T * const __restrict__ workIndex, T * const __restrict__ workPerThread,
    T * const __restrict__ triangles, T found)
{
  if((*uCurr < uLength) && (*vCurr < vLength))
  {
    T comp;
    while(*workIndex < *workPerThread)
    {
      comp = uNodes[*uCurr] - vNodes[*vCurr];
      *triangles += (comp == 0);
      *uCurr += (comp <= 0);
      *vCurr += (comp >= 0);
      *workIndex += (comp == 0) + 1;

      if((*vCurr == vLength) || (*uCurr == uLength))
      {
        break;
      }
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}

// u_len < v_len
template <typename T>
__device__ T count_triangles(T u, T const * const __restrict__ u_nodes, T u_len,
    T v, T const * const __restrict__ v_nodes, T v_len, T threads_per_block,
    volatile T* __restrict__ firstFound, T tId)
{
  // Partitioning the work to the multiple thread of a single GPU processor. 
  // The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
  T work_per_thread, diag_id;
  calcWorkPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
  T triangles = 0;
  T work_index = 0,found=0;
  T u_min,u_max,v_min,v_max,u_curr,v_curr;

  firstFound[tId]=0;

  if(work_per_thread>0)
  {
    // For the binary search, we are figuring out the initial poT of search.
    initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    u_curr = 0; v_curr = 0;

    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    T sum = fixThreadWorkEdges(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    work_index += sum;
    if(tId > 0) firstFound[tId-1] = sum;
    triangles += sum;
    intersectCount(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
        &work_index, &work_per_thread, &triangles, firstFound[tId]);
  }
  return triangles;
}

template <typename T>
__device__ void calcWorkPerBlock(const T numVertices,
    T * const __restrict__ outMpStart,
    T * const __restrict__ outMpEnd)
{
  T verticesPerMp = numVertices/gridDim.x;
  T remainderBlocks = numVertices % gridDim.x;
  T extraVertexBlocks = (blockIdx.x > remainderBlocks)?
    remainderBlocks:blockIdx.x;
  T regularVertexBlocks = (blockIdx.x > remainderBlocks)?
    blockIdx.x - remainderBlocks:0;

  T mpStart = ((verticesPerMp+1)*extraVertexBlocks)
    + (verticesPerMp*regularVertexBlocks);
  *outMpStart = mpStart;
  *outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}

template <typename T, unsigned int blockSize>
__global__ void count_all_trianglesGPU (
    const T nv,
    T const * const __restrict__ d_off,
    T const * const __restrict__ d_ind,
    T * const __restrict__ outPutTriangles,
    const T threads_per_block,
    const T number_blocks, const T shifter)
{
  T tx = threadIdx.x;
  T this_mp_start, this_mp_stop;
  calcWorkPerBlock(nv, &this_mp_start, &this_mp_stop);

  __shared__ T s_triangles[blockSize];
  __shared__ T firstFound[blockSize];

  T adj_offset=tx>>shifter;
  T* firstFoundPos=firstFound + (adj_offset<<shifter);
  for (T src = this_mp_start; src < this_mp_stop; src++)
  {
    T srcLen=d_off[src+1]-d_off[src];
    T tCount = 0;
    for(T iter=d_off[src]+adj_offset; iter<d_off[src+1]; iter+=number_blocks)
    {
      T dest = d_ind[iter];
      T destLen = d_off[dest+1]-d_off[dest];
      bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
      if(avoidCalc)
        continue;

      bool sourceSmaller = (srcLen<destLen);
      T small = sourceSmaller? src : dest;
      T large = sourceSmaller? dest : src;
      T small_len = sourceSmaller? srcLen : destLen;
      T large_len = sourceSmaller? destLen : srcLen;

      T const * const small_ptr = d_ind + d_off[small];
      T const * const large_ptr = d_ind + d_off[large];
      tCount += count_triangles(
          small, small_ptr, small_len,
          large, large_ptr, large_len,
          threads_per_block, firstFoundPos, tx%threads_per_block);
    }
    s_triangles[tx] = tCount;
    blockReduce<T, blockSize>(&outPutTriangles[src],s_triangles);
  }
}

// call triangle count kernel
template <typename T>
void kernelCall(
    unsigned int numberBlocks,
    unsigned int numberThreads,
    const T nv,
    T const * const __restrict__ d_off,
    T const * const __restrict__ d_ind,
    T * const __restrict__ outPutTriangles,
    const T threads_per_block,
    const T number_blocks,
    const T shifter)
{
  switch (numberThreads) {
    case 32: 
      count_all_trianglesGPU<T, 32> <<<numberBlocks, 32>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    case 64: 
      count_all_trianglesGPU<T, 64> <<<numberBlocks, 64>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    case 96: 
      count_all_trianglesGPU<T, 96> <<<numberBlocks, 96>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    case 128: 
      count_all_trianglesGPU<T, 128> <<<numberBlocks, 128>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    case 192: 
      count_all_trianglesGPU<T, 192> <<<numberBlocks, 192>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    case 256: 
      count_all_trianglesGPU<T, 256> <<<numberBlocks, 256>>> (
       nv, d_off, d_ind, outPutTriangles, threads_per_block, number_blocks, shifter);
      break;
    default: ;
  }
}

template <typename T>
void allParamTestGPURun(Param param)
{
  T* offsetVector;
  T* indexVector;
  T vertexCount;
  T edgeCount;

  bool ok = readGraph<T>(param.fileName, offsetVector, indexVector, vertexCount, edgeCount);
  if (!ok) return;

  hipDeviceSynchronize();
  auto memAllocStart = std::chrono::system_clock::now();

  T *dOffset, *dIndex, *dTriangle;

  hipMalloc((void**)&dOffset, (vertexCount + 1) * sizeof(T));
  hipMemcpy(dOffset, offsetVector, (vertexCount + 1) * sizeof(T), hipMemcpyHostToDevice);

  hipMalloc((void**)&dIndex, edgeCount * sizeof(T));
  hipMemcpy(dIndex, indexVector, edgeCount * sizeof(T), hipMemcpyHostToDevice);

  T *triangle = new T[vertexCount];

  hipMalloc((void**)&dTriangle, vertexCount * sizeof(T));

  auto memAllocEnd = std::chrono::system_clock::now();
  std::chrono::duration<float, std::milli> memAllocDuration = memAllocEnd - memAllocStart;

  // output file name 
  std::string separator = std::string(".o.");
  std::string fileOutName = param.fileName + separator + std::to_string(param.blocks);
  std::ofstream writeFile(fileOutName);

  writeFile<<"paramBlockSize"<<"\t"
           <<"paramThreadsPerIntsctn"<<"\t"
           <<"memAllocDuration(ms)"<<"\t"
           <<"execDuration(ms)"<<"\t"
           <<"kernelDuration(ms)"<<"\t"
           <<"execDuration+memAllocDuration(ms)"<<"\t"
           <<"sumTriangles"<<"\n";

  for(auto paramBlockSize : globalParam::blockSizeParam)
  {
    for(auto paramThreadsPerIntsctn : globalParam::threadPerIntersectionParam)
    {
      hipMemset(dTriangle, (T)0, vertexCount * sizeof(T));
      
      // timing data transfer and kernel execution on a device
      auto execStart = std::chrono::system_clock::now();
      unsigned int blocks = param.blocks;
      unsigned int blockSize = paramBlockSize;
      T threadsPerIntsctn = paramThreadsPerIntsctn;
      T intsctnPerBlock = paramBlockSize/paramThreadsPerIntsctn;
      T threadShift = std::log2(paramThreadsPerIntsctn);

      // timing kernel execution on a device
      auto krnlStart = std::chrono::system_clock::now();
      kernelCall<T>(blocks, blockSize, vertexCount, dOffset,
          dIndex, dTriangle, threadsPerIntsctn, intsctnPerBlock, threadShift);
      hipDeviceSynchronize();
      auto krnlEnd = std::chrono::system_clock::now();
      std::chrono::duration<float, std::milli> krnlDuration = krnlEnd - krnlStart;

      hipMemcpy(triangle, dTriangle, vertexCount * sizeof(T), hipMemcpyDeviceToHost);

      auto execEnd = std::chrono::system_clock::now();
      std::chrono::duration<float, std::milli> execDuration = execEnd - execStart;
      
      T sumTriangles = 0;
      for (int i = 0; i < vertexCount; i++)
        sumTriangles += triangle[i];

      writeFile<<paramBlockSize<<"\t"
               <<paramThreadsPerIntsctn<<"\t"
               <<memAllocDuration.count()<<"\t"
               <<execDuration.count()<<"\t"
               <<krnlDuration.count()<<"\t"
               <<execDuration.count()+memAllocDuration.count()<<"\t"
               <<sumTriangles<<"\n";
    }
  }
  writeFile.close();
    
  delete[] offsetVector;
  delete[] indexVector;
  delete[] triangle;
  hipFree(dOffset);
  hipFree(dIndex);
  hipFree(dTriangle);
}

template void allParamTestGPURun<int>(Param param);
template void allParamTestGPURun<long>(Param param);
