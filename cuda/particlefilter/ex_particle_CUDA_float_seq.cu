
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NN 200

#define STRINGIZE_NX(A) #A
#define STRINGIZE(A) STRINGIZE_NX(A)

#define NNS STRINGIZE(NN)
#define threads_per_block 2

#define check_error(X) X

__global__ void normalize_weights_kernel1(double * weights, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double sumWeights;
    
    if(0 == threadIdx.x) {
        sumWeights = 1 / ((double) (Nparticles));
    }
    
    __syncthreads();
    
    if (i < Nparticles) {
        weights[i] = 1 / ((double) (Nparticles)) / sumWeights;
    }
}


__global__ void normalize_weights_kernel2(double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double u1;
    
    if (i == 0) {
        u[0] = 0;
    }
    
    __syncthreads();
    
    if(0 == threadIdx.x) 
        u1 = u[0];
    
    __syncthreads();
        
    if (i < Nparticles) {
        u[i] = u1 + i / ((double) (Nparticles));
    }
}

int main() {

    int Nparticles = atoi(NNS);
    
    int x, y;
    //initial weights are all equal (1/Nparticles)
    double * weights = (double *) malloc(sizeof (double) *Nparticles);
    for (x = 0; x < Nparticles; x++) {
        weights[x] = 1 / ((double) (Nparticles));
    }

    //GPU copies of arrays
    double * weights_GPU;

    double * u_GPU;

    //CUDA memory allocation
    check_error(hipMalloc((void **) &u_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &weights_GPU, sizeof (double) *Nparticles));


    int indX, indY;
    //start send
    check_error(hipMemcpy(weights_GPU, weights, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    int num_blocks = ( Nparticles + threads_per_block - 1) / threads_per_block;
    printf("nb=%d nn=%d\n", num_blocks, num_blocks * threads_per_block);

    {
      normalize_weights_kernel1 <<< num_blocks, threads_per_block >>> (weights_GPU, Nparticles);
        
      normalize_weights_kernel2 <<< num_blocks, threads_per_block >>> (u_GPU, Nparticles);

      for (int i=0; i<Nparticles; i++) {
        int index = -1;
        int x;

        for (x = 0; x < Nparticles; x++) {
            if (weights_GPU[x] >= u_GPU[i]) {
                index = x;
            }
        }

        printf("idx=%d\n", index);
      }
      
    }


    //block till kernels are finished

    hipFree(u_GPU);

    //CUDA freeing of memory
    hipFree(weights_GPU);
    free(weights);
    return 0;
}
