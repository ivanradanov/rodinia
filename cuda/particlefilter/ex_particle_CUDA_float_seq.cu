
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#include <time.h>
#define BLOCK_X 16
#define BLOCK_Y 16
#define PI 3.1415926535897932

// when doing verification
#if 1
#define malloc(X) calloc((X), 1)
#endif

const int threads_per_block = 512;


long long get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000000) +tv.tv_usec;
}
// Returns the number of seconds elapsed between the two specified times

double elapsed_time(long long start_time, long long end_time) {
    return (double) (end_time - start_time) / (1000 * 1000);
}

/*****************************
 * CHECK_ERROR
 * Checks for CUDA errors and prints them to the screen to help with
 * debugging of CUDA related programming
 *****************************/
#define check_error(X) X

/********************************
 * CALC LIKELIHOOD SUM
 * DETERMINES THE LIKELIHOOD SUM BASED ON THE FORMULA: SUM( (IK[IND] - 100)^2 - (IK[IND] - 228)^2)/ 100
 * param 1 I 3D matrix
 * param 2 current ind array
 * param 3 length of ind array
 * returns a double representing the sum
 ********************************/
__device__ double calcLikelihoodSum(unsigned char * I, int * ind, int numOnes, int index) {
    double likelihoodSum = 0.0;
    int x;
    for (x = 0; x < numOnes; x++) {
        likelihoodSum += (pow((double) (I[ind[index * numOnes + x]] - 100), 2) - pow((double) (I[ind[index * numOnes + x]] - 228), 2)) / 50.0;
    }
    return likelihoodSum;
}

/****************************
CDF CALCULATE
CALCULATES CDF
param1 CDF
param2 weights
param3 Nparticles
 *****************************/

/*****************************
 * RANDU
 * GENERATES A UNIFORM DISTRIBUTION
 * returns a double representing a randomily generated number from a uniform distribution with range [0, 1)
 ******************************/
__device__ double d_randu(int * seed, int index) {
  return (1000 + index) / 5000000;

    int M = INT_MAX;
    int A = 1103515245;
    int C = 12345;
    int num = A * seed[index] + C;
    seed[index] = num % M;

    return fabs(seed[index] / ((double) M));
}

__device__ double d_randn(int * seed, int index) {
  return (1000 + index) / 5000000;
    //Box-Muller algortihm
    double pi = 3.14159265358979323846;
    double u = d_randu(seed, index);
    double v = d_randu(seed, index);
    double cosine = cos(2 * pi * v);
    double rt = -2 * log(u);
    return sqrt(rt) * cosine;
}

/** added this function. was missing in original double version.
 * Takes in a double and returns an integer that approximates to that double
 * @return if the mantissa < .5 => return value < input value; else return value > input value
 */
__device__ double dev_round_double(double value) {
    int newValue = (int) (value);
    if (value - newValue < .5f)
        return newValue;
    else
        return newValue++;
}

/*****************************
 * CUDA Find Index Kernel Function to replace FindIndex
 * param1: arrayX
 * param2: arrayY
 * param3: CDF
 * param4: u
 * param5: xj
 * param6: yj
 * param7: weights
 * param8: Nparticles
 *****************************/
__global__ void find_index_kernel(double * CDF, double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;

    if (i < Nparticles) {

        int index = -1;
        int x;

        for (x = 0; x < Nparticles; x++) {
            if (CDF[x] >= u[i]) {
                index = x;
                //break;
            }
        }

        printf("idx=%d\n", index);

    }
    __syncthreads();
}

__global__ void normalize_weights_kernel1(double * weights, int Nparticles, double* partial_sums, double * CDF, double * u, int * seed) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double sumWeights;
    
    if(0 == threadIdx.x) {
        sumWeights = partial_sums[0];
    }
    
    __syncthreads();
    
    if (i < Nparticles) {
        weights[i] = weights[i] / sumWeights;
    }
}


__global__ void normalize_weights_kernel2(double* weights, double * CDF, double * u, int * seed, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double u1;
    
    if (i == 0) {
        CDF[0] = weights[0];
        for (int x = 1; x < Nparticles; x++) {
            CDF[x] = weights[x] + CDF[x - 1];
        }
        u[0] = (1 / ((double) (Nparticles))) * d_randu(seed, 0); // do this to allow all threads in all blocks to use the same u1
    }
    
    __syncthreads();
    
    if(0 == threadIdx.x) 
        u1 = u[0];
    
    __syncthreads();
        
    if (i < Nparticles) {
        u[i] = u1 + i / ((double) (Nparticles));
    }
}


/*****************************
 * CUDA Likelihood Kernel Function to replace FindIndex
 * param1: arrayX
 * param2: arrayY
 * param2.5: CDF
 * param3: ind
 * param4: objxy
 * param5: likelihood
 * param6: I
 * param6.5: u
 * param6.75: weights
 * param7: Nparticles
 * param8: countOnes
 * param9: max_size
 * param10: k
 * param11: IszY
 * param12: Nfr
 *****************************/
__global__ void likelihood_kernel(double * arrayX, double * arrayY, double * xj, double * yj, double * CDF, int * ind, int * objxy, double * likelihood, unsigned char * I, double * u, double * weights, int Nparticles, int countOnes, int max_size, int k, int IszY, int Nfr, int *seed, double* partial_sums) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    int y;
    
    int indX, indY; 
    __shared__ double buffer[512];
    if (i < Nparticles) {
        arrayX[i] = xj[i]; 
        arrayY[i] = yj[i]; 

        //weights[i] = 1 / ((double) (Nparticles)); //Donnie - moved this line from end of find_index_kernel to prevent all weights from being reset before calculating position on final iteration.

        arrayX[i] = arrayX[i] + 1.0 + 5.0 * d_randn(seed, i);
        arrayY[i] = arrayY[i] - 2.0 + 2.0 * d_randn(seed, i);
        
    }

    __syncthreads();

    if (i < Nparticles) {
        for (y = 0; y < countOnes; y++) {
            //added dev_round_double() to be consistent with roundDouble
            indX = dev_round_double(arrayX[i]) + objxy[y * 2 + 1];
            indY = dev_round_double(arrayY[i]) + objxy[y * 2];

            ind[i * countOnes + y] = abs(indX * IszY * Nfr + indY * Nfr + k);
            if (ind[i * countOnes + y] >= max_size)
                ind[i * countOnes + y] = 0;
        }
        likelihood[i] = calcLikelihoodSum(I, ind, countOnes, i);

        likelihood[i] = likelihood[i] / countOnes;

        weights[i] = weights[i] * exp(likelihood[i]); //Donnie Newell - added the missing exponential function call

    }

    buffer[threadIdx.x] = 0.0;

    __syncthreads();

    if (i < Nparticles) {

        buffer[threadIdx.x] = weights[i];
    }

    __syncthreads();

    //this doesn't account for the last block that isn't full
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            buffer[threadIdx.x] += buffer[threadIdx.x + s];
        }

        __syncthreads();

    }
    if (threadIdx.x == 0) {
        partial_sums[blockIdx.x] = buffer[0];
    }

    __syncthreads();


}

/** 
 * Takes in a double and returns an integer that approximates to that double
 * @return if the mantissa < .5 => return value < input value; else return value > input value
 */
double roundDouble(double value) {
    int newValue = (int) (value);
    if (value - newValue < .5)
        return newValue;
    else
        return newValue++;
}

/**
 * Fills a radius x radius matrix representing the disk
 * @param disk The pointer to the disk to be made
 * @param radius  The radius of the disk to be made
 */
void strelDisk(int * disk, int radius) {
    int diameter = radius * 2 - 1;
    int x, y;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            double distance = sqrt(pow((double) (x - radius + 1), 2) + pow((double) (y - radius + 1), 2));
            if (distance < radius)
                disk[x * diameter + y] = 1;
        }
    }
}

/**
 * Fills a 2D array describing the offsets of the disk object
 * @param se The disk object
 * @param numOnes The number of ones in the disk
 * @param neighbors The array that will contain the offsets
 * @param radius The radius used for dilation
 */
void getneighbors(int * se, int numOnes, int * neighbors, int radius) {
    int x, y;
    int neighY = 0;
    int center = radius - 1;
    int diameter = radius * 2 - 1;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            if (se[x * diameter + y]) {
                neighbors[neighY * 2] = (int) (y - center);
                neighbors[neighY * 2 + 1] = (int) (x - center);
                neighY++;
            }
        }
    }
}

/**
 * The implementation of the particle filter using OpenMP for many frames
 * @see http://openmp.org/wp/
 * @note This function is designed to work with a video of several frames. In addition, it references a provided MATLAB function which takes the video, the objxy matrix and the x and y arrays as arguments and returns the likelihoods
 * @param I The video to be run
 * @param IszX The x dimension of the video
 * @param IszY The y dimension of the video
 * @param Nfr The number of frames
 * @param seed The seed array used for random number generation
 * @param Nparticles The number of particles to be used
 */
void particleFilter(unsigned char * I, int IszX, int IszY, int Nfr, int * seed, int Nparticles) {
    int max_size = IszX * IszY*Nfr;
    //original particle centroid
    double xe = roundDouble(IszY / 2.0);
    double ye = roundDouble(IszX / 2.0);

    //expected object locations, compared to center
    int radius = 5;
    int diameter = radius * 2 - 1;
    int * disk = (int*) malloc(diameter * diameter * sizeof (int));
    strelDisk(disk, radius);
    int countOnes = 0;
    int x, y;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            if (disk[x * diameter + y] == 1)
                countOnes++;
        }
    }
    int * objxy = (int *) malloc(countOnes * 2 * sizeof (int));
    getneighbors(disk, countOnes, objxy, radius);
    //initial weights are all equal (1/Nparticles)
    double * weights = (double *) malloc(sizeof (double) *Nparticles);
    for (x = 0; x < Nparticles; x++) {
        weights[x] = 1 / ((double) (Nparticles));
    }

    //initial likelihood to 0.0
    double * likelihood = (double *) malloc(sizeof (double) *Nparticles);
    double * arrayX = (double *) malloc(sizeof (double) *Nparticles);
    double * arrayY = (double *) malloc(sizeof (double) *Nparticles);
    double * xj = (double *) malloc(sizeof (double) *Nparticles);
    double * yj = (double *) malloc(sizeof (double) *Nparticles);
    double * CDF = (double *) malloc(sizeof (double) *Nparticles);

    //GPU copies of arrays
    double * arrayX_GPU;
    double * arrayY_GPU;
    double * xj_GPU;
    double * yj_GPU;
    double * CDF_GPU;
    double * likelihood_GPU;
    unsigned char * I_GPU;
    double * weights_GPU;
    int * objxy_GPU;

    int * ind = (int*) malloc(sizeof (int) *countOnes * Nparticles);
    int * ind_GPU;
    double * u = (double *) malloc(sizeof (double) *Nparticles);
    double * u_GPU;
    int * seed_GPU;
    double* partial_sums;

    //CUDA memory allocation
    check_error(hipMalloc((void **) &arrayX_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &arrayY_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &xj_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &yj_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &CDF_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &u_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &likelihood_GPU, sizeof (double) *Nparticles));
    //set likelihood to zero
    check_error(hipMemset((void *) likelihood_GPU, 0, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &weights_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &I_GPU, sizeof (unsigned char) *IszX * IszY * Nfr));
    check_error(hipMalloc((void **) &objxy_GPU, sizeof (int) *2 * countOnes));
    check_error(hipMalloc((void **) &ind_GPU, sizeof (int) *countOnes * Nparticles));
    check_error(hipMalloc((void **) &seed_GPU, sizeof (int) *Nparticles));
    check_error(hipMalloc((void **) &partial_sums, sizeof (double) *Nparticles));


    //Donnie - this loop is different because in this kernel, arrayX and arrayY
    //  are set equal to xj before every iteration, so effectively, arrayX and 
    //  arrayY will be set to xe and ye before the first iteration.
    for (x = 0; x < Nparticles; x++) {

        xj[x] = xe;
        yj[x] = ye;

    }

    int indX, indY;
    //start send
    check_error(hipMemcpy(I_GPU, I, sizeof (unsigned char) *IszX * IszY*Nfr, hipMemcpyHostToDevice));
    check_error(hipMemcpy(objxy_GPU, objxy, sizeof (int) *2 * countOnes, hipMemcpyHostToDevice));
    check_error(hipMemcpy(weights_GPU, weights, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(xj_GPU, xj, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(yj_GPU, yj, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(seed_GPU, seed, sizeof (int) *Nparticles, hipMemcpyHostToDevice));
    int num_blocks = ceil((double) Nparticles / (double) threads_per_block);


    Nfr = 2;

    {
        int k = 1;
      likelihood_kernel <<< num_blocks, threads_per_block >>> (arrayX_GPU, arrayY_GPU, xj_GPU, yj_GPU, CDF_GPU, ind_GPU, objxy_GPU, likelihood_GPU, I_GPU, u_GPU, weights_GPU, Nparticles, countOnes, max_size, k, IszY, Nfr, seed_GPU, partial_sums);

        int x;
        double sum = 0.0;
        int num_blocks = ceil((double) Nparticles / (double) threads_per_block);
        for (x = 0; x < num_blocks; x++) {
            sum += partial_sums[x];
        }
        partial_sums[0] = sum;

      normalize_weights_kernel1 <<< num_blocks, threads_per_block >>> (weights_GPU, Nparticles, partial_sums, CDF_GPU, u_GPU, seed_GPU);
      normalize_weights_kernel2 <<< num_blocks, threads_per_block >>> (weights_GPU, CDF_GPU, u_GPU, seed_GPU, Nparticles);

      find_index_kernel <<< num_blocks, threads_per_block >>> (CDF_GPU, u_GPU, Nparticles);
      
    }


    //block till kernels are finished

    hipFree(xj_GPU);
    hipFree(yj_GPU);
    hipFree(CDF_GPU);
    hipFree(u_GPU);
    hipFree(likelihood_GPU);
    hipFree(I_GPU);
    hipFree(objxy_GPU);
    hipFree(ind_GPU);
    hipFree(seed_GPU);
    hipFree(partial_sums);

    //CUDA freeing of memory
    hipFree(weights_GPU);
    hipFree(arrayY_GPU);
    hipFree(arrayX_GPU);

    //free regular memory
    free(likelihood);
    free(arrayX);
    free(arrayY);
    free(xj);
    free(yj);
    free(CDF);
    free(ind);
    free(u);
}

int main(int argc, char * argv[]) {

    int IszX, IszY, Nfr, Nparticles;

    //converting a string to a integer
    if (sscanf(argv[2], "%d", &IszX) == EOF) {
    }

    if (IszX <= 0) {
    }

    //converting a string to a integer
    if (sscanf(argv[4], "%d", &IszY) == EOF) {
    }

    if (IszY <= 0) {
    }

    //converting a string to a integer
    if (sscanf(argv[6], "%d", &Nfr) == EOF) {
    }

    if (Nfr <= 0) {
    }

    //converting a string to a integer
    if (sscanf(argv[8], "%d", &Nparticles) == EOF) {
    }

    if (Nparticles <= 0) {
    }
    //establish seed
    int * seed = (int *) malloc(sizeof (int) *Nparticles);
    int i;
    for (i = 0; i < Nparticles; i++)
	    seed[i] = i;
    //malloc matrix
    unsigned char * I = (unsigned char *) malloc(sizeof (unsigned char) *IszX * IszY * Nfr);
    //call video sequence
    //videoSequence(I, IszX, IszY, Nfr, seed);
    //call particle filter
    particleFilter(I, IszX, IszY, Nfr, seed, Nparticles);

    free(seed);
    free(I);
    return 0;
}
