
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#include <time.h>
#define BLOCK_X 16
#define BLOCK_Y 16
#define PI 3.1415926535897932

// when doing verification
#if 1
#define malloc(X) calloc((X), 1)
#endif

const int threads_per_block = 512;

#define check_error(X) X

__device__ double calcLikelihoodSum(unsigned char * I, int * ind, int numOnes, int index) {
    double likelihoodSum = 0.0;
    int x;
    for (x = 0; x < numOnes; x++) {
        likelihoodSum += I[ind[index * numOnes + x]];
    }
    return likelihoodSum;
}

__device__ double d_randu(int * seed, int index) {
  return (1000 + index) / 5000000;

    int M = INT_MAX;
    int A = 1103515245;
    int C = 12345;
    int num = A * seed[index] + C;
    seed[index] = num % M;

    return fabs(seed[index] / ((double) M));
}

__device__ double d_randn(int * seed, int index) {
  return (1000 + index) / 5000000;
    //Box-Muller algortihm
    double pi = 3.14159265358979323846;
    double u = d_randu(seed, index);
    double v = d_randu(seed, index);
    double cosine = cos(2 * pi * v);
    double rt = -2 * log(u);
    return sqrt(rt) * cosine;
}

__device__ double dev_round_double(double value) {
    int newValue = (int) (value);
    if (value - newValue < .5f)
        return newValue;
    else
        return newValue++;
}

__global__ void find_index_kernel(double * CDF, double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;

    if (i < Nparticles) {

        int index = -1;
        int x;

        for (x = 0; x < Nparticles; x++) {
            if (CDF[x] >= u[i]) {
                index = x;
                //break;
            }
        }

        printf("idx=%d\n", index);

    }
    __syncthreads();
}

__global__ void normalize_weights_kernel1(double * weights, int Nparticles, double* partial_sums, double * CDF, double * u, int * seed) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double sumWeights;
    
    if(0 == threadIdx.x) {
        sumWeights = partial_sums[0];
    }
    
    __syncthreads();
    
    if (i < Nparticles) {
        weights[i] = weights[i] / sumWeights;
    }
}


__global__ void normalize_weights_kernel2(double* weights, double * CDF, double * u, int * seed, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double u1;
    
    if (i == 0) {
        u[0] = 0;
    }
    
    __syncthreads();
    
    if(0 == threadIdx.x) 
        u1 = u[0];
    
    __syncthreads();
        
    if (i < Nparticles) {
        u[i] = u1 + i / ((double) (Nparticles));
    }
}

__global__ void likelihood_kernel(double * arrayX, double * arrayY, double * xj, double * yj, double * CDF, int * ind, int * objxy, double * likelihood, unsigned char * I, double * u, double * weights, int Nparticles, int countOnes, int max_size, int k, int IszY, int Nfr, int *seed, double* partial_sums) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    int y;
    
    int indX, indY; 

    if (threadIdx.x == 0) {
        partial_sums[blockIdx.x] = weights[0];
    }

}

/** 
 * Takes in a double and returns an integer that approximates to that double
 * @return if the mantissa < .5 => return value < input value; else return value > input value
 */
double roundDouble(double value) {
    int newValue = (int) (value);
    if (value - newValue < .5)
        return newValue;
    else
        return newValue++;
}

/**
 * Fills a radius x radius matrix representing the disk
 * @param disk The pointer to the disk to be made
 * @param radius  The radius of the disk to be made
 */
void strelDisk(int * disk, int radius) {
    int diameter = radius * 2 - 1;
    int x, y;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            double distance = sqrt(pow((double) (x - radius + 1), 2) + pow((double) (y - radius + 1), 2));
            if (distance < radius)
                disk[x * diameter + y] = 1;
        }
    }
}

/**
 * Fills a 2D array describing the offsets of the disk object
 * @param se The disk object
 * @param numOnes The number of ones in the disk
 * @param neighbors The array that will contain the offsets
 * @param radius The radius used for dilation
 */
void getneighbors(int * se, int numOnes, int * neighbors, int radius) {
    int x, y;
    int neighY = 0;
    int center = radius - 1;
    int diameter = radius * 2 - 1;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            if (se[x * diameter + y]) {
                neighbors[neighY * 2] = (int) (y - center);
                neighbors[neighY * 2 + 1] = (int) (x - center);
                neighY++;
            }
        }
    }
}

/**
 * The implementation of the particle filter using OpenMP for many frames
 * @see http://openmp.org/wp/
 * @note This function is designed to work with a video of several frames. In addition, it references a provided MATLAB function which takes the video, the objxy matrix and the x and y arrays as arguments and returns the likelihoods
 * @param I The video to be run
 * @param IszX The x dimension of the video
 * @param IszY The y dimension of the video
 * @param Nfr The number of frames
 * @param seed The seed array used for random number generation
 * @param Nparticles The number of particles to be used
 */
void particleFilter(unsigned char * I, int IszX, int IszY, int Nfr, int * seed, int Nparticles) {
    int max_size = IszX * IszY*Nfr;
    //original particle centroid
    double xe = roundDouble(IszY / 2.0);
    double ye = roundDouble(IszX / 2.0);

    //expected object locations, compared to center
    int radius = 5;
    int diameter = radius * 2 - 1;
    int * disk = (int*) malloc(diameter * diameter * sizeof (int));
    strelDisk(disk, radius);
    int countOnes = 0;
    int x, y;
    for (x = 0; x < diameter; x++) {
        for (y = 0; y < diameter; y++) {
            if (disk[x * diameter + y] == 1)
                countOnes++;
        }
    }
    int * objxy = (int *) malloc(countOnes * 2 * sizeof (int));
    getneighbors(disk, countOnes, objxy, radius);
    //initial weights are all equal (1/Nparticles)
    double * weights = (double *) malloc(sizeof (double) *Nparticles);
    for (x = 0; x < Nparticles; x++) {
        weights[x] = 1 / ((double) (Nparticles));
    }

    //initial likelihood to 0.0
    double * likelihood = (double *) malloc(sizeof (double) *Nparticles);
    double * arrayX = (double *) malloc(sizeof (double) *Nparticles);
    double * arrayY = (double *) malloc(sizeof (double) *Nparticles);
    double * xj = (double *) malloc(sizeof (double) *Nparticles);
    double * yj = (double *) malloc(sizeof (double) *Nparticles);
    double * CDF = (double *) malloc(sizeof (double) *Nparticles);

    //GPU copies of arrays
    double * arrayX_GPU;
    double * arrayY_GPU;
    double * xj_GPU;
    double * yj_GPU;
    double * CDF_GPU;
    double * likelihood_GPU;
    unsigned char * I_GPU;
    double * weights_GPU;
    int * objxy_GPU;

    int * ind = (int*) malloc(sizeof (int) *countOnes * Nparticles);
    int * ind_GPU;
    double * u = (double *) malloc(sizeof (double) *Nparticles);
    double * u_GPU;
    int * seed_GPU;
    double* partial_sums;

    //CUDA memory allocation
    check_error(hipMalloc((void **) &arrayX_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &arrayY_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &xj_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &yj_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &CDF_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &u_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &likelihood_GPU, sizeof (double) *Nparticles));
    //set likelihood to zero
    check_error(hipMemset((void *) likelihood_GPU, 0, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &weights_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &I_GPU, sizeof (unsigned char) *IszX * IszY * Nfr));
    check_error(hipMalloc((void **) &objxy_GPU, sizeof (int) *2 * countOnes));
    check_error(hipMalloc((void **) &ind_GPU, sizeof (int) *countOnes * Nparticles));
    check_error(hipMalloc((void **) &seed_GPU, sizeof (int) *Nparticles));
    check_error(hipMalloc((void **) &partial_sums, sizeof (double) *Nparticles));


    //Donnie - this loop is different because in this kernel, arrayX and arrayY
    //  are set equal to xj before every iteration, so effectively, arrayX and 
    //  arrayY will be set to xe and ye before the first iteration.
    for (x = 0; x < Nparticles; x++) {

        xj[x] = xe;
        yj[x] = ye;

    }

    int indX, indY;
    //start send
    check_error(hipMemcpy(I_GPU, I, sizeof (unsigned char) *IszX * IszY*Nfr, hipMemcpyHostToDevice));
    check_error(hipMemcpy(objxy_GPU, objxy, sizeof (int) *2 * countOnes, hipMemcpyHostToDevice));
    check_error(hipMemcpy(weights_GPU, weights, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(xj_GPU, xj, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(yj_GPU, yj, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    check_error(hipMemcpy(seed_GPU, seed, sizeof (int) *Nparticles, hipMemcpyHostToDevice));
    int num_blocks = ceil((double) Nparticles / (double) threads_per_block);


    Nfr = 2;

    {
        int k = 1;
      likelihood_kernel <<< num_blocks, threads_per_block >>> (arrayX_GPU, arrayY_GPU, xj_GPU, yj_GPU, CDF_GPU, ind_GPU, objxy_GPU, likelihood_GPU, I_GPU, u_GPU, weights_GPU, Nparticles, countOnes, max_size, k, IszY, Nfr, seed_GPU, partial_sums);

        int x;
        double sum = 0.0;
        int num_blocks = ceil((double) Nparticles / (double) threads_per_block);
        for (x = 0; x < num_blocks; x++) {
            sum += partial_sums[x];
        }
        partial_sums[0] = sum;

      normalize_weights_kernel1 <<< num_blocks, threads_per_block >>> (weights_GPU, Nparticles, partial_sums, CDF_GPU, u_GPU, seed_GPU);
        
        CDF_GPU[0] = weights_GPU[0];
        for (int x = 1; x < Nparticles; x++) {
            CDF_GPU[x] = weights_GPU[x] + CDF_GPU[x - 1];
        }
      normalize_weights_kernel2 <<< num_blocks, threads_per_block >>> (weights_GPU, CDF_GPU, u_GPU, seed_GPU, Nparticles);

      find_index_kernel <<< num_blocks, threads_per_block >>> (CDF_GPU, u_GPU, Nparticles);
      
    }


    //block till kernels are finished

    hipFree(xj_GPU);
    hipFree(yj_GPU);
    hipFree(CDF_GPU);
    hipFree(u_GPU);
    hipFree(likelihood_GPU);
    hipFree(I_GPU);
    hipFree(objxy_GPU);
    hipFree(ind_GPU);
    hipFree(seed_GPU);
    hipFree(partial_sums);

    //CUDA freeing of memory
    hipFree(weights_GPU);
    hipFree(arrayY_GPU);
    hipFree(arrayX_GPU);

    //free regular memory
    free(likelihood);
    free(arrayX);
    free(arrayY);
    free(xj);
    free(yj);
    free(CDF);
    free(ind);
    free(u);
}

int main(int argc, char * argv[]) {

    int IszX = 4096, IszY = 4096, Nfr = 2, Nparticles = 50000;

    //converting a string to a integer
    if (sscanf(argv[8], "%d", &Nparticles) == EOF) {
    }
    
    //establish seed
    int * seed = (int *) malloc(sizeof (int) *Nparticles);
    int i;
    for (i = 0; i < Nparticles; i++)
	    seed[i] = i;
    //malloc matrix
    unsigned char * I = (unsigned char *) malloc(sizeof (unsigned char) *IszX * IszY * Nfr);
    //call video sequence
    //videoSequence(I, IszX, IszY, Nfr, seed);
    //call particle filter
    particleFilter(I, IszX, IszY, Nfr, seed, Nparticles);

    free(seed);
    free(I);
    return 0;
}
