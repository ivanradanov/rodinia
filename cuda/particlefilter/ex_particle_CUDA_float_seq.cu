
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_X 16
#define BLOCK_Y 16

#define threads_per_block 512

#define check_error(X) X

__global__ void normalize_weights_kernel1(double * weights, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double sumWeights;
    
    if(0 == threadIdx.x) {
        sumWeights = 1 / ((double) (Nparticles));
    }
    
    __syncthreads();
    
    if (i < Nparticles) {
        weights[i] = 1 / ((double) (Nparticles)) / sumWeights;
    }
}


__global__ void normalize_weights_kernel2(double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double u1;
    
    if (i == 0) {
        u[0] = 0;
    }
    
    __syncthreads();
    
    if(0 == threadIdx.x) 
        u1 = u[0];
    
    __syncthreads();
        
    if (i < Nparticles) {
        u[i] = u1 + i / ((double) (Nparticles));
    }
}

int main(int argc, char * argv[]) {

    int Nparticles = 50000;

    //converting a string to a integer
    if (sscanf(argv[8], "%d", &Nparticles) == EOF) {
    }
    
    int x, y;
    //initial weights are all equal (1/Nparticles)
    double * weights = (double *) malloc(sizeof (double) *Nparticles);
    for (x = 0; x < Nparticles; x++) {
        weights[x] = 1 / ((double) (Nparticles));
    }

    //GPU copies of arrays
    double * weights_GPU;

    double * u_GPU;

    //CUDA memory allocation
    check_error(hipMalloc((void **) &u_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &weights_GPU, sizeof (double) *Nparticles));


    int indX, indY;
    //start send
    check_error(hipMemcpy(weights_GPU, weights, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    int num_blocks = ceil((double) Nparticles / (double) threads_per_block);

    {
      normalize_weights_kernel1 <<< num_blocks, threads_per_block >>> (weights_GPU, Nparticles);
        
      normalize_weights_kernel2 <<< num_blocks, threads_per_block >>> (u_GPU, Nparticles);

      for (int i=0; i<Nparticles; i++) {
        int index = -1;
        int x;

        for (x = 0; x < Nparticles; x++) {
            if (weights_GPU[x] >= u_GPU[i]) {
                index = x;
            }
        }

        printf("idx=%d\n", index);
      }
      
    }


    //block till kernels are finished

    hipFree(u_GPU);

    //CUDA freeing of memory
    hipFree(weights_GPU);
    free(weights);
    return 0;
}
