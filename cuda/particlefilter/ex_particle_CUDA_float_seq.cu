
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_X 16
#define BLOCK_Y 16

#define threads_per_block 512

#define check_error(X) X

__global__ void find_index_kernel(double * CDF, double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;

    if (i < Nparticles) {

        int index = -1;
        int x;

        for (x = 0; x < Nparticles; x++) {
            if (CDF[x] >= u[i]) {
                index = x;
                //break;
            }
        }

        printf("idx=%d\n", index);

    }
    __syncthreads();
}

__global__ void normalize_weights_kernel1(double * weights, int Nparticles, double* partial_sums) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double sumWeights;
    
    if(0 == threadIdx.x) {
        sumWeights = 1 / ((double) (Nparticles));
    }
    
    __syncthreads();
    
    if (i < Nparticles) {
        weights[i] = 1 / ((double) (Nparticles)) / sumWeights;
    }
}


__global__ void normalize_weights_kernel2(double * u, int Nparticles) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ double u1;
    
    if (i == 0) {
        u[0] = 0;
    }
    
    __syncthreads();
    
    if(0 == threadIdx.x) 
        u1 = u[0];
    
    __syncthreads();
        
    if (i < Nparticles) {
        u[i] = u1 + i / ((double) (Nparticles));
    }
}

int main(int argc, char * argv[]) {

    int Nparticles = 50000;

    //converting a string to a integer
    if (sscanf(argv[8], "%d", &Nparticles) == EOF) {
    }
    
    int x, y;
    //initial weights are all equal (1/Nparticles)
    double * weights = (double *) malloc(sizeof (double) *Nparticles);
    for (x = 0; x < Nparticles; x++) {
        weights[x] = 1 / ((double) (Nparticles));
    }

    //GPU copies of arrays
    double * CDF_GPU;
    double * weights_GPU;

    double * u_GPU;
    double* partial_sums;

    //CUDA memory allocation
    check_error(hipMalloc((void **) &CDF_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &u_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &weights_GPU, sizeof (double) *Nparticles));
    check_error(hipMalloc((void **) &partial_sums, sizeof (double) *Nparticles));


    int indX, indY;
    //start send
    check_error(hipMemcpy(weights_GPU, weights, sizeof (double) *Nparticles, hipMemcpyHostToDevice));
    int num_blocks = ceil((double) Nparticles / (double) threads_per_block);

    {
        for (int i=0; i<Nparticles; i++)
            partial_sums[i] = 1 / ((double) (Nparticles));

        int x;
        double sum = 0.0;
        int num_blocks = ceil((double) Nparticles / (double) threads_per_block);
        for (x = 0; x < num_blocks; x++) {
            sum += partial_sums[x];
        }
        partial_sums[0] = sum;

      normalize_weights_kernel1 <<< num_blocks, threads_per_block >>> (weights_GPU, Nparticles, partial_sums);
        
        CDF_GPU[0] = weights_GPU[0];
        for (int x = 1; x < Nparticles; x++) {
            CDF_GPU[x] = weights_GPU[x] + CDF_GPU[x - 1];
        }
      normalize_weights_kernel2 <<< num_blocks, threads_per_block >>> (u_GPU, Nparticles);

      find_index_kernel <<< num_blocks, threads_per_block >>> (CDF_GPU, u_GPU, Nparticles);
      
    }


    //block till kernels are finished

    hipFree(CDF_GPU);
    hipFree(u_GPU);
    hipFree(partial_sums);

    //CUDA freeing of memory
    hipFree(weights_GPU);
    free(weights);
    return 0;
}
