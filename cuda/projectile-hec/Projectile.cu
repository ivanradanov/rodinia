#include "hip/hip_runtime.h"
//==============================================================
// Copyright © 2020 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================

#include <chrono>
#include <vector>
#include <hip/hip_runtime.h>
#include "Projectile.hpp"

#ifdef DEBUG
static const int num_elements = 100;
#else
static const int num_elements = 10000000;
#endif
const float kPIValue = 3.1415;
const float kGValue = 9.81;
const int BLOCK_SIZE = 256; 

// Function to calculate the range, maximum height and total flight time of a
// projectile

__global__ void CalculateRange(const Projectile *obj, Projectile *pObj) {  
  
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i >= num_elements) return;
  float proj_angle = obj[i].getangle();
  float proj_vel = obj[i].getvelocity();
  float sin_value = sin(proj_angle * kPIValue / 180.0f);
  float cos_value = cos(proj_angle * kPIValue / 180.0f);
  float total_time = fabs((2 * proj_vel * sin_value)) / kGValue;
  float max_range =  fabs(proj_vel * total_time * cos_value);
  float max_height = (proj_vel * proj_vel * sin_value * sin_value) / 2.0f *
                     kGValue;  // h = v^2 * sin^2theta/2g

  pObj[i].setRangeandTime(max_range, total_time, proj_angle, proj_vel, max_height);
}

// in_vect and out_vect are the vectors with N Projectile numbers and are inputs to the
// parallel function
void GpuParallel(std::vector<Projectile>& in_vect,
                 std::vector<Projectile>& out_vect,
                 const int repeat)
{
  Projectile *bufin_vect, *bufout_vect;

  hipMalloc((void**)&bufin_vect, sizeof(Projectile) * num_elements);
  hipMalloc((void**)&bufout_vect, sizeof(Projectile) * num_elements);
  hipMemcpy(bufin_vect, in_vect.data(), sizeof(Projectile) * num_elements, hipMemcpyHostToDevice);

  dim3 grids ((num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 blocks (BLOCK_SIZE);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    CalculateRange <<< grids, blocks >>> (bufin_vect, bufout_vect);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(out_vect.data(), bufout_vect, sizeof(Projectile) * num_elements, hipMemcpyDeviceToHost);
  hipFree(bufin_vect);
  hipFree(bufout_vect);
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  float init_angle = 0.0f;
  float init_vel = 0.0f;
  vector<Projectile> input_vect1, out_parallel_vect2, out_scalar_vect3;

  // Initialize the Input and Output vectors
  srand(2);
  for (int i = 0; i < num_elements; i++) {
    init_angle = rand() % 90 + 10;
    init_vel = rand() % 400 + 10;
    input_vect1.push_back(Projectile(init_angle, init_vel, 1.0f, 1.0f, 1.0f));
    out_parallel_vect2.push_back(Projectile());
    out_scalar_vect3.push_back(Projectile());
  }

  // Call the DpcppParallel with the required inputs and outputs
  GpuParallel(input_vect1, out_parallel_vect2, repeat);
      
#ifdef DEBUG
  for (int i = 0; i < num_elements; i++)
  {
    // Displaying the Parallel computation results.
    cout << "Parallel " << out_parallel_vect2[i];
  } 
#endif
  return 0;
}
