#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

#define p_IJWID 6
#define p_JID   4
#define p_JWID  5
#define p_Np    512
#define p_Nq    8
#define p_Nvgeo 12
#define p_RXID  0
#define p_RYID  1
#define p_RZID  7
#define p_SXID  2
#define p_SYID  3
#define p_SZID  8
#define p_TXID  9
#define p_TYID  10
#define p_TZID  11
#define p_cubNp 4096
#define p_cubNq 16

// kernel
#include "adv.cu"

dfloat *drandAlloc(int N){
  dfloat *v = (dfloat*) calloc(N, sizeof(dfloat));
  for(int n = 0; n < N; ++n) v[n] = drand48();
  return v;
}

int main(int argc, char **argv) {

  if (argc < 4) {
    printf("Usage: ./adv N cubN numElements [nRepetitions]\n");
    exit(-1);
  }

  const int N = atoi(argv[1]);
  const int cubN = atoi(argv[2]);
  const dlong Nelements = atoi(argv[3]);
  int Ntests = 1;

  if(argc >= 5) Ntests = atoi(argv[4]);

  const int Nq = N+1;
  const int cubNq = cubN+1;
  const int Np = Nq*Nq*Nq;
  const int cubNp = cubNq*cubNq*cubNq;
  const dlong offset = Nelements*Np;

  printf("Data type in bytes: %zu\n", sizeof(dfloat));

  srand48(123);
  dfloat *vgeo           = drandAlloc(Np*Nelements*p_Nvgeo);
  dfloat *cubvgeo        = drandAlloc(cubNp*Nelements*p_Nvgeo);
  dfloat *cubDiffInterpT = drandAlloc(3*cubNp*Nelements);
  dfloat *cubInterpT     = drandAlloc(Np*cubNp);
  dfloat *u              = drandAlloc(3*Np*Nelements);
  dfloat *adv            = drandAlloc(3*Np*Nelements);

  dfloat *d_vgeo, *d_cubvgeo, *d_cubDiffInterpT, *d_cubInterpT, *d_u, *d_adv;
  hipMalloc((void**)&d_vgeo, Np*Nelements*p_Nvgeo*sizeof(dfloat));
  hipMalloc((void**)&d_cubvgeo, cubNp*Nelements*p_Nvgeo*sizeof(dfloat));
  hipMalloc((void**)&d_cubDiffInterpT,3*cubNp*Nelements*sizeof(dfloat));
  hipMalloc((void**)&d_cubInterpT, Np*cubNp*sizeof(dfloat));
  hipMalloc((void**)&d_u, 3*Np*Nelements*sizeof(dfloat));
  hipMalloc((void**)&d_adv, 3*Np*Nelements*sizeof(dfloat));

  hipMemcpy(d_vgeo, vgeo, Np*Nelements*p_Nvgeo*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(d_cubvgeo, cubvgeo, cubNp*Nelements*p_Nvgeo*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(d_cubDiffInterpT, cubDiffInterpT, 3*cubNp*Nelements*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(d_cubInterpT, cubInterpT, Np*cubNp*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(d_u, u, 3*Np*Nelements*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(d_adv, adv, 3*Np*Nelements*sizeof(dfloat), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::high_resolution_clock::now();

  // run kernel
  for(int test=0;test<Ntests;++test) 
    advCubatureHex3D<<<dim3(Nelements, 1), dim3(16, 16)>>>( 
        Nelements,
        d_vgeo,
        d_cubvgeo,
        d_cubDiffInterpT,
        d_cubInterpT,
        offset,
        d_u,
        d_adv);

  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  const double elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() / Ntests;

  hipMemcpy(adv, d_adv, 3*Np*Nelements*sizeof(dfloat), hipMemcpyDeviceToHost);

  hipFree(d_vgeo);
  hipFree(d_cubvgeo);
  hipFree(d_cubDiffInterpT);
  hipFree(d_cubInterpT);
  hipFree(d_u);
  hipFree(d_adv);

  double checksum = 0;
  for (int i = 0; i < 3*Np*Nelements; i++) {
    checksum += adv[i];
    #ifdef OUTPUT
    std::cout << adv[i] << "\n";
    #endif
  }
  std::cout << "Checksum=" << checksum << "\n";

  // statistics
  const dfloat GDOFPerSecond = (N*N*N)*Nelements/elapsed;
  std::cout << " NRepetitions=" << Ntests
            << " N=" << N
            << " cubN=" << cubN
            << " Nelements=" << Nelements
            << " elapsed time=" << elapsed
            << " GDOF/s=" << GDOFPerSecond
            << "\n";

  free(vgeo          );
  free(cubvgeo       );
  free(cubDiffInterpT);
  free(cubInterpT    );
  free(u             );
  free(adv           );
  return 0;
}
