#include "hip/hip_runtime.h"
// statistical kernel
__global__ void reduce(	long d_Ne,											// number of elements in array
                        int d_no,											// number of sums to reduce
                        int d_mul,											// increment
                        fp *d_sums,										// pointer to partial sums variable (DEVICE GLOBAL MEMORY)
                        fp *d_sums2){

    // indexes
    int bx = blockIdx.x;													// get current horizontal block index
    int tx = threadIdx.x;													// get current horizontal thread index
    int ei = (bx*NUMBER_THREADS)+tx;										// unique thread id, more threads than actual elements !!!
    int nf = NUMBER_THREADS-(gridDim.x*NUMBER_THREADS-d_no);				// number of elements assigned to last block
    int df = 0;																// divisibility factor for the last block

    // statistical
    __shared__ fp d_psum[NUMBER_THREADS];								// data for block calculations allocated by every block in its shared memory
    __shared__ fp d_psum2[NUMBER_THREADS];

    // counters
    int i;

    // copy data to shared memory
    if(ei<d_no){															// do only for the number of elements, omit extra threads

        d_psum[tx] = d_sums[ei*d_mul];
        d_psum2[tx] = d_sums2[ei*d_mul];

    }

    __syncthreads();
    if(bx != (gridDim.x - 1)){											//
        for(i=2; i<=NUMBER_THREADS; i=2*i){								//
            if((tx+1) % i == 0){										// every ith
                d_psum[tx] = d_psum[tx] + d_psum[tx-i/2];
                d_psum2[tx] = d_psum2[tx] + d_psum2[tx-i/2];
            }
            __syncthreads();											//
        }
        if(tx==(NUMBER_THREADS-1)){										// block result stored in global memory
            d_sums[bx*d_mul*NUMBER_THREADS] = d_psum[tx];
            d_sums2[bx*d_mul*NUMBER_THREADS] = d_psum2[tx];
        }
    }

}
