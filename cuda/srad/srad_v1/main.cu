#include "hip/hip_runtime.h"
//====================================================================================================100
//		UPDATE
//====================================================================================================100

//    2006.03   Rob Janiczek
//        --creation of prototype version
//    2006.03   Drew Gilliam
//        --rewriting of prototype version into current version
//        --got rid of multiple function calls, all code in a  
//         single function (for speed)
//        --code cleanup & commenting
//        --code optimization efforts   
//    2006.04   Drew Gilliam
//        --added diffusion coefficent saturation on [0,1]
//		2009.12 Lukasz G. Szafaryn
//		-- reading from image, command line inputs
//		2010.01 Lukasz G. Szafaryn
//		--comments

//====================================================================================================100
//	DEFINE / INCLUDE
//====================================================================================================100

#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "define.c"
#include "extract_kernel.cu"
#include "prepare_kernel.cu"
#include "reduce_kernel.cu"
#include "srad_kernel.cu"
#include "srad2_kernel.cu"
#include "compress_kernel.cu"
#include "graphics.c"
#include "resize.c"
#include "timer.c"

#include "device.c"				// (in library path specified to compiler)	needed by for device functions

//====================================================================================================100
//	MAIN FUNCTION
//====================================================================================================100

int main(int argc, char *argv []){

    //================================================================================80
    // 	VARIABLES
    //================================================================================80

    // time
    long long time0;
    long long time1;
    long long time2;
    long long time3;
    long long time4;
    long long time5;
    long long time6;
    long long time7;
    long long time8;
    long long time9;
    long long time10;
    long long time11;
    long long time12;

    time0 = get_time();

    // inputs image, input paramenters
    fp* image_ori;																// originalinput image
    int image_ori_rows;
    int image_ori_cols;
    long image_ori_elem;

    // inputs image, input paramenters
    fp* image;															// input image
    int Nr,Nc;													// IMAGE nbr of rows/cols/elements
    long Ne;

    // algorithm parameters
    int niter;																// nbr of iterations
    fp lambda;															// update step size

    // size of IMAGE
    int r1,r2,c1,c2;												// row/col coordinates of uniform ROI
    long NeROI;														// ROI nbr of elements

    // surrounding pixel indicies
    int *iN,*iS,*jE,*jW;    

    // counters
    long i,j;    // image row/col

    // memory sizes
    int mem_size_i;
    int mem_size_j;
    int mem_size_single;

    //================================================================================80
    // 	GPU VARIABLES
    //================================================================================80

    // CUDA kernel execution parameters
    dim3 threads;
    int blocks_x;
    dim3 blocks;
    dim3 blocks2;
    dim3 blocks3;

    // memory sizes
    int mem_size;															// matrix memory size

    // HOST
    int no;
    int mul;
    fp total;
    fp total2;
    fp meanROI;
    fp meanROI2;
    fp varROI;
    fp q0sqr;

    // DEVICE
    fp* d_sums;															// partial sum
    fp* d_sums2;
    int* d_iN;
    int* d_iS;
    int* d_jE;
    int* d_jW;
    fp* d_dN;
    fp* d_dS;
    fp* d_dW;
    fp* d_dE;
    fp* d_I;																// input IMAGE on DEVICE
    fp* d_c;

    time1 = get_time();

    //================================================================================80
    // 	GET INPUT PARAMETERS
    //================================================================================80

    if(argc != 5){
        printf("ERROR: wrong number of arguments\n");
        return 0;
    }
    else{
        niter = atoi(argv[1]);
        lambda = atof(argv[2]);
        Nr = atoi(argv[3]);						// it is 502 in the original image
        Nc = atoi(argv[4]);						// it is 458 in the original image
    }

    time2 = get_time();

    //================================================================================80
    // 	READ IMAGE (SIZE OF IMAGE HAS TO BE KNOWN)
    //================================================================================80

    // read image
    image_ori_rows = 502;
    image_ori_cols = 458;
    image_ori_elem = image_ori_rows * image_ori_cols;

    image_ori = (fp*)malloc(sizeof(fp) * image_ori_elem);

    read_graphics(	"../../../data/srad/image.pgm",
                    image_ori,
                    image_ori_rows,
                    image_ori_cols,
                    1);

    time3 = get_time();

    //================================================================================80
    // 	RESIZE IMAGE (ASSUMING COLUMN MAJOR STORAGE OF image_orig)
    //================================================================================80

    Ne = Nr*Nc;

    image = (fp*)malloc(sizeof(fp) * Ne);

    resize(	image_ori,
            image_ori_rows,
            image_ori_cols,
            image,
            Nr,
            Nc,
            1);

    time4 = get_time();

    //================================================================================80
    // 	SETUP
    //================================================================================80

    r1     = 0;											// top row index of ROI
    r2     = Nr - 1;									// bottom row index of ROI
    c1     = 0;											// left column index of ROI
    c2     = Nc - 1;									// right column index of ROI

    // ROI image size
    NeROI = (r2-r1+1)*(c2-c1+1);											// number of elements in ROI, ROI size

    // allocate variables for surrounding pixels
    mem_size_i = sizeof(int) * Nr;											//
    iN = (int *)malloc(mem_size_i) ;										// north surrounding element
    iS = (int *)malloc(mem_size_i) ;										// south surrounding element
    mem_size_j = sizeof(int) * Nc;											//
    jW = (int *)malloc(mem_size_j) ;										// west surrounding element
    jE = (int *)malloc(mem_size_j) ;										// east surrounding element

    // N/S/W/E indices of surrounding pixels (every element of IMAGE)
    for (i=0; i<Nr; i++) {
        iN[i] = i-1;														// holds index of IMAGE row above
        iS[i] = i+1;														// holds index of IMAGE row below
    }
    for (j=0; j<Nc; j++) {
        jW[j] = j-1;														// holds index of IMAGE column on the left
        jE[j] = j+1;														// holds index of IMAGE column on the right
    }

    // N/S/W/E boundary conditions, fix surrounding indices outside boundary of image
    iN[0]    = 0;															// changes IMAGE top row index from -1 to 0
    iS[Nr-1] = Nr-1;														// changes IMAGE bottom row index from Nr to Nr-1
    jW[0]    = 0;															// changes IMAGE leftmost column index from -1 to 0
    jE[Nc-1] = Nc-1;														// changes IMAGE rightmost column index from Nc to Nc-1

    //================================================================================80
    // 	GPU SETUP
    //================================================================================80

    // allocate memory for entire IMAGE on DEVICE
    mem_size = sizeof(fp) * Ne;																		// get the size of float representation of input IMAGE
    hipMalloc((void **)&d_I, mem_size);														//

    // allocate memory for coordinates on DEVICE
    hipMalloc((void **)&d_iN, mem_size_i);													//
    hipMemcpy(d_iN, iN, mem_size_i, hipMemcpyHostToDevice);				//
    hipMalloc((void **)&d_iS, mem_size_i);													//
    hipMemcpy(d_iS, iS, mem_size_i, hipMemcpyHostToDevice);				//
    hipMalloc((void **)&d_jE, mem_size_j);													//
    hipMemcpy(d_jE, jE, mem_size_j, hipMemcpyHostToDevice);				//
    hipMalloc((void **)&d_jW, mem_size_j);													//
    hipMemcpy(d_jW, jW, mem_size_j, hipMemcpyHostToDevice);			//

    // allocate memory for partial sums on DEVICE
    hipMalloc((void **)&d_sums, mem_size);													//
    hipMalloc((void **)&d_sums2, mem_size);												//

    // allocate memory for derivatives
    hipMalloc((void **)&d_dN, mem_size);														//
    hipMalloc((void **)&d_dS, mem_size);														//
    hipMalloc((void **)&d_dW, mem_size);													//
    hipMalloc((void **)&d_dE, mem_size);														//

    // allocate memory for coefficient on DEVICE
    hipMalloc((void **)&d_c, mem_size);														//

    checkCUDAError("setup");

    //================================================================================80
    // 	KERNEL EXECUTION PARAMETERS
    //================================================================================80

    // all kernels operating on entire matrix
    threads.x = NUMBER_THREADS;												// define the number of threads in the block
    threads.y = 1;
    blocks_x = Ne/threads.x;
    if (Ne % threads.x != 0){												// compensate for division remainder above by adding one grid
        blocks_x = blocks_x + 1;
    }
    blocks.x = blocks_x;													// define the number of blocks in the grid
    blocks.y = 1;

    time5 = get_time();

    //================================================================================80
    // 	COPY INPUT TO CPU
    //================================================================================80

    hipMemcpy(d_I, image, mem_size, hipMemcpyHostToDevice);

    time6 = get_time();

    //================================================================================80
    // 	SCALE IMAGE DOWN FROM 0-255 TO 0-1 AND EXTRACT
    //================================================================================80

    extract<<<blocks, threads>>>(	Ne,
                                    d_I);

    checkCUDAError("extract");

    time7 = get_time();

    //================================================================================80
    // 	COMPUTATION
    //================================================================================80

    // printf("iterations: ");

    // execute main loop
    prepare<<<blocks, threads>>>(	Ne,
                                    d_I,
                                    d_sums,
                                    d_sums2);

    checkCUDAError("prepare");

    // performs subsequent reductions of sums
    blocks2.x = blocks.x;												// original number of blocks
    blocks2.y = blocks.y;
    no = Ne;														// original number of sum elements
    mul = 1;														// original multiplier
    MY_START_CLOCK(srad_v1, reduce);
    reduce<<<blocks2, threads>>>(	Ne,
                                    no,
                                    mul,
                                    d_sums,
                                    d_sums2);
    MY_STOP_CLOCK(srad_v1, reduce);
    //MY_DEVICE_VERIFY_FLOAT(d_sums, mem_size / sizeof(fp));
    return 0;
}

//====================================================================================================100
//	END OF FILE
//====================================================================================================100
