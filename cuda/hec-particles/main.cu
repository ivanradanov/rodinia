#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "particles.h"

#define MAX_EPSILON_ERROR 5.00f
#define THRESHOLD         0.30f
#define GRID_SIZE         64
#define NUM_PARTICLES     16384

// Simulation parameters
const float timestep = 0.5f;              // Time slice for re-computation iteration
//const float gravity = 0.0005f;            // Strength of gravity
//const float damping = 1.0f;
const float fParticleRadius = 0.023f;     // Radius of individual particles
const float fColliderRadius = 0.17f;      // Radius of collider for interacting with particles in 'm' mode
//const float collideSpring = 0.4f;         // Elastic spring constant for impact between particles
//const float collideDamping = 0.025f;      // Inelastic loss component for impact between particles
//const float collideShear = 0.12f;         // Friction constant for particles in contact
//const float collideAttraction = 0.0012f;  // Attraction between particles (~static or Van der Waals) 


inline float frand(void){
  return (float)rand() / (float)RAND_MAX;
}

void initGrid(float *hPos, float *hVel, float particleRadius, float spacing, 
    unsigned int numParticles)
{
  float jitter = particleRadius * 0.01f;
  unsigned int s = (int) ceilf(powf((float) numParticles, 1.0f / 3.0f));
  unsigned int gridSize[3];
  gridSize[0] = gridSize[1] = gridSize[2] = s;

  srand(1973);
  for(unsigned int z=0; z<gridSize[2]; z++) 
  {
    for(unsigned int y=0; y<gridSize[1]; y++) 
    {
      for(unsigned int x=0; x<gridSize[0]; x++) 
      {
        unsigned int i = (z * gridSize[0] * gridSize[1]) + (y * gridSize[1]) + x;
        if (i < numParticles) 
        {
          hPos[i * 4] = (spacing * x) + particleRadius - 1.0f + (frand() * 2.0f - 1.0f) * jitter;
          hPos[i * 4 + 1] = (spacing * y) + particleRadius - 1.0f + (frand() * 2.0f - 1.0f) * jitter;
          hPos[i * 4 + 2] = (spacing * z) + particleRadius - 1.0f + (frand() * 2.0f - 1.0f) * jitter;
          hPos[i * 4 + 3] = 1.0f;
          hVel[i * 4] = 0.0f;
          hVel[i * 4 + 1] = 0.0f;
          hVel[i * 4 + 2] = 0.0f;
          hVel[i * 4 + 3] = 0.0f;
        }
      }
    }
  }
}

// Main program
int main(int argc, char** argv) 
{
  if (argc != 2) {
    printf("Usage: %s <iterations>\n", argv[0]);
    return 1;
  }
  const int iterations = atoi(argv[1]);               // Number of iterations
  unsigned int numParticles = NUM_PARTICLES;
  unsigned int gridDim = GRID_SIZE;

  // Set and log grid size and particle count, after checking optional command-line inputs
  uint3 gridSize;
  gridSize.x = gridSize.y = gridSize.z = gridDim;
  unsigned int numGridCells = gridSize.x * gridSize.y * gridSize.z;

  // set simulation parameters
  simParams_t params;
  params.gridSize = gridSize;
  params.numCells = numGridCells;
  params.numBodies = numParticles;
  params.particleRadius = fParticleRadius; 
  params.colliderPos = {1.2f, -0.8f, 0.8f};
  params.colliderRadius = fColliderRadius;

  params.worldOrigin = {1.0f, -1.0f, -1.0f};
  float cellSize = params.particleRadius * 2.0f;  // cell size equal to particle diameter
  params.cellSize = {cellSize, cellSize, cellSize};

  params.spring = 0.5f;
  params.damping = 0.02f;
  params.shear = 0.1f;
  params.attraction = 0.0f;
  params.boundaryDamping = -0.5f;

  params.gravity = {0.0f, -0.0003f, 0.0f};
  params.globalDamping = 1.0f;

  printf(" grid: %d x %d x %d = %d cells\n", gridSize.x, gridSize.y, gridSize.z, numGridCells);
  printf(" particles: %d\n\n", numParticles);

  float* hPos          = (float*)malloc(numParticles * 4 * sizeof(float));
  float* hVel          = (float*)malloc(numParticles * 4 * sizeof(float));
  float* hReorderedPos = (float*)malloc(numParticles * 4 * sizeof(float));
  float* hReorderedVel = (float*)malloc(numParticles * 4 * sizeof(float));
  unsigned int* hHash      = (unsigned int*)malloc(numParticles * sizeof(unsigned int));
  unsigned int* hIndex     = (unsigned int*)malloc(numParticles * sizeof(unsigned int));
  unsigned int* hCellStart = (unsigned int*)malloc(numGridCells * sizeof(unsigned int));
  unsigned int* hCellEnd   = (unsigned int*)malloc(numGridCells * sizeof(unsigned int));

  // configure grid 
  initGrid(hPos, hVel, params.particleRadius, params.particleRadius * 2.0f, numParticles);

  float4* dPos;
  hipMalloc((void**)&dPos, numParticles * sizeof(float4));
  hipMemcpy(dPos, (float4*)hPos, numParticles * sizeof(float4), hipMemcpyHostToDevice);

  float4* dVel;
  hipMalloc((void**)&dVel, numParticles * sizeof(float4));
  hipMemcpy(dVel, (float4*)hVel, numParticles * sizeof(float4), hipMemcpyHostToDevice);

  float4* dReorderedPos; 
  hipMalloc((void**)&dReorderedPos, numParticles * sizeof(float4));
  hipMemcpy(dReorderedPos, (float4*)hReorderedPos, 
      numParticles * sizeof(float4), hipMemcpyHostToDevice);

  float4* dReorderedVel;
  hipMalloc((void**)&dReorderedVel, numParticles * sizeof(float4));
  hipMemcpy(dReorderedVel, (float4*)hReorderedVel, 
      numParticles * sizeof(float4), hipMemcpyHostToDevice);

  unsigned int* dHash;
  hipMalloc((void**)&dHash, numParticles * sizeof(unsigned int));
  unsigned int* dIndex;
  hipMalloc((void**)&dIndex, numParticles * sizeof(unsigned int));

  unsigned int* dCellStart;
  hipMalloc((void**)&dCellStart, numGridCells * sizeof(unsigned int));

  unsigned int* dCellEnd;
  hipMalloc((void**)&dCellEnd, numGridCells * sizeof(unsigned int));

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < iterations; i++)
  {
    integrateSystem(
        dPos,
        dVel,
        params,
        timestep,
        numParticles);

    calcHash(
        dHash,
        dIndex,
        dPos,
        params,
        numParticles);

    bitonicSort(dHash, dIndex, dHash, dIndex, 1, numParticles, 0);

    //Find start and end of each cell and
    //Reorder particle data for better cache coherency
    findCellBoundsAndReorder(
        dCellStart,
        dCellEnd,
        dReorderedPos,
        dReorderedVel,
        dHash,
        dIndex,
        dPos,
        dVel,
        numParticles,
        numGridCells);

    collide(
        dVel,
        dReorderedPos,
        dReorderedVel,
        dIndex,
        dCellStart,
        dCellEnd,
        params,
        numParticles,
        numGridCells);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Total execution time of %d loop iterations: %f (s)\n", iterations, time * 1e-9f);
  printf("Average execution time of a loop iteration: %f (us)\n", (time * 1e-3f) / iterations);

#ifdef DEBUG
  // results should not differ much from those in CUDA SDK 5_Simulation/particles
  // Note the values of certain simulation parameters must be the same
  hipMemcpy((float4*)hVel, dVel, numParticles * sizeof(float4), hipMemcpyDeviceToHost);
  hipMemcpy((float4*)hPos, dPos, numParticles * sizeof(float4), hipMemcpyDeviceToHost);
  for (unsigned int i = 0; i < numParticles; i++) {
    printf("%d: ", i);
    printf("pos: (%.4f, %.4f, %.4f, %.4f)\n",
        hPos[4*i], hPos[4*i+1], hPos[4*i+2], hPos[4*i+3]);
    printf("vel: (%.4f, %.4f, %.4f, %.4f)\n",
        hVel[4*i], hVel[4*i+1], hVel[4*i+2], hVel[4*i+3]);
  }
#endif

  free(hPos         );
  free(hVel         );
  free(hReorderedPos);
  free(hReorderedVel);
  free(hHash        );
  free(hIndex       );
  free(hCellStart   );
  free(hCellEnd     );
  hipFree(dPos);
  hipFree(dVel);
  hipFree(dReorderedPos); 
  hipFree(dReorderedVel);
  hipFree(dHash);
  hipFree(dIndex);
  hipFree(dCellStart);
  hipFree(dCellEnd);

  return EXIT_SUCCESS;
}
