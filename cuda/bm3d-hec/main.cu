#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>

#include "bm3d.hpp"
#define cimg_display 0
#include "CImg.h"

// Repeat the execution of kernels 100 times
#define REPEAT 100

// Adjust the size of the total shared local memory for different GPUs
// e.g. 48KB on P100
#define TOTAL_SLM     48*1024

// Adjust the thread block size of the block matching kernel for different GPUs. 
// The maximum thread block size is 32 * MAX_NUM_WARPS
#define MAX_NUM_WARPS 16u

using namespace cimg_library;

int main(int argc, char** argv)
{
  if( argc < 4 )
  {
    std::cerr << "Usage: " << argv[0]
      << " NosiyImage DenoisedImage sigma [color] [ReferenceImage]\n"
      << "   color - color image denoising (experimental only)\n"
      << "   ReferenceImage - if provided, computes and prints PSNR " 
      << "between the reference image and denoised image\n";
    return 1;
  }

  //Store a noisy image
  CImg<unsigned char> image(argv[1]);

  float sigma = strtof(argv[3], NULL);

  unsigned int channels = 1;
  if (argc >= 5 && strcmp(argv[4],"color") == 0) channels = 3;

  std::cout << "Sigma = " << sigma << std::endl;

  if (channels > 1)
    std::cout << "Color denoising: yes" << std::endl;
  else
    std::cout << "Color denoising: no" << std::endl;

  std::vector<unsigned int> sigma2(channels);
  sigma2[0] = 25 * 25;

  //Convert color image to YCbCr color space
  if (channels == 3)
  {
    image = image.get_channels(0, 2).RGBtoYCbCr();
    //Convert the sigma^2 variance to the YCbCr color space
    long s = sigma * sigma;
    sigma2[0] = ((66l*66l*s + 129l*129l*s + 25l*25l*s) / (256l*256l));
    sigma2[1] = ((38l*38l*s + 74l*74l*s + 112l*112l*s) / (256l*256l));
    sigma2[2] = ((112l*112l*s + 94l*94l*s + 18l*18l*s) / (256l*256l));
  }

  std::cout << "Noise variance for individual channels (YCrCb if color): ";
  for (unsigned int k = 0; k < sigma2.size(); k++)
    std::cout << sigma2[k] << " ";
  std::cout << std::endl;

  // Check for invalid input
  if(! image.data() )              
  {
    std::cerr << "Could not open or find the image" << std::endl;
    return 1;
  }

  std::cout << "Image width: " << image.width() << " height: " << image.height() << std::endl;

  //Store a denoised image
  CImg<unsigned char> dst_image(image.width(), image.height(), 1, channels, 0);

  // Vector of image channels
  std::vector<uchar*> d_noisy_image;
  std::vector<uchar*> d_denoised_image;
  //Numerator and denominator used for aggregation
  std::vector<float*> d_numerator;  
  std::vector<float*> d_denominator;

  ushort* d_stacks;              //Addresses of similar patches to each reference patch of a batch
  uint* d_num_patches_in_stack;  //Number of similar patches for each referenca patch of a batch that are stored in d_stacks
  float* d_gathered_stacks;      //3D groups of a batch
  float* d_w_P;                  //Weights for aggregation
  float* d_kaiser_window;        //Kaiser window used for aggregation

  uint2 h_batch_size = make_uint2(256, 128);         //h_batch_size.x has to be divisible by properties.warpSize

  //Denoising parameters and their shorthands
  Params h_hard_params(19, 8, 16, 2500, 3, 2.7f);
  const uint k = h_hard_params.k;
  const uint N = h_hard_params.N;
  const uint p = h_hard_params.p;

  //Reserved sizes
  const int width = image.width();
  const int height = image.height();
  size_t image_size = width * height;

  d_noisy_image.resize(channels);
  d_denoised_image.resize(channels);
  d_numerator.resize(channels);
  d_denominator.resize(channels);

  for(auto & it : d_noisy_image)
    cuda_error_check( hipMalloc((void**)&it, sizeof(uchar) * image_size) );

  for(auto & it : d_denoised_image)
    cuda_error_check( hipMalloc((void**)&it, sizeof(uchar) * image_size) );

  for(auto & it : d_numerator) 
    cuda_error_check( hipMalloc((void**)&it, sizeof(float) * image_size) );

  for(auto & it : d_denominator)
    cuda_error_check( hipMalloc((void**)&it, sizeof(float) * image_size) );

  cuda_error_check( hipMalloc((void**)&d_stacks, 
        sizeof(ushort) * h_batch_size.x * h_batch_size.y * N) );

  cuda_error_check( hipMalloc((void**)&d_num_patches_in_stack, 
        sizeof(uint) * h_batch_size.x * h_batch_size.y ) );

  cuda_error_check( hipMalloc((void**)&d_gathered_stacks, 
        sizeof(float) * (N+1) * k * k * h_batch_size.x * h_batch_size.y) );

  cuda_error_check( hipMalloc((void**)&d_w_P,
        sizeof(float) * h_batch_size.x * h_batch_size.y) );

  cuda_error_check( hipMalloc((void**)&d_kaiser_window, 
        sizeof(float) * k * k) );

  //image dimensions
  const uint2 image_dim = make_uint2(width, height);

  //dimensions limiting addresses of reference patches
  const uint2 stacks_dim = make_uint2(width - (k - 1), height - (k - 1));

  int paramN1 = N + 1; //maximal size of a stack with a reference patch

  const uint p_block_width = (warpSize-1) * p + k;
  const uint s_image_p_size = p_block_width * k * sizeof(uchar);

  const uint shared_mem_available = TOTAL_SLM - s_image_p_size;

  //Block-matching shared memory sizes per warp
  const uint s_diff_size = p_block_width * sizeof(uint);
  const uint s_patches_in_stack_size = warpSize * sizeof(uchar);
  const uint s_patch_stacks_size = N * warpSize * sizeof(uint);

  const uint num_warps = std::min(shared_mem_available / 
    (s_diff_size + s_patches_in_stack_size + s_patch_stacks_size), MAX_NUM_WARPS);
  uint lmem_size_bm = ((s_diff_size + s_patches_in_stack_size + s_patch_stacks_size) * num_warps) + 
    s_image_p_size;    

  //Determine launch parameteres for the block match kernel
  dim3 num_threads_bm = dim3(warpSize*num_warps, 1);
  dim3 num_blocks_bm = dim3(h_batch_size.x / warpSize, h_batch_size.y);

  //Determine launch parameteres for the get and aggregate kernels
  const dim3 num_threads(k, k);
  const dim3 num_blocks(h_batch_size.x, h_batch_size.y);

  //Determine launch parameteres for the DCT kernel
  const uint trans_size = k*k*paramN1*h_batch_size.x*h_batch_size.y;
  const dim3 num_blocks_tr((trans_size + (KER2_BLOCK_WIDTH*k) - 1) / (KER2_BLOCK_WIDTH*k), 1, 1);
  const dim3 num_threads_tr(k, KER2_BLOCK_WIDTH/k, 1);

  const uint s_size_t = k*k*(paramN1+1)*sizeof(float); //+1 for avoinding bank conflicts

  //Determine launch parameteres for final division kernel
  const dim3 num_threads_f(64, 4);
  const dim3 num_blocks_f((width + 63)/64, (height + 3)/4);

  //Create an kaiser window (only for k = 8, alpha = 2.0) and copy it to the device.
  std::vector<float> kaiserWindow(k*k);
  if (k == 8) {
    // First quarter of the matrix
    kaiserWindow[0 + k * 0] = 0.1924f; 
    kaiserWindow[0 + k * 1] = 0.2989f;
    kaiserWindow[0 + k * 2] = 0.3846f;
    kaiserWindow[0 + k * 3] = 0.4325f;
    kaiserWindow[1 + k * 0] = 0.2989f;
    kaiserWindow[1 + k * 1] = 0.4642f;
    kaiserWindow[1 + k * 2] = 0.5974f;
    kaiserWindow[1 + k * 3] = 0.6717f;
    kaiserWindow[2 + k * 0] = 0.3846f;
    kaiserWindow[2 + k * 1] = 0.5974f;
    kaiserWindow[2 + k * 2] = 0.7688f;
    kaiserWindow[2 + k * 3] = 0.8644f;
    kaiserWindow[3 + k * 0] = 0.4325f;
    kaiserWindow[3 + k * 1] = 0.6717f;
    kaiserWindow[3 + k * 2] = 0.8644f; 
    kaiserWindow[3 + k * 3] = 0.9718f;

    // Fill the rest of the matrix by symmetry
    for(unsigned i = 0; i < k / 2; i++)
      for (unsigned j = k / 2; j < k; j++)
        kaiserWindow[i + k * j] = kaiserWindow[i + k * (k - j - 1)];

    for (unsigned i = k / 2; i < k; i++)
      for (unsigned j = 0; j < k; j++)
        kaiserWindow[i + k * j] = kaiserWindow[k - i - 1 + k * j];
  }
  else
    for (unsigned i = 0; i < k * k; i++)
      kaiserWindow[i] = 1.0f;

  // Copy images to device
  for(uint i = 0; i < channels; ++i) 
    cuda_error_check( hipMemcpy(d_noisy_image[i],
          image.data()+i*image_size,image_size*sizeof(uchar), hipMemcpyHostToDevice));

  cuda_error_check( hipMemcpy(d_kaiser_window, &kaiserWindow[0],
    k*k*sizeof(float), hipMemcpyHostToDevice));

  hipDeviceSynchronize();

  // start measuring the total time
  auto start = std::chrono::high_resolution_clock::now();

  // repeat the execution of kernels
  for (int n = 0; n < REPEAT; n++) {

    for(auto & it : d_numerator) 
      cuda_error_check( hipMemset(it, 0, image_size * sizeof(float)) );

    for(auto & it : d_denominator)
      cuda_error_check( hipMemset(it, 0, image_size * sizeof(float)) );

    //Batch processing: in each iteration only the batch_size reference patches are processed. 
    uint2 start_point;
    for(start_point.y = 0; start_point.y < stacks_dim.y + p - 1; 
        start_point.y += (h_batch_size.y*p))
    {
      for(start_point.x = 0; start_point.x < stacks_dim.x + p - 1; 
          start_point.x += (h_batch_size.x*p))
      {
        //Finds similar patches for each reference patch of a batch and stores them in d_stacks array
        run_block_matching(
            d_noisy_image[0],      // IN: Image  
            d_stacks,              // OUT: Array of adresses of similar patches
            d_num_patches_in_stack,// OUT: Array containing numbers of these addresses
            image_dim,             // IN: Image dimensions
            stacks_dim,            // IN: Dimensions limiting addresses of reference patches
            h_hard_params,         // IN: Denoising parameters 
            start_point,           // IN: Address of the top-left reference patch of a batch
            num_threads_bm,        // Threads in block 
            num_blocks_bm,         // Blocks in grid
            lmem_size_bm           // Shared memory size
        );

        //cuda_error_check( hipGetLastError() );
        //cuda_error_check( hipDeviceSynchronize() );

        for (uint channel = 0; channel < channels; ++channel)
        {
          //Assembles 3D groups of a batch according to the d_stacks array
          run_get_block(
              start_point,             // IN: First reference patch of a batch
              d_noisy_image[channel],  // IN: Image
              d_stacks,                // IN: Array of adresses of similar patches
              d_num_patches_in_stack,  // IN: Numbers of patches in 3D groups
              d_gathered_stacks,       // OUT: Assembled 3D groups
              image_dim,               // IN: Image dimensions
              stacks_dim,              // IN: Dimensions limiting addresses of reference patches
              h_hard_params,           // IN: Denoising parameters
              num_threads,             // Threads in block
              num_blocks               // Blocks in grid
          );

          //cuda_error_check( hipGetLastError() );
          //cuda_error_check( hipDeviceSynchronize() );

          //Apply the 2D DCT transform to each layer of 3D group
          run_DCT2D8x8(d_gathered_stacks, d_gathered_stacks, trans_size, num_threads_tr, num_blocks_tr);
          //cuda_error_check( hipGetLastError() );
          //cuda_error_check( hipDeviceSynchronize() );

          // 1) 1D Walsh-Hadamard transform of proper size on the 3rd dimension of each 
          //      3D group of a batch to complete the 3D transform.
          // 2) Hard thresholding
          // 3) Inverse 1D Walsh-Hadamard trannsform.
          // 4) Compute the weingt of each 3D group

          run_hard_treshold_block(
              start_point,           // IN: First reference patch of a batch
              d_gathered_stacks,     // IN/OUT: 3D groups with transfomed patches
              d_w_P,                 // OUT: Weight of each 3D group
              d_num_patches_in_stack,// IN: Numbers of patches in 3D groups
              stacks_dim,            // IN: Dimensions limiting addresses of reference patches
              h_hard_params,         // IN: Denoising parameters
              sigma2[channel],       // IN: sigma
              num_threads,           // Threads in block
              num_blocks,            // Blocks in grid
              s_size_t               // Shared memory size
          );

          //cuda_error_check( hipGetLastError() );
          //cuda_error_check( hipDeviceSynchronize() );

          //Apply inverse 2D DCT transform to each layer of 3D group
          run_IDCT2D8x8(d_gathered_stacks, d_gathered_stacks, trans_size, num_threads_tr, num_blocks_tr);

          //cuda_error_check( hipGetLastError() );
          //cuda_error_check( hipDeviceSynchronize() );

          //Aggregates filtered patches of all 3D groups of a batch into numerator and denominator buffers
          run_aggregate_block(
              start_point,           // IN: First reference patch of a batch
              d_gathered_stacks,     // IN: 3D groups with transfomed patches
              d_w_P,                 // IN: Numbers of non zero coeficients after 3D thresholding
              d_stacks,              // IN: Array of adresses of similar patches
              d_kaiser_window,       // IN: Kaiser window
              d_numerator[channel],  // IN/OUT: Numerator aggregation buffer
              d_denominator[channel],// IN/OUT: Denominator aggregation buffer
              d_num_patches_in_stack,// IN: Numbers of patches in 3D groups
              image_dim,             // IN: Image dimensions
              stacks_dim,            // IN: Dimensions limiting addresses of reference patches
              h_hard_params,         // IN: Denoising parameters
              num_threads,           // Threads in block
              num_blocks             // Blocks in grid
          );
          //cuda_error_check( hipGetLastError() );
          //cuda_error_check( hipDeviceSynchronize() );
        }
      }
    }

    //Divide numerator by denominator and save the result in output image
    for (uint channel = 0; channel < channels; ++channel)
    {
      run_aggregate_final(
          d_numerator[channel],      // IN: Numerator aggregation buffer
          d_denominator[channel],    // IN: Denominator aggregation buffer
          image_dim,                 // IN: Image dimensions
          d_denoised_image[channel], // OUT: Image estimate
          num_threads_f,             // Threads in block
          num_blocks_f               // Blocks in grid
      );
    }
  } // REPEAT

  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  double gpuTime = (double)elapsed_seconds.count();
  std::cout << "Average device execution time (s): " << gpuTime / REPEAT << std::endl;

  for (uint channel = 0; channel < channels; ++channel) {
    //cuda_error_check( hipGetLastError() );
    //cuda_error_check( hipDeviceSynchronize() );
    cuda_error_check( hipMemcpy(
          dst_image.data()+channel*image_size,
          d_denoised_image[channel],
          image_size*sizeof(uchar), 
          hipMemcpyDeviceToHost) );
  }

  if (channels == 3) 
    dst_image = dst_image.get_channels(0,2).YCbCrtoRGB();
  else
    dst_image = dst_image.get_channel(0);

  //Save denoised image
  dst_image.save( argv[2] );

  if (argc >= 6) {
    CImg<unsigned char> reference_image(argv[5]);
    std::cout << "PSNR:" << reference_image.PSNR(dst_image) << std::endl;
  }

  cuda_error_check( hipFree(d_stacks) );
  cuda_error_check( hipFree(d_num_patches_in_stack) );
  cuda_error_check( hipFree(d_gathered_stacks) );
  cuda_error_check( hipFree(d_w_P) );
  cuda_error_check( hipFree(d_kaiser_window) );

  for (auto & it : d_noisy_image)
    cuda_error_check( hipFree(it) );
  d_noisy_image.clear();

  for (auto & it : d_denoised_image)
    cuda_error_check( hipFree(it) );
  d_denoised_image.clear();

  for(auto & it : d_numerator)
    cuda_error_check( hipFree(it) );
  d_numerator.clear();

  for(auto & it : d_denominator)
    cuda_error_check( hipFree(it) );
  d_denominator.clear();

  return 0;
}
