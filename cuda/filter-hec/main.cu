/**
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <chrono>
#include <random>
#include <vector>
#include <hip/hip_runtime.h>

__global__ 
void filter_shared (int *__restrict__ dst,
                    int *__restrict__ nres,
                    const int*__restrict__ src,
                    int n)
{
  __shared__ int l_n;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // zero the counter
  if (threadIdx.x == 0)
    l_n = 0;
  __syncthreads();

  // get the value, evaluate the predicate, and
  // increment the counter if needed
  int d, pos;

  if(i < n) {
    d = src[i];
    if(d > 0)
      pos = atomicAdd(&l_n, 1);
  }
  __syncthreads();

  // leader increments the global counter
  if(threadIdx.x == 0)
    l_n = atomicAdd(nres, l_n);
  __syncthreads();

  // threads with true predicates write their elements
  if(i < n && d > 0) {
    pos += l_n; // increment local pos by global counter
    dst[pos] = d;
  }
  __syncthreads();
}

// compare device results with host results
bool check(int *d_nres, int *d_output, int h_nres, std::vector<int> &h_output) {
  int nres;
  hipMemcpy(&nres, d_nres, sizeof(int), hipMemcpyDeviceToHost);

  std::vector<int> output (nres);

  hipMemcpy(output.data(), d_output, sizeof(int) * nres, hipMemcpyDeviceToHost);

  // clear device output
  hipMemset(d_output, 0, sizeof(int) * nres);

  std::sort(output.begin(), output.end());

  bool equal = (h_nres == nres) && 
               std::equal(h_output.begin(),
                          h_output.begin() + h_nres, output.begin());
  return equal;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("Usage: %s <number of elements> <block size> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_elems = atoi(argv[1]);
  const int block_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);
    
  std::vector<int> input (num_elems);

  // Generate input data.
  for (int i = 0; i < num_elems; i++) {
    input[i] = i - num_elems / 2;
  }

  std::mt19937 g;
  g.seed(19937);
  std::shuffle(input.begin(), input.end(), g);

  // Generate host output with host filtering code.
  std::vector<int> h_output (num_elems);

  int h_flt_count = 0;
  for (int i = 0; i < num_elems; i++) {
    if (input[i] > 0) {
      h_output[h_flt_count++] = input[i];
    }
  }
  // Sort the result for comparison
  std::sort(h_output.begin(), h_output.begin() + h_flt_count);

  // Filtering on a device
  int *d_input, *d_output, *d_nres;

  hipMalloc(&d_input, sizeof(int) * num_elems);
  hipMalloc(&d_output, sizeof(int) * num_elems);
  hipMalloc(&d_nres, sizeof(int));

  hipMemcpy(d_input, input.data(),
             sizeof(int) * num_elems, hipMemcpyHostToDevice);

  dim3 dimBlock (block_size);
  dim3 dimGrid ((num_elems + block_size - 1) / block_size);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipMemset(d_nres, 0, sizeof(int));
    filter_shared<<<dimGrid, dimBlock>>>(d_output, d_nres, d_input, num_elems);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of filter (shared memory) %lf (ms)\n",
         (time * 1e-6) / repeat);

  bool match = check(d_nres, d_output, h_flt_count, h_output);
  printf("%s\n", match ? "PASS" : "FAIL");

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_nres);

  return 0;
}
