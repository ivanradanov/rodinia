/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates a combination of Peer-to-Peer (P2P) and
 * Unified Virtual Address Space (UVA) features new to SDK 4.0
 */

#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void SimpleKernel(const float *src, float *dst)
{
  // Just a dummy kernel, doing enough for us to verify that everything
  // worked
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx] * 2.0f;
}

inline bool IsAppBuiltAs64()
{
  return sizeof(void*) == 8;
}

int main(int argc, char **argv)
{
  printf("[%s] - Starting...\n", argv[0]);
  const int repeat = atoi(argv[1]);

  if (!IsAppBuiltAs64())
  {
    printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target.  Test is being waived.\n", argv[0]);
    exit(0);
  }

  // Number of GPUs
  printf("Checking for multiple GPUs...\n");
  int gpu_n;
  hipGetDeviceCount(&gpu_n);
  printf("There are %d GPUs\n", gpu_n);

  if (gpu_n < 2)
  {
    printf("Two or more GPUs with Peer-to-Peer access capability are required for %s.\n", argv[0]);
    printf("Waiving test.\n");
    exit(0);
  }

  // Query device properties
  hipDeviceProp_t prop[64];

  for (int i=0; i < gpu_n; i++)
  {
    hipGetDeviceProperties(&prop[i], i);
  }
  // Check possibility for peer access
  printf("\nChecking GPU(s) for support of peer to peer memory access...\n");

  int can_access_peer;
  int p2pCapableGPUs[2] = {-1, -1}; // We take only 1 pair of P2P capable GPUs

  // Show all the combinations of supported P2P GPUs
  for (int i = 0; i < gpu_n; i++)
  {
    for (int j = 0; j < gpu_n; j++)
    {
      if (i == j)
      {
        continue;
      }
      hipDeviceCanAccessPeer(&can_access_peer, i, j);
      printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[i].name, i,
             prop[j].name, j, can_access_peer ? "Yes" : "No");
      if (can_access_peer && p2pCapableGPUs[0] == -1)
      {
        p2pCapableGPUs[0] = i;
        p2pCapableGPUs[1] = j;
      }
    }
  }

  if (p2pCapableGPUs[0] == -1 || p2pCapableGPUs[1] == -1)
  {
    printf("Two or more GPUs with Peer-to-Peer access capability are required for %s.\n", argv[0]);
    printf("Peer to Peer access is not available amongst GPUs in the system, waiving test.\n");
    exit(0);
  }

  // Use first pair of p2p capable GPUs detected.
  int gpuid[2]; // we want to find the first two GPU's that can support P2P
  gpuid[0] = p2pCapableGPUs[0];
  gpuid[1] = p2pCapableGPUs[1];

  // Enable peer access
  printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0], gpuid[1]);
  hipSetDevice(gpuid[0]);
  hipDeviceEnablePeerAccess(gpuid[1], 0);

  hipSetDevice(gpuid[1]);
  hipDeviceEnablePeerAccess(gpuid[0], 0);

  // Allocate buffers
  const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
  printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n",
         int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);

  // GPU0
  hipSetDevice(gpuid[0]);
  float *g0;
  hipMalloc(&g0, buf_size);

  float *h0;
  hipHostMalloc(&h0, buf_size, hipHostMallocDefault); // Automatically portable with UVA

  // GPU1
  hipSetDevice(gpuid[1]);
  float *g1;
  hipMalloc(&g1, buf_size);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i=0; i<repeat; i++)
  {
    // With UVA we don't need to specify source and target devices, the
    // runtime figures this out by itself from the pointers
    // Ping-pong copy between GPUs
    if (i % 2 == 0)
    {
      hipMemcpy(g1, g0, buf_size, hipMemcpyDefault);
    }
    else
    {
      hipMemcpy(g0, g1, buf_size, hipMemcpyDefault);
    }
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time_memcpy = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  printf("Peer-to-peer copy between GPU%d and GPU%d: %.2fGB/s\n", gpuid[0], gpuid[1],
         1.0f / time_memcpy * (repeat * buf_size));

  // Prepare host buffer and copy to GPU 0
  printf("Preparing host buffer and memcpy to GPU%d...\n", gpuid[0]);

  const int buf_len = buf_size / sizeof(float);
  for (int i=0; i<buf_len; i++)
  {
    h0[i] = float(i % 4096);
  }

  hipSetDevice(gpuid[0]);
  hipMemcpy(g0, h0, buf_size, hipMemcpyDefault);

  // Kernel launch configuration
  const dim3 threads(256);
  const dim3 blocks(buf_len / 256);

  // Run kernel on GPU 1, reading input from the GPU 0 buffer, writing
  // output to the GPU 1 buffer
  printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n",
         gpuid[1], gpuid[0], gpuid[1]);

  hipSetDevice(gpuid[1]);
  SimpleKernel<<<blocks, threads>>>(g0, g1);

  hipDeviceSynchronize();

  // Run kernel on GPU 0, reading input from the GPU 1 buffer, writing
  // output to the GPU 0 buffer
  printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n",
         gpuid[0], gpuid[1], gpuid[0]);

  hipSetDevice(gpuid[0]);
  SimpleKernel<<<blocks, threads>>>(g1, g0);

  hipDeviceSynchronize();

  // Copy data back to host and verify
  printf("Copy data back to host from GPU%d and verify results...\n", gpuid[0]);
  hipMemcpy(h0, g0, buf_size, hipMemcpyDefault);

  int error_count = 0;

  for (int i=0; i<buf_len; i++)
  {
    // Re-generate input data and apply 2x '* 2.0f' computation of both
    // kernel runs
    if (h0[i] != float(i % 4096) * 2.0f * 2.0f)
    {
      printf("Verification error @ element %i: val = %f, ref = %f\n", i, h0[i], (float(i%4096)*2.0f*2.0f));

      if (error_count++ > 10)
      {
        break;
      }
    }
  }

  // Disable peer access (also unregisters memory for non-UVA cases)
  printf("Disabling peer access...\n");
  hipSetDevice(gpuid[0]);
  hipDeviceDisablePeerAccess(gpuid[1]);

  hipSetDevice(gpuid[1]);
  hipDeviceDisablePeerAccess(gpuid[0]);

  // Cleanup and shutdown
  printf("Shutting down...\n");
  hipSetDevice(gpuid[0]);
  hipFree(g0);
  hipHostFree(h0);

  hipSetDevice(gpuid[1]);
  hipFree(g1);

  if (error_count != 0)
  {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }
  else
  {
    printf("Test passed\n");
    exit(EXIT_SUCCESS);
  }
}
