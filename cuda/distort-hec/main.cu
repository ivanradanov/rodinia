#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include "distort.h"

int main(int argc, char **argv)
{
  if (argc != 5) {
    std::cout << "Usage: " << argv[0] <<
      "<input image width> <input image height> <coefficient of distortion> <repeat>\n";
      return 1;
  }

  const int width = atoi(argv[1]);
  const int height = atoi(argv[2]);
  const float K = atof(argv[3]);
  const int repeat = atoi(argv[4]);

  struct Properties prop;
  prop.K = K;
  prop.centerX = width / 2;
  prop.centerY = height / 2;
  prop.width = width;
  prop.height = height;
  prop.thresh = 1.f;

  prop.xshift = calc_shift(0, prop.centerX - 1, prop.centerX, prop.K, prop.thresh);
  float newcenterX = prop.width - prop.centerX;
  float xshift_2 = calc_shift(0, newcenterX - 1, newcenterX, prop.K, prop.thresh);

  prop.yshift = calc_shift(0, prop.centerY - 1, prop.centerY, prop.K, prop.thresh);
  float newcenterY = prop.height - prop.centerY;
  float yshift_2 = calc_shift(0, newcenterY - 1, newcenterY, prop.K, prop.thresh);

  prop.xscale = (prop.width - prop.xshift - xshift_2) / prop.width;
  prop.yscale = (prop.height - prop.yshift - yshift_2) / prop.height;

  const int imageSize = height * width;
  const int imageSize_bytes = imageSize * sizeof(uchar3);

  uchar3* h_src = (uchar3*) malloc (imageSize_bytes);
  uchar3* h_dst = (uchar3*) malloc (imageSize_bytes);
  uchar3* r_dst = (uchar3*) malloc (imageSize_bytes);

  srand(123);
  for (int i = 0; i < imageSize; i++) {
    h_src[i] = make_uchar3(rand() % 256, rand() % 256, rand() % 256);
  }

  uchar3 *d_src, *d_dst;
  hipMalloc((void**)&d_src, imageSize_bytes);
  hipMemcpy(d_src, h_src, imageSize_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dst, imageSize_bytes);

  struct Properties* d_prop;
  hipMalloc((void**)&d_prop, sizeof(Properties));
  hipMemcpy(d_prop, &prop, sizeof(Properties), hipMemcpyHostToDevice);

  const dim3 blocks (16, 16);
  const dim3 grids (width / blocks.x + 1, height / blocks.y + 1);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    barrel_distort<<<grids, blocks>>>(d_src, d_dst, d_prop);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (ms)\n", (time * 1e-6f) / repeat);

  hipMemcpy(h_dst, d_dst, imageSize_bytes, hipMemcpyDeviceToHost);

  // verify
  int ex = 0, ey = 0, ez = 0;
  reference(h_src, r_dst, &prop);
  for (int i = 0; i < imageSize; i++) {
    ex = max(abs(h_dst[i].x - r_dst[i].x), ex);
    ey = max(abs(h_dst[i].y - r_dst[i].y), ey);
    ez = max(abs(h_dst[i].z - r_dst[i].z), ez);
  }

  std::cout << "Max error of each channel: " << ex << " " << ey << " " << ez << std::endl;

  hipFree(d_src);
  hipFree(d_dst);
  hipFree(d_prop);

  free(h_src);
  free(h_dst);

  return 0;
}
