#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <time.h>
#include "bucketsort.h"

// CUDA kernels
#include "kernel_bucketcount.h"
#include "kernel_bucketprefix.h"
#include "kernel_bucketsort.h"
#include "kernel_histogram.h"

////////////////////////////////////////////////////////////////////////////////
// Forward declarations
////////////////////////////////////////////////////////////////////////////////
void calcPivotPoints(float *histogram, int histosize, int listsize,
    int divisions, float min, float max, float *pivotPoints,
    float histo_width);

////////////////////////////////////////////////////////////////////////////////
// Given the input array of floats and the min and max of the distribution,
// sort the elements into float4 aligned buckets of roughly equal size
////////////////////////////////////////////////////////////////////////////////
void bucketSort(float *d_input, float *d_output, int listsize,
    int *sizes, int *nullElements, float minimum, float maximum,
    unsigned int *origOffsets)
{

  const int histosize = 1024;
  //  ////////////////////////////////////////////////////////////////////////////
  //  // First pass - Create 1024 bin histogram
  //  ////////////////////////////////////////////////////////////////////////////
  unsigned int* h_offsets = (unsigned int *) malloc(DIVISIONS * sizeof(unsigned int));
  for(int i = 0; i < DIVISIONS; i++){
    h_offsets[i] = 0;
  }
  float* pivotPoints = (float *)malloc(DIVISIONS * sizeof(float));
  float* historesult = (float *)malloc(histosize * sizeof(float));

  int blocks = ((listsize - 1) / (BUCKET_THREAD_N * BUCKET_BAND)) + 1;

  float *d_input_buff;
  unsigned int* d_offsets_buff;
  hipMalloc((void**)&d_input_buff, sizeof(float)*(listsize + DIVISIONS*4));
  hipMemcpyAsync(d_input_buff, d_input, sizeof(float)*(listsize + DIVISIONS*4), hipMemcpyHostToDevice, 0);
  hipMalloc((void**)&d_offsets_buff, sizeof(unsigned int)*DIVISIONS);

  size_t global_histogram = 6144;

#ifdef HISTO_WG_SIZE_0
  size_t local_histogram = HISTO_WG_SIZE_0;
#else
  size_t local_histogram = 96;
#endif

  hipMemcpyAsync(d_offsets_buff, h_offsets, sizeof(unsigned int)*DIVISIONS, hipMemcpyHostToDevice, 0);
  histogram1024<<<global_histogram/local_histogram, local_histogram>>>(
      d_offsets_buff, d_input_buff, listsize, minimum, maximum);
  hipMemcpy(h_offsets, d_offsets_buff, sizeof(unsigned int)*histosize, hipMemcpyDeviceToHost);

  for(int i=0; i<histosize; i++) {
    historesult[i] = (float)h_offsets[i];
  }


  //  ///////////////////////////////////////////////////////////////////////////
  //  // Calculate pivot points (CPU algorithm)
  //  ///////////////////////////////////////////////////////////////////////////
  calcPivotPoints(historesult, histosize, listsize, DIVISIONS,
      minimum, maximum, pivotPoints,
      (maximum - minimum)/(float)histosize);
  //
  //  ///////////////////////////////////////////////////////////////////////////
  //  // Count the bucket sizes in new divisions
  //  ///////////////////////////////////////////////////////////////////////////


  float* d_pivotPoints_buff;
  int* d_indice_buff;
  unsigned int* d_prefixoffsets_buff;

  hipMalloc((void**)&d_pivotPoints_buff, sizeof(float)*DIVISIONS);
  hipMemcpyAsync(d_pivotPoints_buff, pivotPoints, sizeof(float)*DIVISIONS, hipMemcpyHostToDevice, 0);

  hipMalloc((void**)&d_indice_buff, sizeof(int)*listsize);
  hipMalloc((void**)&d_prefixoffsets_buff, sizeof(unsigned int)* blocks * BUCKET_BLOCK_MEMORY);

  bucketcount<<<blocks, BUCKET_THREAD_N>>>(d_input_buff, d_indice_buff, d_prefixoffsets_buff,
      d_pivotPoints_buff, listsize); 

  //
  //  ///////////////////////////////////////////////////////////////////////////
  //  // Prefix scan offsets and align each division to float4 (required by
  //  // mergesort)
  //  ///////////////////////////////////////////////////////////////////////////
#ifdef BUCKET_WG_SIZE_0
  size_t localpre = BUCKET_WG_SIZE_0;
#else
  size_t localpre = 128;
#endif
  size_t globalpre = DIVISIONS;

  bucketprefix<<<globalpre/localpre, localpre>>>(d_prefixoffsets_buff, d_offsets_buff, blocks);

  // copy the sizes from device to host
  hipMemcpy(h_offsets, d_offsets_buff, sizeof(unsigned int)*DIVISIONS, hipMemcpyDeviceToHost);

  origOffsets[0] = 0;
  for(int i=0; i<DIVISIONS; i++){
    origOffsets[i+1] = h_offsets[i] + origOffsets[i];
    if((h_offsets[i] % 4) != 0){
      nullElements[i] = (h_offsets[i] & ~3) + 4 - h_offsets[i];
    }
    else nullElements[i] = 0;
  }
  for(int i=0; i<DIVISIONS; i++) sizes[i] = (h_offsets[i] + nullElements[i])/4;
  for(int i=0; i<DIVISIONS; i++) {
    if((h_offsets[i] % 4) != 0)  h_offsets[i] = (h_offsets[i] & ~3) + 4;
  }
  for(int i=1; i<DIVISIONS; i++) h_offsets[i] = h_offsets[i-1] + h_offsets[i];
  for(int i=DIVISIONS - 1; i>0; i--) h_offsets[i] = h_offsets[i-1];
  h_offsets[0] = 0;

  //  ///////////////////////////////////////////////////////////////////////////
  //  // Finally, sort the lot
  //  ///////////////////////////////////////////////////////////////////////////

  // update the h_offsets on the device
  hipMemcpyAsync(d_offsets_buff, h_offsets, sizeof(unsigned int)*DIVISIONS, hipMemcpyHostToDevice, 0);

  float* d_bucketOutput;
  hipMalloc((void**)&d_bucketOutput, sizeof(float)*(listsize + DIVISIONS*4)); 
  hipMemcpyAsync(d_bucketOutput, d_output, sizeof(float)*(listsize + DIVISIONS*4), hipMemcpyHostToDevice,0); 

  bucketsort<<<blocks, BUCKET_THREAD_N>>>(d_input_buff, d_indice_buff, d_bucketOutput,
      d_prefixoffsets_buff, d_offsets_buff, listsize);

  hipMemcpy(d_output, d_bucketOutput, sizeof(float)*(listsize + DIVISIONS*4), hipMemcpyDeviceToHost);

  hipFree(d_bucketOutput);
  hipFree(d_input_buff);
  hipFree(d_offsets_buff);
  hipFree(d_pivotPoints_buff);
  hipFree(d_indice_buff);
  hipFree(d_prefixoffsets_buff);
  free(pivotPoints);
  free(historesult);
}
////////////////////////////////////////////////////////////////////////////////
// Given a histogram of the list, figure out suitable pivotpoints that divide
// the list into approximately listsize/divisions elements each
////////////////////////////////////////////////////////////////////////////////
void calcPivotPoints(float *histogram, int histosize, int listsize,
    int divisions, float min, float max, float *pivotPoints, float histo_width)
{
  float elemsPerSlice = listsize/(float)divisions;
  float startsAt = min;
  float endsAt = min + histo_width;
  float we_need = elemsPerSlice;
  int p_idx = 0;
  for(int i=0; i<histosize; i++)
  {
    if(i == histosize - 1){
      if(!(p_idx < divisions)){
        pivotPoints[p_idx++] = startsAt + (we_need/histogram[i]) * histo_width;
      }
      break;
    }
    while(histogram[i] > we_need){
      if(!(p_idx < divisions)){
        printf("i=%d, p_idx = %d, divisions = %d\n", i, p_idx, divisions);
        exit(0);
      }
      pivotPoints[p_idx++] = startsAt + (we_need/histogram[i]) * histo_width;
      startsAt += (we_need/histogram[i]) * histo_width;
      histogram[i] -= we_need;
      we_need = elemsPerSlice;
    }
    // grab what we can from what remains of it
    we_need -= histogram[i];

    startsAt = endsAt;
    endsAt += histo_width;
  }
  while(p_idx < divisions){
    pivotPoints[p_idx] = pivotPoints[p_idx-1];
    p_idx++;
  }
}
