#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

// limits of integration
#define A 0
#define B 15

// row size is related to accuracy
#define ROW_SIZE 17
#define EPS      1e-7

__host__ __device__ inline double f(double x)
{
  return exp(x)*sin(x);
}

__device__ inline unsigned int getFirstSetBitPos(int n)
{
  return log2((float)(n&-n))+1;
}

__global__ void romberg(double a, double b, double *result)  
{
  extern __shared__ double smem[];
  double diff = (b-a)/gridDim.x, step;
  int k;
  int max_eval = (1<<(ROW_SIZE-1));
  b = a + (blockIdx.x+1)*diff;
  a += blockIdx.x*diff;

  step = (b-a)/max_eval;

  double local_col[ROW_SIZE];  // specific to the row size
  for(int i = 0; i < ROW_SIZE; i++) local_col[i] = 0.0;
  if(!threadIdx.x)
  {
    k = blockDim.x;
    local_col[0] = f(a) + f(b);
  }
  else
    k = threadIdx.x;

  for(; k < max_eval; k += blockDim.x)
    local_col[ROW_SIZE - getFirstSetBitPos(k)] += 2.0*f(a + step*k);

  for(int i = 0; i < ROW_SIZE; i++)
    smem[ROW_SIZE*threadIdx.x + i] = local_col[i];
  __syncthreads();

  if(threadIdx.x < ROW_SIZE)
  {
    double sum = 0.0;
    for(int i = threadIdx.x; i < blockDim.x*ROW_SIZE; i+=ROW_SIZE)
      sum += smem[i];
    smem[threadIdx.x] = sum;
  }

  if(!threadIdx.x)
  {
    double *table = local_col;
    table[0] = smem[0];

    for(int k = 1; k < ROW_SIZE; k++)
      table[k] = table[k-1] + smem[k];

    for(int k = 0; k < ROW_SIZE; k++)  
      table[k]*= (b-a)/(1<<(k+1));

    for(int col = 0 ; col < ROW_SIZE-1 ; col++)
      for(int row = ROW_SIZE-1; row > col; row--)
        table[row] = table[row] + (table[row] - table[row-1])/((1<<(2*col+1))-1);

    result[blockIdx.x] = table[ROW_SIZE-1];
  }
}


int main( int argc, char** argv)
{
  if (argc != 4) {
    printf("Usage: %s <number of work-groups> ", argv[0]);
    printf("<work-group size> <repeat>\n");
    return 1;
  }
  const int nwg = atoi(argv[1]);
  const int wgs = atoi(argv[2]);
  const int repeat = atoi(argv[3]);

  const int result_size_byte = nwg * sizeof(double);
  double *h_result = (double*) malloc (result_size_byte);

  double *d_result;
  hipMalloc((void**) &d_result, result_size_byte);

  dim3 grids (nwg);
  dim3 blocks (wgs);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    romberg <<< grids, blocks, ROW_SIZE*wgs*sizeof(double) >>> (A,B,d_result);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", time * 1e-9f / repeat);

  // verify

  hipMemcpy(h_result, d_result, result_size_byte, hipMemcpyDeviceToHost);
  double sum = 0.0;
  for(int k = 0; k < nwg; k++) sum += h_result[k];

  double ref_sum = reference(f, A, B, ROW_SIZE, EPS);
  printf("%s\n", (fabs(sum - ref_sum) > EPS) ? "FAIL" : "PASS");

  hipFree(d_result);
  free(h_result);
  return 0;
}
