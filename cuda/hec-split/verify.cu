
#include <hip/hip_runtime.h>
#include <string.h>

bool verify(const unsigned int* sorted_keys, const unsigned int* keys, 
            const unsigned int threads, const int N) 
{

  unsigned int m1[16], m2[16];

  int n = threads * 4;   // n elements are expected to be sorted
  for (int i = 0; i < N; i = i+n) {
    for (int j = 0; j < n-1; j++)
      if (sorted_keys[i+j] > sorted_keys[i+j+1]) return false;
  }

  for (int i = 0; i < N; i++) {
    if (sorted_keys[i] >= 16) return false;
  }

  for (int i = 0; i < N; i = i+n) {
    memset(m1, 0, 64);
    memset(m2, 0, 64);
    for (int j = 0; j < n; j++) {
      m1[keys[i+j]]++;
      m2[sorted_keys[i+j]]++;
    }
    if (memcmp(m1, m2, 64)) return false;
  }
  return true;
}

