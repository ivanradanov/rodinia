
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////////
// File: xchebyshev_Tn_series.c                                               //
// Routine(s):                                                                //
//    xChebyshev_Tn_Series                                                    //
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// double xChebyshev_Tn_Series(double x, double a[],int degree)//
//                                                                            //
//  Description:                                                              //
//     This routine uses Clenshaw's recursion algorithm to evaluate a given   //
//     polynomial p(x) expressed as a linear combination of Chebyshev         //
//     polynomials of the first kind, Tn, at a point x,                       //
//       p(x) = a[0] + a[1]*T[1](x) + a[2]*T[2](x) + ... + a[deg]*T[deg](x).  //
//                                                                            //
//     Clenshaw's recursion formula applied to Chebyshev polynomials of the   //
//     first kind is:                                                         //
//     Set y[degree + 2] = 0, y[degree + 1] = 0, then for k = degree, ..., 1  //
//     set y[k] = 2 * x * y[k+1] - y[k+2] + a[k].  Finally                    //
//     set y[0] = x * y[1] - y[2] + a[0].  Then p(x) = y[0].                  //
//                                                                            //
//  Arguments:                                                                //
//     double x                                                          //
//        The point at which to evaluate the polynomial.                      //
//     double a[]                                                        //
//        The coefficients of the expansion in terms of Chebyshev polynomials,//
//        i.e. a[k] is the coefficient of T[k](x).  Note that in the calling  //
//        routine a must be defined double a[N] where N >= degree + 1.        //
//     int    degree                                                          //
//        The degree of the polynomial p(x).                                  //
//                                                                            //
//  Return Value:                                                             //
//     The value of the polynomial at x.                                      //
//     If degree is negative, then 0.0 is returned.                           //
//                                                                            //
//  Example:                                                                  //
//     double x, a[N], p;                                                //
//     int    deg = N - 1;                                                    //
//                                                                            //
//     ( code to initialize x, and a[i] i = 0, ... , a[deg] )                 //
//                                                                            //
//     p = xChebyshev_Tn_Series(x, a, deg);                                   //
////////////////////////////////////////////////////////////////////////////////

extern "C" __host__ __device__
double xChebyshev_Tn_Series(double x, double a[], int degree)
{
  double yp2 = 0.0;
  double yp1 = 0.0;
  double y = 0.0;
  double two_x = x + x;
  int k;

  // Check that degree >= 0.  If not, then return 0. //

  if ( degree < 0 ) return 0.0;

  // Apply Clenshaw's recursion save the last iteration. //

  for (k = degree; k >= 1; k--, yp2 = yp1, yp1 = y) 
    y = two_x * yp1 - yp2 + a[k];

  // Now apply the last iteration and return the result. //

  return x * yp1 - yp2 + a[0];
}
