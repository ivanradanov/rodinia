/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:
  * Redistributions of source code must retain the above copyright
    notice, this list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright
    notice, this list of conditions and the following disclaimer in the
    documentation and/or other materials provided with the distribution.
  * Neither the name of NVIDIA CORPORATION nor the names of its
    contributors may be used to endorse or promote products derived
    from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *__restrict__ a,
                             float *__restrict__ b,
                             float *__restrict__ c,
                             int N) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) (((size_t)x + (size - 1)) & (~(size - 1)))

void eval (bool warmup, bool bPinGenericMemory, const int repeat) {
  int n, nelem;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;       // Pinned memory allocated on the CPU
  float *a_UA = nullptr,
        *b_UA = nullptr,
        *c_UA = nullptr;  // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c; // Device pointers for mapped memory
  float errorNorm, refNorm, ref, diff;

#if defined(__APPLE__) || defined(MACOSX)
  bPinGenericMemory = false;
  printf("Warning: Generic Pinning of System Paged memory is not support on MacOS\n");
#endif

  if (bPinGenericMemory) {
    printf("> Using Generic System Paged Memory (malloc)\n");
  } else {
    printf("> Using Host Allocated (cudaHostAlloc)\n");
  }

  if (warmup) printf("Warmup...\n");

  // Allocate mapped CPU memory

  for (nelem = 1024*1024; nelem <= (1024*1024*64); nelem = nelem*2) {

    if (!warmup)
      printf("\nvector length = %d\n", nelem);

    bytes = nelem * sizeof(float);

    if (bPinGenericMemory) {
      auto start = std::chrono::steady_clock::now();

      // Allocate generic memory with malloc() and pin it later 
      // instead of using cudaHostAlloc()
      a_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
      b_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
      c_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);

      // We need to ensure memory is aligned to 4K,
      // so we will need to pad memory accordingly
      a = (float *)ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
      b = (float *)ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
      c = (float *)ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

      hipHostRegister(a, bytes, hipHostRegisterMapped);
      hipHostRegister(b, bytes, hipHostRegisterMapped);
      hipHostRegister(c, bytes, hipHostRegisterMapped);

      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      if (!warmup)
        printf("Memory allocation (cudaHostRegister): %lf ms\n", time * 1e-6);
    } else {
      auto start = std::chrono::steady_clock::now();
      flags = hipHostMallocMapped;
      hipHostAlloc((void **)&a, bytes, flags);
      hipHostAlloc((void **)&b, bytes, flags);
      hipHostAlloc((void **)&c, bytes, flags);
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      if (!warmup)
        printf("Memory allocation (cudaHostAlloc): %lf ms\n", time * 1e-6);
    }

    // Initialize the vectors
    for (n = 0; n < nelem; n++) {
      a[n] = rand() / (float)RAND_MAX;
      b[n] = rand() / (float)RAND_MAX;
    }

    // Get the device pointers for the pinned CPU memory mapped into the GPU
    // memory space
    auto start = std::chrono::steady_clock::now();
    hipHostGetDevicePointer((void **)&d_a, (void *)a, 0);
    hipHostGetDevicePointer((void **)&d_b, (void *)b, 0);
    hipHostGetDevicePointer((void **)&d_c, (void *)c, 0);
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    if (!warmup)
      printf("cudaHostGetDevicePointer: %lf ms\n", time * 1e-6);

    // Call the GPU kernel using the pointers residing in CPU mapped memory
    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem / (float)block.x));

    start = std::chrono::steady_clock::now();
    for (n = 0; n < repeat; n++) {
      vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    if (!warmup)
      printf("Average kernel execution time: %lf ms\n", time * 1e-6 / repeat);

    // Compare the results
    if (warmup) {
      errorNorm = 0.f;
      refNorm = 0.f;

      for (n = 0; n < nelem; n++) {
        ref = a[n] + b[n];
        diff = c[n] - ref;
        errorNorm += diff * diff;
        refNorm += ref * ref;
      }

      errorNorm = (float)sqrt((double)errorNorm);
      refNorm = (float)sqrt((double)refNorm);

      printf("%s\n", (errorNorm / refNorm < 1.e-6f) ? "SUCCESS" : "FAILURE");
    }

    // Memory clean up

    if (bPinGenericMemory) {
      auto start = std::chrono::steady_clock::now();
      hipHostUnregister(a);
      hipHostUnregister(b);
      hipHostUnregister(c);
      free(a_UA);
      free(b_UA);
      free(c_UA);
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      if (!warmup)
        printf("Memory deallocation (cudaHostUnregister): %lf ms\n", time * 1e-6);
    } else {
      auto start = std::chrono::steady_clock::now();
      hipHostFree(a);
      hipHostFree(b);
      hipHostFree(c);
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      if (!warmup)
        printf("Memory deallocation (cudaFreeHost): %lf ms\n", time * 1e-6);
    }
  }
  if (warmup) printf("Done.\n");
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  bool bPinGenericMemory;

  bPinGenericMemory = false;
  eval(true, bPinGenericMemory, repeat); 
  eval(false, bPinGenericMemory, repeat); 

  bPinGenericMemory = true;
  eval(true, bPinGenericMemory, repeat); 
  eval(false, bPinGenericMemory, repeat); 
  return 0;
}
