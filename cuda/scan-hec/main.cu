#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

template<typename T>
void verify(const T* cpu_out, const T* gpu_out, int n)
{
  int error = memcmp(cpu_out, gpu_out, n * sizeof(T));
  printf("%s\n", error ? "FAIL" : "PASS");
}

// bank conflict aware optimization

#define LOG_MEM_BANKS 5
#define OFFSET(n) ((n) >> LOG_MEM_BANKS)

// N is the number of elements to scan in a thread block
template<typename T, int N>
__global__ void scan_bcao (
        T *__restrict__ g_odata,
  const T *__restrict__ g_idata)
{
  __shared__ T temp[2*N];

  int bid = blockIdx.x;
  g_idata += bid * N;
  g_odata += bid * N;

  int thid = threadIdx.x; 
  int a = thid;
  int b = a + (N/2);
  int oa = OFFSET(a);
  int ob = OFFSET(b);

  temp[a + oa] = g_idata[a];
  temp[b + ob] = g_idata[b];

  int offset = 1;
  for (int d = N >> 1; d > 0; d >>= 1) 
  {
    __syncthreads();
    if (thid < d) 
    {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      ai += OFFSET(ai);
      bi += OFFSET(bi);
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }

  if (thid == 0) temp[N-1+OFFSET(N-1)] = 0; // clear the last elem
  for (int d = 1; d < N; d *= 2) // traverse down
  {
    offset >>= 1;
    __syncthreads();
    if (thid < d)
    {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      ai += OFFSET(ai);
      bi += OFFSET(bi);
      T t = temp[ai];
      temp[ai] = temp[bi];
      temp[bi] += t;
    }
  }
  __syncthreads(); // required

  g_odata[a] = temp[a + oa];
  g_odata[b] = temp[b + ob];
}

template<typename T, int N>
__global__ void scan(
        T *__restrict__ g_odata,
  const T *__restrict__ g_idata)
{
  __shared__ T temp[N];

  int bid = blockIdx.x;
  g_idata += bid * N;
  g_odata += bid * N;

  int thid = threadIdx.x; 
  int offset = 1;
  temp[2*thid]   = g_idata[2*thid];
  temp[2*thid+1] = g_idata[2*thid+1];
  for (int d = N >> 1; d > 0; d >>= 1) 
  {
    __syncthreads();
    if (thid < d)
    {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }

  if (thid == 0) temp[N-1] = 0; // clear the last elem
  for (int d = 1; d < N; d *= 2) // traverse down
  {
    offset >>= 1;
    __syncthreads();      
    if (thid < d)
    {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      float t = temp[ai];
      temp[ai] = temp[bi];
      temp[bi] += t;
    }
  }
  g_odata[2*thid] = temp[2*thid];
  g_odata[2*thid+1] = temp[2*thid+1];
}

template <typename T, int N>
void runTest (const size_t n, const int repeat, bool timing = false) 
{
  const size_t num_blocks = (n + N - 1) / N;

  const size_t nelems = num_blocks * N; // actual total number of elements

  size_t bytes = nelems * sizeof(T);

  T *in = (T*) malloc (bytes);
  T *cpu_out = (T*) malloc (bytes);
  T *gpu_out = (T*) malloc (bytes);

  srand(123);
  for (size_t n = 0; n < nelems; n++) in[n] = rand() % 5 + 1;

  T *t_in = in;
  T *t_out = cpu_out;
  for (size_t n = 0; n < num_blocks; n++) { 
    t_out[0] = 0;
    for (int i = 1; i < N; i++) 
      t_out[i] = t_out[i-1] + t_in[i-1];
    t_out += N;
    t_in += N;
  }

  T *d_in, *d_out;

  hipMalloc((void**)&d_in, bytes);
  hipMemcpy(d_in, in, bytes, hipMemcpyHostToDevice); 

  hipMalloc((void**)&d_out, bytes);

  dim3 grids (num_blocks);
  dim3 blocks (N/2);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    scan<T, N><<<grids, blocks>>>(d_out, d_in);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  if (timing) {
    printf("Element size in bytes is %zu. Average execution time of scan (w/  bank conflicts): %f (us)\n",
           sizeof(T), (time * 1e-3f) / repeat);
  }
  hipMemcpy(gpu_out, d_out, bytes, hipMemcpyDeviceToHost);
  if (!timing) verify(cpu_out, gpu_out, nelems);

  // bcao
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    scan_bcao<T, N><<<grids, blocks>>>(d_out, d_in);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  auto bcao_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  if (timing) {
    printf("Element size in bytes is %zu. Average execution time of scan (w/o bank conflicts): %f (us). ",
           sizeof(T), (bcao_time * 1e-3f) / repeat);
    printf("Reduce the time by %.1f%%\n", (time - bcao_time) * 1.0 / time * 100);
  }
  hipMemcpy(gpu_out, d_out, bytes, hipMemcpyDeviceToHost);
  if (!timing) verify(cpu_out, gpu_out, nelems);

  hipFree(d_in);
  hipFree(d_out);
  free(in);
  free(cpu_out);
  free(gpu_out);
}

template<int N>
void run (const int n, const int repeat) {
  for (int i = 0; i < 2; i++) {
    bool report_timing = i > 0;
    printf("\nThe number of elements to scan in a thread block: %d\n", N);
    runTest< char, N>(n, repeat, report_timing);
    runTest<short, N>(n, repeat, report_timing);
    runTest<  int, N>(n, repeat, report_timing);
    runTest< long, N>(n, repeat, report_timing);
  }
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

  run< 128>(n, repeat);  
  run< 256>(n, repeat);  
  run< 512>(n, repeat);  
  run<1024>(n, repeat);  
  run<2048>(n, repeat);  

  return 0; 
}
