#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "constants.h"

#define R 4
#define NDIM 8

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[NDIM+2*R][NDIM+2*R][NDIM+2*R];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;
    
    const int ti = threadIdx.z;
    const int tj = threadIdx.y;
    const int tk = threadIdx.x;

    const llint i = i0 + ti;
    const llint j = j0 + tj;
    const llint k = k0 + tk;

    s_u[ti][tj][tk] = 0.f;

    if (ti < 2*R && tj < 2*R && tk< 2*R)
      s_u[NDIM+ti][NDIM+tj][NDIM+tk] = 0.f;

    __syncthreads();

    const llint sui = ti + R;
    const llint suj = tj + R;
    const llint suk = tk + R;

    const int z_side = ti / R;
    s_u[ti+z_side*NDIM][suj][suk] = u[IDX3_l(i+(z_side*2-1)*R,j,k)];
    const int y_side = tj / R;
    s_u[sui][tj+y_side*NDIM][suk] = u[IDX3_l(i,j+(y_side*2-1)*R,k)];
    s_u[sui][suj][tk] = u[IDX3_l(i,j,k-R)];
    s_u[sui][suj][tk+NDIM] = u[IDX3_l(i,j,k+R)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = coef0 * s_u[sui][suj][suk] + 
              coefx_1 * (s_u[sui+1][suj][suk] + s_u[sui-1][suj][suk]) +
              coefy_1 * (s_u[sui][suj+1][suk] + s_u[sui][suj-1][suk]) +
              coefz_1 * (s_u[sui][suj][suk+1] + s_u[sui][suj][suk-1]) +
              coefx_2 * (s_u[sui+2][suj][suk] + s_u[sui-2][suj][suk]) +
              coefy_2 * (s_u[sui][suj+2][suk] + s_u[sui][suj-2][suk]) +
              coefz_2 * (s_u[sui][suj][suk+2] + s_u[sui][suj][suk-2]) +
              coefx_3 * (s_u[sui+3][suj][suk] + s_u[sui-3][suj][suk]) +
              coefy_3 * (s_u[sui][suj+3][suk] + s_u[sui][suj-3][suk]) +
              coefz_3 * (s_u[sui][suj][suk+3] + s_u[sui][suj][suk-3]) +
              coefx_4 * (s_u[sui+4][suj][suk] + s_u[sui-4][suj][suk]) +
              coefy_4 * (s_u[sui][suj+4][suk] + s_u[sui][suj-4][suk]) +
              coefz_4 * (s_u[sui][suj][suk+4] + s_u[sui][suj][suk-4]);
    v[IDX3_l(i,j,k)] = 2.f * s_u[sui][suj][suk] + vp[IDX3(i,j,k)] * lap - v[IDX3_l(i,j,k)];
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[NDIM+2*R][NDIM+2*R][NDIM+2*R];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const int ti = threadIdx.z;
    const int tj = threadIdx.y;
    const int tk = threadIdx.x;

    const llint i = i0 + ti;
    const llint j = j0 + tj;
    const llint k = k0 + tk;

    s_u[ti][tj][tk] = 0.f;

    if (ti < 2*R && tj < 2*R && tk< 2*R)
      s_u[NDIM+ti][NDIM+tj][NDIM+tk] = 0.f;

    __syncthreads();

    const llint sui = ti + R;
    const llint suj = tj + R;
    const llint suk = tk + R;

    const int z_side = ti / R;
    s_u[ti+z_side*NDIM][suj][suk] = u[IDX3_l(i+(z_side*2-1)*R,j,k)];
    const int y_side = tj / R;
    s_u[sui][tj+y_side*NDIM][suk] = u[IDX3_l(i,j+(y_side*2-1)*R,k)];
    s_u[sui][suj][tk] = u[IDX3_l(i,j,k-R)];
    s_u[sui][suj][tk+NDIM] = u[IDX3_l(i,j,k+R)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = coef0 * s_u[sui][suj][suk] + 
              coefx_1 * (s_u[sui+1][suj][suk] + s_u[sui-1][suj][suk]) +
              coefy_1 * (s_u[sui][suj+1][suk] + s_u[sui][suj-1][suk]) +
              coefz_1 * (s_u[sui][suj][suk+1] + s_u[sui][suj][suk-1]) +
              coefx_2 * (s_u[sui+2][suj][suk] + s_u[sui-2][suj][suk]) +
              coefy_2 * (s_u[sui][suj+2][suk] + s_u[sui][suj-2][suk]) +
              coefz_2 * (s_u[sui][suj][suk+2] + s_u[sui][suj][suk-2]) +
              coefx_3 * (s_u[sui+3][suj][suk] + s_u[sui-3][suj][suk]) +
              coefy_3 * (s_u[sui][suj+3][suk] + s_u[sui][suj-3][suk]) +
              coefz_3 * (s_u[sui][suj][suk+3] + s_u[sui][suj][suk-3]) +
              coefx_4 * (s_u[sui+4][suj][suk] + s_u[sui-4][suj][suk]) +
              coefy_4 * (s_u[sui][suj+4][suk] + s_u[sui][suj-4][suk]) +
              coefz_4 * (s_u[sui][suj][suk+4] + s_u[sui][suj][suk-4]);

    const float s_eta_c = eta[IDX3_eta1(i,j,k)];

    v[IDX3_l(i,j,k)] = ((2.f*s_eta_c + 2.f - s_eta_c*s_eta_c)*s_u[sui][suj][suk] + 
		    (vp[IDX3(i,j,k)] * (lap + phi[IDX3(i,j,k)]) - v[IDX3_l(i,j,k)])) / 
	    (2.f*s_eta_c+1.f);

    phi[IDX3(i,j,k)] = 
     (phi[IDX3(i,j,k)] - 
     ((eta[IDX3_eta1(i+1,j,k)]-eta[IDX3_eta1(i-1,j,k)]) * 
     (s_u[sui+1][suj][suk]-s_u[sui-1][suj][suk]) * hdx_2 + 
     (eta[IDX3_eta1(i,j+1,k)]-eta[IDX3_eta1(i,j-1,k)]) *
     (s_u[sui][suj+1][suk]-s_u[sui][suj-1][suk]) * hdy_2 +
     (eta[IDX3_eta1(i,j,k+1)]-eta[IDX3_eta1(i,j,k-1)]) *
     (s_u[sui][suj][suk+1]-s_u[sui][suj][suk-1]) * hdz_2)) / (1.f + s_eta_c);
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);

    const llint size_u_ext = ((((nx+NDIM-1) / NDIM + 1) * NDIM) + 2 * lx)
                           * ((((ny+NDIM-1) / NDIM + 1) * NDIM) + 2 * ly)
                           * ((((nz+NDIM-1) / NDIM + 1) * NDIM) + 2 * lz);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u);
    hipMalloc(&d_v, sizeof(float) * size_u);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(NDIM, NDIM, NDIM);

    #ifdef DEBUG
    const uint npo = 100;
    #endif

    hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME, &start);

    for (uint istep = 1; istep <= nsteps; ++istep) {

        dim3 n_block_front(
            (z2-z1+NDIM-1) / NDIM,
            (ny+NDIM-1) / NDIM,
            (nx+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+NDIM-1) / NDIM,
            (y2-y1+NDIM-1) / NDIM,
            (nx+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+NDIM-1) / NDIM,
            (y4-y3+NDIM-1) / NDIM,
            (x2-x1+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+NDIM-1) / NDIM,
            (y4-y3+NDIM-1) / NDIM,
            (x4-x3+NDIM-1) / NDIM);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+NDIM-1) / NDIM,
            (y4-y3+NDIM-1) / NDIM,
            (x6-x5+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+NDIM-1) / NDIM,
            (y6-y5+NDIM-1) / NDIM,
            (nx+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+NDIM-1) / NDIM,
            (ny+NDIM-1) / NDIM,
            (nx+NDIM-1) / NDIM);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        #ifdef DEBUG
        if (istep % npo == 0) printf("time step %u / %u\n", istep, nsteps);
        #endif
    }

    hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME, &end);
    *time_kernel = (end.tv_sec  - start.tv_sec) +
                   (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}
