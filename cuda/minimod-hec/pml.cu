#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "pml.h"
#include "constants.h"

/**
 * @param profile has dimension [i_min,i_max]
 */
void pml_profile_init(float *profile, llint i_min, llint i_max, llint n_first, llint n_last,
                      float scale)
{
    llint n = i_max-i_min+1;
    llint shift = i_min-1;

    llint first_beg = 1 + shift;
    llint first_end = n_first + shift;
    llint last_beg  = n - n_last+1 + shift;
    llint last_end  = n + shift;

    for (llint i = i_min; i <= i_max; ++i) {
        profile[i] = 0.f;
    }

    float tmp = scale / POW2(first_end-first_beg+1);
    for (llint i = 1; i <= first_end-first_beg+1; ++i) {
        profile[first_end-i+1] = POW2(i)*tmp;
    }

    for (llint i = 1; i <= last_end-last_beg+1; ++i) {
        profile[last_beg+i-1] = POW2(i)*tmp;
    }
}

void pml_profile_extend(llint nx, llint ny, llint nz,
                        float *eta, const float *etax, const float *etay, const float *etaz,
                        llint xbeg, llint xend, llint ybeg, llint yend, llint zbeg, llint zend)
{
    const llint n_ghost = 1;
    for (llint ix = xbeg-n_ghost; ix <= xend+n_ghost; ++ix) {
        for (llint iy = ybeg-n_ghost; iy <= yend+n_ghost; ++iy) {
            for (llint iz = zbeg-n_ghost; iz <= zend+n_ghost; ++iz) {
                eta[IDX3_eta0(ix,iy,iz)] = etax[ix] + etay[iy] + etaz[iz];
            }
        }
    }
}

void pml_profile_extend_all(llint nx, llint ny, llint nz,
                            float *eta, const float *etax, const float *etay, const float *etaz,
                            llint xmin, llint xmax, llint ymin, llint ymax,
                            llint x1, llint x2, llint x5, llint x6,
                            llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
                            llint z1, llint z2, llint z3, llint z4, llint z5, llint z6)
{
    // Top.
    if (z1 != -1)
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,xmin,xmax,ymin,ymax,z1,z2);
    // Bottom.
    if (z5 != -5)
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,xmin,xmax,ymin,ymax,z5,z6);
    // Front.
    if ((y1!=-1) && (z3!=-3))
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,xmin,xmax,y1,y2,z3,z4);
    // Back.
    if ((y6!=-6) && (z3!=-3))
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,xmin,xmax,y5,y6,z3,z4);
    // Left.
    if ((x1!=-1) && (y3!=-3) && (z3!=-3))
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,x1,x2,y3,y4,z3,z4);
    // Right.
    if ((x6!=-6) && (y3!=-3) && (z3!=-3))
    pml_profile_extend(nx,ny,nz,eta,etax,etay,etaz,x5,x6,y3,y4,z3,z4);
}

void init_eta(llint nx, llint ny, llint nz, struct grid_t grid,
              float dt_sch,
              float *eta)
{
    for (llint i = -1; i < nx+1; ++i) {
        for (llint j = -1; j < ny+1; ++j) {
            for (llint k = -1; k < nz+1; ++k) {
                eta[IDX3_eta1(i,j,k)] = 0.f;
            }
        }
    }

    /* etax */
    float param = dt_sch * 3.f * vmax * logf(1000.f)/(2.f*grid.ndampx*grid.dx);
    float *etax = (float*) malloc(sizeof(float)*(nx+2));
    pml_profile_init(etax, 0, grid.nx+1, grid.ndampx, grid.ndampx, param);

    /* etay */
    param = dt_sch*3.f*vmax*logf(1000.f)/(2.f*grid.ndampy*grid.dy);
    float *etay = (float*) malloc(sizeof(float)*(ny+2));
    pml_profile_init(etay, 0, grid.ny+1, grid.ndampy, grid.ndampy, param);

    /* etaz */
    param = dt_sch*3.f*vmax*logf(1000.f)/(2.f*grid.ndampz*grid.dz);
    float *etaz = (float*) malloc(sizeof(float)*(nz+2));
    pml_profile_init(etaz, 0, grid.nz+1, grid.ndampz, grid.ndampz, param);

    (void)pml_profile_extend_all(nx, ny, nz,
                eta, etax, etay, etaz,
                1, nx, 1, ny,
                grid.x1+1, grid.x2, grid.x5+1, grid.x6,
                grid.y1+1, grid.y2, grid.y3+1, grid.y4, grid.y5+1, grid.y6,
                grid.z1+1, grid.z2, grid.z3+1, grid.z4, grid.z5+1, grid.z6);

    free(etax);
    free(etay);
    free(etaz);
}
