#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#ifdef TIMING
#include "timing.h"
#endif

#define MAX_THREADS_PER_BLOCK 512

FILE *fp;

#ifdef TIMING
struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

#include "kernel.cu"
#include "kernel2.cu"

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	BFSGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv) 
{
	int no_of_nodes=0;
	int edge_list_size=0;
    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}

	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);

	int start, edgeno;   
	// initialize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].starting = start;
		h_graph_nodes[i].no_of_edges = edgeno;
		h_graph_mask[i]=false;
		h_updating_graph_mask[i]=false;
		h_graph_visited[i]=false;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;

	//set the source node as true in the mask
	h_graph_mask[source]=true;
	h_graph_visited[source]=true;

	fscanf(fp,"%d",&edge_list_size);

	int id,cost;
	int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i] = id;
	}

	if(fp)
		fclose(fp);    

	printf("Read File\n");

#ifdef  TIMING
    gettimeofday(&tv_total_start, NULL);
#endif
	//Copy the Node list to device memory
	Node* d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	//Copy the Edge List to device Memory
	int* d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

	//Copy the Mask to device memory
	bool* d_graph_mask;
	hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

	bool* d_updating_graph_mask;
	hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
	hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes) ;
	hipMemcpy( d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

	// allocate mem for the result on host side
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
	for(int i=0;i<no_of_nodes;i++)
		h_cost[i]=-1;
	h_cost[source]=0;
	
	// allocate device memory for result
	int* d_cost;
	hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes);
	hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

	//make a bool to check if the execution is over
	bool *d_over;
	hipMalloc( (void**) &d_over, sizeof(bool));
#ifdef  TIMING
    gettimeofday(&tv_mem_alloc_end, NULL);
    tvsub(&tv_mem_alloc_end, &tv_total_start, &tv);
    h2d_time = tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

	printf("Copied Everything to GPU memory\n");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	printf("Start traversing the tree\n");
	bool stop;
	//Call the Kernel untill all the elements of Frontier are not false

	MY_START_CLOCK(bfs, );
	do
	{
		//if no thread changes this value then the loop stops
		stop=false;
#ifdef  TIMING
		gettimeofday(&tv_h2d_start, NULL);
#endif
		hipMemcpy( d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;
#ifdef  TIMING
		gettimeofday(&tv_h2d_end, NULL);
		tvsub(&tv_h2d_end, &tv_h2d_start, &tv);
		h2d_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

		Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
		// check if kernel execution generated and error

		Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
		// check if kernel execution generated and error

#ifdef  TIMING
		hipDeviceSynchronize();
		gettimeofday(&tv_kernel_end, NULL);
		tvsub(&tv_kernel_end, &tv_h2d_end, &tv);
		kernel_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

		hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
#ifdef  TIMING
		gettimeofday(&tv_d2h_end, NULL);
		tvsub(&tv_d2h_end, &tv_kernel_end, &tv);
		d2h_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

		k++;
	}
	while(stop);
	MY_STOP_CLOCK(bfs, );


	printf("Kernel Executed %d times\n",k);

	// copy result from device to host
#ifdef  TIMING
	gettimeofday(&tv_d2h_start, NULL);
#endif
	hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;
#ifdef  TIMING
	gettimeofday(&tv_d2h_end, NULL);
	tvsub(&tv_d2h_end, &tv_d2h_start, &tv);
	d2h_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif




	MY_VERIFY_INT(h_cost, no_of_nodes);

	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) cost:%d\n",i,h_cost[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");


	// cleanup memory
	free( h_graph_nodes);
	free( h_graph_edges);
	free( h_graph_mask);
	free( h_updating_graph_mask);
	free( h_graph_visited);
	free( h_cost);
#ifdef  TIMING
    gettimeofday(&tv_close_start, NULL);
#endif
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);
	hipFree(d_cost);

#ifdef  TIMING
	gettimeofday(&tv_close_end, NULL);
	tvsub(&tv_close_end, &tv_close_start, &tv);
	close_time = tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
	tvsub(&tv_close_end, &tv_total_start, &tv);
	total_time = tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;

	printf("Init: %f\n", init_time);
	printf("MemAlloc: %f\n", mem_alloc_time);
	printf("HtoD: %f\n", h2d_time);
	printf("Exec: %f\n", kernel_time);
	printf("DtoH: %f\n", d2h_time);
	printf("Close: %f\n", close_time);
	printf("Total: %f\n", total_time);
#endif
}
