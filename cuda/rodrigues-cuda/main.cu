#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ 
void rotate (const int n, const float angle, const float3 w, float3 *d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  float s, c;
  sincosf(angle, &s,&c);
  
  const float3 p = d[i];
  const float mc = 1.f - c;

  // Rodrigues' formula:
  float m1 = c+(w.x)*(w.x)*(mc);
  float m2 = (w.z)*s+(w.x)*(w.y)*(mc);
  float m3 =-(w.y)*s+(w.x)*(w.z)*(mc);
  
  float m4 =-(w.z)*s+(w.x)*(w.y)*(mc);
  float m5 = c+(w.y)*(w.y)*(mc);
  float m6 = (w.x)*s+(w.y)*(w.z)*(mc);
  
  float m7 = (w.y)*s+(w.x)*(w.z)*(mc);
  float m8 =-(w.x)*s+(w.y)*(w.z)*(mc);
  float m9 = c+(w.z)*(w.z)*(mc);

  float ox = p.x*m1 + p.y*m2 + p.z*m3;
  float oy = p.x*m4 + p.y*m5 + p.z*m6;
  float oz = p.x*m7 + p.y*m8 + p.z*m9;
  d[i] = {ox, oy, oz};
}

__global__ 
void rotate2 (const int n, const float angle, const float3 w, float4 *d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  float s, c;
  sincosf(angle, &s,&c);
  
  const float4 p = d[i];
  const float mc = 1.f - c;

  // Rodrigues' formula:
  float m1 = c+(w.x)*(w.x)*(mc);
  float m2 = (w.z)*s+(w.x)*(w.y)*(mc);
  float m3 =-(w.y)*s+(w.x)*(w.z)*(mc);
  
  float m4 =-(w.z)*s+(w.x)*(w.y)*(mc);
  float m5 = c+(w.y)*(w.y)*(mc);
  float m6 = (w.x)*s+(w.y)*(w.z)*(mc);
  
  float m7 = (w.y)*s+(w.x)*(w.z)*(mc);
  float m8 =-(w.x)*s+(w.y)*(w.z)*(mc);
  float m9 = c+(w.z)*(w.z)*(mc);

  float ox = p.x*m1 + p.y*m2 + p.z*m3;
  float oy = p.x*m4 + p.y*m5 + p.z*m6;
  float oz = p.x*m7 + p.y*m8 + p.z*m9;
  d[i] = {ox, oy, oz, 0.f};
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <number of points> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);
    
  float3 w = make_float3(-0.33f, -0.66, 0.66);
  float angle = 0.5f;

  float3 *h = (float3*) malloc (sizeof(float3) * n);
  float4 *h2 = (float4*) malloc (sizeof(float4) * n);

  srand(123);
  for (int i = 0; i < n; i++) {
    float a = rand();
    float b = rand();
    float c = rand();
    float d = sqrtf(a*a + b*b + c*c);
    h[i] = make_float3(a/d, b/d, c/d);
    h2[i] = make_float4(a/d, b/d, c/d, 0.f);
  }

  dim3 grids ((n + 255) / 256);
  dim3 blocks (256);
 
  float3 *d;
  hipMalloc((void**)&d, sizeof(float3) * n);
  hipMemcpy(d, h, sizeof(float3) * n, hipMemcpyHostToDevice);

  float4 *d2;
  hipMalloc((void**)&d2, sizeof(float4) * n);
  hipMemcpy(d2, h2, sizeof(float4) * n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    rotate <<<grids, blocks>>> (n, angle, w, d);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (float3): %f (us)\n", (time * 1e-3f) / repeat);

  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    rotate2 <<<grids, blocks>>> (n, angle, w, d2);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (float4): %f (us)\n", (time * 1e-3f) / repeat);

  hipFree(d);
  hipFree(d2);
  free(h);
  free(h2);
  return 0;
}
