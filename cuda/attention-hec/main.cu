#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

__global__ 
void kernel1 (
    const float*__restrict__ key, 
    const float*__restrict__ query, 
    float*__restrict__ dot_product, 
    float*__restrict__ exp_sum, 
    const int n,
    const int d) 
{

  int i = blockIdx.x * blockDim.x + threadIdx.x;  
  if (i < n) {
    float sum = 0;
    for (int j = 0; j < d; j++)
      sum += key[i * d + j] * query[j];
    dot_product[i] = sum;
    atomicAdd(exp_sum, expf(sum));
  }
}

__global__ 
void kernel2 (
    const float*__restrict__ exp_sum, 
    const float*__restrict__ dot_product, 
    float*__restrict__ score, 
    const int n)
{

  int i = blockIdx.x * blockDim.x + threadIdx.x;  
  if (i < n)
    score[i] = expf(dot_product[i]) / exp_sum[0];
}

__global__ 
void kernel3 (
    const float*__restrict__ score, 
    const float*__restrict__ value, 
    float*__restrict__ output, 
    const int n,
    const int d) 
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;  
  if (j < d) {
    float sum = 0;
    for (int i = 0; i < n; i++)
      sum += score[i] * value[i * d + j];
    output[j] = sum;
  }
}

float* attention_device(const float* key, const float* value, const float* query,
                        const int n, const int d, const int repeat) 
{
  // input
  float *d_key;
  hipMalloc((void**)&d_key, n * d * sizeof(float)); 
  hipMemcpy(d_key, key, n * d * sizeof(float), hipMemcpyHostToDevice); 

  float *d_value;
  hipMalloc((void**)&d_value, n * d * sizeof(float)); 
  hipMemcpy(d_value, value, n * d * sizeof(float), hipMemcpyHostToDevice); 

  float *d_query;
  hipMalloc((void**)&d_query, d * sizeof(float)); 
  hipMemcpy(d_query, query, d * sizeof(float), hipMemcpyHostToDevice); 

  // intermediate
  float *d_dot_product;
  hipMalloc((void**)&d_dot_product, n * sizeof(float));

  float *d_score;
  hipMalloc((void**)&d_score, n * sizeof(float));

  float *d_exp_sum;
  hipMalloc((void**)&d_exp_sum, sizeof(float));

  // result
  float *output = (float*) malloc (d * sizeof(float));
  float *d_output;
  hipMalloc((void**)&d_output, d * sizeof(float));

  dim3 n_grid((n+255)/256);
  dim3 n_block(256);
  dim3 d_grid((d+255)/256);
  dim3 d_block(256);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int k = 0; k < repeat; k++) {
    hipMemset(d_exp_sum, 0, 4);

    kernel1<<<n_grid, n_block>>>(d_key, d_query, d_dot_product, d_exp_sum, n, d);

    kernel2<<<n_grid, n_block>>>(d_exp_sum, d_dot_product, d_score, n);

    kernel3<<<d_grid, d_block>>>(d_score, d_value, d_output, n, d);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of kernels %f (ms)\n", time * 1e-6f / repeat);

  hipMemcpy(output, d_output, d * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_score);
  hipFree(d_value);
  hipFree(d_output);
  hipFree(d_key);
  hipFree(d_dot_product);
  hipFree(d_exp_sum);
  return output;
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Usage: %s <rows> <columns> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int d = atoi(argv[2]);
  const int r = atoi(argv[3]);

  // input
  float* key = (float*) malloc (n * d * sizeof(float));
  float* value = (float*) malloc (n * d * sizeof(float));
  float* query = (float*) malloc (d * sizeof(float));

  srand(2);
  for (int i = 0; i < n * d; i++) {
    key[i] = 0.1;
    value[i] = 0.3;
    if (rand() % 2)
      query[i % d] = value[i] + key[i] ;
    else
      query[i % d] = value[i] - key[i] ;
  }

  float* hout = attention_host(key, value, query, n, d);

  float* dout = attention_device(key, value, query, n, d, r);

  float rmse = 0;
  for (int i = 0; i < d; i++) 
    rmse += (hout[i] - dout[i]) * (hout[i] - dout[i]);
  printf("RMSE = %f\n", sqrtf(rmse / d));

  free(key);
  free(value);
  free(query);
  free(dout);
  free(hout);
  return 0;
}
