/*
 ** Code to implement a d2q9-bgk lattice boltzmann scheme.
 ** 'd2' inidates a 2-dimensional grid, and
 ** 'q9' indicates 9 velocities per grid cell.
 ** 'bgk' refers to the Bhatnagar-Gross-Krook collision step.
 **
 ** The 'speeds' in each cell are numbered as follows:
 **
 ** 6 2 5
 **  \|/
 ** 3-0-1
 **  /|\
 ** 7 4 8
 **
 ** A 2D grid:
 **
 **           cols
 **       --- --- ---
 **      | D | E | F |
 ** rows  --- --- ---
 **      | A | B | C |
 **       --- --- ---
 **
 ** 'unwrapped' in row major order to give a 1D array:
 **
 **  --- --- --- --- --- ---
 ** | A | B | C | D | E | F |
 **  --- --- --- --- --- ---
 **
 ** Grid indicies are:
 **
 **          ny
 **          ^       cols(ii)
 **          |  ----- ----- -----
 **          | | ... | ... | etc |
 **          |  ----- ----- -----
 ** rows(jj) | | 1,0 | 1,1 | 1,2 |
 **          |  ----- ----- -----
 **          | | 0,0 | 0,1 | 0,2 |
 **          |  ----- ----- -----
 **          ----------------------> nx
 **
 ** Note the names of the input parameter and obstacle files
 ** are passed on the command line, e.g.:
 **
 **   ./d2q9-bgk input.params obstacles.dat
 **
 ** Be sure to adjust the grid dimensions in the parameter file
 ** if you choose a different obstacle file.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define WARMUPS         1000
#define NSPEEDS         9
#define LOCALSIZEX      128
#define LOCALSIZEY      1

/* dump output files for verification */
#define FINALSTATEFILE  "final_state.dat"
#define AVVELSFILE      "av_vels.dat"

/* struct to hold the parameter values */
typedef struct
{
  int   nx;            /* no. of cells in x-direction */
  int   ny;            /* no. of cells in y-direction */
  int   maxIters;      /* no. of iterations */
  int   reynolds_dim;  /* dimension for Reynolds number */
  float density;       /* density per link */
  float accel;         /* density redistribution */
  float omega;         /* relaxation parameter */
} t_param;

/* struct to hold the 'speed' values */
typedef struct
{
  float speeds[NSPEEDS];
} t_speed;

/*
 ** function prototypes
 */

/* load params, allocate memory, load obstacles & initialise fluid particle densities */
int initialise(const char* paramfile, const char* obstaclefile,
    t_param* params, t_speed** cells_ptr, t_speed** tmp_cells_ptr,
    int** obstacles_ptr, float** av_vels_ptr);

/*
 ** The main calculation methods.
 ** timestep calls, in order, the functions:
 ** accelerate_flow(), propagate(), rebound() & collision()
 */
int write_values(const t_param params, t_speed* cells, int* obstacles, float* av_vels);

/* finalise, including freeing up allocated memory */
int finalise(t_speed* cells_ptr, t_speed* tmp_cells_ptr,
    int* obstacles_ptr, float* av_vels_ptr);

/* Sum all the densities in the grid.
 ** The total should remain constant from one timestep to the next. */
float total_density(const t_param params, t_speed* cells);

/* compute average velocity */
float av_velocity(const t_param params, t_speed* cells, int* obstacles);

/* calculate Reynolds number */
float calc_reynolds(const t_param params, t_speed* cells, int* obstacles);

/* utility functions */
void die(const char* message, const int line, const char* file);
void usage(const char* exe);

__device__ bool 
isGreater(const float x, const float y) 
{
  return x > y ? 1 : 0;
}

__global__ void d2q9_bgk(
  const float* __restrict__ Speed0A,
  const float* __restrict__ Speed1A,
  const float* __restrict__ Speed2A,
  const float* __restrict__ Speed3A,
  const float* __restrict__ Speed4A,
  const float* __restrict__ Speed5A,
  const float* __restrict__ Speed6A,
  const float* __restrict__ Speed7A,
  const float* __restrict__ Speed8A,
  float* __restrict__ Tmp0A,
  float* __restrict__ Tmp1A,
  float* __restrict__ Tmp2A,
  float* __restrict__ Tmp3A,
  float* __restrict__ Tmp4A,
  float* __restrict__ Tmp5A,
  float* __restrict__ Tmp6A,
  float* __restrict__ Tmp7A,
  float* __restrict__ Tmp8A,
  const int* __restrict__ ObstaclesA,
  float* __restrict__ Partial_Sum,
  int* __restrict__ Partial_Sum2,
  const float densityaccel,
  const float omega,
  const int nx,
  const int ny,
  const int tt)
{
  //setup local memory
  __shared__ int local_sum2[LOCALSIZEX*LOCALSIZEY];
  __shared__ float local_sum[LOCALSIZEX*LOCALSIZEY];

  /* get column and row indices */
  const int ii = blockIdx.x * blockDim.x + threadIdx.x;
  const int jj = blockIdx.y * blockDim.y + threadIdx.y;

  const float c_sq_inv = 3.f;
  const float c_sq = 1.f/c_sq_inv; /* square of speed of sound */
  const float temp1 = 4.5f;
  const float w1 = 1.f/9.f;
  const float w0 = 4.f * w1;  /* weighting factor */
  const float w2 = 1.f/36.f; /* weighting factor */
  const float w11 = densityaccel * w1;
  const float w21 = densityaccel * w2;

  /* determine indices of axis-direction neighbours
   ** respecting periodic boundary conditions (wrap around) */
  const int y_n = (jj + 1) % ny;
  const int x_e = (ii + 1) % nx;
  const int y_s = (jj == 0) ? (jj + ny - 1) : (jj - 1);
  const int x_w = (ii == 0) ? (ii + nx - 1) : (ii - 1);

  /* propagate densities from neighbouring cells, following
   ** appropriate directions of travel and writing into
   ** scratch space grid */

  float tmp_s0 = Speed0A[ii + jj*nx];
  float tmp_s1 = (jj == ny-2 && (!ObstaclesA[x_w + jj*nx] && isGreater((Speed3A[x_w + jj*nx] - w11) , 0.f) && isGreater((Speed6A[x_w + jj*nx] - w21) , 0.f) && isGreater((Speed7A[x_w + jj*nx] - w21) , 0.f))) ? Speed1A[x_w + jj*nx]+w11 : Speed1A[x_w + jj*nx];
  float tmp_s2 = Speed2A[ii + y_s*nx];
  float tmp_s3 = (jj == ny-2 && (!ObstaclesA[x_e + jj*nx] && isGreater((Speed3A[x_e + jj*nx] - w11) , 0.f) && isGreater((Speed6A[x_e + jj*nx] - w21) , 0.f) && isGreater((Speed7A[x_e + jj*nx] - w21) , 0.f))) ? Speed3A[x_e + jj*nx]-w11 : Speed3A[x_e + jj*nx];
  float tmp_s4 = Speed4A[ii + y_n*nx];
  float tmp_s5 = (y_s == ny-2 && (!ObstaclesA[x_w + y_s*nx] && isGreater((Speed3A[x_w + y_s*nx] - w11) , 0.f) && isGreater((Speed6A[x_w + y_s*nx] - w21) , 0.f) && isGreater((Speed7A[x_w + y_s*nx] - w21) , 0.f))) ? Speed5A[x_w + y_s*nx]+w21 : Speed5A[x_w + y_s*nx];
  float tmp_s6 = (y_s == ny-2 && (!ObstaclesA[x_e + y_s*nx] && isGreater((Speed3A[x_e + y_s*nx] - w11) , 0.f) && isGreater((Speed6A[x_e + y_s*nx] - w21) , 0.f) && isGreater((Speed7A[x_e + y_s*nx] - w21) , 0.f))) ? Speed6A[x_e + y_s*nx]-w21 : Speed6A[x_e + y_s*nx];
  float tmp_s7 = (y_n == ny-2 && (!ObstaclesA[x_e + y_n*nx] && isGreater((Speed3A[x_e + y_n*nx] - w11) , 0.f) && isGreater((Speed6A[x_e + y_n*nx] - w21) , 0.f) && isGreater((Speed7A[x_e + y_n*nx] - w21) , 0.f))) ? Speed7A[x_e + y_n*nx]-w21 : Speed7A[x_e + y_n*nx];
  float tmp_s8 = (y_n == ny-2 && (!ObstaclesA[x_w + y_n*nx] && isGreater((Speed3A[x_w + y_n*nx] - w11) , 0.f) && isGreater((Speed6A[x_w + y_n*nx] - w21) , 0.f) && isGreater((Speed7A[x_w + y_n*nx] - w21) , 0.f))) ? Speed8A[x_w + y_n*nx]+w21 : Speed8A[x_w + y_n*nx];

  /* compute local density total */
  float local_density = tmp_s0 + tmp_s1 + tmp_s2 + tmp_s3 + tmp_s4  + tmp_s5  + tmp_s6  + tmp_s7  + tmp_s8;
  const float local_density_recip = 1.f/(local_density);
  /* compute x velocity component */
  float u_x = (tmp_s1
      + tmp_s5
      + tmp_s8
      - tmp_s3
      - tmp_s6
      - tmp_s7)
    * local_density_recip;
  /* compute y velocity component */
  float u_y = (tmp_s2
      + tmp_s5
      + tmp_s6
      - tmp_s4
      - tmp_s8
      - tmp_s7)
    * local_density_recip;

  /* velocity squared */
  const float temp2 = - (u_x * u_x + u_y * u_y)/(2.f * c_sq);

  /* equilibrium densities */
  float d_equ[NSPEEDS];
  /* zero velocity density: weight w0 */
  d_equ[0] = w0 * local_density
    * (1.f + temp2);
  /* axis speeds: weight w1 */
  d_equ[1] = w1 * local_density * (1.f + u_x * c_sq_inv
      + (u_x * u_x) * temp1
      + temp2);
  d_equ[2] = w1 * local_density * (1.f + u_y * c_sq_inv
      + (u_y * u_y) * temp1
      + temp2);
  d_equ[3] = w1 * local_density * (1.f - u_x * c_sq_inv
      + (u_x * u_x) * temp1
      + temp2);
  d_equ[4] = w1 * local_density * (1.f - u_y * c_sq_inv
      + (u_y * u_y) * temp1
      + temp2);
  /* diagonal speeds: weight w2 */
  d_equ[5] = w2 * local_density * (1.f + (u_x + u_y) * c_sq_inv
      + ((u_x + u_y) * (u_x + u_y)) * temp1
      + temp2);
  d_equ[6] = w2 * local_density * (1.f + (-u_x + u_y) * c_sq_inv
      + ((-u_x + u_y) * (-u_x + u_y)) * temp1
      + temp2);
  d_equ[7] = w2 * local_density * (1.f + (-u_x - u_y) * c_sq_inv
      + ((-u_x - u_y) * (-u_x - u_y)) * temp1
      + temp2);
  d_equ[8] = w2 * local_density * (1.f + (u_x - u_y) * c_sq_inv
      + ((u_x - u_y) * (u_x - u_y)) * temp1
      + temp2);

  float tmp;
  int expression = ObstaclesA[ii + jj*nx];
  tmp_s0 = expression ? tmp_s0 : (tmp_s0 + omega * (d_equ[0] - tmp_s0));
  tmp = tmp_s1;
  tmp_s1 = expression ? tmp_s3 : (tmp_s1 + omega * (d_equ[1] - tmp_s1));
  tmp_s3 = expression ? tmp : (tmp_s3 + omega * (d_equ[3] - tmp_s3));
  tmp = tmp_s2;
  tmp_s2 = expression ? tmp_s4 : (tmp_s2 + omega * (d_equ[2] - tmp_s2));
  tmp_s4 = expression ? tmp : (tmp_s4 + omega * (d_equ[4] - tmp_s4));
  tmp = tmp_s5;
  tmp_s5 = expression ? tmp_s7 : (tmp_s5 + omega * (d_equ[5] - tmp_s5));
  tmp_s7 = expression ? tmp : (tmp_s7 + omega * (d_equ[7] - tmp_s7));
  tmp = tmp_s6;
  tmp_s6 = expression ? tmp_s8 : (tmp_s6 + omega * (d_equ[6] - tmp_s6));
  tmp_s8 = expression ? tmp : (tmp_s8 + omega * (d_equ[8] - tmp_s8));

  /* local density total */
  local_density = 1.f/(tmp_s0 + tmp_s1 + tmp_s2 + tmp_s3 + tmp_s4 + tmp_s5 + tmp_s6 + tmp_s7 + tmp_s8);

  /* x-component of velocity */
  u_x = (tmp_s1
      + tmp_s5
      + tmp_s8
      - tmp_s3
      - tmp_s6
      - tmp_s7)
    * local_density;
  /* compute y velocity component */
  u_y = (tmp_s2
      + tmp_s5
      + tmp_s6
      - tmp_s4
      - tmp_s7
      - tmp_s8)
    * local_density;

  Tmp0A[ii + jj*nx] = tmp_s0;
  Tmp1A[ii + jj*nx] = tmp_s1;
  Tmp2A[ii + jj*nx] = tmp_s2;
  Tmp3A[ii + jj*nx] = tmp_s3;
  Tmp4A[ii + jj*nx] = tmp_s4;
  Tmp5A[ii + jj*nx] = tmp_s5;
  Tmp6A[ii + jj*nx] = tmp_s6;
  Tmp7A[ii + jj*nx] = tmp_s7;
  Tmp8A[ii + jj*nx] = tmp_s8;


  int local_idi = threadIdx.x;
  int local_idj = threadIdx.y;
  int local_sizei = blockDim.x;
  int local_sizej = blockDim.y;

  /* accumulate the norm of x- and y- velocity components */
  local_sum[local_idi + local_idj*local_sizei] = (ObstaclesA[ii + jj*nx]) ? 0 : hypotf(u_x,u_y);
  /* increase counter of inspected cells */
  local_sum2[local_idi + local_idj*local_sizei] = (ObstaclesA[ii + jj*nx]) ? 0 : 1 ;

  __syncthreads();

  int group_id = blockIdx.x;
  int group_id2 = blockIdx.y; 
  int group_size = gridDim.x;
  int group_size2 = gridDim.y;
  if(local_idi == 0 && local_idj == 0){
    float sum = 0.0f;
    int sum2 = 0;
    for(int i = 0; i<local_sizei*local_sizej; i++){
      sum += local_sum[i];
      sum2 += local_sum2[i];
    }
    Partial_Sum[group_id+group_id2*group_size+tt*group_size*group_size2] = sum;
    Partial_Sum2[group_id+group_id2*group_size+tt*group_size*group_size2] = sum2;
  }
}

int main(int argc, char* argv[])
{
  char*    paramfile = NULL;    /* input parameter file */
  char*    obstaclefile = NULL; /* input obstacle file */
  t_param  params;              /* struct to hold parameter values */
  t_speed* cells     = NULL;    /* grid containing fluid densities */
  t_speed* tmp_cells = NULL;    /* scratch space */
  int*     obstaclesHost = NULL;/* grid indicating which cells are blocked */
  float*   av_vels   = NULL;    /* a record of the av. velocity computed for each timestep */
  struct timeval timstr;        /* structure to hold elapsed time */
  double tic, toc;              /* floating point numbers to calculate elapsed wallclock time */

  /* parse the command line */
  if (argc != 3)
  {
    usage(argv[0]);
  }
  else
  {
    paramfile = argv[1];
    obstaclefile = argv[2];
  }

  /* initialise our data structures and load values from file */
  initialise(paramfile, obstaclefile, &params, &cells, 
      &tmp_cells, &obstaclesHost, &av_vels);

  // declare host arrays
  int Ny = params.ny;
  int Nx = params.nx;
  int MaxIters = params.maxIters;

  float *speedsHostS0 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS1 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS2 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS3 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS4 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS5 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS6 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS7 = (float*) malloc (sizeof(float)*Ny*Nx);
  float *speedsHostS8 = (float*) malloc (sizeof(float)*Ny*Nx);

  float *tot_up = (float*) malloc (sizeof(float) * (Ny/LOCALSIZEY) * (Nx/LOCALSIZEX) * MaxIters);
  int *tot_cellsp = (int*) malloc (sizeof(int) * (Ny/LOCALSIZEY) * (Nx/LOCALSIZEX) * MaxIters);

  // Init arrays
  /* loop over _all_ cells */
  for (int jj = 0; jj < Ny; jj++)
  {
    for (int ii = 0; ii < Nx; ii++)
    {
      speedsHostS0[ii + jj*Nx] = cells[ii + jj*Nx].speeds[0];
      speedsHostS1[ii + jj*Nx] = cells[ii + jj*Nx].speeds[1];
      speedsHostS2[ii + jj*Nx] = cells[ii + jj*Nx].speeds[2];
      speedsHostS3[ii + jj*Nx] = cells[ii + jj*Nx].speeds[3];
      speedsHostS4[ii + jj*Nx] = cells[ii + jj*Nx].speeds[4];
      speedsHostS5[ii + jj*Nx] = cells[ii + jj*Nx].speeds[5];
      speedsHostS6[ii + jj*Nx] = cells[ii + jj*Nx].speeds[6];
      speedsHostS7[ii + jj*Nx] = cells[ii + jj*Nx].speeds[7];
      speedsHostS8[ii + jj*Nx] = cells[ii + jj*Nx].speeds[8];
    }
  }

  // Creating buffers which are bound to host arrays
  float *speeds0, *speeds1, *speeds2, *speeds3, *speeds4,
    *speeds5, *speeds6, *speeds7, *speeds8;

  float *tmp_speeds0, *tmp_speeds1, *tmp_speeds2, *tmp_speeds3, *tmp_speeds4,
    *tmp_speeds5, *tmp_speeds6, *tmp_speeds7, *tmp_speeds8;

  hipMalloc((void**)&speeds0, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds0, speedsHostS0, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds1, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds1, speedsHostS1, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds2, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds2, speedsHostS2, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds3, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds3, speedsHostS3, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds4, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds4, speedsHostS4, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds5, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds5, speedsHostS5, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds6, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds6, speedsHostS6, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds7, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds7, speedsHostS7, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);
  hipMalloc((void**)&speeds8, sizeof(float)*Ny*Nx);
  hipMemcpy(speeds8, speedsHostS8, sizeof(float)*Ny*Nx, hipMemcpyHostToDevice);

  hipMalloc((void**)&tmp_speeds0, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds1, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds2, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds3, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds4, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds5, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds6, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds7, sizeof(float)*Ny*Nx);
  hipMalloc((void**)&tmp_speeds8, sizeof(float)*Ny*Nx);

  int *obstacles, *partial_sum2; 
  float *partial_sum; 

  hipMalloc((void**)&obstacles, sizeof(int)*Ny*Nx);
  hipMalloc((void**)&partial_sum, sizeof(float)*(Ny/LOCALSIZEY)*(Nx/LOCALSIZEX)*MaxIters);
  hipMalloc((void**)&partial_sum2, sizeof(int)*(Ny/LOCALSIZEY)*(Nx/LOCALSIZEX)*MaxIters);

  hipMemcpy(obstacles, obstaclesHost, sizeof(int)*Ny*Nx, hipMemcpyHostToDevice);

  // parameters for kernel 
  float omega = params.omega;
  float densityaccel = params.density*params.accel;

  dim3 grids(Nx/LOCALSIZEX, Ny/LOCALSIZEY);
  dim3 threads(LOCALSIZEX, LOCALSIZEY);

  for (int tt = 0; tt < MaxIters; tt++){
    if (tt == WARMUPS - 1) {
      //start timer after warmup
      hipDeviceSynchronize();
      gettimeofday(&timstr, NULL);
      tic = timstr.tv_sec * 1e6 + timstr.tv_usec;
    }
    d2q9_bgk<<<grids, threads>>>(
        speeds0, 
        speeds1,
        speeds2,
        speeds3,
        speeds4,
        speeds5,
        speeds6,
        speeds7,
        speeds8,
        tmp_speeds0, 
        tmp_speeds1,
        tmp_speeds2,
        tmp_speeds3,
        tmp_speeds4,
        tmp_speeds5,
        tmp_speeds6,
        tmp_speeds7,
        tmp_speeds8,
        obstacles,
        partial_sum,
        partial_sum2,
        densityaccel,
        omega,
        Nx,
        Ny,
        tt );

    // swap the buffers
    float* speed_tmp = speeds0;
    speeds0 = tmp_speeds0;
    tmp_speeds0 = speed_tmp;

    speed_tmp = speeds1;
    speeds1 = tmp_speeds1;
    tmp_speeds1 = speed_tmp;

    speed_tmp = speeds2;
    speeds2 = tmp_speeds2;
    tmp_speeds2 = speed_tmp;

    speed_tmp = speeds3;
    speeds3 = tmp_speeds3;
    tmp_speeds3 = speed_tmp;

    speed_tmp = speeds4;
    speeds4 = tmp_speeds4;
    tmp_speeds4 = speed_tmp;

    speed_tmp = speeds5;
    speeds5 = tmp_speeds5;
    tmp_speeds5 = speed_tmp;

    speed_tmp = speeds6;
    speeds6 = tmp_speeds6;
    tmp_speeds6 = speed_tmp;

    speed_tmp = speeds7;
    speeds7 = tmp_speeds7;
    tmp_speeds7 = speed_tmp;

    speed_tmp = speeds8;
    speeds8 = tmp_speeds8;
    tmp_speeds8 = speed_tmp;
  }

  //end timer
  hipDeviceSynchronize();
  gettimeofday(&timstr, NULL);
  toc = timstr.tv_sec * 1e6 + timstr.tv_usec;
  printf("After warmup for %d iterations, ", WARMUPS);
  printf("average kernel execution time over %d iterations:\t\t\t%.6lf (us)\n",
         MaxIters - WARMUPS, (toc - tic) / (MaxIters - WARMUPS));

  hipMemcpy(tot_up, partial_sum, sizeof(float)*(Ny/LOCALSIZEY)*(Nx/LOCALSIZEX)*MaxIters, hipMemcpyDeviceToHost);
  hipMemcpy(tot_cellsp, partial_sum2, sizeof(int)*(Ny/LOCALSIZEY)*(Nx/LOCALSIZEX)*MaxIters, hipMemcpyDeviceToHost);

  hipMemcpy(speedsHostS0, speeds0, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS1, speeds1, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS2, speeds2, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS3, speeds3, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS4, speeds4, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS5, speeds5, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS6, speeds6, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS7, speeds7, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);
  hipMemcpy(speedsHostS8, speeds8, sizeof(float)*Ny*Nx, hipMemcpyDeviceToHost);

  hipFree(speeds0); 
  hipFree(speeds1); 
  hipFree(speeds2);
  hipFree(speeds3);
  hipFree(speeds4);
  hipFree(speeds5);
  hipFree(speeds6); 
  hipFree(speeds7);
  hipFree(speeds8);
  hipFree(tmp_speeds0); 
  hipFree(tmp_speeds1); 
  hipFree(tmp_speeds2);
  hipFree(tmp_speeds3);
  hipFree(tmp_speeds4);
  hipFree(tmp_speeds5);
  hipFree(tmp_speeds6); 
  hipFree(tmp_speeds7);
  hipFree(tmp_speeds8);
  hipFree(obstacles);
  hipFree(partial_sum2); 
  hipFree(partial_sum); 

  float tot_u = 0;
  int tot_cells = 0;
  for (int tt = 0; tt < params.maxIters; tt++){
    tot_u = 0;
    tot_cells = 0;
    for(int i = 0; i < params.nx/LOCALSIZEX*params.ny/LOCALSIZEY; i++){
      tot_u += tot_up[i+tt*params.nx/LOCALSIZEX*params.ny/LOCALSIZEY];
      tot_cells += tot_cellsp[i+tt*params.nx/LOCALSIZEX*params.ny/LOCALSIZEY];
      //printf("%d %f %d\n", i, tot_u, tot_cells);
    }
    av_vels[tt] = tot_u/tot_cells;
  }

  // put answers back into cells
  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      cells[ii + jj*params.nx].speeds[0] = speedsHostS0[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[1] = speedsHostS1[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[2] = speedsHostS2[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[3] = speedsHostS3[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[4] = speedsHostS4[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[5] = speedsHostS5[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[6] = speedsHostS6[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[7] = speedsHostS7[ii + jj*params.nx];
      cells[ii + jj*params.nx].speeds[8] = speedsHostS8[ii + jj*params.nx];
    }
  }

  /* write final values and free memory */
  printf("==done==\n");
  printf("Reynolds number:\t\t%.12E\n", calc_reynolds(params, cells, obstaclesHost));
  write_values(params, cells, obstaclesHost, av_vels);
  finalise(cells, tmp_cells, obstaclesHost, av_vels);

  free(speedsHostS0);
  free(speedsHostS1);
  free(speedsHostS2);
  free(speedsHostS3);
  free(speedsHostS4);
  free(speedsHostS5);
  free(speedsHostS6);
  free(speedsHostS7);
  free(speedsHostS8);
  free(tot_up);
  free(tot_cellsp);

  return EXIT_SUCCESS;
}

float av_velocity(const t_param params, t_speed* cells, int* obstacles)
{
  int    tot_cells = 0;  /* no. of cells used in calculation */
  float tot_u;          /* accumulated magnitudes of velocity for each cell */

  /* initialise */
  tot_u = 0.f;

  /* loop over all non-blocked cells */
  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      /* ignore occupied cells */
      if (!obstacles[ii + jj*params.nx])
      {
        /* local density total */
        float local_density = 0.f;

        for (int kk = 0; kk < NSPEEDS; kk++)
        {
          local_density += cells[ii + jj*params.nx].speeds[kk];
        }

        /* x-component of velocity */
        float u_x = (cells[ii + jj*params.nx].speeds[1]
            + cells[ii + jj*params.nx].speeds[5]
            + cells[ii + jj*params.nx].speeds[8]
            - (cells[ii + jj*params.nx].speeds[3]
              + cells[ii + jj*params.nx].speeds[6]
              + cells[ii + jj*params.nx].speeds[7]))
          / local_density;
        /* compute y velocity component */
        float u_y = (cells[ii + jj*params.nx].speeds[2]
            + cells[ii + jj*params.nx].speeds[5]
            + cells[ii + jj*params.nx].speeds[6]
            - (cells[ii + jj*params.nx].speeds[4]
              + cells[ii + jj*params.nx].speeds[7]
              + cells[ii + jj*params.nx].speeds[8]))
          / local_density;
        /* accumulate the norm of x- and y- velocity components */
        tot_u += sqrtf((u_x * u_x) + (u_y * u_y));
        /* increase counter of inspected cells */
        ++tot_cells;
      }
    }
  }

  return tot_u / (float)tot_cells;
}

int initialise(const char* paramfile, const char* obstaclefile,
    t_param* params, t_speed** cells_ptr, t_speed** tmp_cells_ptr,
    int** obstacles_ptr, float** av_vels_ptr){
  char   message[1024];  /* message buffer */
  FILE*  fp;             /* file pointer */
  int    xx, yy;         /* generic array indices */
  int    blocked;        /* indicates whether a cell is blocked by an obstacle */
  int    retval;         /* to hold return value for checking */

  /* open the parameter file */
  fp = fopen(paramfile, "r");

  if (fp == NULL)
  {
    sprintf(message, "could not open input parameter file: %s", paramfile);
    die(message, __LINE__, __FILE__);
  }
  /* read in the parameter values */
  retval = fscanf(fp, "%d\n", &(params->nx));

  if (retval != 1) die("could not read param file: nx", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->ny));

  if (retval != 1) die("could not read param file: ny", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->maxIters));

  if (retval != 1) die("could not read param file: maxIters", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->reynolds_dim));

  if (retval != 1) die("could not read param file: reynolds_dim", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->density));

  if (retval != 1) die("could not read param file: density", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->accel));

  if (retval != 1) die("could not read param file: accel", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->omega));

  if (retval != 1) die("could not read param file: omega", __LINE__, __FILE__);

  /* and close up the file */
  fclose(fp);

  /*
   ** allocate memory.
   **
   ** remember C is pass-by-value, so we need to
   ** pass pointers into the initialise function.
   **
   ** nb we are allocating a 1D array, so that the
   ** memory will be contiguous.  We still want to
   ** index this memory as if it were a (row major
   ** ordered) 2D array, however.  We will perform
   ** some arithmetic using the row and column
   ** coordinates, inside the square brackets, when
   ** we want to access elements of this array.
   **
   ** note also that we are using a structure to
   ** hold an array of 'speeds'.  We will allocate
   ** a 1D array of these structs.
   */

  /* main grid */
  *cells_ptr = (t_speed*)malloc(sizeof(t_speed) * (params->ny * params->nx));

  if (*cells_ptr == NULL) die("cannot allocate memory for cells", __LINE__, __FILE__);

  /* 'helper' grid, used as scratch space */
  *tmp_cells_ptr = (t_speed*)malloc(sizeof(t_speed) * (params->ny * params->nx));

  if (*tmp_cells_ptr == NULL) die("cannot allocate memory for tmp_cells", __LINE__, __FILE__);

  /* the map of obstacles */
  *obstacles_ptr = (int*) malloc (sizeof(int) * params->ny * params->nx);

  if (*obstacles_ptr == NULL) die("cannot allocate column memory for obstacles", __LINE__, __FILE__);

  /* initialise densities */
  float w0 = params->density * 4.f / 9.f;
  float w1 = params->density      / 9.f;
  float w2 = params->density      / 36.f;

  for (int jj = 0; jj < params->ny; jj++)
  {
    for (int ii = 0; ii < params->nx; ii++)
    {
      /* centre */
      (*cells_ptr)[ii + jj*params->nx].speeds[0] = w0;
      /* axis directions */
      (*cells_ptr)[ii + jj*params->nx].speeds[1] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[2] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[3] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[4] = w1;
      /* diagonals */
      (*cells_ptr)[ii + jj*params->nx].speeds[5] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[6] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[7] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[8] = w2;
    }
  }

  /* first set all cells in obstacle array to zero */
  for (int jj = 0; jj < params->ny; jj++)
  {
    for (int ii = 0; ii < params->nx; ii++)
    {
      (*obstacles_ptr)[ii + jj*params->nx] = 0;
    }
  }

  /* open the obstacle data file */
  fp = fopen(obstaclefile, "r");

  if (fp == NULL)
  {
    sprintf(message, "could not open input obstacles file: %s", obstaclefile);
    die(message, __LINE__, __FILE__);
  }

  /* read-in the blocked cells list */
  while ((retval = fscanf(fp, "%d %d %d\n", &xx, &yy, &blocked)) != EOF)
  {
    /* some checks */
    if (retval != 3) die("expected 3 values per line in obstacle file", __LINE__, __FILE__);

    if (xx < 0 || xx > params->nx - 1) die("obstacle x-coord out of range", __LINE__, __FILE__);

    if (yy < 0 || yy > params->ny - 1) die("obstacle y-coord out of range", __LINE__, __FILE__);

    if (blocked != 1) die("obstacle blocked value should be 1", __LINE__, __FILE__);

    /* assign to array */
    (*obstacles_ptr)[xx + yy*params->nx] = blocked;
  }

  /* and close the file */
  fclose(fp);

  /*
   ** allocate space to hold a record of the avarage velocities computed
   ** at each timestep
   */
  *av_vels_ptr = (float*)malloc(sizeof(float) * params->maxIters);

  return EXIT_SUCCESS;
}

int finalise(t_speed* cells_ptr, t_speed* tmp_cells_ptr,
    int* obstacles_ptr, float* av_vels_ptr)
{
  /*
   ** free up allocated memory
   */
  free(cells_ptr);
  free(tmp_cells_ptr);
  free(obstacles_ptr);
  free(av_vels_ptr);

  return EXIT_SUCCESS;
}


float calc_reynolds(const t_param params, t_speed* cells, int* obstacles)
{
  const float viscosity = 1.f / 6.f * (2.f / params.omega - 1.f);

  return av_velocity(params, cells, obstacles) * params.reynolds_dim / viscosity;
}

float total_density(const t_param params, t_speed* cells)
{
  float total = 0.f;  /* accumulator */

  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      for (int kk = 0; kk < NSPEEDS; kk++)
      {
        total += cells[ii + jj*params.nx].speeds[kk];
      }
    }
  }

  return total;
}

int write_values(const t_param params, t_speed* cells, int* obstacles, float* av_vels)
{
  FILE* fp;                     /* file pointer */
  const float c_sq = 1.f / 3.f; /* sq. of speed of sound */
  float local_density;         /* per grid cell sum of densities */
  float pressure;              /* fluid pressure in grid cell */
  float u_x;                   /* x-component of velocity in grid cell */
  float u_y;                   /* y-component of velocity in grid cell */
  float u;                     /* norm--root of summed squares--of u_x and u_y */

  fp = fopen(FINALSTATEFILE, "w");

  if (fp == NULL)
  {
    die("could not open file output file", __LINE__, __FILE__);
  }

  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      /* an occupied cell */
      if (obstacles[ii + jj*params.nx])
      {
        u_x = u_y = u = 0.f;
        pressure = params.density * c_sq;
      }
      /* no obstacle */
      else
      {
        local_density = 0.f;

        for (int kk = 0; kk < NSPEEDS; kk++)
        {
          local_density += cells[ii + jj*params.nx].speeds[kk];
        }

        /* compute x velocity component */
        u_x = (cells[ii + jj*params.nx].speeds[1]
            + cells[ii + jj*params.nx].speeds[5]
            + cells[ii + jj*params.nx].speeds[8]
            - (cells[ii + jj*params.nx].speeds[3]
              + cells[ii + jj*params.nx].speeds[6]
              + cells[ii + jj*params.nx].speeds[7]))
          / local_density;
        /* compute y velocity component */
        u_y = (cells[ii + jj*params.nx].speeds[2]
            + cells[ii + jj*params.nx].speeds[5]
            + cells[ii + jj*params.nx].speeds[6]
            - (cells[ii + jj*params.nx].speeds[4]
              + cells[ii + jj*params.nx].speeds[7]
              + cells[ii + jj*params.nx].speeds[8]))
          / local_density;
        /* compute norm of velocity */
        u = sqrtf((u_x * u_x) + (u_y * u_y));
        /* compute pressure */
        pressure = local_density * c_sq;
      }

      /* write to file */
      fprintf(fp, "%d %d %.12E %.12E %.12E %.12E %d\n", ii, jj, u_x, u_y, u, pressure, obstacles[ii * params.nx + jj]);
    }
  }

  fclose(fp);

  fp = fopen(AVVELSFILE, "w");

  if (fp == NULL)
  {
    die("could not open file output file", __LINE__, __FILE__);
  }

  for (int ii = 0; ii < params.maxIters; ii++)
  {
    fprintf(fp, "%d:\t%.12E\n", ii, av_vels[ii]);
  }

  fclose(fp);

  return EXIT_SUCCESS;
}

void die(const char* message, const int line, const char* file)
{
  fprintf(stderr, "Error at line %d of file %s:\n", line, file);
  fprintf(stderr, "%s\n", message);
  fflush(stderr);
  exit(EXIT_FAILURE);
}

void usage(const char* exe)
{
  fprintf(stderr, "Usage: %s <paramfile> <obstaclefile>\n", exe);
  exit(EXIT_FAILURE);
}
