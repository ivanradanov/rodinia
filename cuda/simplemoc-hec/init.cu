#include "hip/hip_runtime.h"
#include "SimpleMOC-kernel_header.h"

// Gets I from user and sets defaults
Input * set_default_input( void )
{
  Input * I = (Input *) malloc(sizeof(Input));

  I->source_2D_regions = 5000;
  I->coarse_axial_intervals = 27;
  I->fine_axial_intervals = 5;
  I->decomp_assemblies_ax = 20; // Number of subdomains per assembly axially
#ifdef VERIFY
  I->segments = 1;
#else
  I->segments = 50000000;
#endif
  I->egroups = 128;
  I->repeat = 1;
  return I;
}

Source * initialize_sources( Input * I )
{
  I->nbytes = 0;

  // Source Data Structure Allocation
  Source * sources = (Source *) malloc( I->source_3D_regions * sizeof(Source));
  I->nbytes += I->source_3D_regions * sizeof(Source);

  // Allocate Fine Source Data
  float * data = (float *) malloc(
      I->source_3D_regions * I->fine_axial_intervals *
      I->egroups * sizeof(float));
  I->nbytes += I->source_3D_regions * I->fine_axial_intervals * I->egroups * sizeof(float);
  for( int i = 0; i < I->source_3D_regions; i++ )
    sources[i].fine_source = &data[i*I->fine_axial_intervals*I->egroups];

  // Allocate Fine Flux Data
  data = (float *) malloc(
      I->source_3D_regions * I->fine_axial_intervals * I->egroups * sizeof(float));
  I->nbytes += I->source_3D_regions * I->fine_axial_intervals * I->egroups * sizeof(float);
  for( int i = 0; i < I->source_3D_regions; i++ )
    sources[i].fine_flux = &data[i*I->fine_axial_intervals*I->egroups];

  // Allocate SigT
  data = (float *) malloc( I->source_3D_regions * I->egroups * sizeof(float));
  I->nbytes += I->source_3D_regions * I->egroups * sizeof(float);
  for( int i = 0; i < I->source_3D_regions; i++ )
    sources[i].sigT = &data[i * I->egroups];

  // Initialize fine source and flux to random numbers
  for( int i = 0; i < I->source_3D_regions; i++ )
    for( int j = 0; j < I->fine_axial_intervals; j++ )
      for( int k = 0; k < I->egroups; k++ )
      {
        sources[i].fine_source[j * I->egroups + k] = rand() / (float)RAND_MAX;
        sources[i].fine_flux[j * I->egroups + k] = rand() / (float)RAND_MAX;
      }

  // Initialize SigT Values
  for( int i = 0; i < I->source_3D_regions; i++ )
    for( int j = 0; j < I->egroups; j++ )
      sources[i].sigT[j] = rand() / (float) RAND_MAX;

  return sources;
}



SIMD_Vectors allocate_simd_vectors(Input * I)
{
  SIMD_Vectors A;
  float * ptr = (float * ) malloc( I->egroups * 14 * sizeof(float));
  A.q0 = ptr;
  ptr += I->egroups;
  A.q1 = ptr;
  ptr += I->egroups;
  A.q2 = ptr;
  ptr += I->egroups;
  A.sigT = ptr;
  ptr += I->egroups;
  A.tau = ptr;
  ptr += I->egroups;
  A.sigT2 = ptr;
  ptr += I->egroups;
  A.expVal = ptr;
  ptr += I->egroups;
  A.reuse = ptr;
  ptr += I->egroups;
  A.flux_integral = ptr;
  ptr += I->egroups;
  A.tally = ptr;
  ptr += I->egroups;
  A.t1 = ptr;
  ptr += I->egroups;
  A.t2 = ptr;
  ptr += I->egroups;
  A.t3 = ptr;
  ptr += I->egroups;
  A.t4 = ptr;

  return A;
}

// Timer function. Depends on if compiled with MPI, openmp, or vanilla
double get_time(void)
{
#ifdef OPENMP
  return omp_get_wtime();
#endif

  time_t time;
  time = clock();

  return (double) time / (double) CLOCKS_PER_SEC;
}

Source* copy_sources( Input * I, Source *S ) 
{
  Source * sources = (Source *) malloc( sizeof(Source));

  sources->fine_source = NULL;
  sources->fine_flux = NULL;
  sources->sigT = NULL;

  // Allocate Fine Source Data
  posix_memalign((void**)&(sources->fine_source), 1024,  I->source_3D_regions * I->fine_axial_intervals *
      I->egroups * sizeof(float));

  // Allocate Fine Flux Data
  posix_memalign((void**)&(sources->fine_flux), 1024,  I->source_3D_regions * I->fine_axial_intervals *
      I->egroups * sizeof(float));

  // Allocate SigT
  posix_memalign((void**)&(sources->sigT), 1024,  I->source_3D_regions * I->egroups * sizeof(float));

  // Initialize fine source and flux 
  for( int i = 0; i < I->source_3D_regions; i++ )
    for( int j = 0; j < I->fine_axial_intervals; j++ )
      for( int k = 0; k < I->egroups; k++ )
      {
        sources->fine_source[i * (I->egroups) * (I->fine_axial_intervals) + j * I->egroups + k] 
          = S[i].fine_source[j * I->egroups + k];

        sources->fine_flux[i * (I->egroups) * (I->fine_axial_intervals) + j * I->egroups + k] 
          = S[i].fine_flux[j * I->egroups + k];
      }

  // Initialize 1-D SigT Values
  for( int i = 0; i < I->source_3D_regions; i++ )
    for( int j = 0; j < I->egroups; j++ )
      sources->sigT[i*(I->egroups)+j] = S[i].sigT[j]; 

  return sources;

}

