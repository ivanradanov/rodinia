#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "SimpleMOC-kernel_header.h"

// host 
void attenuate_segment(
    Input *I, Source *S,
    int QSR_id, int FAI_id, float *state_flux,
    SIMD_Vectors *simd_vecs)
{
  // Unload local vector vectors
  float *q0 =            simd_vecs->q0;
  float *q1 =            simd_vecs->q1;
  float *q2 =            simd_vecs->q2;
  float *sigT =          simd_vecs->sigT;
  float *tau =           simd_vecs->tau;
  float *sigT2 =         simd_vecs->sigT2;
  float *expVal =        simd_vecs->expVal;
  float *reuse =         simd_vecs->reuse;
  float *flux_integral = simd_vecs->flux_integral;
  float *tally =         simd_vecs->tally;
  float *t1 =            simd_vecs->t1;
  float *t2 =            simd_vecs->t2;
  float *t3 =            simd_vecs->t3;
  float *t4 =            simd_vecs->t4;

  // Some placeholder constants - In the full app some of these are
  // calculated based off position in geometry. This treatment
  // shaves off a few FLOPS, but is not significant compared to the
  // rest of the function.
  const float dz = 0.1f;
  const float zin = 0.3f; 
  const float weight = 0.5f;
  const float mu = 0.9f;
  const float mu2 = 0.3f;
  const float ds = 0.7f;

  const int egroups = I->egroups;

  // load fine source region flux vector
  float * FSR_flux = &S[QSR_id].fine_flux[FAI_id * egroups];

  if( FAI_id == 0 )
  {
    float * f2 = &S[QSR_id].fine_source[FAI_id*egroups]; 
    float * f3 = &S[QSR_id].fine_source[(FAI_id+1)*egroups]; 
    // cycle over energy groups
#ifdef INTEL
#pragma vector
#elif defined IBM
#pragma vector_level(10)
#endif
    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y2 = f2[g];
      const float y3 = f3[g];

      // do linear "fitting"
      const float c0 = y2;
      const float c1 = (y3 - y2) / dz;

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin;
      q1[g] = c1;
      q2[g] = 0;
    }
  }
  else if ( FAI_id == I->fine_axial_intervals - 1 )
  {
    float * f1 = &S[QSR_id].fine_source[(FAI_id-1)*egroups]; 
    float * f2 = &S[QSR_id].fine_source[FAI_id*egroups]; 
    // cycle over energy groups
#ifdef INTEL
#pragma vector
#elif defined IBM
#pragma vector_level(10)
#endif
    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y1 = f1[g];
      const float y2 = f2[g];

      // do linear "fitting"
      const float c0 = y2;
      const float c1 = (y2 - y1) / dz;

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin;
      q1[g] = c1;
      q2[g] = 0;
    }
  }
  else
  {
    float * f1 = &S[QSR_id].fine_source[(FAI_id-1)*egroups]; 
    float * f2 = &S[QSR_id].fine_source[FAI_id*egroups]; 
    float * f3 = &S[QSR_id].fine_source[(FAI_id+1)*egroups]; 
    // cycle over energy groups
#ifdef INTEL
#pragma vector
#elif defined IBM
#pragma vector_level(10)
#endif
    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y1 = f1[g]; 
      const float y2 = f2[g];
      const float y3 = f3[g];

      // do quadratic "fitting"
      const float c0 = y2;
      const float c1 = (y1 - y3) / (2.f*dz);
      const float c2 = (y1 - 2.f*y2 + y3) / (2.f*dz*dz);

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin + c2*zin*zin;
      q1[g] = c1 + 2.f*c2*zin;
      q2[g] = c2;
    }
  }


  // cycle over energy groups
#ifdef INTEL
#pragma vector
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    // load total cross section
    sigT[g] = S[QSR_id].sigT[g];

    // calculate common values for efficiency
    tau[g] = sigT[g] * ds;
    sigT2[g] = sigT[g] * sigT[g];
  }


  // cycle over energy groups
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    expVal[g] = 1.f - expf( -tau[g] ); // exp is faster on many architectures
  }

  // Flux Integral

  // Re-used Term
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    reuse[g] = tau[g] * (tau[g] - 2.f) + 2.f * expVal[g] 
      / (sigT[g] * sigT2[g]); 
  }


  //#pragma vector alignednontemporal
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    // add contribution to new source flux
    flux_integral[g] = (q0[g] * tau[g] + (sigT[g] * state_flux[g] - q0[g])
        * expVal[g]) / sigT2[g] + q1[g] * mu * reuse[g] + q2[g] * mu2 
      * (tau[g] * (tau[g] * (tau[g] - 3.f) + 6.f) - 6.f * expVal[g]) 
      / (3.f * sigT2[g] * sigT2[g]);
  }


#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    // Prepare tally
    tally[g] = weight * flux_integral[g];
  }

#ifdef INTEL
#pragma vector
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    FSR_flux[g] += tally[g];
  }


  // Term 1
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    t1[g] = q0[g] * expVal[g] / sigT[g];  
  }
  // Term 2
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    t2[g] = q1[g] * mu * (tau[g] - expVal[g]) / sigT2[g]; 
  }
  // Term 3
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    t3[g] =  q2[g] * mu2 * reuse[g];
  }
  // Term 4
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    t4[g] = state_flux[g] * (1.f - expVal[g]);
  }

#ifdef VERIFY
  for( int g = 0; g < egroups; g++) {
    printf("q0[%d] = %f\n", g, q0[g]);
    printf("q1[%d] = %f\n", g, q1[g]);
    printf("q2[%d] = %f\n", g, q2[g]);
    printf("sigT[%d] = %f\n", g, sigT[g]);
    printf("tau[%d] = %f\n", g, tau[g]);
    printf("sigT2[%d] = %f\n", g, sigT2[g]);
    printf("expVal[%d] = %f\n", g, expVal[g]);
    printf("reuse[%d] = %f\n", g, reuse[g]);
    printf("flux_integral[%d] = %f\n", g, flux_integral[g]);
    printf("tally[%d] = %f\n", g, tally[g]);
    printf("t1[%d] = %f\n", g, t1[g]);
    printf("t2[%d] = %f\n", g, t2[g]);
    printf("t3[%d] = %f\n", g, t3[g]);
    printf("t4[%d] = %f\n", g, t4[g]);
  }
#endif

  // Total psi
#ifdef INTEL
#pragma vector aligned
#elif defined IBM
#pragma vector_level(10)
#endif
  for( int g = 0; g < egroups; g++)
  {
    state_flux[g] = t1[g] + t2[g] + t3[g] + t4[g];
  }
}  

__global__
void att (
  const int*__restrict__ QSR_id_acc,
  const int*__restrict__ FAI_id_acc,
  float*__restrict__ fine_flux_acc,
  float*__restrict__ fine_source_acc,
  float*__restrict__ sigT_acc,
  float*__restrict__ state_flux_acc,
  float*__restrict__ v_acc,
  const int fine_axial_intervals,
  const int egroups,
  const int segments )
{
  int gid = blockIdx.x*blockDim.x+threadIdx.x;
  if (gid >= segments) return; 

  const float dz = 0.1f;
  const float zin = 0.3f; 
  const float weight = 0.5f;
  const float mu = 0.9f;
  const float mu2 = 0.3f;
  const float ds = 0.7f;

  int QSR_id = QSR_id_acc[gid];
  int FAI_id = FAI_id_acc[gid];

  // load fine source region flux vector
  int offset = QSR_id * fine_axial_intervals * egroups;

  float *FSR_flux = fine_flux_acc + offset + FAI_id * egroups;

  float* q0 = v_acc;
  float* q1 = v_acc + egroups;
  float* q2 = v_acc + egroups * 2;
  float* sigT = v_acc + egroups * 3;
  float* tau = v_acc + egroups * 4;
  float* sigT2 = v_acc + egroups * 5;
  float* expVal = v_acc + egroups * 6;
  float* reuse = v_acc + egroups * 7;
  float* flux_integral = v_acc + egroups * 8;
  float* tally = v_acc + egroups * 9;
  float* t1 = v_acc + egroups * 10;
  float* t2 = v_acc + egroups * 11;
  float* t3 = v_acc + egroups * 12;
  float* t4 = v_acc + egroups * 13;

  if( FAI_id == 0 )
  {
    float * f2 = fine_source_acc + offset + FAI_id*egroups;
    float * f3 = fine_source_acc + offset + (FAI_id+1)*egroups; 
    // cycle over energy groups
    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y2 = f2[g];
      const float y3 = f3[g];

      // do linear "fitting"
      const float c0 = y2;
      const float c1 = (y3 - y2) / dz;

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin;
      q1[g] = c1;
      q2[g] = 0;
    }
  }
  else if ( FAI_id == fine_axial_intervals - 1 )
  {
    float * f1 = fine_source_acc + offset + (FAI_id-1)*egroups; 
    float * f2 = fine_source_acc + offset + FAI_id*egroups; 

    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y1 = f1[g];
      const float y2 = f2[g];

      // do linear "fitting"
      const float c0 = y2;
      const float c1 = (y2 - y1) / dz;

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin;
      q1[g] = c1;
      q2[g] = 0;
    }
  }
  else
  {
    float * f1 = fine_source_acc + offset + (FAI_id-1)*egroups; 
    float * f2 = fine_source_acc + offset + FAI_id*egroups; 
    float * f3 = fine_source_acc + offset + (FAI_id+1)*egroups; 
    // cycle over energy groups
    for( int g = 0; g < egroups; g++)
    {
      // load neighboring sources
      const float y1 = f1[g]; 
      const float y2 = f2[g];
      const float y3 = f3[g];

      // do quadratic "fitting"
      const float c0 = y2;
      const float c1 = (y1 - y3) / (2.f*dz);
      const float c2 = (y1 - 2.f*y2 + y3) / (2.f*dz*dz);

      // calculate q0, q1, q2
      q0[g] = c0 + c1*zin + c2*zin*zin;
      q1[g] = c1 + 2.f*c2*zin;
      q2[g] = c2;
    }
  }


  // cycle over energy groups
  offset = QSR_id * egroups;
  for( int g = 0; g < egroups; g++)
  {
    // load total cross section
    sigT[g] = sigT_acc[offset + g];

    // calculate common values for efficiency
    tau[g] = sigT[g] * ds;
    sigT2[g] = sigT[g] * sigT[g];

    expVal[g] = 1.f - exp( -tau[g] ); // exp is faster on many architectures
    reuse[g] = tau[g] * (tau[g] - 2.f) + 2.f * expVal[g] / (sigT[g] * sigT2[g]); 

    // add contribution to new source flux
    flux_integral[g] = (q0[g] * tau[g] + (sigT[g] * state_flux_acc[g] - q0[g])
        * expVal[g]) / sigT2[g] + q1[g] * mu * reuse[g] + q2[g] * mu2 
      * (tau[g] * (tau[g] * (tau[g] - 3.f) + 6.f) - 6.f * expVal[g]) 
      / (3.f * sigT2[g] * sigT2[g]);

    tally[g] = weight * flux_integral[g];
    FSR_flux[g] += tally[g];
    t1[g] = q0[g] * expVal[g] / sigT[g];  
    t2[g] = q1[g] * mu * (tau[g] - expVal[g]) / sigT2[g]; 
    t3[g] = q2[g] * mu2 * reuse[g];
    t4[g] = state_flux_acc[g] * (1.f - expVal[g]);
    state_flux_acc[g] = t1[g]+t2[g]+t3[g]+t4[g];
  }
}

int main( int argc, char * argv[] )
{
  unsigned int seed = 2;

  srand(seed);

  // Get Inputs
  Input * I = set_default_input();
  read_CLI( argc, argv, I );

  // Calculate Number of 3D Source Regions
  I->source_3D_regions = (int) ceil((double)I->source_2D_regions *
      I->coarse_axial_intervals / I->decomp_assemblies_ax);

  logo(4); // Based on the 4th version

  // Build Source data (needed when verification is disabled)
  Source *S = initialize_sources(I); 

  // Build Device data from Source data
  Source *S2 = copy_sources(I, S); 

  print_input_summary(I);

  center_print("SIMULATION", 79);
  border_print();
  printf("Attentuating fluxes across segments...\n");

  // Run Simulation Kernel Loop

  // Host allocation
  SIMD_Vectors simd_vecs = allocate_simd_vectors(I);

  float * state_flux = (float *) malloc(I->egroups * sizeof(float));

  // Device allocation
  float * state_flux_device = NULL;
  posix_memalign( (void**)&state_flux_device, 1024, I->egroups * sizeof(float));

  int* QSR_id_arr = NULL;
  int* FAI_id_arr = NULL;
  posix_memalign( (void**)&QSR_id_arr, 1024, sizeof(int) * I->segments );
  posix_memalign( (void**)&FAI_id_arr, 1024, sizeof(int) * I->segments );

  // initialize the state flux 
  for( int i = 0; i < I->egroups; i++ ) {
    state_flux_device[i] = rand_r(&seed) / (float) RAND_MAX;
    state_flux[i] = state_flux_device[i];
  }

  // Verification is performed for one segment;
  // Attentate segment is not run on CPU to reduce simulation time
  for( long i = 0; i < I->segments; i++ )
  {
    // Pick Random QSR
    int QSR_id = rand_r(&seed) % I->source_3D_regions;

    // for device
    QSR_id_arr[i] = QSR_id;

    // Pick Random Fine Axial Interval
    int FAI_id = rand_r(&seed) % I->fine_axial_intervals;

    // for device
    FAI_id_arr[i] = FAI_id;

    // Attenuate Segment for one segment
#ifdef VERIFY
    attenuate_segment( I, S, QSR_id, FAI_id, state_flux, &simd_vecs);
#endif
  }

#ifdef VERIFY
  float* simd_vecs_debug = (float*) malloc (sizeof(float)*I->egroups*14);
#endif

  double start = get_time();

  int fine_axial_intervals = I->fine_axial_intervals;
  int egroups = I->egroups;
  int segments = I->segments;

  int *d_QSR_id;
  int *d_FAI_id;
  float *d_fine_flux;
  float *d_fine_source;
  float *d_sigT;
  float* d_state_flux;
  float* d_simd_vecs;

  hipMalloc((void**)&d_QSR_id, sizeof(int)*I->segments);
  hipMemcpy(d_QSR_id, QSR_id_arr, sizeof(int)*I->segments, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_FAI_id, sizeof(int)*I->segments);
  hipMemcpy(d_FAI_id, FAI_id_arr, sizeof(int)*I->segments, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_fine_flux, 
      sizeof(float) * I->source_3D_regions * I->fine_axial_intervals * I->egroups);

  hipMemcpy(d_fine_flux, S2->fine_flux, 
      sizeof(float) * I->source_3D_regions * I->fine_axial_intervals * I->egroups,
      hipMemcpyHostToDevice);

  hipMalloc((void**)&d_fine_source, sizeof(float)*
      I->source_3D_regions * I->fine_axial_intervals * I->egroups);

  hipMemcpy(d_fine_source, S2->fine_source, 
      sizeof(float) * I->source_3D_regions * I->fine_axial_intervals * I->egroups,
      hipMemcpyHostToDevice);

  hipMalloc((void**)&d_sigT, sizeof(float)*I->source_3D_regions * I->egroups);
  hipMemcpy(d_sigT, S2->sigT, 
      sizeof(float) * I->source_3D_regions * I->egroups,
      hipMemcpyHostToDevice);

  hipMalloc((void**)&d_state_flux, sizeof(float)*I->egroups);
  hipMemcpy(d_state_flux, state_flux_device,
      sizeof(float) * I->egroups,
      hipMemcpyHostToDevice);

  hipMalloc((void**)&d_simd_vecs, sizeof(float)*I->egroups*14);

  dim3 grids ((segments+127)/128*128);
  dim3 threads (128);

  double kstart = get_time();

  for (int n = 0; n < I->repeat; n++) 
    att<<<grids, threads>>>(
        d_QSR_id,
        d_FAI_id,
        d_fine_flux,
        d_fine_source,
        d_sigT,
        d_state_flux,
        d_simd_vecs,
        fine_axial_intervals,
        egroups,
        segments );

  hipDeviceSynchronize();
  double kstop = get_time();

  hipMemcpy(state_flux_device, d_state_flux, 
      sizeof(float) * I->egroups,
      hipMemcpyDeviceToHost);

  hipMemcpy(S2->fine_flux, d_fine_flux, 
      sizeof(float) * I->source_3D_regions * I->fine_axial_intervals * I->egroups,
      hipMemcpyDeviceToHost);

#ifdef VERIFY
  hipMemcpy(simd_vecs_debug, d_simd_vecs, 
    sizeof(float) * I->egroups * 14,
    hipMemcpyDeviceToHost);
#endif

  hipFree(d_QSR_id);
  hipFree(d_FAI_id);
  hipFree(d_fine_flux);
  hipFree(d_fine_source);
  hipFree(d_sigT);
  hipFree(d_state_flux);
  hipFree(d_simd_vecs);
  
  double stop = get_time();
  printf("Simulation Complete.\n");

#ifdef VERIFY
  const float* q0 = simd_vecs_debug;
  const float* q1 = simd_vecs_debug + egroups;
  const float* q2 = simd_vecs_debug + egroups * 2;
  const float* sigT = simd_vecs_debug + egroups * 3;
  const float* tau = simd_vecs_debug + egroups * 4;
  const float* sigT2 = simd_vecs_debug + egroups * 5;
  const float* expVal = simd_vecs_debug + egroups * 6;
  const float* reuse = simd_vecs_debug + egroups * 7;
  const float* flux_integral = simd_vecs_debug + egroups * 8;
  const float* tally = simd_vecs_debug + egroups * 9;
  const float* t1 = simd_vecs_debug + egroups * 10;
  const float* t2 = simd_vecs_debug + egroups * 11;
  const float* t3 = simd_vecs_debug + egroups * 12;
  const float* t4 = simd_vecs_debug + egroups * 13;
  for (int g = 0; g < I->egroups; g++) {
    printf("q0[%d] = %f\n", g, q0[g]);
    printf("q1[%d] = %f\n", g, q1[g]);
    printf("q2[%d] = %f\n", g, q2[g]);
    printf("sigT[%d] = %f\n", g, sigT[g]);
    printf("tau[%d] = %f\n", g, tau[g]);
    printf("sigT2[%d] = %f\n", g, sigT2[g]);
    printf("expVal[%d] = %f\n", g, expVal[g]);
    printf("reuse[%d] = %f\n", g, reuse[g]);
    printf("flux_integral[%d] = %f\n", g, flux_integral[g]);
    printf("tally[%d] = %f\n", g, tally[g]);
    printf("t1[%d] = %f\n", g, t1[g]);
    printf("t2[%d] = %f\n", g, t2[g]);
    printf("t3[%d] = %f\n", g, t3[g]);
    printf("t4[%d] = %f\n", g, t4[g]);
  }
  
  bool error = false;
  for (int i = 0; i < I->egroups; i++) {
    if ( fabs(state_flux_device[i] - state_flux[i]) > 1e-1 ) {
      printf("%f %f\n", state_flux_device[i], state_flux[i]);
      error = true;
      break;
    }
  }
  if (error)
    printf("Fail\n");
  else
    printf("Success\n");
  
 #endif
  
  border_print();
  center_print("RESULTS SUMMARY", 79);
  border_print();

  printf("%-25s%.3lf seconds\n", "Total kernel time:", kstop-kstart);
  printf("%-25s%.3lf seconds\n", "Device offload time:", stop-start);

  double tpi = ((double) (kstop - kstart) / (I->repeat) /
                (double)I->segments / (double) I->egroups) * 1.0e9;
  printf("%-25s%.3lf ns\n", "Time per Intersection:", tpi);
  border_print();

  free(simd_vecs.q0);
  free(state_flux);
  free(QSR_id_arr);
  free(FAI_id_arr);
  free(state_flux_device);

  free(S2->fine_source);
  free(S2->fine_flux);
  free(S2->sigT);
  free(S2);
  free(S[0].fine_source);
  free(S[0].fine_flux);
  free(S[0].sigT);
  free(S);
  free(I);
  return 0;
}
