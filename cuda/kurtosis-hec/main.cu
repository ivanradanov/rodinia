#include <chrono>
#include <cstdio>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include "StoreTypedefs.h"
#include "StoreElement.h"
#include "kurtosis.h"

int main(int argc, char* argv[]) {
  if (argc != 3) {
    std::cout << "Usage ./main <elemCount> <repeat>\n";
    return 1;
  }

  const int elemCount = atoi(argv[1]);
  const int repeat = atoi(argv[2]);
    
  storeElement *elem = new storeElement[elemCount];

  std::mt19937 gen(19937);
  std::uniform_real_distribution<float> dis(1.f, 2.f);
  for (int i = 0; i < elemCount; i++)
    elem[i] = {i, 0, (unsigned long long int)i, dis(gen)};

  storeElement *d_elem;
  hipMalloc((void**)&d_elem, sizeof(storeElement) * elemCount);
  hipMemcpy(d_elem, elem, sizeof(storeElement) * elemCount, hipMemcpyHostToDevice);

  auto start = std::chrono::steady_clock::now();

  kurtosisResult* result;
  const size_t s = kurtosis(d_elem, elemCount, repeat, (void**)&result);

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Total device compute time: " << time * 1e-9 << " (s)\n";
  
  hipFree(d_elem);

  std::cout << "Results:" << std::endl;
  std::cout << s << " "
            << result->count << " "
            << result->m2 << " "
            << result->m3 << " "
            << result->m4 << std::endl;

  delete [] elem;
  delete result;
  return 0;
}
